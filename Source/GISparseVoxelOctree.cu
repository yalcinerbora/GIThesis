#include "hip/hip_runtime.h"
#include "GISparseVoxelOctree.h"
#include <cuda_gl_interop.h>
#include "GICudaAllocator.h"
#include "GIKernels.cuh"
#include "CudaTimer.h"
#include "Macros.h"
#include "Camera.h"
#include "Globals.h"
#include "IEUtility/IEMath.h"

GISparseVoxelOctree::GISparseVoxelOctree()
	: svoNodeBuffer(512)
	, svoMaterialBuffer(512)
	, dSVONodeAllocator(1)
	, dSVOConstants(1)
	, tSVODense(0)
	, computeVoxTraceWorld(ShaderType::COMPUTE, "Shaders/VoxTraceWorld.glsl")
	, svoTraceData(1)
{
	svoTraceData.AddData({});
}

GISparseVoxelOctree::~GISparseVoxelOctree()
{
	if(svoNodeResource) CUDA_CHECK(hipGraphicsUnregisterResource(svoNodeResource));
	if(svoMaterialResource) CUDA_CHECK(hipGraphicsUnregisterResource(svoMaterialResource));
	if(tSVODense) CUDA_CHECK(hipDestroyTextureObject(tSVODense));
	if(denseArray) CUDA_CHECK(hipFreeArray(denseArray));
}

__global__ void fastreadkernel3D(hipTextureObject_t texture,
							   unsigned int* gOut)
{
	uint3 globalId;
	globalId.x = threadIdx.x + blockIdx.x * blockDim.x;
	globalId.y = threadIdx.y + blockIdx.y * blockDim.y;
	globalId.z = threadIdx.z + blockIdx.z * blockDim.z;

	float3 texUV;
	texUV.x = static_cast<float>(globalId.x);
	texUV.y = static_cast<float>(globalId.y);
	texUV.z = static_cast<float>(globalId.z);

	unsigned int currentNode = tex3D<unsigned int>(texture, texUV.x, texUV.y, texUV.z);
	gOut[globalId.z * GI_DENSE_SIZE * GI_DENSE_SIZE +
		 globalId.y	* GI_DENSE_SIZE + 
		 globalId.x] = currentNode;
}

__global__ void fastreadkernel(hipTextureObject_t texture,
								 unsigned int* gOut)
{
	unsigned int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	
	float3 texUV;
	texUV.x = static_cast<float>(globalId % GI_DENSE_SIZE);
	texUV.y = static_cast<float>((globalId / GI_DENSE_SIZE) % GI_DENSE_SIZE);
	texUV.z = static_cast<float>(globalId / GI_DENSE_SIZE / GI_DENSE_SIZE);

	unsigned int currentNode = tex3D<unsigned int>(texture, texUV.x, texUV.y, texUV.z);
	printf("%d TexCoordXYZ %f, %f, %f\n", globalId, texUV.x, texUV.y, texUV.z);

	gOut[globalId] = currentNode;
}

void GISparseVoxelOctree::LinkAllocators(GICudaAllocator** newAllocators,
										 size_t allocatorSize,
										 float sceneMultiplier)
{
	allocatorGrids.clear();
	allocators.resize(allocatorSize);
	allocatorGrids.resize(allocatorSize);

	assert(allocatorSize > 0);
	assert(newAllocators != nullptr);

	std::copy(newAllocators, newAllocators + allocatorSize, allocators.data());
	for(unsigned int i = 0; i < allocatorSize; i++)
		allocatorGrids[i] = &(newAllocators[i]->GetVoxelGridHost());

	// TODO: More Dynamic Allocation Scheme
	size_t totalAlloc = static_cast<size_t>(sceneMultiplier * 1024.0f * 1024.0f);
	svoNodeBuffer.Resize(totalAlloc + GI_DENSE_SIZE * GI_DENSE_SIZE * GI_DENSE_SIZE);

	dSVODense = nullptr;
	dSVOSparse = nullptr;

	// Mat Tree holds up to level 0
	matSparseOffset = static_cast<unsigned int>((1.0 - std::pow(8.0f, GI_DENSE_LEVEL + 1)) / (1.0f - 8.0f));
	svoMaterialBuffer.Resize(totalAlloc + matSparseOffset);

	hSVOLevelOffsets.resize(allocatorGrids[0]->depth + allocatorSize - GI_DENSE_LEVEL + 2);
	hSVOLevelSizes.resize(allocatorGrids[0]->depth + allocatorSize - GI_DENSE_LEVEL);
	dSVOLevelSizes.Resize(allocatorGrids[0]->depth + allocatorSize - GI_DENSE_LEVEL);
	
	CUDA_CHECK(hipGraphicsGLRegisterBuffer(&svoNodeResource, 
											svoNodeBuffer.getGLBuffer(), 
											cudaGLMapFlagsWriteDiscard));

	CUDA_CHECK(hipGraphicsGLRegisterBuffer(&svoMaterialResource, 
											svoMaterialBuffer.getGLBuffer(), 
											cudaGLMapFlagsWriteDiscard));

	// Clear All Data
	GLuint allOne = 0xFFFFFFFF;
	GLuint zero = 0;

	glBindBuffer(GL_COPY_WRITE_BUFFER, svoNodeBuffer.getGLBuffer());
	glClearBufferData(GL_COPY_WRITE_BUFFER, GL_R32UI, GL_RED_INTEGER, GL_UNSIGNED_INT, &allOne);

	glBindBuffer(GL_COPY_WRITE_BUFFER, svoMaterialBuffer.getGLBuffer());
	glClearBufferData(GL_COPY_WRITE_BUFFER, GL_R32UI, GL_RED_INTEGER, GL_UNSIGNED_INT, &zero);

	dSVONodeAllocator.Memset(0x00, 0, 1);
	dSVOLevelSizes.Memset(0x00, 0, dSVOLevelSizes.Size());
	std::fill(hSVOLevelSizes.begin(), hSVOLevelSizes.end(), 0);
	std::fill(hSVOLevelOffsets.begin(), hSVOLevelOffsets.end(), 0);
	
	unsigned int totalLevel = static_cast<unsigned int>(allocatorGrids[0]->depth + allocatorSize - 1);

	hSVOConstants.denseDepth = GI_DENSE_LEVEL;
	hSVOConstants.denseDim = GI_DENSE_SIZE;
	hSVOConstants.totalDepth = totalLevel;
	hSVOConstants.numCascades = static_cast<unsigned int>(allocatorSize);

	// Copy to device
	CUDA_CHECK(hipMemcpy(dSVOConstants.Data(), 
						  &hSVOConstants, 
						  sizeof(CSVOConstants), 
						  hipMemcpyHostToDevice));


	hipChannelFormatDesc fd = hipCreateChannelDesc<unsigned int>();
	if(denseArray) CUDA_CHECK(hipFreeArray(denseArray));
	CUDA_CHECK(hipMalloc3DArray(&denseArray,
								 &fd,
								 {GI_DENSE_SIZE, GI_DENSE_SIZE, GI_DENSE_SIZE},
								 hipArrayDefault));

	// Texture of SVO Dense
	hipResourceDesc resDesc = {};
	hipTextureDesc texDesc = {};
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = denseArray;
	
	texDesc.addressMode[0] = hipAddressModeWrap;
	texDesc.addressMode[1] = hipAddressModeWrap;
	texDesc.addressMode[2] = hipAddressModeWrap;
	texDesc.filterMode = hipFilterModePoint;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = 0;

	if(tSVODense != 0) CUDA_CHECK(hipDestroyTextureObject(tSVODense));
	CUDA_CHECK(hipCreateTextureObject(&tSVODense, &resDesc, &texDesc, nullptr));
}

void GISparseVoxelOctree::ConstructDense()
{
	// Level 0 does not gurantee lower cascade parents
	// Each Allocator tries to allocate its parent
	for(unsigned int i = 0; i < allocators.size(); i++)
	{
		uint32_t gridSize = ((allocators[i]->NumPages() * GI_PAGE_SIZE) + 
							GI_THREAD_PER_BLOCK - 1) /
							GI_THREAD_PER_BLOCK;
		SVOReconstructDetermineNode<<<gridSize, GI_THREAD_PER_BLOCK>>>
		(
			dSVODense,
			allocators[i]->GetVoxelPagesDevice(),
		
			i,
			*dSVOConstants.Data()
		);
		CUDA_KERNEL_CHECK();
	}

	uint32_t gridSize = ((GI_DENSE_SIZE * GI_DENSE_SIZE * GI_DENSE_SIZE) + GI_THREAD_PER_BLOCK - 1) /
						  GI_THREAD_PER_BLOCK;
	SVOReconstructAllocateLevel<<<gridSize, GI_THREAD_PER_BLOCK>>>
	(
		dSVODense,
		dSVOLevelSizes.Data(),
		*dSVONodeAllocator.Data(),

		0,
		static_cast<unsigned int>(svoNodeBuffer.Capacity() - GI_DENSE_SIZE * GI_DENSE_SIZE * GI_DENSE_SIZE),
		GI_DENSE_LEVEL,
		GI_DENSE_SIZE * GI_DENSE_SIZE * GI_DENSE_SIZE,
		*dSVOConstants.Data()
	);
	CUDA_KERNEL_CHECK();

	// Copy Level Start Location to array
	CUDA_CHECK(hipMemcpy(hSVOLevelOffsets.data() + 2,
							dSVONodeAllocator.Data(),
							sizeof(unsigned int), 
							hipMemcpyDeviceToHost));
}

void GISparseVoxelOctree::ConstructLevel(unsigned int currentLevel,
										 unsigned int allocatorOffset)
{
	// Early Bail check 
	unsigned int currentLevelIndex = currentLevel - GI_DENSE_LEVEL;
	CUDA_CHECK(hipMemcpy(hSVOLevelSizes.data() + currentLevelIndex,
						  dSVOLevelSizes.Data() + currentLevelIndex,
						  sizeof(unsigned int),
						  hipMemcpyDeviceToHost));
	if(hSVOLevelSizes[currentLevelIndex] == 0) return;

	// ChildBitSet your Level (with next level's child)
	// Allocate next level
	// Memcopy next level start location to array
	// Only ChildBitSet Upper Level
	// Then Allocate your level
	// Average Color to the level
	for(unsigned int i = allocatorOffset; i < allocators.size(); i++)
	{
		uint32_t gridSize = ((allocators[i]->NumPages() * GI_PAGE_SIZE) + 
							 GI_THREAD_PER_BLOCK - 1) /
							 GI_THREAD_PER_BLOCK;

		SVOReconstructDetermineNode<<<gridSize, GI_THREAD_PER_BLOCK>>>
		(
			dSVOSparse,
			tSVODense,
			allocators[i]->GetVoxelPagesDevice(),

			i,
			currentLevel,
			*dSVOConstants.Data()
		);
		CUDA_KERNEL_CHECK();
	}
	
	uint32_t gridSize = (hSVOLevelSizes[currentLevelIndex] + GI_THREAD_PER_BLOCK - 1) / GI_THREAD_PER_BLOCK;
	SVOReconstructAllocateLevel<<<gridSize, GI_THREAD_PER_BLOCK>>>
	(
		dSVOSparse,
		dSVOLevelSizes.Data(),
		*dSVONodeAllocator.Data(),

		hSVOLevelOffsets[currentLevelIndex],
		static_cast<unsigned int>(svoNodeBuffer.Capacity() - GI_DENSE_SIZE * GI_DENSE_SIZE * GI_DENSE_SIZE),
		currentLevel,
		hSVOLevelSizes[currentLevelIndex],
		*dSVOConstants.Data()
	);
	CUDA_KERNEL_CHECK();

	// Copy Level Start Location to array
	CUDA_CHECK(hipMemcpy(hSVOLevelOffsets.data() + ((currentLevel - GI_DENSE_LEVEL) + 2),
						  dSVONodeAllocator.Data(),
						  sizeof(unsigned int),
						  hipMemcpyDeviceToHost));
}

void GISparseVoxelOctree::ConstructFullAtomic()
{
	// Fully Atomic Version
	for(unsigned int i = 0; i < allocators.size(); i++)
	{
		uint32_t gridSize = (allocators[i]->NumPages() * GI_PAGE_SIZE + GI_THREAD_PER_BLOCK - 1) /
							GI_THREAD_PER_BLOCK;
		SVOReconstruct<<<gridSize, GI_THREAD_PER_BLOCK>>>
		(
			dSVOMaterial,
			dSVOSparse,
			dSVODense,
			dSVOLevelSizes.Data(),
			*dSVONodeAllocator.Data(),

			allocators[i]->GetVoxelPagesDevice(),
			allocators[i]->GetObjRenderCacheDevice(),

			matSparseOffset,
			static_cast<unsigned int>(svoNodeBuffer.Capacity() - (GI_DENSE_SIZE * GI_DENSE_SIZE * GI_DENSE_SIZE)),
			i,
			*dSVOConstants.Data()
		);
		CUDA_KERNEL_CHECK();
	}

	// Leafs have averaged colors
	// Reduce Colors to lower levels
	// This is tricky since we do not know which depth is where
	// TODO:
	



	
	// Copy Dense to Texture
	hipMemcpy3DParms params = { 0 };
	params.dstArray = denseArray;
	params.srcPtr =
	{
		dSVODense,
		GI_DENSE_SIZE * sizeof(unsigned int),
		GI_DENSE_SIZE,
		GI_DENSE_SIZE
	};
	params.extent = { GI_DENSE_SIZE, GI_DENSE_SIZE, GI_DENSE_SIZE };
	params.kind = hipMemcpyDeviceToDevice;
	CUDA_CHECK(hipMemcpy3D(&params));
}

void GISparseVoxelOctree::ConstructLevelByLevel()
{
	// Start with constructing dense
	ConstructDense();

	// Copy Dense to Texture
	hipMemcpy3DParms params = { 0 };
	params.dstArray = denseArray;
	params.srcPtr =
	{
		dSVODense,
		GI_DENSE_SIZE * sizeof(unsigned int),
		GI_DENSE_SIZE,
		GI_DENSE_SIZE
	};
	params.extent = { GI_DENSE_SIZE, GI_DENSE_SIZE, GI_DENSE_SIZE };
	params.kind = hipMemcpyDeviceToDevice;
	CUDA_CHECK(hipMemcpy3D(&params));

	// Construct Levels
	for(unsigned int i = GI_DENSE_LEVEL + 1; i < allocatorGrids[0]->depth; i++)
	{
		ConstructLevel(i, 0);
	}

	// Now adding cascade levels
	for(unsigned int i = 1; i < allocators.size(); i++)
	{
		unsigned int currentLevel = allocatorGrids[0]->depth + i - 1;
		ConstructLevel(currentLevel, i);
	}

	// Last memcpy of the leaf cascade size
	CUDA_CHECK(hipMemcpy(hSVOLevelSizes.data() + (allocatorGrids[0]->depth - GI_DENSE_LEVEL),
		dSVOLevelSizes.Data() + (allocatorGrids[0]->depth - GI_DENSE_LEVEL),
		sizeof(unsigned int),
		hipMemcpyDeviceToHost));
}

void GISparseVoxelOctree::AverageNodes(bool orderedNodes)
{
	// Leaf Nodes Already ordered
	
	// First Average Leafs atomically	
	//for(unsigned int i = 0; i < allocators.size(); i++)
	//{
	//	assert(allocators[i]->IsGLMapped() == true);
	//	uint32_t gridSize = (allocators[i]->NumPages() * GI_PAGE_SIZE +  GI_THREAD_PER_BLOCK - 1) / 
	//						GI_THREAD_PER_BLOCK;
	//			
	//	// Average Leaf Node
	//	SVOReconstructAverageLeaf<<<gridSize, GI_THREAD_PER_BLOCK>>>
	//	(
	//		dSVOMaterial.Data(),
	//		dSVOSparse,
	//		tSVODense,
	//		allocators[i]->GetVoxelPagesDevice(),
	//		dSVOLevelStartIndices.Data(),
	//		allocators[i]->GetObjRenderCacheDevice(),
	//		matSparseOffset,
	//		i,
	//		hSVOConstants.totalDepth - (hSVOConstants.numCascades - i),
	//		*dSVOConstants.Data()
	//	);
	//	CUDA_KERNEL_CHECK();
	//}

	// Now use leaf nodes to average upper nodes
	// Start bottom up (dont average until inner averages itself
	// TODO
}

double GISparseVoxelOctree::UpdateSVO()
{
	CUDA_CHECK(hipGraphicsMapResources(1, &svoMaterialResource));
	CUDA_CHECK(hipGraphicsMapResources(1, &svoNodeResource));
	
	size_t size;
	CUDA_CHECK(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&dSVODense), 
													 &size, svoNodeResource));
	CUDA_CHECK(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&dSVOMaterial),
													 &size, svoMaterialResource));
	dSVOSparse = dSVODense + GI_DENSE_SIZE * GI_DENSE_SIZE * GI_DENSE_SIZE;

	CudaTimer timer;
	timer.Start();

	// Reset Atomic Counter since we reconstruct every frame
	unsigned int usedNodeCount;
	CUDA_CHECK(hipMemcpy(&usedNodeCount, dSVONodeAllocator.Data(), sizeof(unsigned int),
						  hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemset(dSVODense, 0xFF, sizeof(CSVONode) * (usedNodeCount + GI_DENSE_SIZE * GI_DENSE_SIZE * GI_DENSE_SIZE)));
	CUDA_CHECK(hipMemset(dSVOMaterial + matSparseOffset, 0x00, sizeof(CSVOMaterial) * usedNodeCount));

	dSVONodeAllocator.Memset(0x00, 0, 1);
	dSVOLevelSizes.Memset(0x00, 0, dSVOLevelSizes.Size());
	std::fill(hSVOLevelSizes.begin(), hSVOLevelSizes.end(), 0);
	std::fill(hSVOLevelOffsets.begin(), hSVOLevelOffsets.end(), 0);

	// Maxwell is faster with fully atomic code (CAS Locks etc.)
	// However kepler sucks (100ms compared to 5ms) 
	if(CudaInit::CapabilityMajor() >= 5)
	{
		// Since fully atomic construction does not 
		// create level nodes in ordered manner
		// for each level we need to traverse node
		ConstructFullAtomic();
		AverageNodes(false);
	}
	else
	{
		ConstructLevelByLevel();
		AverageNodes(true);
	}

	//// DEBUG
	//GI_LOG("-------------------------------------------");
	//GI_LOG("Tree Node Data");
	//CUDA_CHECK(hipMemcpy(hSVOLevelSizes.data(),
	//					  dSVOLevelSizes.Data(),
	//					  dSVOLevelSizes.Size() * sizeof(unsigned int),
	//					  hipMemcpyDeviceToHost));
	//unsigned int i;
	//for(i = 0; i <= allocatorGrids[0].depth - GI_DENSE_LEVEL + allocators.size() - 1; i++)
	//{
	//	if(i == 0) GI_LOG("#%d Dense : %d", GI_DENSE_LEVEL + i, GI_DENSE_SIZE * GI_DENSE_SIZE * GI_DENSE_SIZE);
	//	else GI_LOG("#%d Level : %d", GI_DENSE_LEVEL + i, hSVOLevelSizes[i]);
	//}
	//unsigned int total;
	//CUDA_CHECK(hipMemcpy(&total, dSVONodeAllocator.Data(), sizeof(unsigned int),
	//					  hipMemcpyDeviceToHost));
	//GI_LOG("Total : %d", total);
	//GI_LOG("-------------------------------------------");

	timer.Stop();
	
	CUDA_CHECK(hipGraphicsUnmapResources(1, &svoMaterialResource));
	CUDA_CHECK(hipGraphicsUnmapResources(1, &svoNodeResource));
	return timer.ElapsedMilliS();
}

double GISparseVoxelOctree::ConeTrace(GLuint depthBuffer,
									  GLuint normalBuffer,
									  GLuint colorBuffer,
									  const Camera& camera)
{
	return 0.0;
}

double GISparseVoxelOctree::DebugTraceSVO(GLuint writeImage,
										  StructuredBuffer<InvFrameTransform>& invFT,
										  FrameTransformBuffer& ft,
										  const uint2& imgDim)
{
	// Timing Voxelization Process
	GLuint queryID;
	glGenQueries(1, &queryID);
	glBeginQuery(GL_TIME_ELAPSED, queryID);

	// Set Cascade Trace Data
	float3 pos = allocatorGrids[0]->position;
	uint32_t dim = allocatorGrids[0]->dimension.x * (0x1 << (allocators.size() - 1));
	uint32_t depth = allocatorGrids[0]->depth + static_cast<uint32_t>(allocators.size()) - 1;
	svoTraceData.CPUData()[0] = 
	{
		{pos.x, pos.y, pos.z, allocatorGrids.back()->span},
		{dim, depth, GI_DENSE_SIZE, GI_DENSE_LEVEL},
		{
			static_cast<unsigned int>(allocators.size()), 
			GI_DENSE_SIZE * GI_DENSE_SIZE * GI_DENSE_SIZE,
			matSparseOffset,
			0
		}
	};
	svoTraceData.SendData();

	// Shaders
	computeVoxTraceWorld.Bind();

	// Buffers
	svoNodeBuffer.BindAsShaderStorageBuffer(LU_SVO_NODE);
	svoMaterialBuffer.BindAsShaderStorageBuffer(LU_SVO_MATERIAL);
	invFT.BindAsUniformBuffer(U_INVFTRANSFORM);
	ft.Bind();
	svoTraceData.BindAsUniformBuffer(U_SVO_CONSTANTS);

	// Images
	glBindImageTexture(I_COLOR_FB, writeImage, 0, false, 0, GL_WRITE_ONLY, GL_RGBA8);

	// Dispatch
	uint2 gridSize;
	gridSize.x = (imgDim.x + 16 - 1) / 16;
	gridSize.y = (imgDim.y + 16 - 1) / 16;
	glDispatchCompute(gridSize.x, gridSize.y, 1);
	
	// Timer
	GLuint64 timeElapsed = 0;
	glEndQuery(GL_TIME_ELAPSED);
	glGetQueryObjectui64v(queryID, GL_QUERY_RESULT, &timeElapsed);

	glMemoryBarrier(GL_SHADER_IMAGE_ACCESS_BARRIER_BIT);
	return timeElapsed / 1000000.0;
}

uint64_t GISparseVoxelOctree::MemoryUsage() const
{
	uint64_t totalBytes = 0;
	totalBytes += svoNodeBuffer.Capacity() * sizeof(CSVONode);
	totalBytes += svoMaterialBuffer.Capacity() * sizeof(CSVOMaterial);
	totalBytes += dSVOLevelSizes.Size() * sizeof(unsigned int);
	totalBytes += sizeof(unsigned int);
	return totalBytes;
}