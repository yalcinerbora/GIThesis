#include "GISparseVoxelOctree.h"
#include <cuda_gl_interop.h>
#include "GICudaAllocator.h"
#include "GIKernels.cuh"
#include "CudaTimer.h"

const unsigned int GISparseVoxelOctree::TPBWithHelperWarp = GI_THREAD_PER_BLOCK_PRIME + (32 - (GI_THREAD_PER_BLOCK_PRIME % 32));

GISparseVoxelOctree::GISparseVoxelOctree(GLuint lightIntensityTex)
	: dSVO()
	, lightIntensityTexLink(nullptr)
	, dSVONodeCountAtomic(1)
	, dSVOConstants(1)
{
	

	CUDA_CHECK(hipGraphicsGLRegisterImage(&lightIntensityTexLink, lightIntensityTex,
											GL_TEXTURE_2D,
											cudaGraphicsMapFlagsWriteDiscard));

}

GISparseVoxelOctree::~GISparseVoxelOctree()
{
	if(lightIntensityTexLink) CUDA_CHECK(hipGraphicsUnregisterResource(lightIntensityTexLink));
	if(lightBufferLink) CUDA_CHECK(hipGraphicsUnregisterResource(lightBufferLink));
	if(shadowMapArrayTexLink) CUDA_CHECK(hipGraphicsUnregisterResource(shadowMapArrayTexLink));
}

void GISparseVoxelOctree::LinkAllocators(GICudaAllocator** newAllocators,
										 size_t allocatorSize)
{
	allocatorGrids.clear();
	allocators.resize(allocatorSize);
	allocatorGrids.resize(allocatorSize);

	assert(allocatorSize > 0);
	assert(newAllocators != nullptr);

	std::copy(newAllocators, newAllocators + allocatorSize, allocators.data());
	for(unsigned int i = 0; i < allocatorSize; i++)
		allocatorGrids[i] = newAllocators[i]->GetVoxelGridHost();

	// TODO: More Dynamic Allocation Scheme
	size_t totalAlloc = GI_DENSE_SIZE * GI_DENSE_SIZE * GI_DENSE_SIZE;
	for(unsigned int i = 0; i < allocatorSize; i++)
	{
		uint32_t depthMultiplier = 1;
		if(i == 0) depthMultiplier = (allocatorGrids[i].depth - GI_DENSE_LEVEL);
		totalAlloc += allocators[i]->NumPages() * GI_PAGE_SIZE * depthMultiplier;
	}
	dSVO.Resize(totalAlloc);
	dSVOColor.Resize(totalAlloc);

	dSVODense = dSVO.Data();
	dSVOSparse = dSVO.Data() + (GI_DENSE_SIZE * GI_DENSE_SIZE * GI_DENSE_SIZE);

	dSVOLevelStartIndices.Resize(allocatorGrids[0].depth + allocatorSize - 1);
	dSVOLevelStartIndices.Memset(0x00, 0, dSVOLevelStartIndices.Size());

	unsigned int totalLevel = static_cast<unsigned int>(allocatorGrids[0].depth + allocatorSize - 1);

	hSVOConstants.denseDepth = GI_DENSE_LEVEL;
	hSVOConstants.denseDim = GI_DENSE_SIZE;
	hSVOConstants.totalDepth = totalLevel;
	hSVOConstants.numCascades = static_cast<unsigned int>(allocatorSize);

	// Copy to device
	CUDA_CHECK(hipMemcpy(dSVOConstants.Data(), 
						  &hSVOConstants, 
						  sizeof(CSVOConstants), 
						  hipMemcpyHostToDevice));
}

void GISparseVoxelOctree::ConstructDense()
{
	// Level 0 is special it constructs the upper levels in addition to its level
	uint32_t gridSize = ((allocators[0]->NumPages() * GI_PAGE_SIZE) + TPBWithHelperWarp - 1) /
						 TPBWithHelperWarp;
	SVOReconstructChildSet<<<gridSize, TPBWithHelperWarp>>>
	(
		dSVODense,
		allocators[0]->GetVoxelPagesDevice(),
		
		0u,
		*dSVOConstants.Data()
	);
	CUDA_KERNEL_CHECK();

	dSVO.DumpToFile("svoDump", 0, GI_DENSE_SIZE * GI_DENSE_SIZE * GI_DENSE_SIZE);

	gridSize = ((GI_DENSE_SIZE * GI_DENSE_SIZE * GI_DENSE_SIZE) + GI_THREAD_PER_BLOCK - 1) /
				GI_THREAD_PER_BLOCK;
	SVOReconstructAllocateNext<<<gridSize, GI_THREAD_PER_BLOCK>>>
	(
		dSVO.Data(),
		*dSVONodeCountAtomic.Data(),
		*dSVOLevelStartIndices.Data(),
		GI_DENSE_SIZE * GI_DENSE_SIZE * GI_DENSE_SIZE
	);
	CUDA_KERNEL_CHECK();

	// Copy Level Start Location to array
	CUDA_CHECK(hipMemcpy(dSVOLevelStartIndices.Data() + 1,
						  dSVONodeCountAtomic.Data(),
						  sizeof(unsigned int), hipMemcpyDeviceToDevice));

	dSVO.DumpToFile("svoDump", 0, GI_DENSE_SIZE * GI_DENSE_SIZE * GI_DENSE_SIZE);
}

void GISparseVoxelOctree::ConstructLevel(unsigned int currentLevel,
										 unsigned int allocatorIndex,
										 unsigned int cascadeNo)
{
	// ChildBitSet your Level
	// Allocate next level
	// Memcopy next level start location to array
	// Only ChildBitSet Upper Level
	// Then Allocate your level
	// Average Color to the level
	unsigned int currentLevelIndex = currentLevel - GI_DENSE_LEVEL;
	uint32_t gridSize = ((allocators[allocatorIndex]->NumPages() * GI_PAGE_SIZE) + TPBWithHelperWarp - 1) /
						 TPBWithHelperWarp;

	SVOReconstructChildSet<<<gridSize, TPBWithHelperWarp>>>
	(
		dSVOSparse,
		dSVODense,
		allocators[allocatorIndex]->GetVoxelPagesDevice(),
		dSVOLevelStartIndices.Data(),

		cascadeNo,
		currentLevel,
		*dSVOConstants.Data()
	);
	CUDA_KERNEL_CHECK();

	// Call count is on GPU
	uint32_t levelNodeCount, levelNodeStarts[2];
	CUDA_CHECK(hipMemcpy(levelNodeStarts, 
						  dSVOLevelStartIndices.Data() + currentLevelIndex - 1, 
						  sizeof(unsigned int) * 2,
						  hipMemcpyDeviceToHost));
	levelNodeCount = levelNodeStarts[1] - levelNodeStarts[0];


	dSVO.DumpToFile("svoDump", 0, levelNodeStarts[1] +
					GI_DENSE_SIZE * GI_DENSE_SIZE * GI_DENSE_SIZE);


	gridSize = ((levelNodeCount) + GI_THREAD_PER_BLOCK - 1) / GI_THREAD_PER_BLOCK;
	SVOReconstructAllocateNext<<<gridSize, GI_THREAD_PER_BLOCK>>>
	(
		dSVOSparse,
		*dSVONodeCountAtomic.Data(),
		*(dSVOLevelStartIndices.Data() + currentLevelIndex - 1),
		levelNodeCount
	);
	CUDA_KERNEL_CHECK();

	dSVO.DumpToFile("svoDump", 0, levelNodeStarts[1] + 
					GI_DENSE_SIZE * GI_DENSE_SIZE * GI_DENSE_SIZE);

	// Copy Level Start Location to array
	CUDA_CHECK(hipMemcpy(dSVOLevelStartIndices.Data() + currentLevelIndex + 1, dSVONodeCountAtomic.Data(),
						  sizeof(unsigned int), hipMemcpyDeviceToDevice));
}

double GISparseVoxelOctree::UpdateSVO()
{
	CudaTimer timer;
	timer.Start();

	// Reset Atomic Counter since we reconstruct every frame
	unsigned int usedNodeCount;
	CUDA_CHECK(hipMemcpy(&usedNodeCount, dSVONodeCountAtomic.Data(), sizeof(unsigned int),
						  hipMemcpyDeviceToHost));
	dSVO.Memset(0x00, 0, usedNodeCount + GI_DENSE_SIZE * GI_DENSE_SIZE * GI_DENSE_SIZE);
	dSVONodeCountAtomic.Memset(0x00, 0, 1);
	dSVOLevelStartIndices.Memset(0x00, 0, dSVOLevelStartIndices.Size());

	// Start with constructing dense
	ConstructDense();
	
	//DEBUG
	dSVO.DumpToFile("svoDump", 0, GI_DENSE_SIZE * GI_DENSE_SIZE * GI_DENSE_SIZE);
	dSVOLevelStartIndices.DumpToFile("startIndices");

	// Construct Levels
	for(unsigned int i = GI_DENSE_LEVEL + 1; i < allocatorGrids[0].depth; i++)
	{
		ConstructLevel(i, 0, 0);

		//DEBUG
		dSVOLevelStartIndices.DumpToFile("startIndices");
	}

	//// Now adding cascade levels
	//for(unsigned int i = 1; i < allocators.size(); i++)
	//{
	//	unsigned int currentLevel = allocatorGrids[0].depth + i;
	//	ConstructLevel(currentLevel, i, i);
	//}

	timer.Stop();
	return timer.ElapsedMilliS();
}

double GISparseVoxelOctree::ConeTrace(GLuint depthBuffer,
									  GLuint normalBuffer,
									  GLuint colorBuffer,
									  const Camera& camera)
{
	return 0.0;
}

void GISparseVoxelOctree::LinkScene(GLuint lightBuffer,
									GLuint shadowMapArrayTexture)
{

}

uint64_t GISparseVoxelOctree::MemoryUsage() const
{
	uint64_t totalBytes = 0;
	totalBytes += dSVO.Size() * sizeof(CSVONode);
	totalBytes += dSVOColor.Size() * sizeof(CSVOColor);
	totalBytes += dSVOLevelStartIndices.Size() * sizeof(unsigned int);
	totalBytes += sizeof(unsigned int);
	return totalBytes;
}

//void GICudaAllocator::LinkSceneShadowMapArray(GLuint shadowMapArray)
//{
//	//CUDA_CHECK(hipGraphicsGLRegisterImage(&sceneShadowMapLink,
//	//									   shadowMapArray,
//	//									   GL_TEXTURE_2D_ARRAY,
//	//									   hipGraphicsRegisterFlagsReadOnly));
//}
//
//void GICudaAllocator::LinkSceneGBuffers(GLuint depthTex,
//										GLuint normalTex,
//										GLuint lightIntensityTex)
//{
//	//CUDA_CHECK(hipGraphicsGLRegisterImage(&depthBuffLink,
//	//										depthTex,
//	//										GL_TEXTURE_2D,
//	//										hipGraphicsRegisterFlagsReadOnly));
//	//CUDA_CHECK(hipGraphicsGLRegisterImage(&normalBuffLink,
//	//										normalTex,
//	//										GL_TEXTURE_2D,
//	//										hipGraphicsRegisterFlagsReadOnly));
//	//CUDA_CHECK(hipGraphicsGLRegisterImage(&lightIntensityLink,
//	//										lightIntensityTex,
//	//										GL_TEXTURE_2D,
//	//										hipGraphicsRegisterFlagsSurfaceLoadStore));
//}
//
//void GICudaAllocator::UnLinkGBuffers()
//{
//	//CUDA_CHECK(hipGraphicsUnregisterResource(depthBuffLink));
//	//CUDA_CHECK(hipGraphicsUnregisterResource(normalBuffLink));
//	//CUDA_CHECK(hipGraphicsUnregisterResource(lightIntensityLink));
//}

// Textures
//hipArray_t texArray;
//hipMipmappedArray_t mipArray;
//hipResourceDesc resDesc = {};
//hipTextureDesc texDesc = {};

//resDesc.resType = hipResourceTypeMipmappedArray;

//texDesc.addressMode[0] = hipAddressModeWrap;
//texDesc.addressMode[1] = hipAddressModeWrap;
//texDesc.filterMode = hipFilterModePoint;
//texDesc.readMode = hipReadModeElementType;
//texDesc.normalizedCoords = 1;

//CUDA_CHECK(hipGraphicsMapResources(1, &sceneShadowMapLink));
//CUDA_CHECK(cudaGraphicsResourceGetMappedMipmappedArray(&mipArray, sceneShadowMapLink));
//resDesc.res.mipmap.mipmap = mipArray;
//CUDA_CHECK(hipCreateTextureObject(&shadowMaps, &resDesc, &texDesc, nullptr));

//texDesc.normalizedCoords = 1;
//resDesc.resType = hipResourceTypeArray;

//CUDA_CHECK(hipGraphicsMapResources(1, &depthBuffLink));
//CUDA_CHECK(hipGraphicsSubResourceGetMappedArray(&texArray, depthBuffLink, 0, 0));
//resDesc.res.array.array = texArray;
//CUDA_CHECK(hipCreateTextureObject(&depthBuffer, &resDesc, &texDesc, nullptr));

//CUDA_CHECK(hipGraphicsMapResources(1, &normalBuffLink));
//CUDA_CHECK(hipGraphicsSubResourceGetMappedArray(&texArray, normalBuffLink, 0, 0));
//resDesc.res.array.array = texArray;
//CUDA_CHECK(hipCreateTextureObject(&normalBuffer, &resDesc, &texDesc, nullptr));

//CUDA_CHECK(hipGraphicsMapResources(1, &lightIntensityLink));
//CUDA_CHECK(hipGraphicsSubResourceGetMappedArray(&texArray, lightIntensityLink, 0, 0));
//resDesc.res.array.array = texArray;
//CUDA_CHECK(hipCreateSurfaceObject(&lightIntensityBuffer, &resDesc));

