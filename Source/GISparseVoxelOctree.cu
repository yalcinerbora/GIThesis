#include "hip/hip_runtime.h"
#include "GISparseVoxelOctree.h"
#include <cuda_gl_interop.h>
#include "GICudaAllocator.h"
#include "GIKernels.cuh"
#include "CudaTimer.h"
#include "Macros.h"

GISparseVoxelOctree::GISparseVoxelOctree(GLuint lightIntensityTex)
	: dSVO()
	, lightIntensityTexLink(nullptr)
	, dSVONodeCountAtomic(1)
	, dSVOConstants(1)
	, tSVODense(0)
	, vaoNormPosData(512)
	, vaoColorData(512)
{
	CUDA_CHECK(hipGraphicsGLRegisterImage(&lightIntensityTexLink, lightIntensityTex,
											GL_TEXTURE_2D,
											cudaGraphicsMapFlagsWriteDiscard));
}

GISparseVoxelOctree::~GISparseVoxelOctree()
{
	if(lightIntensityTexLink) CUDA_CHECK(hipGraphicsUnregisterResource(lightIntensityTexLink));
	if(lightBufferLink) CUDA_CHECK(hipGraphicsUnregisterResource(lightBufferLink));
	if(shadowMapArrayTexLink) CUDA_CHECK(hipGraphicsUnregisterResource(shadowMapArrayTexLink));
	if(tSVODense) CUDA_CHECK(hipDestroyTextureObject(tSVODense));
	if(denseArray) CUDA_CHECK(hipFreeArray(denseArray));
}

__global__ void fastreadkernel3D(hipTextureObject_t texture,
							   unsigned int* gOut)
{
	uint3 globalId;
	globalId.x = threadIdx.x + blockIdx.x * blockDim.x;
	globalId.y = threadIdx.y + blockIdx.y * blockDim.y;
	globalId.z = threadIdx.z + blockIdx.z * blockDim.z;

	float3 texUV;
	texUV.x = static_cast<float>(globalId.x);
	texUV.y = static_cast<float>(globalId.y);
	texUV.z = static_cast<float>(globalId.z);

	unsigned int currentNode = tex3D<unsigned int>(texture, texUV.x, texUV.y, texUV.z);
	gOut[globalId.z * GI_DENSE_SIZE * GI_DENSE_SIZE +
		 globalId.y	* GI_DENSE_SIZE + 
		 globalId.x] = currentNode;
}

__global__ void fastreadkernel(hipTextureObject_t texture,
								 unsigned int* gOut)
{
	unsigned int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	
	float3 texUV;
	texUV.x = static_cast<float>(globalId % GI_DENSE_SIZE);
	texUV.y = static_cast<float>((globalId / GI_DENSE_SIZE) % GI_DENSE_SIZE);
	texUV.z = static_cast<float>(globalId / GI_DENSE_SIZE / GI_DENSE_SIZE);

	/*x = idx % (max_x)
		idx /= (max_x)
		y = idx % (max_y)
		idx /= (max_y)
		z = idx
		return (x, y, z)*/


	unsigned int currentNode = tex3D<unsigned int>(texture, texUV.x, texUV.y, texUV.z);
	printf("%d TexCoordXYZ %f, %f, %f\n", globalId, texUV.x, texUV.y, texUV.z);

	gOut[globalId] = currentNode;
}

void GISparseVoxelOctree::LinkAllocators(GICudaAllocator** newAllocators,
										 size_t allocatorSize)
{
	allocatorGrids.clear();
	allocators.resize(allocatorSize);
	allocatorGrids.resize(allocatorSize);

	assert(allocatorSize > 0);
	assert(newAllocators != nullptr);

	std::copy(newAllocators, newAllocators + allocatorSize, allocators.data());
	for(unsigned int i = 0; i < allocatorSize; i++)
		allocatorGrids[i] = newAllocators[i]->GetVoxelGridHost();

	// TODO: More Dynamic Allocation Scheme
	size_t totalAlloc = GI_DENSE_SIZE * GI_DENSE_SIZE * GI_DENSE_SIZE;
	for(unsigned int i = 0; i < allocatorSize; i++)
	{
		uint32_t depthMultiplier = 1;
		if(i == 0) depthMultiplier = (allocatorGrids[i].depth - GI_DENSE_LEVEL);
		totalAlloc += allocators[i]->NumPages() * GI_PAGE_SIZE * depthMultiplier;
	}
	dSVO.Resize(totalAlloc);
	dSVOColor.Resize(totalAlloc);

	dSVODense = dSVO.Data();
	dSVOSparse = dSVO.Data() + (GI_DENSE_SIZE * GI_DENSE_SIZE * GI_DENSE_SIZE);

	dSVOLevelStartIndices.Resize(allocatorGrids[0].depth + allocatorSize);
	dSVOLevelStartIndices.Memset(0x00, 0, dSVOLevelStartIndices.Size());

	unsigned int totalLevel = static_cast<unsigned int>(allocatorGrids[0].depth + allocatorSize - 1);

	hSVOConstants.denseDepth = GI_DENSE_LEVEL;
	hSVOConstants.denseDim = GI_DENSE_SIZE;
	hSVOConstants.totalDepth = totalLevel;
	hSVOConstants.numCascades = static_cast<unsigned int>(allocatorSize);

	// Copy to device
	CUDA_CHECK(hipMemcpy(dSVOConstants.Data(), 
						  &hSVOConstants, 
						  sizeof(CSVOConstants), 
						  hipMemcpyHostToDevice));


	hipChannelFormatDesc fd = hipCreateChannelDesc<unsigned int>();
	if(denseArray) CUDA_CHECK(hipFreeArray(denseArray));
	CUDA_CHECK(hipMalloc3DArray(&denseArray,
								 &fd,
								 {GI_DENSE_SIZE, GI_DENSE_SIZE, GI_DENSE_SIZE},
								 hipArrayDefault));

	// Texture of SVO Dense
	hipResourceDesc resDesc = {};
	hipTextureDesc texDesc = {};
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = denseArray;
	
	texDesc.addressMode[0] = hipAddressModeWrap;
	texDesc.addressMode[1] = hipAddressModeWrap;
	texDesc.addressMode[2] = hipAddressModeWrap;
	texDesc.filterMode = hipFilterModePoint;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = 0;

	if(tSVODense != 0) CUDA_CHECK(hipDestroyTextureObject(tSVODense));
	CUDA_CHECK(hipCreateTextureObject(&tSVODense, &resDesc, &texDesc, nullptr));
}

void GISparseVoxelOctree::ConstructDense()
{
	//double childSet, alloc;
	//CudaTimer timer;
	//timer.Start();

	// Level 0 is special it constructs the upper levels in addition to its level
	uint32_t gridSize = ((allocators[0]->NumPages() * GI_PAGE_SIZE) + 
						 GI_THREAD_PER_BLOCK - 1) /
						 GI_THREAD_PER_BLOCK;
	SVOReconstructChildSet<<<gridSize, GI_THREAD_PER_BLOCK>>>
	(
		dSVODense,
		allocators[0]->GetVoxelPagesDevice(),
		
		0u,
		*dSVOConstants.Data()
	);
	CUDA_KERNEL_CHECK();

	//timer.Stop();
	//childSet = timer.ElapsedMilliS();
	//timer.Start();

	gridSize = ((GI_DENSE_SIZE * GI_DENSE_SIZE * GI_DENSE_SIZE) + GI_THREAD_PER_BLOCK - 1) /
				GI_THREAD_PER_BLOCK;
	SVOReconstructAllocateNext<<<gridSize, GI_THREAD_PER_BLOCK>>>
	(
		dSVO.Data(),
		*dSVONodeCountAtomic.Data(),
		*dSVOLevelStartIndices.Data(),
		*dSVOLevelStartIndices.Data(),
		GI_DENSE_SIZE * GI_DENSE_SIZE * GI_DENSE_SIZE
	);
	CUDA_KERNEL_CHECK();

	//timer.Stop();
	//alloc = timer.ElapsedMilliS();
	//timer.Start();

	// Copy Level Start Location to array
	CUDA_CHECK(hipMemcpy(dSVOLevelStartIndices.Data() + 1,
						  dSVONodeCountAtomic.Data(),
						  sizeof(unsigned int), hipMemcpyDeviceToDevice));

	//GI_LOG("---------------------------------------");
	//GI_LOG("Level %d", GI_DENSE_LEVEL);
	//GI_LOG("Child %f ms", childSet);
	//GI_LOG("Alloc %f ms", alloc);
	//GI_LOG("");
}

void GISparseVoxelOctree::ConstructLevel(unsigned int currentLevel,
										 unsigned int allocatorIndex,
										 unsigned int cascadeNo)
{
	//double childSet, memCopy, alloc;
	//CudaTimer timer;
	//timer.Start();

	// Early Bail check 
	unsigned int currentLevelIndex = currentLevel - GI_DENSE_LEVEL;
	uint32_t levelNodeCount, levelNodeStarts[2];
	CUDA_CHECK(hipMemcpy(levelNodeStarts,
						  dSVOLevelStartIndices.Data() + currentLevelIndex - 1,
						  sizeof(unsigned int) * 2,
						  hipMemcpyDeviceToHost));
	levelNodeCount = levelNodeStarts[1] - levelNodeStarts[0];

	if(levelNodeCount == 0) return;

	// ChildBitSet your Level (with next level's child)
	// Allocate next level
	// Memcopy next level start location to array
	// Only ChildBitSet Upper Level
	// Then Allocate your level
	// Average Color to the level
	uint32_t gridSize = ((allocators[allocatorIndex]->NumPages() * GI_PAGE_SIZE) + 
						 GI_THREAD_PER_BLOCK - 1) /
						 GI_THREAD_PER_BLOCK;

	SVOReconstructChildSet<<<gridSize, GI_THREAD_PER_BLOCK>>>
	(
		dSVOSparse,
		tSVODense,
		allocators[allocatorIndex]->GetVoxelPagesDevice(),
		dSVOLevelStartIndices.Data(),

		cascadeNo,
		currentLevel,
		*dSVOConstants.Data()
	);
	CUDA_KERNEL_CHECK();

	//timer.Stop();
	//memCopy = timer.ElapsedMilliS();
	//timer.Start();

	gridSize = ((levelNodeCount) + GI_THREAD_PER_BLOCK - 1) / GI_THREAD_PER_BLOCK;
	SVOReconstructAllocateNext<<<gridSize, GI_THREAD_PER_BLOCK>>>
	(
		dSVOSparse,
		*dSVONodeCountAtomic.Data(),
		*(dSVOLevelStartIndices.Data() + currentLevelIndex - 1),
		*(dSVOLevelStartIndices.Data() + currentLevelIndex),
		levelNodeCount
	);
	CUDA_KERNEL_CHECK();

	//timer.Stop();
	//alloc = timer.ElapsedMilliS();
	//timer.Start();

	// Copy Level Start Location to array
	CUDA_CHECK(hipMemcpy(dSVOLevelStartIndices.Data() + currentLevelIndex + 1, dSVONodeCountAtomic.Data(),
						  sizeof(unsigned int), hipMemcpyDeviceToDevice));

//	dSVO.DumpToFile("svoDump", 0, levelNodeStarts[1] + GI_DENSE_SIZE * GI_DENSE_SIZE * GI_DENSE_SIZE);
//	dSVOLevelStartIndices.DumpToFile("lvlDump");

	//GI_LOG("Level %d", currentLevel);
	//GI_LOG("Child %f ms", childSet);
	//GI_LOG("Alloc %f ms", alloc);
	//GI_LOG("Memcpy %f ms", memCopy);
	//GI_LOG("");
}

double GISparseVoxelOctree::UpdateSVO()
{
	CudaTimer timer;
	timer.Start();

	// Reset Atomic Counter since we reconstruct every frame
	unsigned int usedNodeCount;
	CUDA_CHECK(hipMemcpy(&usedNodeCount, dSVONodeCountAtomic.Data(), sizeof(unsigned int),
						  hipMemcpyDeviceToHost));
	dSVO.Memset(0x00, 0, usedNodeCount + GI_DENSE_SIZE * GI_DENSE_SIZE * GI_DENSE_SIZE);
	dSVONodeCountAtomic.Memset(0x00, 0, 1);
	dSVOLevelStartIndices.Memset(0x00, 0, dSVOLevelStartIndices.Size());

	// Start with constructing dense
	ConstructDense();
	
	// Copy to dense
	hipMemcpy3DParms params = {0};
	params.dstArray = denseArray;
	params.srcPtr = 
	{
		dSVODense, 
		GI_DENSE_SIZE * sizeof(unsigned int), 
		GI_DENSE_SIZE, 
		GI_DENSE_SIZE
	};
	params.extent = {GI_DENSE_SIZE, GI_DENSE_SIZE, GI_DENSE_SIZE};
	params.kind = hipMemcpyDeviceToDevice;
	CUDA_CHECK(hipMemcpy3D(&params));
		
	//CudaVector<unsigned int> texRead;
	//texRead.Resize(GI_DENSE_SIZE * GI_DENSE_SIZE * GI_DENSE_SIZE);
	//
	////dim3 blockSize(GI_DENSE_SIZE, GI_DENSE_SIZE, GI_DENSE_SIZE);
	////fastreadkernel3d<<<1, blockSize>>>
	////(
	////	tSVODense,
	////	texRead.Data()
	////);

	//fastreadkernel<<<1, GI_DENSE_SIZE * GI_DENSE_SIZE * GI_DENSE_SIZE>>>
	//(
	//	tSVODense,
	//	texRead.Data()
	//);

//	texRead.DumpToFile("texDump");
//	dSVO.DumpToFile("svoDump", 0, GI_DENSE_SIZE * GI_DENSE_SIZE * GI_DENSE_SIZE);
//	dSVOLevelStartIndices.DumpToFile("lvlDump");

	// Construct Levels
	for(unsigned int i = GI_DENSE_LEVEL + 1; i < allocatorGrids[0].depth; i++)
	{
		ConstructLevel(i, 0, 0);
	}

	//// Now adding cascade levels
	//for(unsigned int i = 1; i < allocators.size(); i++)
	//{
	//	unsigned int currentLevel = allocatorGrids[0].depth + i;
	//	ConstructLevel(currentLevel, i, i);
	//}

	//DEBUG
	std::vector<unsigned int> nodeCounts;
	nodeCounts.resize(dSVOLevelStartIndices.Size());
	CUDA_CHECK(hipMemcpy(nodeCounts.data(), dSVOLevelStartIndices.Data(),
		sizeof(unsigned int) * dSVOLevelStartIndices.Size(), hipMemcpyDeviceToHost));

	GI_LOG("-------------------------------------------");
	GI_LOG("Tree Node Data");
	for(unsigned int i = 0; i <= allocatorGrids[0].depth - GI_DENSE_LEVEL; i++)
	{
		if(i == 0) GI_LOG("#%d Dense : %d", GI_DENSE_LEVEL + i, GI_DENSE_SIZE * GI_DENSE_SIZE * GI_DENSE_SIZE);
		else GI_LOG("#%d Level : %d", GI_DENSE_LEVEL + i, nodeCounts[i] - nodeCounts[i - 1]);
	}
	GI_LOG("-------------------------------------------");

	timer.Stop();
	return timer.ElapsedMilliS();
}

double GISparseVoxelOctree::ConeTrace(GLuint depthBuffer,
									  GLuint normalBuffer,
									  GLuint colorBuffer,
									  const Camera& camera)
{
	return 0.0;
}

void GISparseVoxelOctree::LinkScene(GLuint lightBuffer,
									GLuint shadowMapArrayTexture)
{

}

//VoxelDebugVAO GISparseVoxelOctree::VoxelDataForRendering(double& transferTime,
//														 unsigned int& voxelCount,
//														 unsigned int level)
//{
//	// Find Node count
//	unsigned int currentLevelIndex = level - GI_DENSE_LEVEL;
//	uint32_t levelNodeCount, levelNodeStarts[2];
//	CUDA_CHECK(hipMemcpy(levelNodeStarts,
//						  dSVOLevelStartIndices.Data() + currentLevelIndex - 1,
//						  sizeof(unsigned int) * 2,
//						  hipMemcpyDeviceToHost));
//	levelNodeCount = levelNodeStarts[1] - levelNodeStarts[0];
//
//	//
//	vaoNormPosData.Resize(levelNodeCount);
//	vaoColorData.Resize(levelNodeCount);
//
//	//// Cuda stuff;
//
//	////
//	//uint32_t gridSize = ((levelNodeCount) + GI_THREAD_PER_BLOCK - 1) / GI_THREAD_PER_BLOCK;
//	//SVOVoxelFetch<<<gridSize, GI_THREAD_PER_BLOCK>>>
//	//(
//	//	dSVOSparse,
//	//	*dSVONodeCountAtomic.Data(),
//	//	*(dSVOLevelStartIndices.Data() + currentLevelIndex - 1),
//	//	*(dSVOLevelStartIndices.Data() + currentLevelIndex),
//	//	levelNodeCount
//	//);
//	//CUDA_KERNEL_CHECK();
//	
//}

uint64_t GISparseVoxelOctree::MemoryUsage() const
{
	uint64_t totalBytes = 0;
	totalBytes += dSVO.Size() * sizeof(CSVONode);
	totalBytes += dSVOColor.Size() * sizeof(CSVOColor);
	totalBytes += dSVOLevelStartIndices.Size() * sizeof(unsigned int);
	totalBytes += sizeof(unsigned int);
	return totalBytes;
}