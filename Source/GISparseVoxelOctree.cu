#include "hip/hip_runtime.h"
#include "GISparseVoxelOctree.h"
#include "SceneI.h"
#include "Globals.h"
#include "SceneLights.h"
#include "DeferredRenderer.h"
#include "GLSLBindPoints.h"
#include "GIVoxelPages.h"
#include "GIVoxelCache.h"
#include "SVOKernels.cuh"
#include "CudaTimer.h"
#include <numeric>
#include <cuda_gl_interop.h>

//#include "SVOKernels.cuh"
//#include "CudaTimer.h"
//#include "Macros.h"
//#include "Camera.h"
//#include "Globals.h"
//#include "CDebug.cuh"
//#include "IEUtility/IEMath.h"

//#include "GLSLBindPoints.h"

GISparseVoxelOctree::ShadowMapsCUDA::ShadowMapsCUDA()
	: lightCount(0)
	, matrixOffset(0)
	, lightOffset(0)
	, shadowMapResource(nullptr)
	, lightBufferResource(nullptr)
	, shadowMapArray(nullptr)
	, tShadowMapArray(0)
	, dLightParamArray(nullptr)
	, dLightVPMatrixArray(nullptr)
{}

GISparseVoxelOctree::ShadowMapsCUDA::ShadowMapsCUDA(const SceneLights& sLights)
	: lightCount(sLights.getLightCount())
	, matrixOffset(sLights.getMatrixOffset())
	, lightOffset(sLights.getLightOffset())
	, shadowMapResource(nullptr)
	, lightBufferResource(nullptr)
	, shadowMapArray(nullptr)
	, tShadowMapArray(0)
	, dLightParamArray(nullptr)
	, dLightVPMatrixArray(nullptr)
{
	// Here i gurantee that these buffers are read only, thus const_cast
	GLuint glBuffer = const_cast<SceneLights&>(sLights).getGLBuffer();
	GLuint shadowMaps = const_cast<SceneLights&>(sLights).getShadowTextureArrayView();

	CUDA_CHECK(hipGraphicsGLRegisterBuffer(&lightBufferResource, glBuffer,
											cudaGraphicsMapFlagsReadOnly));
	CUDA_CHECK(hipGraphicsGLRegisterImage(&shadowMapResource, shadowMaps,
										   GL_TEXTURE_2D_ARRAY,
										   cudaGraphicsMapFlagsReadOnly));
}

GISparseVoxelOctree::ShadowMapsCUDA::ShadowMapsCUDA(ShadowMapsCUDA&& other)
	: lightCount(other.lightCount)
	, matrixOffset(other.matrixOffset)
	, lightOffset(other.lightOffset)
	, shadowMapResource(other.shadowMapResource)
	, lightBufferResource(other.lightBufferResource)
	, shadowMapArray(other.shadowMapArray)
	, tShadowMapArray(other.tShadowMapArray)
	, dLightParamArray(other.dLightParamArray)
	, dLightVPMatrixArray(other.dLightVPMatrixArray)
{
	other.shadowMapResource = nullptr;
	other.lightBufferResource = nullptr;
}

GISparseVoxelOctree::ShadowMapsCUDA& GISparseVoxelOctree::ShadowMapsCUDA::operator=(ShadowMapsCUDA&& other)
{
	assert(&other != this);
	if(lightBufferResource)
		CUDA_CHECK(hipGraphicsUnregisterResource(lightBufferResource));
	if(shadowMapResource)
		CUDA_CHECK(hipGraphicsUnregisterResource(shadowMapResource));

	lightCount = other.lightCount;
	matrixOffset = other.matrixOffset;
	lightOffset = other.lightOffset;
	shadowMapResource = other.shadowMapResource;
	lightBufferResource = other.lightBufferResource;
	shadowMapArray = other.shadowMapArray;
	tShadowMapArray = other.tShadowMapArray;
	dLightParamArray = other.dLightParamArray;
	dLightVPMatrixArray = other.dLightVPMatrixArray;

	other.shadowMapResource = nullptr;
	other.lightBufferResource = nullptr;
	return *this;
}

GISparseVoxelOctree::ShadowMapsCUDA::~ShadowMapsCUDA()
{
	if(lightBufferResource)
		CUDA_CHECK(hipGraphicsUnregisterResource(lightBufferResource));
	if(shadowMapResource)
		CUDA_CHECK(hipGraphicsUnregisterResource(shadowMapResource));
}

void GISparseVoxelOctree::ShadowMapsCUDA::Map()
{
	assert(dLightParamArray == nullptr);
	assert(dLightVPMatrixArray == nullptr);
	assert(shadowMapArray == nullptr);
	assert(tShadowMapArray == 0);

	CUDA_CHECK(hipGraphicsMapResources(1, &shadowMapResource));
	CUDA_CHECK(cudaGraphicsResourceGetMappedMipmappedArray(&shadowMapArray, shadowMapResource));

	// Texture Generation From Cuda Array
	hipResourceDesc resDesc = {};
	hipTextureDesc texDesc = {};
	resDesc.resType = hipResourceTypeMipmappedArray;
	resDesc.res.mipmap.mipmap = shadowMapArray;

	texDesc.addressMode[0] = hipAddressModeWrap;
	texDesc.addressMode[1] = hipAddressModeWrap;
	texDesc.addressMode[2] = hipAddressModeWrap;
	texDesc.filterMode = hipFilterModePoint;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = 0;

	CUDA_CHECK(hipCreateTextureObject(&tShadowMapArray, &resDesc, &texDesc, nullptr));

	// Buffer
	size_t size;
	uint8_t* glBufferCUDA = nullptr;
	CUDA_CHECK(hipGraphicsMapResources(1, &lightBufferResource));
	CUDA_CHECK(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&glBufferCUDA),
													&size, lightBufferResource));

	dLightParamArray = reinterpret_cast<const CLight*>(glBufferCUDA + lightOffset);
	dLightVPMatrixArray = reinterpret_cast<const CMatrix4x4*>(glBufferCUDA + matrixOffset);
}

void GISparseVoxelOctree::ShadowMapsCUDA::Unmap()
{
	assert(dLightParamArray != nullptr);
	assert(dLightVPMatrixArray != nullptr);
	assert(shadowMapArray != nullptr);
	assert(tShadowMapArray != 0);

	// Unmap Texture
	CUDA_CHECK(hipGraphicsUnmapResources(1, &shadowMapResource));
	CUDA_CHECK(hipDestroyTextureObject(tShadowMapArray));
	tShadowMapArray = 0;
	shadowMapArray = nullptr;

	// Unmap Buffer
	CUDA_CHECK(hipGraphicsUnmapResources(1, &lightBufferResource));
	dLightParamArray = nullptr;
	dLightVPMatrixArray = nullptr;
}

const CLight* GISparseVoxelOctree::ShadowMapsCUDA::LightParamArray() const
{
	return dLightParamArray;
}

const CMatrix4x4* GISparseVoxelOctree::ShadowMapsCUDA::LightVPMatrices() const
{
	return dLightVPMatrixArray;
}

hipTextureObject_t GISparseVoxelOctree::ShadowMapsCUDA::ShadowMapArray() const
{
	return tShadowMapArray;
}

uint32_t GISparseVoxelOctree::ShadowMapsCUDA::LightCount() const
{
	return lightCount;
}

GISparseVoxelOctree::GISparseVoxelOctree()
	: octreeParams(nullptr)
	, scene(nullptr)
	, octreeUniformsOffset(0)
	, indirectUniformsOffset(0)
	, illumOffsetsOffset(0)
	, nodeOffset(0)
	, illumOffset(0)
	, gpuResource(nullptr)
	, dLevelCapacities(nullptr)
	, dLevelSizes(nullptr)
	, dOctreeLevels(nullptr)
	, nodeIllumDifference(0)
{}

GISparseVoxelOctree::GISparseVoxelOctree(const OctreeParameters& octreeParams,
										 const SceneI* currentScene,
										 const size_t sizes[])
	: octreeParams(&octreeParams)
	, scene(currentScene)
	, octreeUniformsOffset(0)
	, indirectUniformsOffset(0)
	, illumOffsetsOffset(0)
	, nodeOffset(0)
	, illumOffset(0)
	, gpuResource(nullptr)
	, dLevelCapacities(nullptr)
	, dLevelSizes(nullptr)
	, dOctreeLevels(nullptr)
	, nodeIllumDifference(0)
	, shadowMaps(currentScene->getSceneLights())
	, compVoxTraceWorld(ShaderType::COMPUTE, "Shaders/VoxTraceWorld.comp")
	, compVoxSampleWorld(ShaderType::COMPUTE, "Shaders/VoxTraceDeferred.comp")
	, compGI(ShaderType::COMPUTE, "Shaders/VoxGI.comp")
{	
	// Generate Initial Sizes for each level
	std::vector<uint32_t> levelCapacities(octreeParams.MaxSVOLevel + 1, 0);
	std::vector<uint32_t> internalOffsets(octreeParams.MaxSVOLevel + 1, 0);
	size_t offset = 0;
	for(uint32_t i = octreeParams.MinSVOLevel; i < octreeParams.MaxSVOLevel + 1; i++)
	{
		if(i == octreeParams.DenseLevel) nodeIllumDifference = offset;
		size_t levelSize = (i <= octreeParams.DenseLevel)
								? ((1 << i) * (1 << i) * (1 << i))
								: sizes[i];
		internalOffsets[i] = static_cast<uint32_t>(offset);
		levelCapacities[i] = static_cast<uint32_t>(levelSize);
		offset += levelSize;
	}
	size_t totalIllumSize = offset;
	size_t totalNodeSize = offset - nodeIllumDifference;
	hIllumOffsets = internalOffsets;

	// Allocation of OpenGL Side
	offset = 0;
	// OctreeUniforms
	octreeUniformsOffset = offset;
	offset += sizeof(OctreeUniforms);
	// IndirectUniforms	
	offset = DeviceOGLParameters::UBOAlignOffset(offset);
	indirectUniformsOffset = offset;
	offset += sizeof(IndirectUniforms);
	// IllumOffsets
	offset = DeviceOGLParameters::SSBOAlignOffset(offset);
	illumOffsetsOffset = offset;
	offset += (octreeParams.MaxSVOLevel + 1) * sizeof(uint32_t);
	// Nodes
	offset = DeviceOGLParameters::SSBOAlignOffset(offset);
	nodeOffset = offset;
	offset += (totalNodeSize) * sizeof(CSVONode);
	// Illum Data
	offset = DeviceOGLParameters::SSBOAlignOffset(offset);
	illumOffset = offset;
	offset += (totalIllumSize) * sizeof(CSVOIllumination);
	
	// Offsets Generated Allocate
	oglData.Resize(offset, false);

	// Now CUDA
	size_t totalSize = (octreeParams.MaxSVOLevel + 1) * (sizeof(uint32_t) * 2 + 
														 sizeof(CSVOLevel));
	cudaData.Resize(totalSize);

	// Allocation of CUDA Side
	offset = 0;
	// Level Capacities
	dLevelCapacities = reinterpret_cast<uint32_t*>(cudaData.Data() + offset);
	offset += (sizeof(uint32_t) * (octreeParams.MaxSVOLevel + 1));
	// Level Sizes
	dLevelSizes = reinterpret_cast<uint32_t*>(cudaData.Data() + offset);
	offset += (sizeof(uint32_t) * (octreeParams.MaxSVOLevel + 1));
	// Octree Levels
	dOctreeLevels = reinterpret_cast<CSVOLevel*>(cudaData.Data() + offset);
	offset += (sizeof(CSVOLevel) * (octreeParams.MaxSVOLevel + 1));
	assert(offset == totalSize);

	// Load Level Capacities
	CUDA_CHECK(hipMemcpy(const_cast<uint32_t*>(dLevelCapacities),
						  levelCapacities.data(),
						  (octreeParams.MaxSVOLevel + 1) * sizeof(uint32_t),
						  hipMemcpyHostToDevice));

	// Register CUDA Resource
	CUDA_CHECK(hipGraphicsGLRegisterBuffer(&gpuResource, oglData.getGLBuffer(),
											cudaGraphicsMapFlagsNone));
}

GISparseVoxelOctree::GISparseVoxelOctree(GISparseVoxelOctree&& other)
	: octreeParams(other.octreeParams)
	, scene(other.scene)
	, oglData(std::move(other.oglData))
	, octreeUniformsOffset(other.octreeUniformsOffset)
	, indirectUniformsOffset(other.indirectUniformsOffset)
	, illumOffsetsOffset(other.illumOffsetsOffset)
	, nodeOffset(other.nodeOffset)
	, illumOffset(other.illumOffset)
	, gpuResource(other.gpuResource)
	, cudaData(std::move(other.cudaData))
	, dLevelCapacities(other.dLevelCapacities)
	, dLevelSizes(other.dLevelSizes)
	, dOctreeLevels(other.dOctreeLevels)
	, hIllumOffsets(std::move(other.hIllumOffsets))
	, nodeIllumDifference(other.nodeIllumDifference)
	, shadowMaps(std::move(other.shadowMaps))
	, compVoxTraceWorld(std::move(other.compVoxTraceWorld))
	, compVoxSampleWorld(std::move(other.compVoxSampleWorld))
	, compGI(std::move(other.compGI))
{
	other.gpuResource = nullptr;
}

GISparseVoxelOctree& GISparseVoxelOctree::operator=(GISparseVoxelOctree&& other)
{
	assert(&other != this);
	if(gpuResource) CUDA_CHECK(hipGraphicsUnregisterResource(gpuResource));

	octreeParams = other.octreeParams;
	scene = other.scene;
	oglData = std::move(other.oglData);
	octreeUniformsOffset = other.octreeUniformsOffset;
	indirectUniformsOffset = other.indirectUniformsOffset;
	illumOffsetsOffset = other.illumOffsetsOffset;
	nodeOffset = other.nodeOffset;
	illumOffset = other.illumOffset;
	gpuResource = other.gpuResource;
	cudaData = std::move(other.cudaData);
	dLevelCapacities = other.dLevelCapacities;
	dLevelSizes = other.dLevelSizes;
	dOctreeLevels = other.dOctreeLevels;
	hIllumOffsets = std::move(other.hIllumOffsets);
	nodeIllumDifference = other.nodeIllumDifference;
	shadowMaps = std::move(other.shadowMaps);
	compVoxTraceWorld = std::move(other.compVoxTraceWorld);
	compVoxSampleWorld = std::move(other.compVoxSampleWorld);
	compGI = std::move(other.compGI);

	other.gpuResource = nullptr;
	return *this;
}

GISparseVoxelOctree::~GISparseVoxelOctree()
{
	if(gpuResource) CUDA_CHECK(hipGraphicsUnregisterResource(gpuResource));
}

void GISparseVoxelOctree::MapOGLData()
{
	// Get Node Pointer
	CUDA_CHECK(hipGraphicsMapResources(1, &gpuResource));
	size_t size; uint8_t* oglCudaPtr;
	CUDA_CHECK(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&oglCudaPtr),
													&size, gpuResource));
	assert(size == oglData.Capacity());
	
	// Recieve Used Pointer Sizes
	std::vector<uint32_t> levelAllocators(octreeParams->MaxSVOLevel + 1);
	CUDA_CHECK(hipMemcpy(levelAllocators.data(), dLevelSizes,
						  (octreeParams->MaxSVOLevel + 1) * sizeof(uint32_t),
						  hipMemcpyDeviceToHost));

	std::vector<CSVOLevel> svoLevels(octreeParams->MaxSVOLevel + 1, {nullptr, nullptr});
	for(uint32_t i = octreeParams->MinSVOLevel; i < octreeParams->MaxSVOLevel + 1; i++)
	{
		CSVONode* nodePtr = nullptr;
		CSVOIllumination* illumPtr = reinterpret_cast<CSVOIllumination*>(oglCudaPtr + illumOffset)
																		 + hIllumOffsets[i];
		if(i < octreeParams->DenseLevel)
		{
			nodePtr = reinterpret_cast<CSVONode*>(oglCudaPtr + nodeOffset) 
												  + (hIllumOffsets[i] - nodeIllumDifference);

			// Clear used node pointers
			//CUDA_CHECK(hipMemset(nodePtr, 0xFF, levelAllocators[i]));
		}

		svoLevels[i].gLevelNodes = nodePtr;
		svoLevels[i].gLevelIllum = illumPtr;

		// Clear used illum
		//CUDA_CHECK(hipMemset(illumPtr, 0x00, levelAllocators[i]));
	}

	// Clear level allocators
	CUDA_CHECK(hipMemset(dLevelSizes, 0x00, octreeParams->MaxSVOLevel + 1));

	// Copy Generated Pointers
	CUDA_CHECK(hipMemcpy(dOctreeLevels, svoLevels.data(),
						  (octreeParams->MaxSVOLevel + 1) * sizeof(CSVOLevel),
						  hipMemcpyHostToDevice));
}

void GISparseVoxelOctree::UnmapOGLData()
{
	CUDA_CHECK(hipGraphicsUnmapResources(1, &gpuResource));
}

double GISparseVoxelOctree::GenerateHierarchy(bool doTiming,
											  // Page System
											  const GIVoxelPages& pages,
											  // Cache System
											  const GIVoxelCache& caches,
											  // Constants
											  uint32_t batchCount,
											  const LightInjectParameters& injectParams,
											  bool injectOn)
{
	CudaTimer t;
	if(doTiming) t.Start();

	// Gen LI Params
	CLightInjectParameters liParams = 
	{
		injectOn,
		injectParams.camPos,
		injectParams.camDir,

		shadowMaps.LightVPMatrices(),
		shadowMaps.LightParamArray(),

		injectParams.depthNear,
		injectParams.depthFar,
		shadowMaps.ShadowMapArray(),
		shadowMaps.LightCount()
	};
	
	//// KC
	//int gridSize = 1;// CudaInit::GenBlockSize(static_cast<int>(pages.PageCount() * GIVoxelPages::PageSize));
	//int blockSize = CudaInit::TBP;
	//SVOReconstruct<<<gridSize, blockSize>>>(// SVO
	//									    dOctreeLevels,
	//									    reinterpret_cast<const CSVOLevelConst*>(dOctreeLevels),
	//									    dLevelSizes,
	//									    dLevelCapacities,
	//									    // Voxel Pages
	//									    pages.getVoxelPages(),
	//									    pages.getVoxelGrids(),
	//									    // Cache Data (for Voxel Albedo)
	//									    caches.getDeviceCascadePointersDevice().Data(),
	//									    // Light Injection Related
	//									    liParams,
	//									    // Limits
	//									    *octreeParams,
	//									    batchCount);
	//CUDA_KERNEL_CHECK();
	//hipDeviceSynchronize();
	//GI_LOG("-----");

	if(doTiming)
	{
		t.Stop();
		return t.ElapsedMilliS();
	}
	return 0.0;
}

double GISparseVoxelOctree::AverageNodes(bool doTiming)
{
	CudaTimer t;
	if(doTiming) t.Start();


	// Work

	if(doTiming)
	{
		t.Stop();
		return t.ElapsedMilliS();
	}
	return 0.0;
}

void GISparseVoxelOctree::UpdateSVO(// Timing Related
									double& reconstructTime,
									double& averageTime,
									bool doTiming,
									// Page System
									const GIVoxelPages& pages,
									// Cache System
									const GIVoxelCache& caches,
									// Constants
									uint32_t batchCount,
									const LightInjectParameters& injectParams,
									bool injectOn)
{
	MapOGLData();
	shadowMaps.Map();

	reconstructTime = GenerateHierarchy(doTiming,
										pages, caches, batchCount,
										injectParams, injectOn);
	averageTime = AverageNodes(doTiming);

	shadowMaps.Unmap();
	UnmapOGLData();
}

void GISparseVoxelOctree::UpdateOctreeUniforms(const IEVector3& outerCascadePos)
{
	// Octree Uniforms
	OctreeUniforms u = {};
	u.worldPos = outerCascadePos;
	u.baseSpan = octreeParams->BaseSpan;
	u.gridSize = octreeParams->CascadeBaseLevelSize;
	u.minSVOLevel = octreeParams->MinSVOLevel;
	u.maxSVOLevel = octreeParams->MaxSVOLevel;
	u.denseLevel = octreeParams->DenseLevel;
	u.cascadeCount = octreeParams->CascadeCount;
	u.nodeOffsetDifference = static_cast<uint32_t>(nodeIllumDifference);
	std::memcpy(oglData.CPUData().data() + octreeUniformsOffset,
				&u, sizeof(OctreeUniforms));
	oglData.SendSubData(static_cast<uint32_t>(octreeUniformsOffset), sizeof(OctreeUniforms));
}

void GISparseVoxelOctree::UpdateIndirectUniforms(const IndirectUniforms& indirectUniforms)
{
	std::memcpy(oglData.CPUData().data() + indirectUniformsOffset,
				&indirectUniforms,
				sizeof(IndirectUniforms));
	oglData.SendSubData(static_cast<uint32_t>(indirectUniformsOffset), sizeof(IndirectUniforms));
}

double GISparseVoxelOctree::GlobalIllumination(GLuint outputTexture,
											   const DeferredRenderer& dRenderer,
											   const Camera& camera,
											   const IndirectUniforms&,
											   bool giOn,
											   bool aoOn,
											   bool specularOn)
{
	// Light Intensity Texture
	static const GLubyte ff[4] = {0xFF, 0xFF, 0xFF, 0xFF};
	glClearTexImage(outputTexture, 0, GL_RGBA, GL_UNSIGNED_BYTE, &ff);
	
	// Timing Voxelization Process
	GLuint queryID;
	glGenQueries(1, &queryID);
	glBeginQuery(GL_TIME_ELAPSED, queryID);
		
	// Shaders
	compGI.Bind();
		
	// Uniforms
	glUniform1ui(U_CAST_SPECULAR_CONE, (specularOn) ? 1u : 0u);
	
	// Uniform Buffers
	// Frame transform already bound
	dRenderer.BindInvFrameTransform(U_INVFTRANSFORM);
	oglData.BindAsUniformBuffer(U_OCTREE_UNIFORMS, static_cast<uint32_t>(octreeUniformsOffset), 
								sizeof(OctreeUniforms));
	oglData.BindAsUniformBuffer(U_INDIRECT_UNIFORMS, static_cast<uint32_t>(indirectUniformsOffset), 
								sizeof(IndirectUniforms));

	// SSBO Buffers
	oglData.BindAsUniformBuffer(LU_SVO_LEVEL_OFFSET, 
								static_cast<uint32_t>(illumOffsetsOffset), 
								sizeof(uint32_t) * (octreeParams->MaxSVOLevel + 1));
	oglData.BindAsUniformBuffer(LU_SVO_NODE, static_cast<uint32_t>(nodeOffset), 
								static_cast<uint32_t>(illumOffset - nodeOffset));
	oglData.BindAsUniformBuffer(LU_SVO_ILLUM, static_cast<uint32_t>(illumOffset), 
								static_cast<uint32_t>(oglData.Count() - illumOffset));

	// Textures
	dRenderer.getGBuffer().BindAsTexture(T_COLOR, RenderTargetLocation::COLOR);
	dRenderer.getGBuffer().BindAsTexture(T_DEPTH, RenderTargetLocation::DEPTH);
	dRenderer.getGBuffer().BindAsTexture(T_NORMAL, RenderTargetLocation::NORMAL);

	// Images
	glBindImageTexture(I_OUT_TEXTURE, outputTexture, 0, false, 0, GL_WRITE_ONLY, GL_RGBA8);
	
	// Dispatch
	uint2 gridSize;
	gridSize.x = (TraceWidth + 16 - 1) / 16;
	gridSize.y = (TraceHeight + 16 - 1) / 16;
	glDispatchCompute(gridSize.x, gridSize.y, 1);
	glMemoryBarrier(GL_SHADER_IMAGE_ACCESS_BARRIER_BIT);
	
	// Timer
	GLuint64 timeElapsed = 0;
	glEndQuery(GL_TIME_ELAPSED);
	glGetQueryObjectui64v(queryID, GL_QUERY_RESULT, &timeElapsed);
	
	// I have to unbind the compute shader or weird things happen
	Shader::Unbind(ShaderType::COMPUTE);
	return timeElapsed / 1000000.0;
}

double GISparseVoxelOctree::DebugTraceSVO(GLuint outputTexture,
										  const DeferredRenderer&,
										  const Camera& camera,
										  uint32_t renderLevel,
										  OctreeRenderType)
{
	return 0.0f;
}

double GISparseVoxelOctree::DebugSampleSVO(GLuint& outputTexture,
										   const DeferredRenderer&,
										   const Camera& camera,
										   uint32_t renderLevel,
										   OctreeRenderType)
{
	return 0.0f;
}

size_t GISparseVoxelOctree::MemoryUsage() const
{
	return oglData.Capacity() + cudaData.Size();
}

//GISparseVoxelOctree::GISparseVoxelOctree(const OctreeParameters& octreeParams)
//	: octreeParams(octreeParams)
//	, dSVOConstants(1)
//	, computeVoxTraceWorld(ShaderType::COMPUTE, "Shaders/VoxTraceWorld.comp")
//	, computeVoxTraceDeferredLerp(ShaderType::COMPUTE, "Shaders/VoxTraceDeferredLerp.comp")
//	, computeVoxTraceDeferred(ShaderType::COMPUTE, "Shaders/VoxTraceDeferred.comp")
//	, computeAO(ShaderType::COMPUTE, "Shaders/VoxAO.comp")
//	, computeGI(ShaderType::COMPUTE, "Shaders/VoxGI.comp")
//	, computeGauss32(ShaderType::COMPUTE, "Shaders/Gauss32.comp")
//	, computeEdge(ShaderType::COMPUTE, "Shaders/EdgeDetect.comp")
//	, computeAOSurf(ShaderType::COMPUTE, "Shaders/SurfAO.comp")
//	, computeLIApply(ShaderType::COMPUTE, "Shaders/ApplyVoxLI.comp")
//	, svoTraceData(1)
//	, svoConeParams(1)
//	, svoNodeResource(nullptr)
//	, svoLevelOffsetResource(nullptr)
//	, svoMaterialResource(nullptr)
//	, svoDenseNodeResource(nullptr)
//	, sceneShadowMapResource(nullptr)
//	, sceneLightParamResource(nullptr)
//	, sceneVPMatrixResource(nullptr)
//	, tSVODenseNode(0)
//	, sSVODenseNode(0)
//	, tShadowMapArray(0)
//	, dSVODenseNodeArray(nullptr)
//	, traceTexture(0)
//	, gaussTex(0)
//	, edgeTex(0)
//	, svoDenseMat(0)
//	, sSVODenseMat(octreeParams.DenseLevelCount, 0)
//	, dSVODenseMatArray(octreeParams.DenseLevelCount, nullptr)
//	, nodeSampler(0)
//	, materialSampler(0)
//	, gaussSampler(0)
//{
//	svoTraceData.AddData({});
//	svoConeParams.AddData({});
//
//	// Light Intensity Tex
//	glGenTextures(1, &traceTexture);
//	glBindTexture(GL_TEXTURE_2D, traceTexture);
//	glTexStorage2D(GL_TEXTURE_2D, 1, GL_RGBA8/*GL_RGBA16*/, TraceWidth, TraceHeight);
//
//	// Gauss Intermediary Tex
//	glGenTextures(1, &gaussTex);
//	glBindTexture(GL_TEXTURE_2D, gaussTex);
//	glTexStorage2D(GL_TEXTURE_2D, 1, GL_RGBA8/*GL_RGBA16*/, TraceWidth, TraceHeight);
//
//	// Edge Map Tex
//	glGenTextures(1, &edgeTex);
//	glBindTexture(GL_TEXTURE_2D, edgeTex);
//	glTexStorage2D(GL_TEXTURE_2D, 1, GL_RG8, TraceWidth, TraceHeight);
//	
//	//// Dense Tex
//	//glGenTextures(1, &svoDenseNode);
//	//glBindTexture(GL_TEXTURE_3D, svoDenseNode);
//	//glTexStorage3D(GL_TEXTURE_3D, 1, GL_R32UI, GI_DENSE_SIZE, GI_DENSE_SIZE, GI_DENSE_SIZE);
//	//CUDA_CHECK(hipGraphicsGLRegisterImage(&svoDenseNodeResource, svoDenseNode, GL_TEXTURE_3D, 
//	//									   hipGraphicsRegisterFlagsSurfaceLoadStore)); //|
//	//									   //hipGraphicsRegisterFlagsWriteDiscard*/));
//
//	//// Mat Texture Binds
//	//// Mipped 3D tex
//	//glGenTextures(1, &svoDenseMat);	
//	//glBindTexture(GL_TEXTURE_3D, svoDenseMat);
//	//glTexStorage3D(GL_TEXTURE_3D, GI_DENSE_TEX_COUNT, GL_RGBA32UI, GI_DENSE_SIZE, GI_DENSE_SIZE, GI_DENSE_SIZE);
//	//CUDA_CHECK(hipGraphicsGLRegisterImage(&svoDenseTexResource, svoDenseMat, GL_TEXTURE_3D,
//	//									   hipGraphicsRegisterFlagsSurfaceLoadStore)); //|
//	//									   //hipGraphicsRegisterFlagsWriteDiscard));
//	
//	// Flat Sampler for Node Index Fetch
//	glGenSamplers(1, &nodeSampler);
//	glSamplerParameteri(nodeSampler, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
//	glSamplerParameteri(nodeSampler, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
//	glSamplerParameteri(nodeSampler, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
//	glSamplerParameteri(nodeSampler, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
//	glSamplerParameteri(nodeSampler, GL_TEXTURE_WRAP_R, GL_CLAMP_TO_EDGE);
//
//	// Nearest Sample for Material Fetch since its interger tex no interpolation
//	glGenSamplers(1, &materialSampler);
//	glSamplerParameteri(materialSampler, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
//	glSamplerParameteri(materialSampler, GL_TEXTURE_MIN_FILTER, GL_NEAREST_MIPMAP_NEAREST);
//	glSamplerParameteri(materialSampler, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
//	glSamplerParameteri(materialSampler, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
//	glSamplerParameteri(materialSampler, GL_TEXTURE_WRAP_R, GL_CLAMP_TO_EDGE);
//
//	// Bilinear Sample for Gauss Fetch (Out of bounds are zero)
//	GLfloat col[] = {0.0f, 0.0f, 0.0f, 0.0f};
//	glGenSamplers(1, &gaussSampler);
//	glSamplerParameteri(gaussSampler, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
//	glSamplerParameteri(gaussSampler, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
//	glSamplerParameteri(gaussSampler, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_BORDER);
//	glSamplerParameteri(gaussSampler, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_BORDER);
//	glSamplerParameteri(gaussSampler, GL_TEXTURE_WRAP_R, GL_CLAMP_TO_BORDER);
//	glSamplerParameterfv(gaussSampler, GL_TEXTURE_BORDER_COLOR, col);
//}
//
//GISparseVoxelOctree::~GISparseVoxelOctree()
//{
//	if(svoNodeResource) CUDA_CHECK(hipGraphicsUnregisterResource(svoNodeResource));
//	if(svoMaterialResource) CUDA_CHECK(hipGraphicsUnregisterResource(svoMaterialResource));
//	if(svoLevelOffsetResource) CUDA_CHECK(hipGraphicsUnregisterResource(svoLevelOffsetResource));
//	if(svoDenseTexResource) CUDA_CHECK(hipGraphicsUnregisterResource(svoDenseTexResource));
//	for(unsigned int i = 0; i < octreeParams.DenseLevelCount; i++)
//	{
//		if(sSVODenseMat[i]) CUDA_CHECK(hipDestroySurfaceObject(sSVODenseMat[i]));
//	}
//	if(svoDenseNodeResource) CUDA_CHECK(hipGraphicsUnregisterResource(svoDenseNodeResource));
//	if(tSVODenseNode) CUDA_CHECK(hipDestroyTextureObject(tSVODenseNode));
//	if(sSVODenseNode) CUDA_CHECK(hipDestroySurfaceObject(sSVODenseNode));
//
//	if(traceTexture) glDeleteTextures(1, &traceTexture);
//	if(gaussTex) glDeleteTextures(1, &gaussTex);
//	if(edgeTex) glDeleteTextures(1, &edgeTex);
//	if(svoDenseNode) glDeleteTextures(1, &svoDenseNode);
//	if(svoDenseMat) glDeleteTextures(1, &svoDenseMat);
//	if(nodeSampler) glDeleteSamplers(1, &nodeSampler);
//	if(materialSampler) glDeleteSamplers(1, &materialSampler);
//	if(materialSampler) glDeleteSamplers(1, &gaussSampler);
//}

//void GISparseVoxelOctree::LinkAllocators(std::vector<GICudaAllocator*> newAllocators,
//										 const uint32_t levelCounts[])
//{
//	//allocatorGrids.clear();
//	//allocators.resize(newAllocators.size());
//	//allocatorGrids.resize(newAllocators.size());
//
//	//assert(newAllocators.size() > 0);
//
//	//std::copy(newAllocators.data(), newAllocators.data() + newAllocators.size(), allocators.data());
//	//for(unsigned int i = 0; i < newAllocators.size(); i++)
//	//	allocatorGrids[i] = &(newAllocators[i]->GetVoxelGridHost());
//
//	//size_t sparseNodeCount = allocatorGrids[0]->depth + newAllocators.size() - GI_DENSE_LEVEL;
//	//uint32_t totalLevel = allocatorGrids[0]->depth + static_cast<uint32_t>(newAllocators.size() - 1);
//
// //   size_t totalAlloc = 0;
// //   for(unsigned int i = GI_DENSE_LEVEL + 1; i <= totalLevel; i++)
// //   {
// //       totalAlloc += levelCounts[i];
// //   }
//
//	//// TODO: More Dynamic Allocation Scheme
//	//hSVOLevelTotalSizes.resize(sparseNodeCount);
//	//dSVOLevelTotalSizes.Resize(sparseNodeCount);
//	//dSVOLevelSizes.Resize(sparseNodeCount);
//	//hSVOLevelSizes.resize(sparseNodeCount);
//	////svoLevelOffsets.Resize(sparseNodeCount);
//
//	////// Sparse Portion
//	////svoNodeBuffer.Resize(totalAlloc + DenseSizeCube);
//	////svoMaterialBuffer.Resize(totalAlloc);
//	//
//	//// Register
//	//if(svoNodeResource) CUDA_CHECK(hipGraphicsUnregisterResource(svoNodeResource));
//	//if(svoMaterialResource) CUDA_CHECK(hipGraphicsUnregisterResource(svoMaterialResource));
//	//if(svoLevelOffsetResource) CUDA_CHECK(hipGraphicsUnregisterResource(svoLevelOffsetResource));
//	//CUDA_CHECK(hipGraphicsGLRegisterBuffer(&svoNodeResource, 
//	//										svoNodeBuffer.getGLBuffer(), 
//	//										cudaGLMapFlagsWriteDiscard));
//	//CUDA_CHECK(hipGraphicsGLRegisterBuffer(&svoMaterialResource, 
//	//										svoMaterialBuffer.getGLBuffer(), 
//	//										cudaGLMapFlagsWriteDiscard));
//	//CUDA_CHECK(hipGraphicsGLRegisterBuffer(&svoLevelOffsetResource,
//	//										svoLevelOffsets.getGLBuffer(),
//	//										cudaGLMapFlagsReadOnly));
//
//	//// Actual Data Init
//	//GLuint allOne = 0xFFFFFFFF;
//	//GLuint zero = 0;
//
//	//glBindBuffer(GL_COPY_WRITE_BUFFER, svoNodeBuffer.getGLBuffer());
//	//glClearBufferData(GL_COPY_WRITE_BUFFER, GL_R32UI, GL_RED_INTEGER, GL_UNSIGNED_INT, &allOne);
//
//	//glBindBuffer(GL_COPY_WRITE_BUFFER, svoMaterialBuffer.getGLBuffer());
//	//glClearBufferData(GL_COPY_WRITE_BUFFER, GL_R32UI, GL_RED_INTEGER, GL_UNSIGNED_INT, &zero);
//
//	//dSVOLevelSizes.Memset(0x00, 0, dSVOLevelSizes.Size());
//	//std::fill(hSVOLevelSizes.begin(), hSVOLevelSizes.end(), 0);
//	//std::copy(levelCounts + GI_DENSE_LEVEL, 
//	//		  levelCounts + GI_DENSE_LEVEL + sparseNodeCount, 
//	//		  hSVOLevelTotalSizes.data());
//	//hSVOLevelTotalSizes[0] = DenseSizeCube;
//	//dSVOLevelTotalSizes = hSVOLevelTotalSizes;
//
//	//// SVO Constants set
//	//hSVOConstants.denseDepth = DenseLevel;
//	//hSVOConstants.denseDim = DenseSize;
//	//hSVOConstants.totalDepth = totalLevel;
//	//hSVOConstants.numCascades = static_cast<uint32_t>(newAllocators.size());
//
//	//// Offset Set
//	//uint32_t levelOffset = 0;
//	//svoLevelOffsets.CPUData().clear();
//	//for(unsigned int i = GI_DENSE_LEVEL; i <= totalLevel; i++)
//	//{
//	//	svoLevelOffsets.AddData(levelOffset);
//	//	levelOffset += (i != GI_DENSE_LEVEL) ? levelCounts[i] : 0;
//	//}
//	//svoLevelOffsets.SendData();
//	//assert(levelOffset <= totalAlloc);
//
//	//// Copy to device
//	//CUDA_CHECK(hipMemcpy(dSVOConstants.Data(), 
//	//					  &hSVOConstants, 
//	//					  sizeof(CSVOConstants), 
//	//					  hipMemcpyHostToDevice));
//}
//
//void GISparseVoxelOctree::LinkSceneShadowMaps(SceneI* scene)
//{
//	//GLuint lightParamBuffer = scene->getSceneLights().getGLBuffer();
//	//GLuint shadowMapTexture = scene->getSceneLights().getShadowArrayGL();
//
//
//	//if(sceneShadowMapResource) CUDA_CHECK(hipGraphicsUnregisterResource(sceneShadowMapResource));
//	//if(sceneLightParamResource) CUDA_CHECK(hipGraphicsUnregisterResource(sceneLightParamResource));
//	//if(sceneVPMatrixResource) CUDA_CHECK(hipGraphicsUnregisterResource(sceneVPMatrixResource));
//	//CUDA_CHECK(hipGraphicsGLRegisterImage(&sceneShadowMapResource, shadowMapTexture, GL_TEXTURE_2D_ARRAY,
//	//									   hipGraphicsRegisterFlagsReadOnly));
//	//CUDA_CHECK(hipGraphicsGLRegisterBuffer(&sceneLightParamResource, lightParamBuffer,
//	//										hipGraphicsRegisterFlagsReadOnly));
//	//CUDA_CHECK(hipGraphicsGLRegisterBuffer(&sceneVPMatrixResource, lightVPBuffer,
//	//										hipGraphicsRegisterFlagsReadOnly));
//}
//
//void GISparseVoxelOctree::CreateSurfFromArray(hipArray_t& arr, hipSurfaceObject_t& surf)
//{
//	// Texture of SVO Dense
//	hipResourceDesc resDesc = {};
//	resDesc.resType = hipResourceTypeArray;
//	resDesc.res.array.array = arr;
//
//	if(surf != 0) CUDA_CHECK(hipDestroySurfaceObject(surf));
//	CUDA_CHECK(hipCreateSurfaceObject(&surf, &resDesc));
//}

//void GISparseVoxelOctree::CreateTexFromArray(hipArray_t& arr, hipTextureObject_t& tex)
//{
//	// Texture of SVO Dense
//	hipResourceDesc resDesc = {};
//	hipTextureDesc texDesc = {};
//	resDesc.resType = hipResourceTypeArray;
//	resDesc.res.array.array = arr;
//
//	texDesc.addressMode[0] = hipAddressModeWrap;
//	texDesc.addressMode[1] = hipAddressModeWrap;
//	texDesc.addressMode[2] = hipAddressModeWrap;
//	texDesc.filterMode = hipFilterModePoint;
//	texDesc.readMode = hipReadModeElementType;
//	texDesc.normalizedCoords = 0;
//
//	if(tex != 0) CUDA_CHECK(hipDestroyTextureObject(tex));
//	CUDA_CHECK(hipCreateTextureObject(&tex, &resDesc, &texDesc, nullptr));
//}

//void GISparseVoxelOctree::CopyFromBufferToTex(hipArray_t& arr, unsigned int* devPtr)
//{
//	// Copy Dense to Texture
//	hipMemcpy3DParms params = {0};
//	params.dstArray = arr;
//	params.srcPtr =
//	{
//		devPtr,
//		octreeParams.DenseSize * sizeof(unsigned int),
//		octreeParams.DenseSize,
//		octreeParams.DenseSize
//	};
//	params.extent = {octreeParams.DenseSize, octreeParams.DenseSize, octreeParams.DenseSize};
//	params.kind = hipMemcpyDeviceToDevice;
//	CUDA_CHECK(hipMemcpy3D(&params));
//}
//
//void GISparseVoxelOctree::CreateTexLayeredFromArray(hipMipmappedArray_t& arr, 
//													hipTextureObject_t& tex)
//{
//	// Texture of SVO Dense
//	hipResourceDesc resDesc = {};
//	hipTextureDesc texDesc = {};
//	resDesc.resType = hipResourceTypeMipmappedArray;
//	resDesc.res.mipmap.mipmap = arr;
//
//	texDesc.addressMode[0] = hipAddressModeWrap;
//	texDesc.addressMode[1] = hipAddressModeWrap;
//	texDesc.addressMode[2] = hipAddressModeWrap;
//	texDesc.filterMode = hipFilterModePoint;
//	texDesc.readMode = hipReadModeElementType;
//	texDesc.normalizedCoords = 1;
//
//	if(tex != 0) CUDA_CHECK(hipDestroyTextureObject(tex));
//	CUDA_CHECK(hipCreateTextureObject(&tex, &resDesc, &texDesc, nullptr));
//}

//void GISparseVoxelOctree::ConstructDense()
//{
//	//// Level 0 does not gurantee lower cascade parents
//	//// Each Allocator tries to allocate its parent
//	//for(unsigned int i = 0; i < allocators.size(); i++)
//	//{
//	//	uint32_t gridSize = ((allocators[i]->NumPages() * GI_PAGE_SIZE) + 
//	//							GI_THREAD_PER_BLOCK - 1) /
//	//							GI_THREAD_PER_BLOCK;
//	//	SVOReconstructDetermineNode<<<gridSize, GI_THREAD_PER_BLOCK>>>
//	//	(
//	//		dSVODense,
//	//		allocators[i]->GetVoxelPagesDevice(),
//	//	
//	//		i,
//	//		*dSVOConstants.Data()
//	//	);
//	//	CUDA_KERNEL_CHECK();
//	//}
//
//	//uint32_t gridSize = ((GI_DENSE_SIZE_CUBE) + GI_THREAD_PER_BLOCK - 1) / GI_THREAD_PER_BLOCK;
//	//SVOReconstructAllocateLevel<<<gridSize, GI_THREAD_PER_BLOCK>>>
//	//(
//	//	dSVODense,
//	//	*(dSVOLevelSizes.Data() + 1),
//	//	*(dSVOLevelTotalSizes.Data() + 1),
//	//	*(dSVOLevelTotalSizes.Data()),
//	//	*dSVOConstants.Data()
//	//);
//	//CUDA_KERNEL_CHECK();
//}
//
//void GISparseVoxelOctree::ConstructLevel(unsigned int currentLevel,
//										 unsigned int allocatorOffset)
//{
//	// Early Bail check 
//	unsigned int currentLevelIndex = currentLevel - octreeParams.DenseLevel;
//	CUDA_CHECK(hipMemcpy(hSVOLevelSizes.data() + currentLevelIndex,
//						  dSVOLevelSizes.Data() + currentLevelIndex,
//						  sizeof(unsigned int),
//						  hipMemcpyDeviceToHost));
//	if(hSVOLevelSizes[currentLevelIndex] == 0) return;
//
//	// ChildBitSet your Level (with next level's child)
//	// Allocate next level
//	// Memcopy next level start location to array
//	// Only ChildBitSet Upper Level
//	// Then Allocate your level
//	// Average Color to the level
//	//for(unsigned int i = allocatorOffset; i < allocators.size(); i++)
//	//{
//	//	//uint32_t gridSize = ((allocators[i]->NumPages() * GI_PAGE_SIZE) + 
//	//	//					 GI_THREAD_PER_BLOCK - 1) /
//	//	//					 GI_THREAD_PER_BLOCK;
//
//	//	//SVOReconstructDetermineNode<<<gridSize, GI_THREAD_PER_BLOCK>>>
//	//	//(
//	//	//	dSVOSparse,
//	//	//	tSVODenseNode,
//	//	//	allocators[i]->GetVoxelPagesDevice(),
//	//	//	dSVOOffsets,
//
//	//	//	i,
//	//	//	currentLevel,
//	//	//	*dSVOConstants.Data()
//	//	//);
//	//	//CUDA_KERNEL_CHECK();
//	//}
//	
//	//uint32_t gridSize = (hSVOLevelSizes[currentLevelIndex] + GI_THREAD_PER_BLOCK - 1) / GI_THREAD_PER_BLOCK;
//	//SVOReconstructAllocateLevel<<<gridSize, GI_THREAD_PER_BLOCK>>>
//	//(
//	//	dSVOSparse + svoLevelOffsets.CPUData()[currentLevelIndex],
//	//	*(dSVOLevelSizes.Data() + currentLevelIndex + 1),
//	//	*(dSVOLevelTotalSizes.Data() + currentLevelIndex + 1),
//	//	*(dSVOLevelSizes.Data() + currentLevelIndex),
//	//	*dSVOConstants.Data()
//	//);
//	//CUDA_KERNEL_CHECK();
//}
//
//double GISparseVoxelOctree::ConstructFullAtomic(const IEVector3& ambientColor, const InjectParams& p)
//{
//	//CudaTimer timer;
//	//timer.Start();
//
//	//// Fully Atomic Version
//	//for(unsigned int i = 0; i < allocators.size(); i++)
//	//{
//	//	uint32_t nodeCount = allocators[i]->NumPages() * GI_PAGE_SIZE;
//	//	uint32_t gridSize = (nodeCount + GI_THREAD_PER_BLOCK - 1) / GI_THREAD_PER_BLOCK;
//	//	SVOReconstruct<<<gridSize, GI_THREAD_PER_BLOCK>>>
//	//	(
//	//		dSVOMaterial,
// //           //dSVOLight,
//	//		dSVOSparse,
//	//		dSVODense,
//	//		dSVOLevelSizes.Data(),
//
//	//		dSVOOffsets,
//	//		dSVOLevelTotalSizes.Data(),
//	//			
//	//		// VoxelSystem Data
//	//		allocators[i]->GetVoxelPagesDevice(),
//	//		allocators[i]->GetObjRenderCacheDevice(),
//
//	//		//{ambientColor.getX(), ambientColor.getY(), ambientColor.getZ()},
//	//		0,
//	//		i,
//	//		*dSVOConstants.Data(),
//
// //           p.inject,
// //           p.span,
// //           p.outerCascadePos,
// //           float3{ambientColor.getX(), ambientColor.getY(), ambientColor.getZ()},
//
// //           p.camPos,
// //           p.camDir,
//
// //           dLightVPArray,
// //           dLightParamArray,
//
// //           p.depthNear,
// //           p.depthFar,
//
// //           tShadowMapArray,
// //           1
//	//	);
//	//	CUDA_KERNEL_CHECK();
//	//}
//	//// Copy Level Sizes
//	//CUDA_CHECK(hipMemcpy(hSVOLevelSizes.data(),
//	//					  dSVOLevelSizes.Data(),
//	//					  hSVOLevelSizes.size() * sizeof(uint32_t),
//	//					  hipMemcpyDeviceToHost));
//
//	//CopyFromBufferToTex(dSVODenseNodeArray, dSVODense);
//	//timer.Stop();
//	//return timer.ElapsedMilliS();
//	return 0;
//}
//
//double GISparseVoxelOctree::ConstructLevelByLevel(const IEVector3& ambientColor, const InjectParams& p)
//{
//	//CudaTimer timer;
//	//timer.Start();
//
//	//// Start with constructing dense
//	//ConstructDense();
//	//CopyFromBufferToTex(dSVODenseNodeArray, dSVODense);
//
//	//// Construct Levels
//	//for(unsigned int i = GI_DENSE_LEVEL + 1; i < allocatorGrids[0]->depth; i++)
//	//{
//	//	ConstructLevel(i, 0);
//	//}
//
//	//// Now adding cascade levels
//	//for(unsigned int i = 1; i < allocators.size(); i++)
//	//{
//	//	unsigned int currentLevel = allocatorGrids[0]->depth + i - 1;
//	//	ConstructLevel(currentLevel, i);
//	//}
//
//	//// Memcpy Last Total Size
//	//CUDA_CHECK(hipMemcpy(hSVOLevelSizes.data() + (hSVOConstants.totalDepth - GI_DENSE_LEVEL),
//	//					  dSVOLevelSizes.Data() + (hSVOConstants.totalDepth - GI_DENSE_LEVEL),
//	//					  sizeof(uint32_t),
//	//					  hipMemcpyDeviceToHost));
//
//	//// Average Leafs
//	//for(unsigned int i = 0; i < allocators.size(); i++)
//	//{
//	//	assert(allocators[i]->IsGLMapped() == true);
//	//	uint32_t gridSize = (allocators[i]->NumPages() * GI_PAGE_SIZE +  GI_THREAD_PER_BLOCK - 1) / 
//	//						GI_THREAD_PER_BLOCK;
//	//			
//	//	// Average Leaf Node
//	//	SVOReconstructMaterialLeaf<<<gridSize, GI_THREAD_PER_BLOCK>>>
//	//	(
//	//		dSVOMaterial,
//
//	//		// Const SVO Data
//	//		dSVOSparse,
//	//		dSVOOffsets,
//	//		tSVODenseNode,
//
//	//		// Page Data
//	//		allocators[i]->GetVoxelPagesDevice(),
//	//									  
//	//		// For Color Lookup
//	//		allocators[i]->GetObjRenderCacheDevice(),
//
//	//		// Constants
//	//		0,
//	//		i,
//	//		*dSVOConstants.Data(),
//
// //           p.inject,
// //           p.span,
// //           p.outerCascadePos,
// //           float3{ambientColor.getX(), ambientColor.getY(), ambientColor.getZ()},
//
// //           p.camPos,
// //           p.camDir,
//
// //           dLightVPArray,
// //           dLightParamArray,
//
// //           p.depthNear,
// //           p.depthFar,
//
// //           tShadowMapArray,
// //           1
//	//	);
//	//	CUDA_KERNEL_CHECK();
//	//}
//
//	//timer.Stop();
//	//return timer.ElapsedMilliS();
//	return 0.0;
//}
//
//double GISparseVoxelOctree::LightInject(InjectParams params,
//										const std::vector<IEMatrix4x4>& projMatrices,
//										const std::vector<IEMatrix4x4>& invViewProj)
//{
//    return 0.0;
//}
//
//double GISparseVoxelOctree::AverageNodes()
//{
//	////CudaTimer timer;
//	////timer.Start();
//
//	////// Now use leaf nodes to average upper nodes
//	////// Start bottom up
//	////for(int i = hSVOConstants.totalDepth - 1; i >= static_cast<int>(hSVOConstants.denseDepth); i--)
//	////{
//	////	unsigned int arrayIndex = i - GI_DENSE_LEVEL;
//	////	unsigned int levelDim = GI_DENSE_SIZE >> (GI_DENSE_LEVEL - i);
//	////	unsigned int levelSize = (i > GI_DENSE_LEVEL) ? hSVOLevelSizes[arrayIndex]: 
//	////													levelDim * levelDim * levelDim;
//	////	if(levelSize == 0) continue;
//
//	////	uint32_t gridSize = (levelSize * 2 + GI_THREAD_PER_BLOCK - 1) / GI_THREAD_PER_BLOCK;
//	////	// Average Level
//	////	SVOReconstructAverageNode<<<gridSize, GI_THREAD_PER_BLOCK>>>
//	////	(
//	////		dSVOMaterial,
//	////		sSVODenseMat[0],
//
// ////           //dSVOLight,
//	////		dSVODense,
//	////		dSVOSparse,
//
//	////		dSVOOffsets,
//	////		*(dSVOOffsets + arrayIndex),
//	////		*(dSVOOffsets + arrayIndex + 1),
//
//	////		levelSize,
//	////		0,
//	////		i,
//	////		*dSVOConstants.Data()
//	////	);
//	////	CUDA_KERNEL_CHECK();
//	////}
//	////
//	////// Dense Reduction
//	////for(int i = 1; i < GI_DENSE_TEX_COUNT; i++)
//	////{
//	////	uint32_t levelSize = GI_DENSE_SIZE >> i;
//	////	uint32_t levelSizeCube = levelSize * levelSize * levelSize;
//	////		
//	////	uint32_t grid = ((levelSizeCube * GI_DENSE_WORKER_PER_PARENT) + GI_THREAD_PER_BLOCK - 1) / 
//	////					GI_THREAD_PER_BLOCK;
//	////	
//	////	SVOReconstructAverageNode<<<grid, GI_THREAD_PER_BLOCK>>>
//	////	(
//	////		sSVODenseMat[i - 1],
//	////		sSVODenseMat[i],
//	////		levelSize
//	////	);
//	////}
//
//	////timer.Stop();
//	////return timer.ElapsedMilliS();
//	//return 0.0;
//	return 0;
//}
//
//void GISparseVoxelOctree::UpdateSVO(double& reconstTime,
//									double& injectTime,
//									double& averageTime,
//									const IEVector3& ambientColor,
//									const InjectParams& p,
//									const std::vector<IEMatrix4x4>& projMatrices,
//									const std::vector<IEMatrix4x4>& invViewProj)
//{
//	//// Clear Mat Texture
//	//GLuint ff[4] = {0x0, 0x0, 0x0, 0x0};
//	//for(unsigned int i = 0; i < DenseLevelCount; i++)
//	//	glClearTexImage(svoDenseMat, i, GL_RGBA_INTEGER, GL_UNSIGNED_INT, &ff);
//
//	//// Shadow Maps
//	//CUDA_CHECK(hipGraphicsMapResources(1, &sceneLightParamResource));
//	//CUDA_CHECK(hipGraphicsMapResources(1, &sceneVPMatrixResource));
//	//CUDA_CHECK(hipGraphicsMapResources(1, &sceneShadowMapResource));
//
//	//// SVO Nodes
//	//CUDA_CHECK(hipGraphicsMapResources(1, &svoMaterialResource));
//	//CUDA_CHECK(hipGraphicsMapResources(1, &svoNodeResource));
//	//CUDA_CHECK(hipGraphicsMapResources(1, &svoLevelOffsetResource));
//	//CUDA_CHECK(hipGraphicsMapResources(1, &svoDenseNodeResource));
//	//CUDA_CHECK(hipGraphicsMapResources(1, &svoDenseTexResource));
//	//
//	//size_t size;
//	//CUDA_CHECK(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&dSVODense), 
//	//												 &size, svoNodeResource));
//	//CUDA_CHECK(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&dSVOMaterial),
//	//												 &size, svoMaterialResource));
//	//CUDA_CHECK(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&dSVOOffsets),
//	//												&size, svoLevelOffsetResource));
//	//CUDA_CHECK(hipGraphicsSubResourceGetMappedArray(&dSVODenseNodeArray, svoDenseNodeResource, 0, 0));
//	//CreateSurfFromArray(dSVODenseNodeArray, sSVODenseNode);
//	//CreateTexFromArray(dSVODenseNodeArray, tSVODenseNode);
//	//for(unsigned int i = 0; i < DenseLevelCount; i++)
//	//{
//	//	CUDA_CHECK(hipGraphicsSubResourceGetMappedArray(&dSVODenseMatArray[i], svoDenseTexResource, 0, i));
//	//	CreateSurfFromArray(dSVODenseMatArray[i], sSVODenseMat[i]);
//	//}
//	//dSVOSparse = dSVODense + DenseSizeCube;
//
//	//// Shadow Related
//	//CUDA_CHECK(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&dLightParamArray),
//	//												&size, sceneLightParamResource));
//	//CUDA_CHECK(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&dLightVPArray),
//	//												&size, sceneVPMatrixResource));
//	//CUDA_CHECK(cudaGraphicsResourceGetMappedMipmappedArray(&shadowMapArray, sceneShadowMapResource));
//	//CreateTexLayeredFromArray(shadowMapArray, tShadowMapArray);
//
//	//// Reset Atomic Counter since we reconstruct every frame
//	//uint32_t usedNodeCount = hSVOLevelSizes.back() + svoLevelOffsets.CPUData().back();
//	//CUDA_CHECK(hipMemset(dSVODense, 0xFF, sizeof(CSVONode) * (usedNodeCount + GI_DENSE_SIZE_CUBE)));
//	//CUDA_CHECK(hipMemset(dSVOMaterial, 0x00, sizeof(CSVOMaterial) * (usedNodeCount)));
//
//	//dSVOLevelSizes.Memset(0x00, 0, dSVOLevelSizes.Size());
//	//std::fill(hSVOLevelSizes.begin(), hSVOLevelSizes.end(), 0);
//
//	//// Maxwell is faster with fully atomic code (CAS Locks etc.)
//	//// However kepler sucks(660ti) (100ms compared to 5ms) 
// //   IEVector3 aColor = (false) ? IEVector3::ZeroVector : ambientColor;
//	//if(CudaInit::CapabilityMajor() >= 5)
//	//	reconstTime = ConstructFullAtomic(aColor, p);
//	//else
//	//	reconstTime = ConstructLevelByLevel(aColor, p);
// //   injectTime = 0.0;// LightInject(p, projMatrices, invViewProj);
//	//averageTime = AverageNodes();
//
//	////// DEBUG
//	////GI_LOG("-------------------------------------------");
//	////GI_LOG("Tree Node Data");
//	////unsigned int i;
//	////for(i = 0; i <= allocatorGrids[0]->depth - GI_DENSE_LEVEL + allocators.size() - 1; i++)
//	////{
//	////	if(i == 0) GI_LOG("#%d Dense : %d", GI_DENSE_LEVEL + i, GI_DENSE_SIZE_CUBE);
//	////	else GI_LOG("#%d Level : %d", GI_DENSE_LEVEL + i, hSVOLevelSizes[i]);
//	////}
//	////unsigned int total = std::accumulate(hSVOLevelSizes.begin(),
//	////									 hSVOLevelSizes.end(), 0);
//	////GI_LOG("Total : %d", total);
//	//
//	//CUDA_CHECK(hipGraphicsUnmapResources(1, &sceneLightParamResource));
//	//CUDA_CHECK(hipGraphicsUnmapResources(1, &sceneShadowMapResource));
//	//CUDA_CHECK(hipGraphicsUnmapResources(1, &sceneVPMatrixResource));
//
//	//CUDA_CHECK(hipGraphicsUnmapResources(1, &svoMaterialResource));
//	//CUDA_CHECK(hipGraphicsUnmapResources(1, &svoNodeResource));
//	//CUDA_CHECK(hipGraphicsUnmapResources(1, &svoLevelOffsetResource));
//	//CUDA_CHECK(hipGraphicsUnmapResources(1, &svoDenseNodeResource));
//	//CUDA_CHECK(hipGraphicsUnmapResources(1, &svoDenseTexResource));
//}
//
//double GISparseVoxelOctree::GlobalIllumination(DeferredRenderer& dRenderer,
//											   const Camera& camera,
//											   SceneI& scene,
//											   float coneAngle,
//											   float maxDistance,
//											   float falloffFactor,
//											   float sampleDistanceRatio,
//											   float intensityFactorAO,
//											   float intensityFactorGI,
//											   bool giOn,
//											   bool aoOn,
//											   bool specular)
//{
////	// Light Intensity Texture
////	static const GLubyte ff[4] = {0xFF, 0xFF, 0xFF, 0xFF};
////	glClearTexImage(traceTexture, 0, GL_RGBA, GL_UNSIGNED_BYTE, &ff);
////
////	// Update FrameTransform Matrices 
////	// And its inverse realted buffer
////	//assert(TraceWidth == DeferredRenderer::gBuffWidth);
////	//assert(TraceHeight == DeferredRenderer::gBuffHeight);
////	dRenderer.RefreshInvFTransform(camera, TraceWidth, TraceHeight);
////	dRenderer.GetFTransform().Update(camera.generateTransform());
////
////	// Timing Voxelization Process
////	GLuint queryID;
////	glGenQueries(1, &queryID);
////	glBeginQuery(GL_TIME_ELAPSED, queryID);
////
////	// Set Cascade Trace Data
////	float3 pos = allocatorGrids[0]->position;
////	uint32_t dim = allocatorGrids[0]->dimension.x * (0x1 << (allocators.size() - 1));
////	uint32_t depth = allocatorGrids[0]->depth + static_cast<uint32_t>(allocators.size()) - 1;
////	svoTraceData.CPUData()[0] =
////	{
////		{pos.x, pos.y, pos.z, allocatorGrids.back()->span},
////		{dim, depth, GI_DENSE_SIZE, GI_DENSE_LEVEL},
////		{
////			static_cast<unsigned int>(allocators.size()),
////			GI_DENSE_SIZE_CUBE,
////			0,
////			0
////		}
////	};
////	svoTraceData.SendData();
////
////    //TEST
////    //// Convert Diameter to interpolation weight and levels
////    //float diameter = std::tan(coneAngle) * maxDistance;
////    //float diameterRatio = diameter / allocatorGrids.back()->span;
////    //diameterRatio = std::max(diameterRatio, 1.0f);
////    //unsigned int closestPow = static_cast<unsigned int>(std::floor(std::log2(diameterRatio)));
////    //float interp = (diameterRatio - float(0x1 << closestPow)) / float(0x1 << closestPow);
////    //unsigned int nodeLevel = depth - closestPow;
////    ////nodeDepth = 8;
////
////    //GI_LOG("(D%f, C%d)(%f, %d, %d)", diameterRatio, closestPow, interp, nodeLevel, nodeLevel - 1);
////
////	// Set Cone Trace Data
////	svoConeParams.CPUData()[0] =
////	{
////		{maxDistance, std::tan(coneAngle), std::tan(coneAngle * 0.5f), sampleDistanceRatio},
////		{intensityFactorAO, intensityFactorGI, IEMath::Sqrt3, falloffFactor}
////	};
////	svoConeParams.SendData();
////
////	// Shaders
////	computeGI.Bind();
////
////	// Shadow Related
////	dRenderer.BindShadowMaps(scene);
////	dRenderer.BindLightBuffers(scene);
////
////	// Uniforms
////	glUniform1ui(U_LIGHT_INDEX, static_cast<GLuint>(0));
////	glUniform1ui(U_ON_OFF_SWITCH, specular ? 1u : 0u);
////
////	// Buffers
////	svoNodeBuffer.BindAsShaderStorageBuffer(LU_SVO_NODE);
////	svoMaterialBuffer.BindAsShaderStorageBuffer(LU_SVO_MATERIAL);
////	svoLevelOffsets.BindAsShaderStorageBuffer(LU_SVO_LEVEL_OFFSET);
////	dRenderer.GetInvFTransfrom().BindAsUniformBuffer(U_INVFTRANSFORM);
////	dRenderer.GetFTransform().Bind();
////	svoTraceData.BindAsUniformBuffer(U_SVO_CONSTANTS);
////	svoConeParams.BindAsUniformBuffer(U_CONE_PARAMS);
////
////	// Images
////	dRenderer.GetGBuffer().BindAsTexture(T_COLOR, RenderTargetLocation::COLOR);
////	dRenderer.GetGBuffer().BindAsTexture(T_DEPTH, RenderTargetLocation::DEPTH);
////	dRenderer.GetGBuffer().BindAsTexture(T_NORMAL, RenderTargetLocation::NORMAL);
////	glBindImageTexture(I_LIGHT_INENSITY, traceTexture, 0, false, 0, GL_WRITE_ONLY, GL_RGBA8/*GL_RGBA16*/);
////	glActiveTexture(GL_TEXTURE0 + T_DENSE_NODE);
////	glBindTexture(GL_TEXTURE_3D, svoDenseNode);
////	glBindSampler(T_DENSE_NODE, nodeSampler);
////	glActiveTexture(GL_TEXTURE0 + T_DENSE_MAT);
////	glBindTexture(GL_TEXTURE_3D, svoDenseMat);
////	glBindSampler(T_DENSE_MAT, materialSampler);
////
////	// Dispatch
////	uint2 gridSize;
////    gridSize.x = (TraceWidth + 16 - 1) / 16;
////    gridSize.y = (TraceHeight + 16 - 1) / 16;
////	glDispatchCompute(gridSize.x, gridSize.y, 1);
////	glMemoryBarrier(GL_SHADER_IMAGE_ACCESS_BARRIER_BIT);
////
////    // Detect Edge
////    computeEdge.Bind();
////    glUniform2f(U_TRESHOLD, 0.007f, IEMath::CosF(IEMath::ToRadians(10.0f)));
////    glUniform2f(U_NEAR_FAR, camera.near, camera.far);
////    dRenderer.GetGBuffer().BindAsTexture(T_DEPTH, RenderTargetLocation::DEPTH);
////    dRenderer.GetGBuffer().BindAsTexture(T_NORMAL, RenderTargetLocation::NORMAL);
////    glBindImageTexture(I_OUT, edgeTex, 0, false, 0, GL_WRITE_ONLY, GL_RG8);
////    //glBindImageTexture(I_OUT, traceTexture, 0, false, 0, GL_WRITE_ONLY, GL_RGBA16F);
////
////    gridSize.x = (TraceWidth + 16 - 1) / 16;
////    gridSize.y = (TraceHeight + 16 - 1) / 16;
////    glDispatchCompute(gridSize.x, gridSize.y, 1);
////    glMemoryBarrier(GL_SHADER_IMAGE_ACCESS_BARRIER_BIT);
////
////    // Edge Aware Gauss
////    computeGauss32.Bind();
////    glActiveTexture(GL_TEXTURE0 + T_EDGE);
////    glBindTexture(GL_TEXTURE_2D, edgeTex);
////    glBindSampler(T_EDGE, nodeSampler);
////
////    // Call #1 (Vertical)
////    GLuint inTex = traceTexture;
////    GLuint outTex = gaussTex;
////    for(unsigned int i = 0; i < 4; i++)
////    {
////        glActiveTexture(GL_TEXTURE0 + T_IN);
////        glBindTexture(GL_TEXTURE_2D, inTex);
////        glBindSampler(T_IN, gaussSampler);
////        glBindImageTexture(I_OUT, outTex, 0, false, 0, GL_WRITE_ONLY, GL_RGBA8/*GL_RGBA16*/);
////        glUniform1ui(U_DIRECTION, 0);
////        glDispatchCompute(gridSize.x, gridSize.y, 1);
////        glMemoryBarrier(GL_SHADER_IMAGE_ACCESS_BARRIER_BIT);
////
////        // Call #2 (Horizontal)
////        glActiveTexture(GL_TEXTURE0 + T_IN);
////        glBindTexture(GL_TEXTURE_2D, outTex);
////        glBindSampler(T_IN, gaussSampler);
////        glBindImageTexture(I_OUT, inTex, 0, false, 0, GL_WRITE_ONLY, GL_RGBA8/*GL_RGBA16*/);
////        glUniform1ui(U_DIRECTION, 1);
////        glDispatchCompute(gridSize.x, gridSize.y, 1);
////
////    }
////    glMemoryBarrier(GL_SHADER_IMAGE_ACCESS_BARRIER_BIT);
////
////	// Apply to DRenderer Li Tex
////	computeLIApply.Bind();
////	
////	// Uniform
////	glUniform2ui(U_ON_OFF_SWITCH, aoOn ? 1u : 0u, giOn ? 1u : 0u);
////
////	// Textures
////	GLuint gBufferLITex = dRenderer.GetLightIntensityBufferGL();
////	glBindImageTexture(I_LIGHT_INENSITY, gBufferLITex, 0, false, 0, GL_READ_WRITE, GL_RGBA16F);
////	glActiveTexture(GL_TEXTURE0 + T_COLOR);
////	glBindTexture(GL_TEXTURE_2D, traceTexture);
////	glBindSampler(T_COLOR, nodeSampler);
////
////	gridSize.x = (DeferredRenderer::gBuffWidth + 16 - 1) / 16;
////	gridSize.y = (DeferredRenderer::gBuffHeight + 16 - 1) / 16;
////	glDispatchCompute(gridSize.x, gridSize.y, 1);
////	glMemoryBarrier(GL_SHADER_IMAGE_ACCESS_BARRIER_BIT);
////
////	// Timer
////	GLuint64 timeElapsed = 0;
////	glEndQuery(GL_TIME_ELAPSED);
////	glGetQueryObjectui64v(queryID, GL_QUERY_RESULT, &timeElapsed);
////
////	// I have to unbind the compute shader or weird things happen
////	Shader::Unbind(ShaderType::COMPUTE);
////	return timeElapsed / 1000000.0;
////}
////
////double GISparseVoxelOctree::AmbientOcclusion(DeferredRenderer& dRenderer,
////											 const Camera& camera,
////											 float coneAngle,
////											 float maxDistance,
////											 float falloffFactor,
////											 float sampleDistanceRatio,
////											 float intensityFactor)
////{
////	// Light Intensity Texture
////	static const GLubyte ff[4] = {0xFF, 0xFF, 0xFF, 0xFF};
////	glClearTexImage(traceTexture, 0, GL_RGBA, GL_UNSIGNED_BYTE, &ff);
////
////	// Update FrameTransform Matrices 
////	// And its inverse realted buffer
////	//assert(TraceWidth == DeferredRenderer::gBuffWidth);
////	//assert(TraceHeight == DeferredRenderer::gBuffHeight);
////	dRenderer.RefreshInvFTransform(camera, TraceWidth, TraceHeight);
////	dRenderer.GetFTransform().Update(camera.generateTransform());
////
////	// Timing Voxelization Process
////	GLuint queryID;
////	glGenQueries(1, &queryID);
////	glBeginQuery(GL_TIME_ELAPSED, queryID);
////
////	// Set Cascade Trace Data
////	float3 pos = allocatorGrids[0]->position;
////	uint32_t dim = allocatorGrids[0]->dimension.x * (0x1 << (allocators.size() - 1));
////	uint32_t depth = allocatorGrids[0]->depth + static_cast<uint32_t>(allocators.size()) - 1;
////	svoTraceData.CPUData()[0] =
////	{
////		{pos.x, pos.y, pos.z, allocatorGrids.back()->span},
////		{dim, depth, GI_DENSE_SIZE, GI_DENSE_LEVEL},
////		{
////			static_cast<unsigned int>(allocators.size()),
////			GI_DENSE_SIZE_CUBE,
////			0,
////			0
////		}
////	};
////	svoTraceData.SendData();
////
////	// Set Cone Trace Data
////	svoConeParams.CPUData()[0] =
////	{
////		{maxDistance, std::tan(coneAngle), std::tan(coneAngle * 0.5f), sampleDistanceRatio},
////		{intensityFactor, IEMath::Sqrt2, IEMath::Sqrt3, falloffFactor}
////	};
////	svoConeParams.SendData();
////
////	// Shaders
////	computeAO.Bind();
////	//computeAOSurf.Bind();
////
////	// Buffers
////	svoNodeBuffer.BindAsShaderStorageBuffer(LU_SVO_NODE);
////	svoMaterialBuffer.BindAsShaderStorageBuffer(LU_SVO_MATERIAL);
////	svoLevelOffsets.BindAsShaderStorageBuffer(LU_SVO_LEVEL_OFFSET);
////	dRenderer.GetInvFTransfrom().BindAsUniformBuffer(U_INVFTRANSFORM);
////	dRenderer.GetFTransform().Bind();
////	svoTraceData.BindAsUniformBuffer(U_SVO_CONSTANTS);
////	svoConeParams.BindAsUniformBuffer(U_CONE_PARAMS);
////
////	// Images
////	dRenderer.GetGBuffer().BindAsTexture(T_DEPTH, RenderTargetLocation::DEPTH);
////	dRenderer.GetGBuffer().BindAsTexture(T_NORMAL, RenderTargetLocation::NORMAL);
////	glBindImageTexture(I_LIGHT_INENSITY, traceTexture, 0, false, 0, GL_WRITE_ONLY, GL_RGBA8);
////	glActiveTexture(GL_TEXTURE0 + T_DENSE_NODE);
////	glBindTexture(GL_TEXTURE_3D, svoDenseNode);
////	glBindSampler(T_DENSE_NODE, nodeSampler);
////	glActiveTexture(GL_TEXTURE0 + T_DENSE_MAT);
////	glBindTexture(GL_TEXTURE_3D, svoDenseMat);
////	glBindSampler(T_DENSE_MAT, materialSampler);
////	
////	// Dispatch
////	uint2 gridSize;
////	gridSize.x = (TraceWidth + 16 - 1) / 16;
////	gridSize.y = (TraceHeight + 16 - 1) / 16;
////	glDispatchCompute(gridSize.x, gridSize.y, 1);
////
////	//uint2 gridSize;
////	//gridSize.x = (TraceWidth + 16 - 1) / 16;
////	//gridSize.y = (TraceHeight + 16 - 1) / 16;
////	//glDispatchCompute(gridSize.x, gridSize.y, 1);
////
////	//// Detect Edge
////	//computeEdge.Bind();
////	//glUniform2f(U_TRESHOLD, 0.007f, IEMath::CosF(IEMath::ToRadians(20.0f)));
////	//glUniform2f(U_NEAR_FAR, camera.near, camera.far);
////	//dRenderer.GetGBuffer().BindAsTexture(T_DEPTH, RenderTargetLocation::DEPTH);
////	//dRenderer.GetGBuffer().BindAsTexture(T_NORMAL, RenderTargetLocation::NORMAL);
////	//glBindImageTexture(I_OUT, edgeTex, 0, false, 0, GL_WRITE_ONLY, GL_RG8);
////	//
////	//gridSize.x = (TraceWidth + 16 - 1) / 16;
////	//gridSize.y = (TraceHeight + 16 - 1) / 16;
////	//glDispatchCompute(gridSize.x, gridSize.y, 1);
////	//glMemoryBarrier(GL_SHADER_IMAGE_ACCESS_BARRIER_BIT);
////
////	////dRenderer.ShowTexture(camera, edgeTex);
////
////	//// Edge Aware Gauss
////	//computeGauss32.Bind();
////	//glActiveTexture(GL_TEXTURE0 + T_EDGE);
////	//glBindTexture(GL_TEXTURE_2D, svoDenseMat);
////	//glBindSampler(T_EDGE, gaussSampler);
////
////	//// Call #1 (Vertical)
////	//GLuint inTex = liTexture;
////	//GLuint outTex = gaussTex;
////	//for(unsigned int i = 0; i < 32; i++)
////	//{
////	//	glActiveTexture(GL_TEXTURE0 + T_IN);
////	//	glBindTexture(GL_TEXTURE_2D, inTex);
////	//	glBindSampler(T_EDGE, gaussSampler);
////	//	glBindImageTexture(I_OUT, outTex, 0, false, 0, GL_WRITE_ONLY, GL_RGBA8);
////	//	glUniform1ui(U_DIRECTION, 0);
////	//	glDispatchCompute(gridSize.x, gridSize.y, 1);
////	//	glMemoryBarrier(GL_SHADER_IMAGE_ACCESS_BARRIER_BIT);
////
////	//	// Call #2 (Horizontal)
////	//	glActiveTexture(GL_TEXTURE0 + T_IN);
////	//	glBindTexture(GL_TEXTURE_2D, outTex);
////	//	glBindSampler(T_EDGE, gaussSampler);
////	//	glBindImageTexture(I_OUT, inTex, 0, false, 0, GL_WRITE_ONLY, GL_RGBA8);
////	//	glUniform1ui(U_DIRECTION, 1);
////	//	glDispatchCompute(gridSize.x, gridSize.y, 1);
////
////	//	GLuint temp = inTex;
////	//	inTex = outTex;
////	//	outTex = temp;
////	//}
////
////	// Render to window
////	glMemoryBarrier(GL_SHADER_IMAGE_ACCESS_BARRIER_BIT);
////	dRenderer.ShowTexture(camera, traceTexture);
////
////	// Timer
////	GLuint64 timeElapsed = 0;
////	glEndQuery(GL_TIME_ELAPSED);
////	glGetQueryObjectui64v(queryID, GL_QUERY_RESULT, &timeElapsed);
////
////	// I have to unbind the compute shader or weird things happen
////	Shader::Unbind(ShaderType::COMPUTE);
////	return timeElapsed / 1000000.0;
//	return 0.0;
//}
//
//double GISparseVoxelOctree::DebugDeferredSVO(DeferredRenderer& dRenderer,
//											 const Camera& camera,
//											 uint32_t renderLevel,
//											 SVOTraceType type)
//{
//	//// Update FrameTransform Matrices 
//	//// And its inverse realted buffer
//	//assert(TraceWidth == DeferredRenderer::gBuffWidth);
//	//assert(TraceHeight == DeferredRenderer::gBuffHeight);
//	//dRenderer.RefreshInvFTransform(camera, TraceWidth, TraceHeight);
//	//dRenderer.GetFTransform().Update(camera.generateTransform());
//
//	//// Timing Voxelization Process
//	//GLuint queryID;
//	//glGenQueries(1, &queryID);
//	//glBeginQuery(GL_TIME_ELAPSED, queryID);
//
//	//// Set Cascade Trace Data
//	//float3 pos = allocatorGrids[0]->position;
//	//uint32_t dim = allocatorGrids[0]->dimension.x * (0x1 << (allocators.size() - 1));
//	//uint32_t depth = allocatorGrids[0]->depth + static_cast<uint32_t>(allocators.size()) - 1;
//	//svoTraceData.CPUData()[0] =
//	//{
//	//	{pos.x, pos.y, pos.z, allocatorGrids.back()->span},
//	//	{dim, depth, GI_DENSE_SIZE, GI_DENSE_LEVEL},
//	//	{
//	//		static_cast<unsigned int>(allocators.size()),
//	//		GI_DENSE_SIZE * GI_DENSE_SIZE * GI_DENSE_SIZE,
//	//		0,
//	//		GI_DENSE_LEVEL - GI_DENSE_TEX_COUNT + 1
//	//	}
//	//};
//	//svoTraceData.SendData();
//
//	//// Shaders
//	////computeVoxTraceDeferred.Bind();
//	//computeVoxTraceDeferredLerp.Bind();
//	//glUniform1ui(U_RENDER_TYPE, static_cast<GLuint>(type));
//	//glUniform1ui(U_FETCH_LEVEL, static_cast<GLuint>(renderLevel));
//
//	//// Buffers
//	//svoNodeBuffer.BindAsShaderStorageBuffer(LU_SVO_NODE);
//	//svoMaterialBuffer.BindAsShaderStorageBuffer(LU_SVO_MATERIAL);
//	//svoLevelOffsets.BindAsShaderStorageBuffer(LU_SVO_LEVEL_OFFSET);
//	//dRenderer.GetInvFTransfrom().BindAsUniformBuffer(U_INVFTRANSFORM);
//	//dRenderer.GetFTransform().Bind();
//	//svoTraceData.BindAsUniformBuffer(U_SVO_CONSTANTS);
//
//	//// Images
//	//dRenderer.GetGBuffer().BindAsTexture(T_DEPTH, RenderTargetLocation::DEPTH);
//	//glBindImageTexture(I_COLOR_FB, traceTexture, 0, false, 0, GL_WRITE_ONLY, GL_RGBA8);
//	//glActiveTexture(GL_TEXTURE0 + T_DENSE_NODE);
//	//glBindTexture(GL_TEXTURE_3D, svoDenseNode);
//	//glBindSampler(T_DENSE_NODE, nodeSampler);
//	//glActiveTexture(GL_TEXTURE0 + T_DENSE_MAT);
//	//glBindTexture(GL_TEXTURE_3D, svoDenseMat);
//	//glBindSampler(T_DENSE_MAT, materialSampler);
//
//	//// Dispatch
//	//uint2 gridSize;
//	//gridSize.x = (TraceWidth + 16 - 1) / 16;
//	//gridSize.y = (TraceHeight + 16 - 1) / 16;
//	//glDispatchCompute(gridSize.x, gridSize.y, 1);
//
//	//// Render to window
//	//glMemoryBarrier(GL_SHADER_IMAGE_ACCESS_BARRIER_BIT);
//	//dRenderer.ShowTexture(camera, traceTexture);
//
//	//// Timer
//	//GLuint64 timeElapsed = 0;
//	//glEndQuery(GL_TIME_ELAPSED);
//	//glGetQueryObjectui64v(queryID, GL_QUERY_RESULT, &timeElapsed);
//
//	//// I have to unbind the compute shader or weird things happen
//	//Shader::Unbind(ShaderType::COMPUTE);
//	//return static_cast<double>(timeElapsed) / 1000000.0;
//	return 0.0;
//}
//
//double GISparseVoxelOctree::DebugTraceSVO(DeferredRenderer& dRenderer,
//										  const Camera& camera,
//										  uint32_t renderLevel,
//										  SVOTraceType type)
//{
//	//// Update FrameTransform Matrices 
//	//// And its inverse realted buffer
//	//dRenderer.RefreshInvFTransform(camera, TraceWidth, TraceHeight);
//	//dRenderer.GetFTransform().Update(camera.generateTransform());
//
//	//// Timing Voxelization Process
//	//GLuint queryID;
//	//glGenQueries(1, &queryID);
//	//glBeginQuery(GL_TIME_ELAPSED, queryID);
//	//glBindFramebuffer(GL_FRAMEBUFFER, 0);
//
//	//// Set Cascade Trace Data
//	//float3 pos = allocatorGrids[0]->position;
//	//uint32_t dim = allocatorGrids[0]->dimension.x * (0x1 << (allocators.size() - 1));
//	//uint32_t depth = allocatorGrids[0]->depth + static_cast<uint32_t>(allocators.size()) - 1;
//	//svoTraceData.CPUData()[0] = 
//	//{
//	//	{pos.x, pos.y, pos.z, allocatorGrids.back()->span},
//	//	{dim, depth, GI_DENSE_SIZE, GI_DENSE_LEVEL},
//	//	{
//	//		static_cast<unsigned int>(allocators.size()), 
//	//		GI_DENSE_SIZE_CUBE,
//	//		0,
//	//		GI_DENSE_LEVEL - GI_DENSE_TEX_COUNT + 1
//	//	}
//	//};
//	//svoTraceData.SendData();
//
//	//// Shaders
//	//computeVoxTraceWorld.Bind();
//	//glUniform1ui(U_RENDER_TYPE, static_cast<GLuint>(type));
//	//glUniform1ui(U_FETCH_LEVEL, static_cast<GLuint>(renderLevel));
//
//	//// Buffers
//	//svoNodeBuffer.BindAsShaderStorageBuffer(LU_SVO_NODE);
//	//svoMaterialBuffer.BindAsShaderStorageBuffer(LU_SVO_MATERIAL);
//	//svoLevelOffsets.BindAsShaderStorageBuffer(LU_SVO_LEVEL_OFFSET);
//	//dRenderer.GetInvFTransfrom().BindAsUniformBuffer(U_INVFTRANSFORM);
//	//dRenderer.GetFTransform().Bind();
//	//svoTraceData.BindAsUniformBuffer(U_SVO_CONSTANTS);
//
//	//// Images
//	//glBindImageTexture(I_COLOR_FB, traceTexture, 0, false, 0, GL_WRITE_ONLY, GL_RGBA8);
//	//glActiveTexture(GL_TEXTURE0 + T_DENSE_NODE);
//	//glBindTexture(GL_TEXTURE_3D, svoDenseNode);
//	//glBindSampler(T_DENSE_NODE, nodeSampler);
//	//glActiveTexture(GL_TEXTURE0 + T_DENSE_MAT);
//	//glBindTexture(GL_TEXTURE_3D, svoDenseMat);
//	//glBindSampler(T_DENSE_MAT, materialSampler);
//
//	//// Dispatch
//	//uint2 gridSize;
//	//gridSize.x = (TraceWidth + 16 - 1) / 16;
//	//gridSize.y = (TraceHeight + 16 - 1) / 16;
//	//glDispatchCompute(gridSize.x, gridSize.y, 1);
//	//
//	//// Render to window
//	//glMemoryBarrier(GL_SHADER_IMAGE_ACCESS_BARRIER_BIT);
//	//dRenderer.ShowTexture(camera, traceTexture);
//
//	//// Timer
//	//GLuint64 timeElapsed = 0;
//	//glEndQuery(GL_TIME_ELAPSED);
//	//glGetQueryObjectui64v(queryID, GL_QUERY_RESULT, &timeElapsed);
//	//
//	//// I have to unbind the compute shader or weird things happen
//	//Shader::Unbind(ShaderType::COMPUTE);
//	//return timeElapsed / 1000000.0;
//	return 0.0;
//}
//
//uint64_t GISparseVoxelOctree::MemoryUsage() const
//{
//	//uint64_t totalBytes = 0;
//	//totalBytes += svoNodeBuffer.Capacity() * sizeof(CSVONode);
//	//totalBytes += svoMaterialBuffer.Capacity() * sizeof(CSVOMaterial);
//	//totalBytes += dSVOLevelSizes.Size() * sizeof(unsigned int);
//	//totalBytes += sizeof(unsigned int);
//	//totalBytes += GI_DENSE_SIZE_CUBE * sizeof(CSVONode);	// Dense Tex
//	//for(unsigned int i = 0; i < GI_DENSE_TEX_COUNT; i++)
//	//{
//	//	size_t texSize = GI_DENSE_SIZE >> i;
//	//	totalBytes += sizeof(CSVOMaterial) * texSize * texSize * texSize;
//	//}
//	//return totalBytes;
//	return 0;
//}
//
//uint32_t GISparseVoxelOctree::MinLevel() const
//{
//	//return hSVOConstants.denseDepth - GI_DENSE_TEX_COUNT + 1;
//	return 0;
//}
//
//uint32_t GISparseVoxelOctree::MaxLevel() const
//{
//	return hSVOConstants.totalDepth;
//}
//
//const CSVOConstants& GISparseVoxelOctree::SVOConsts() const
//{
//	return hSVOConstants;
//}
