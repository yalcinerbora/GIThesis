#include "hip/hip_runtime.h"
#include "SVOKernels.cuh"
#include "GISparseVoxelOctree.h"
#include "GIVoxelPages.h"
#include "CSVOHash.cuh"
#include "CVoxelFunctions.cuh"
#include "CSVOLightInject.cuh"
#include "CSVOIllumAverage.cuh"
#include "CSVONodeAlloc.cuh"
#include <hip/hip_runtime.h>

//
//
//
//	unsigned int location;
//	CSVONode* node = nullptr;
//	for(unsigned int i = octreeParams.DenseLevel; i <= level; i++)
//	{
//		CSVONode* node = nullptr;
//		if(i == svoConstants.denseDepth)
//		{
//			uint3 levelVoxId = CalculateLevelVoxId(voxelPos, i, svoConstants.totalDepth);
//			node = gSVODense +
//				svoConstants.denseDim * svoConstants.denseDim * levelVoxId.z +
//				svoConstants.denseDim * levelVoxId.y +
//				levelVoxId.x;
//		}
//		else
//		{
//			node = gSVOSparse + gLevelOffsets[levelIndex] + location;
//		}
//
//		// Allocate (or acquire) next location
//		location = AtomicAllocateNode(node, gLevelAllocators[levelIndex + 1]);
//		assert(location < gLevelTotalSizes[levelIndex + 1]);
//
//		// Offset child
//		unsigned int childId = CalculateLevelChildId(currentVoxPos, i + 1, svoConstants.totalDepth);
//		location += childId;
//	}
//}
//
////inline __device__ unsigned int FindDenseChildren(const uint3& parentIndex,
////                                                 const unsigned int childId,
////                                                 const unsigned int levelDim)
////{
////    // Go down 1 lvl
////    uint3 childIndex = parentIndex;
////    childIndex.x *= 2;
////    childIndex.y *= 2;
////    childIndex.z *= 2;
////
////    uint3 offsetIndex =
////    {
////        childId % 2,
////        childId / 2,
////        childId / 4
////    };
////    childIndex.x += offsetIndex.x;
////    childIndex.y += offsetIndex.y;
////    childIndex.z += offsetIndex.z;
////
////    unsigned int childLvlDim = levelDim << 1;
////    unsigned int linearChildId = childIndex.z * childLvlDim * childLvlDim +
////        childIndex.y * childLvlDim +
////        childIndex.z;
////    return linearChildId;
////}
////
//
//__global__ void SVOReconstructAverageNode(CSVOMaterial* gSVOMat,
//										  hipSurfaceObject_t sDenseMat,
//
//										  const CSVONode* gSVODense,
//										  const CSVONode* gSVOSparse,
//
//										  const unsigned int* gLevelOffsets,
//										  const unsigned int& gSVOLevelOffset,
//										  const unsigned int& gSVONextLevelOffset,
//
//										  const unsigned int levelNodeCount,
//										  const unsigned int matOffset,
//										  const unsigned int currentLevel,
//										  const CSVOConstants& svoConstants)
//{
//	unsigned int globalId = threadIdx.x + blockIdx.x * blockDim.x;
//	unsigned int nodeId = globalId / 2;
//
//	// Cull if out of range
//	if(nodeId > levelNodeCount) return;
//
//	// Read Sibling Materials
//	const CSVONode* n = (currentLevel == svoConstants.denseDepth) ? gSVODense : gSVOSparse;
//	CSVONode node = n[gSVOLevelOffset + nodeId];
//
//	// Cull if there is no node no need to average
//	if(node == 0xFFFFFFFF) return;
//
//	// Only fetch parent when there a potential to have one
//	bool fetchParentMat = ((svoConstants.totalDepth - currentLevel) < svoConstants.numCascades);
//
//	uint64_t parentMat;
//	if(globalId % 2 == 0) parentMat = fetchParentMat ? gSVOMat[matOffset + gSVOLevelOffset + nodeId].colorPortion : 0x0;
//	else parentMat = fetchParentMat ? gSVOMat[matOffset + gSVOLevelOffset + nodeId].normalPortion : 0x0;
//
//	// Average Portion
//	// Material Data
//	unsigned int count = 0;
//	float4 avgSegment1 = {0.0f, 0.0f, 0.0f, 0.0f};
//	float4 avgSegment2 = {0.0f, 0.0f, 0.0f, 0.0f};
//
//	// Parent Incorporate
//	if(parentMat != 0x0)
//	{
//		if(globalId % 2 == 0)
//		{
//			CSVOColor colorPacked = UnpackSVOMaterialColorOrNormal(parentMat);
//			float4 color = UnpackSVOColor(colorPacked);
//		
//			avgSegment1.x = 8 * color.x;
//			avgSegment1.y = 8 * color.y;
//			avgSegment1.z = 8 * color.z;
//			avgSegment1.w = 8 * color.w;
//		}
//		else
//		{
//			CVoxelNorm normalPacked = UnpackSVOMaterialColorOrNormal(parentMat);
//			float4 normal = UnpackSVONormal(normalPacked);
//
//			avgSegment2.x = 8 * normal.x;
//			avgSegment2.y = 8 * normal.y;
//			avgSegment2.z = 8 * normal.z;
//			avgSegment2.w = 8 * normal.w;
//		}
//		count += 8;
//	}
//
//	// Average
//	if(node != 0xFFFFFFFF)
//	{
//		#pragma unroll
//		for(unsigned int i = 0; i < 8; i++)
//		{
//			unsigned int currentNodeId = node + i;
//			if(globalId % 2 == 0)
//			{
//				uint64_t mat = gSVOMat[matOffset + gSVONextLevelOffset + currentNodeId].colorPortion;
//				if(mat == 0x0) continue;
//
//				CSVOColor colorPacked = UnpackSVOMaterialColorOrNormal(mat);
//				float4 color = UnpackSVOColor(colorPacked);
//
//				avgSegment1.x += color.x;
//				avgSegment1.y += color.y;
//				avgSegment1.z += color.z;
//				avgSegment1.w += color.w;
//			}
//			else
//			{
//				uint64_t mat = gSVOMat[matOffset + gSVONextLevelOffset + currentNodeId].normalPortion;
//				if(mat == 0x0) continue;
//
//				CVoxelNorm normalPacked = UnpackSVOMaterialColorOrNormal(mat);
//				float4 normal = UnpackSVONormal(normalPacked);
//
//				avgSegment2.x += normal.x;
//				avgSegment2.y += normal.y;
//				avgSegment2.z += normal.z;
//				avgSegment2.w += normal.w;
//			}
//			count++;
//		}
//	}
//
//	// Divide by Count
//	if(count == 0) count = 1.0f;
//	float countInv = 1.0f / static_cast<float>(count);
//	avgSegment1.x *= countInv;
//	avgSegment1.y *= countInv;
//	avgSegment1.z *= countInv;
//	avgSegment1.w *= countInv;
//
//	avgSegment2.x *= countInv;
//	avgSegment2.y *= countInv;
//	avgSegment2.z *= countInv;
//	avgSegment2.w *= (count > 8) ? 0.0625f : 0.125f;
//
//	// Pack and Store	
//	uint64_t averageValue;
//	if(globalId % 2 == 0)
//	{
//		CSVOColor colorPacked = PackSVOColor(avgSegment1);		
//		averageValue = PackSVOMaterialPortion(colorPacked, 0x0);
//	}
//	else
//	{		
//		CVoxelNorm normPacked = PackSVONormal(avgSegment2);
//		averageValue = PackSVOMaterialPortion(normPacked, 0x0);
//	}
//	
//    if(currentLevel == svoConstants.denseDepth)
//    {
//        int3 dim =
//        {
//            static_cast<int>(nodeId % svoConstants.denseDim),
//            static_cast<int>((nodeId / svoConstants.denseDim) % svoConstants.denseDim),
//            static_cast<int>(nodeId / (svoConstants.denseDim * svoConstants.denseDim))
//        };
//        uint2 data =
//        {
//            static_cast<unsigned int>(averageValue & 0x00000000FFFFFFFF),
//            static_cast<unsigned int>(averageValue >> 32)
//        };
//		int dimX = (globalId % 2 == 0) ? (dim.x * sizeof(uint4)) : (dim.x * sizeof(uint4) + sizeof(uint2));
//        surf3Dwrite(data, sDenseMat, dimX, dim.y, dim.z);
//    }
//    else
//    {
//		if(globalId % 2 == 0) gSVOMat[matOffset + gSVOLevelOffset + nodeId].colorPortion = averageValue;
//		else gSVOMat[matOffset + gSVOLevelOffset + nodeId].normalPortion = averageValue;
//    }
//}
//
//__global__ void SVOReconstructAverageNode(hipSurfaceObject_t sDenseMatChild,
//                                          hipSurfaceObject_t sDenseMatParent,
//
//                                          const unsigned int parentSize)
//{
//    // Linear Id
//    unsigned int globalId = threadIdx.x + blockIdx.x * blockDim.x;
//    unsigned int parentId = globalId / GI_DENSE_WORKER_PER_PARENT;
//
//    // 3D Id
//    char3 idMap = voxLookup[globalId % GI_DENSE_WORKER_PER_PARENT];
//    uint3 parentId3D =
//    {
//        static_cast<unsigned int>(parentId % parentSize),
//        static_cast<unsigned int>((parentId / parentSize) % parentSize),
//        static_cast<unsigned int>(parentId / (parentSize * parentSize))
//    };
//    uint3 childId3D =
//    {
//        parentId3D.x * 2 + idMap.x,
//        parentId3D.y * 2 + idMap.y,
//        parentId3D.z * 2 + idMap.z
//    };
//
//    // 3D Fetch
//    uint4 data;
//    surf3Dread(&data, sDenseMatChild,
//               childId3D.x * sizeof(uint4),
//               childId3D.y,
//               childId3D.z);
//
//    // Data
//    unsigned int count = (data.x == 0 && 
//						  data.y == 0 && 
//						  data.z == 0 && 
//						  data.w == 0) ? 0 : 1;
//    float4 color = UnpackSVOColor(data.x);
//    float4 normal = UnpackSVONormal(data.w);
//
//    // Average	
//    #pragma unroll
//    for(int offset = GI_DENSE_WORKER_PER_PARENT / 2; offset > 0; offset /= 2)
//    {
//        color.x += __shfl_down(color.x, offset, GI_DENSE_WORKER_PER_PARENT);
//        color.y += __shfl_down(color.y, offset, GI_DENSE_WORKER_PER_PARENT);
//        color.z += __shfl_down(color.z, offset, GI_DENSE_WORKER_PER_PARENT);
//        color.w += __shfl_down(color.w, offset, GI_DENSE_WORKER_PER_PARENT);
//
//        normal.x += __shfl_down(normal.x, offset, GI_DENSE_WORKER_PER_PARENT);
//        normal.y += __shfl_down(normal.y, offset, GI_DENSE_WORKER_PER_PARENT);
//        normal.z += __shfl_down(normal.z, offset, GI_DENSE_WORKER_PER_PARENT);
//        normal.w += __shfl_down(normal.w, offset, GI_DENSE_WORKER_PER_PARENT);
//
//        count += __shfl_down(count, offset, GI_DENSE_WORKER_PER_PARENT);
//    }
//
//    // Division
//    float countInv = 1.0f / ((count != 0) ? float(count) : 1.0f);
//    color.x *= countInv;
//    color.y *= countInv;
//    color.z *= countInv;
//    color.w *= countInv;
//
//    normal.x *= countInv;
//    normal.y *= countInv;
//    normal.z *= countInv;
//	normal.w *= 0.125f;
//
//    data.x = PackSVOColor(color);
//    data.w = PackSVONormal(normal);
//
//    if(globalId % GI_DENSE_WORKER_PER_PARENT == 0 && count != 0)
//    {
//        surf3Dwrite(data, sDenseMatParent,
//                    parentId3D.x * sizeof(uint4),
//                    parentId3D.y,
//                    parentId3D.z);
//    }
//}
//


//inline __device__ unsigned int& Index(uint4& low, 
//									  uint4& high,
//									  const int& index)
//{
//	uint4& loc = (index < 4) ? low : high;
//	return reinterpret_cast<unsigned int*>(&loc)[index % 4];
//}


__global__ void AverageLevel(CSVOLevel& gSVOLevel,
							 const uint32_t nodeCount,
							 const OctreeParameters octreeParams)
{
	//unsigned int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	//if(globalId >= nodeCount) return;


	//#pragma unroll
	//for(int i = 0; i )
}

__global__ void ResetIllumCounter(CSVOLevel& gSVOLevel,
								  const uint32_t nodeCount)
{
	// Two Threads per load
	unsigned int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int illumId = globalId / 2;
	unsigned int wordId = globalId % 2;
	if(illumId >= nodeCount) return;

	// Load only required portion (compiler may load entire 128-bit illum)
	uint32_t* wordPartitionIllum = reinterpret_cast<uint32_t*>(gSVOLevel.gLevelIllum);
	uint32_t* portionWithCounter = wordPartitionIllum + illumId * 4 + wordId * 2 + 1;
	uint32_t portion = *portionWithCounter;
	if(portion != 0x0)
	{
		// Make it as if it has single node
		portion &= 0x01FFFFFF;
		*portionWithCounter = portion;


		//if(liParams.injectOn)
		//{
		//	// Gen Illumination
		//	float4 irradiance;
		//	float3 lightDir = {0.0f, 0.0f, 0.0f};

		//	// World Space Position Reconstruction
		//	const float3 edgePos = gGridInfos[cascadeId].position;
		//	const float span = gGridInfos[cascadeId].span;
		//	const uint3	voxPos = ExpandVoxPos(voxelPosPacked);
		//	float3 weights = ExpandOccupancy(voxOccupPacked);

		//	float3 worldPos;
		//	worldPos.x = edgePos.x + (static_cast<float>(voxPos.x) + weights.x) * span;
		//	worldPos.y = edgePos.y + (static_cast<float>(voxPos.y) + weights.y) * span;
		//	worldPos.z = edgePos.z + (static_cast<float>(voxPos.z) + weights.z) * span;

		//	// Normal
		//	float3 normal = ExpandVoxNormal(voxelNormPacked);

		//	// Generated Irradiance
		//	float3 irradianceDiffuse = LightInject(lightDir,
		//										   // Node Params
		//										   worldPos,
		//										   voxAlbedo,
		//										   normal,
		//										   // Light Parameters
		//										   liParams);

		//	irradiance.x = irradianceDiffuse.x;
		//	irradiance.y = irradianceDiffuse.y;
		//	irradiance.z = irradianceDiffuse.z;
		//	irradiance.w = voxAlbedo.w;

		//	irradPacked = PackSVOIrradiance(irradiance);
		//	lightDirPacked = PackVoxNormal(lightDir);
		//}
		//else
		//{
		//	irradPacked = PackSVOIrradiance(voxAlbedo);
		//}
	}	
}

__global__ void SVOReconstruct(// SVO
							   CSVOLevel* gSVOLevels,
							   uint32_t* gLevelAllocators,
							   const uint32_t* gLevelCapacities,
							   // Voxel Pages
							   const CVoxelPageConst* gVoxelPages,
							   const CVoxelGrid* gGridInfos,
							   // Cache Data (for Voxel Albedo)
							   const BatchVoxelCache* gBatchVoxelCache,
							   // Light Injection Related
							   const CLightInjectParameters liParams,
							   // Limits
							   const OctreeParameters octreeParams,
							   const uint32_t batchCount)
{
	// Shared Memory for generic data
	__shared__ CSegmentInfo sSegInfo;
	__shared__ CMeshVoxelInfo sMeshVoxelInfo;

	// Local Ids
	unsigned int blockLocalId = threadIdx.x;
	//unsigned int nodeLocalId = blockLocalId % 2;

	unsigned int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int pageId = globalId / GIVoxelPages::PageSize;
	unsigned int pageLocalId = globalId % GIVoxelPages::PageSize;
	unsigned int pageLocalSegmentId = pageLocalId / GIVoxelPages::SegmentSize;
	unsigned int segmentLocalVoxId = pageLocalId % GIVoxelPages::SegmentSize;

	// Get Segments Obj Information Struct
	CObjectType objType;
	CSegmentOccupation occupation;
	uint8_t cascadeId;
	bool firstOccurance;
	if(blockLocalId == 0)
	{
		// Load to smem
		// Todo split this into the threadss
		sSegInfo = gVoxelPages[pageId].dSegmentInfo[pageLocalSegmentId];
		ExpandSegmentInfo(cascadeId, objType, occupation, firstOccurance, sSegInfo.packed);
	}
	__syncthreads();
	if(blockLocalId != 0)
	{
		ExpandSegmentInfo(cascadeId, objType, occupation, firstOccurance, sSegInfo.packed);
	}
	// Full Block Cull
	if(occupation == CSegmentOccupation::EMPTY) return;
	assert(occupation != CSegmentOccupation::MARKED_FOR_CLEAR);
	if(blockLocalId == 0)
	{
		sMeshVoxelInfo = gBatchVoxelCache[cascadeId * batchCount + sSegInfo.batchId].dMeshVoxelInfo[sSegInfo.objId];
	}
	__syncthreads();

	// Now we can cull invalid nodes
	const CVoxelNorm voxelNormPacked = gVoxelPages[pageId].dGridVoxNorm[pageLocalId];
	if(voxelNormPacked == 0xFFFFFFFF) return;

	// Unpack Vox Pos
	const CVoxelPos voxelPosPacked = gVoxelPages[pageId].dGridVoxPos[pageLocalId];

	// Unpack Occupancy
	const CVoxelOccupancy voxOccupPacked = gVoxelPages[pageId].dGridVoxOccupancy[pageLocalId];
	
	// Get Albedo (it is dependant)
	// Find your opengl data and voxel cache
	const uint16_t& batchId = sSegInfo.batchId;
	const BatchVoxelCache& batchCache = gBatchVoxelCache[cascadeId * batchCount + batchId];
	// Voxel Ids
	const uint32_t objectLocalVoxelId = sSegInfo.objectSegmentId * GIVoxelPages::SegmentSize + segmentLocalVoxId;
	const uint32_t batchLocalVoxelId = objectLocalVoxelId + sMeshVoxelInfo.voxOffset;
	// Voxel Albedo
	const VoxelAlbedo albedoPacked = batchCache.dVoxelAlbedo[batchLocalVoxelId];

	// Now we will start allocating all nodes
	// Each node will generate multiple neigbouring nodes (8-neigbour filtering)
	// Two threads are responsible for single node, they store the alocated positions
	// Then each thread will average upper or lower half of the node
	#pragma unroll
	for(int a = 0; a < 8; a++)
	{
		// Convert Linear Loop to 3D		
		int i = (a >> 0) & 0x1;
		int j = (a >> 1) & 0x1;
		int k = (a >> 2) & 0x1;

		const uint3	voxPos = ExpandVoxPos(voxelPosPacked);
		
		uint3 myMetaNeigbour;
		myMetaNeigbour.x = static_cast<int>(voxPos.x) + i;
		myMetaNeigbour.y = static_cast<int>(voxPos.y) + j;
		myMetaNeigbour.z = static_cast<int>(voxPos.z) + k;

		// Boundary Check
		bool validNode = (myMetaNeigbour.x < octreeParams.CascadeBaseLevelSize ||
						  myMetaNeigbour.y < octreeParams.CascadeBaseLevelSize ||
						  myMetaNeigbour.z < octreeParams.CascadeBaseLevelSize);

		const uint3 nodePos = ExpandToSVODepth(myMetaNeigbour,
											   cascadeId,
											   octreeParams.CascadeCount,
											   octreeParams.CascadeBaseLevel);

		// Allocate and Average Illumination Values
		if(validNode)
		{
			uint32_t cascadeMaxLevel = octreeParams.MaxSVOLevel - cascadeId;
			CSVOIllumination* illumNode = TraverseAndAllocate(// SVO
															  gLevelAllocators,
															  gLevelCapacities,
															  gSVOLevels,
															  // Node Related
															  nodePos,
															  // Constants
															  octreeParams,
															  cascadeMaxLevel);
			
			// Calculte this nodes occupancy
			float3 weights = ExpandOccupancy(voxOccupPacked);
			float3 volume;		
			volume.x = (i == 1) ? weights.x : (1.0f - weights.x);
			volume.y = (j == 1) ? weights.y : (1.0f - weights.y);
			volume.z = (k == 1) ? weights.z : (1.0f - weights.z);
			float occupancy = volume.x * volume.y * volume.z;

			float4 unpackAlbedo = UnpackSVOIrradiance(albedoPacked);
			AtomicIllumLeafAvg(reinterpret_cast<uint64_t*>(illumNode), unpackAlbedo, occupancy);

			/*unpackAlbedo = UnpackSVONormal(voxelNormPacked);
			AtomicIllumLeafAvg(reinterpret_cast<uint64_t*>(illumNode + 1), unpackAlbedo, occupancy);*/
		}
	}

	//// Now Each Thread will load single node
	//#pragma unroll
	//for(int i = 0; i < 8; i++)
	//{
	//	uint32_t shuffleValidBits = __shfl(validBits, i / 4, 2);
	//	if((shuffleValidBits >> (i % 4)) == 0) continue;

	//	uint32_t cascadeMaxLevel = octreeParams.MaxSVOLevel - cascadeId;
	//	uint32_t nodeShare = nodeLocations[i % 4];
	//	uint32_t nodeOffset = __shfl(nodeShare, i / 4, 2); 
	//	uint64_t* illumNodePartial = reinterpret_cast<uint64_t*>(gSVOLevels[cascadeMaxLevel].gLevelIllum + nodeOffset);

	//	// Calculte this nodes occupancy
	//	float3 weights = ExpandOccupancy(voxOccupPacked);
	//	float3 volume;		
	//	volume.x = (voxLookup8[i].x == 1) ? weights.x : (1.0f - weights.x);
	//	volume.y = (voxLookup8[i].y == 1) ? weights.y : (1.0f - weights.y);
	//	volume.z = (voxLookup8[i].z == 1) ? weights.z : (1.0f - weights.z);
	//	float occupancy = volume.x * volume.y * volume.z;

	//	// Portions
	//	float4 upperPortion = {0.0f};
	//	float3 lowerPortion = {0.0f};

	//	// Determine your data
	//	if(nodeLocalId == 0)
	//	{
	//		upperPortion = UnpackSVOIrradiance(irradPacked);
	//		lowerPortion = ExpandVoxNormal(voxelNormPacked);

	//		upperPortion.x *= occupancy;
	//		upperPortion.y *= occupancy;
	//		upperPortion.z *= occupancy;
	//		upperPortion.w *= occupancy;
	//	}
	//	else
	//	{
	//		// TODO: Anisotropic Occupancy
	//		upperPortion = float4{occupancy, occupancy, occupancy, occupancy};
	//		lowerPortion = ExpandVoxNormal(lightDirPacked);
	//	}
	//	lowerPortion.x *= occupancy;
	//	lowerPortion.y *= occupancy;
	//	lowerPortion.z *= occupancy;

	//	// Portion Average (Code Invariant Average)
	//	AtomicIllumPortionAvg(illumNodePartial + nodeLocalId, upperPortion, lowerPortion);
	//}
}

//
//__global__ void SVOIllumInject(// SVO
//							   CSVOLevel* gSVOLevels,
//							   uint32_t* gLevelAllocators,
//							   const uint32_t* gLevelCapacities,
//							   // Voxel Pages
//							   const CVoxelPageConst* gVoxelPages,
//							   const CVoxelGrid* gGridInfos,
//							   // Cache Data (for Voxel Albedo)
//							   const BatchVoxelCache* gBatchVoxelCache,
//							   // Light Injection Related
//							   const CLightInjectParameters liParams,
//							   // Limits
//							   const OctreeParameters octreeParams,
//							   const uint32_t batchCount)
//{
//	// Shared Memory for generic data
//	__shared__ CSegmentInfo sSegInfo;
//	__shared__ CMeshVoxelInfo sMeshVoxelInfo;
//
//	// Local Ids
//	unsigned int blockLocalId = threadIdx.x;
//	unsigned int nodeLocalId = blockLocalId % 2;
//
//	unsigned int globalId = (threadIdx.x + blockIdx.x * blockDim.x) / 2;
//	unsigned int pageId = globalId / GIVoxelPages::PageSize;
//	unsigned int pageLocalId = globalId % GIVoxelPages::PageSize;
//	unsigned int pageLocalSegmentId = pageLocalId / GIVoxelPages::SegmentSize;
//	unsigned int segmentLocalVoxId = pageLocalId % GIVoxelPages::SegmentSize;
//
//	// Get Segments Obj Information Struct
//	CObjectType objType;
//	CSegmentOccupation occupation;
//	uint8_t cascadeId;
//	bool firstOccurance;
//	if(blockLocalId == 0)
//	{
//		// Load to smem
//		// Todo split this into the threadss
//		sSegInfo = gVoxelPages[pageId].dSegmentInfo[pageLocalSegmentId];
//		ExpandSegmentInfo(cascadeId, objType, occupation, firstOccurance, sSegInfo.packed);
//	}
//	__syncthreads();
//	if(blockLocalId != 0)
//	{
//		ExpandSegmentInfo(cascadeId, objType, occupation, firstOccurance, sSegInfo.packed);
//	}
//	// Full Block Cull
//	if(occupation == CSegmentOccupation::EMPTY) return;
//	assert(occupation != CSegmentOccupation::MARKED_FOR_CLEAR);
//	if(blockLocalId == 0)
//	{
//		sMeshVoxelInfo = gBatchVoxelCache[cascadeId * batchCount + sSegInfo.batchId].dMeshVoxelInfo[sSegInfo.objId];
//	}
//	__syncthreads();
//
//	// Fetch Position and Normal
//	// Generate Light Direction and Irradiance
//	const CVoxelPos voxelPosPacked = gVoxelPages[pageId].dGridVoxPos[pageLocalId];
//	const CVoxelNorm voxelNormPacked = gVoxelPages[pageId].dGridVoxNorm[pageLocalId];
//
//	// Unpack Occupancy
//	const CVoxelOccupancy voxOccupPacked = gVoxelPages[pageId].dGridVoxOccupancy[pageLocalId];
//	float3 weights = ExpandOccupancy(voxOccupPacked);
//
//	// Now we can cull invalid nodes
//	if(voxelNormPacked == 0xFFFFFFFF) return;
//
//	// Illum Data Packed
//	VoxelAlbedo irradPacked = 0;
//	VoxelNormal lightDirPacked = 0;
//
//	//// Light Injection
//	//if(nodeLocalId == 0)
//	//{
//	//	// Now load albeo etc and average those on leaf levels
//	//	// Find your opengl data and voxel cache
//	//	const uint16_t& batchId = sSegInfo.batchId;
//	//	const BatchVoxelCache& batchCache = gBatchVoxelCache[cascadeId * batchCount + batchId];
//
//	//	// Voxel Ids
//	//	const uint32_t objectLocalVoxelId = sSegInfo.objectSegmentId * GIVoxelPages::SegmentSize + segmentLocalVoxId;
//	//	const uint32_t batchLocalVoxelId = objectLocalVoxelId + sMeshVoxelInfo.voxOffset;
//
//	//	// Voxel Albedo
//	//	const CVoxelAlbedo voxAlbedoPacked = batchCache.dVoxelAlbedo[batchLocalVoxelId];
//	//	float4 voxAlbedo = UnpackSVOIrradiance(*reinterpret_cast<const CSVOIrradiance*>(&voxAlbedoPacked));
//
//	//	if(liParams.injectOn)
//	//	{
//	//		// Gen Illumination
//	//		float4 irradiance;
//	//		float3 lightDir = {0.0f, 0.0f, 0.0f};
//
//	//		// World Space Position Reconstruction
//	//		const float3 edgePos = gGridInfos[cascadeId].position;
//	//		const float span = gGridInfos[cascadeId].span;
//	//		const uint3	voxPos = ExpandVoxPos(voxelPosPacked);
//
//	//		float3 worldPos;
//	//		worldPos.x = edgePos.x + (static_cast<float>(voxPos.x) + weights.x) * span;
//	//		worldPos.y = edgePos.y + (static_cast<float>(voxPos.y) + weights.y) * span;
//	//		worldPos.z = edgePos.z + (static_cast<float>(voxPos.z) + weights.z) * span;
//
//	//		// Normal
//	//		float3 normal = ExpandVoxNormal(voxelNormPacked);
//
//	//		// Generated Irradiance
//	//		float3 irradianceDiffuse = LightInject(lightDir,
//	//											   // Node Params
//	//											   worldPos,
//	//											   voxAlbedo,
//	//											   normal,
//	//											   // Light Parameters
//	//											   liParams);
//
//	//		irradiance.x = irradianceDiffuse.x;
//	//		irradiance.y = irradianceDiffuse.y;
//	//		irradiance.z = irradianceDiffuse.z;
//	//		irradiance.w = voxAlbedo.w;
//
//	//		irradPacked = PackSVOIrradiance(irradiance);
//	//		lightDirPacked = PackVoxNormal(lightDir);
//	//	}
//	//	else
//	//	{
//	//		irradPacked = PackSVOIrradiance(voxAlbedo);
//	//	}
//	//}
//	// Transfer illum data to neigbour
//	//unsigned int warpLocalNodeId = globalId % (warpSize >> 1);
//	irradPacked = __shfl(irradPacked, 0, 2);
//	lightDirPacked = __shfl(lightDirPacked, 0, 2);
//
//	// Now we will start allocating all nodes
//	// Each node will generate multiple neigbouring nodes (8-neigbour filtering)
//	// Two threads are responsible for single node, they store the alocated positions
//	// Then each thread will average upper or lower half of the node
//	uint4 nodeLocations; uint8_t validBits = 0x00;
//	#pragma unroll
//	for(int j = 0; j < 4; j++)
//	{
//		const uint3	voxPos = ExpandVoxPos(voxelPosPacked);
//
//		int3 myMetaNeigbour;
//		myMetaNeigbour.x = static_cast<int>(voxPos.x) + voxLookup8[nodeLocalId * 4 + j].x;
//		myMetaNeigbour.y = static_cast<int>(voxPos.y) + voxLookup8[nodeLocalId * 4 + j].y;
//		myMetaNeigbour.z = static_cast<int>(voxPos.z) + voxLookup8[nodeLocalId * 4 + j].z;
//
//		// Boundary Check
//		bool validNode = (myMetaNeigbour.x >= 0 || myMetaNeigbour.x < octreeParams.CascadeBaseLevelSize ||
//						  myMetaNeigbour.y >= 0 || myMetaNeigbour.y < octreeParams.CascadeBaseLevelSize ||
//						  myMetaNeigbour.z >= 0 || myMetaNeigbour.z < octreeParams.CascadeBaseLevelSize);
//		validBits |= ((validNode) ? 1 : 0) << j;
//
//		uint3 uMyMetaNeigbour;
//		uMyMetaNeigbour.x = static_cast<unsigned int>(myMetaNeigbour.x);
//		uMyMetaNeigbour.y = static_cast<unsigned int>(myMetaNeigbour.y);
//		uMyMetaNeigbour.z = static_cast<unsigned int>(myMetaNeigbour.z);
//		const uint3 nodePos = ExpandToSVODepth(uMyMetaNeigbour,
//											   cascadeId,
//											   octreeParams.CascadeCount,
//											   octreeParams.CascadeBaseLevel);
//
//		// Allocate and Average Illumination Values
//		if(validNode)
//		{
//			uint32_t cascadeMaxLevel = octreeParams.MaxSVOLevel - cascadeId;
//			uint32_t test;
//			const CSVONode* illumNode = TraverseNode(test,
//													  // SVO
//													  reinterpret_cast<const CSVOLevelConst*>(gSVOLevels),
//													  // Node Related
//													  nodePos,
//													  // Constants
//													  octreeParams,
//													  cascadeMaxLevel);
//
//			uint32_t nodeOffset = illumNode - gSVOLevels[cascadeMaxLevel].gLevelNodes;
//			reinterpret_cast<unsigned int*>(&nodeLocations)[j] = nodeOffset;
//		}
//	}
//
//	// Now Each Thread will load single node
//	#pragma unroll
//	for(int i = 0; i < 8; i++)
//	{
//		uint32_t shuffleValidBits = __shfl(validBits, (i < 4) ? 0 : 1, 2);
//		if((shuffleValidBits >> (i % 4)) == 0) continue;
//
//		uint32_t cascadeMaxLevel = octreeParams.MaxSVOLevel - cascadeId;
//		uint32_t nodeShare = reinterpret_cast<unsigned int*>(&nodeLocations)[i % 4];
//		uint32_t nodeOffset = __shfl(nodeShare, (i < 4) ? 0 : 1, 2); 
//		uint64_t* illumNodePartial = reinterpret_cast<uint64_t*>(gSVOLevels[cascadeMaxLevel].gLevelIllum + nodeOffset);
//
//		float4 upperPortion = {0.0f};
//		float3 lowerPortion = {0.0f};
//
//		// Calculte this nodes occupancy
//		float3 volume;
//		volume.x = (voxLookup8[i].x == 1) ? weights.x : (1.0f - weights.x);
//		volume.y = (voxLookup8[i].y == 1) ? weights.y : (1.0f - weights.y);
//		volume.z = (voxLookup8[i].z == 1) ? weights.z : (1.0f - weights.z);
//		float occupancy = 0; volume.x * volume.y * volume.z;
//
//		// Determine your data
//		if(nodeLocalId == 0)
//		{
//			upperPortion = UnpackSVOIrradiance(irradPacked);
//			lowerPortion = ExpandVoxNormal(voxelNormPacked);
//
//			upperPortion.x *= occupancy;
//			upperPortion.y *= occupancy;
//			upperPortion.z *= occupancy;
//			upperPortion.w *= occupancy;
//		}
//		else
//		{
//			// TODO: Anisotropic Occupancy
//			upperPortion = float4{occupancy, occupancy, occupancy, occupancy};
//			lowerPortion = ExpandVoxNormal(lightDirPacked);
//		}
//		lowerPortion.x *= occupancy;
//		lowerPortion.y *= occupancy;
//		lowerPortion.z *= occupancy;
//
//		// Portion Average (Code Invariant Average)
//		AtomicIllumPortionAvg(illumNodePartial + nodeLocalId, upperPortion, lowerPortion);
//	}
//}