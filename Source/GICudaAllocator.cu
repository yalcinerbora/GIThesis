#include "GICudaAllocator.h"
#include "GICudaStructMatching.h"
#include <cuda_gl_interop.h>

GICudaAllocator::GICudaAllocator()
	: totalObjectCount(0)
{
	cudaGLSetGLDevice(0);
}

void GICudaAllocator::LinkOGLVoxelCache(GLuint batchAABBBuffer,
										GLuint batchTransformBuffer,
										GLuint relativeTransformBuffer,
										GLuint infoBuffer,
										GLuint voxelBuffer,
										GLuint voxelRenderBuffer,
										size_t objCount)
{
	rTransformLinks.emplace_back(nullptr);
	transformLinks.emplace_back(nullptr);
	aabbLinks.emplace_back(nullptr);
	objectInfoLinks.emplace_back(nullptr);
	cacheLinks.emplace_back(nullptr);
	cacheRenderLinks.emplace_back(nullptr);

	hipGraphicsGLRegisterBuffer(&rTransformLinks.back(), relativeTransformBuffer, cudaGraphicsMapFlagsReadOnly);
	hipGraphicsGLRegisterBuffer(&transformLinks.back(), batchTransformBuffer, cudaGraphicsMapFlagsReadOnly);
	hipGraphicsGLRegisterBuffer(&aabbLinks.back(), batchAABBBuffer, cudaGraphicsMapFlagsReadOnly);
	hipGraphicsGLRegisterBuffer(&objectInfoLinks.back(), infoBuffer, cudaGraphicsMapFlagsReadOnly);

	hipGraphicsGLRegisterBuffer(&cacheLinks.back(), voxelBuffer, cudaGraphicsMapFlagsReadOnly);
	hipGraphicsGLRegisterBuffer(&cacheRenderLinks.back(), voxelRenderBuffer, cudaGraphicsMapFlagsReadOnly);

	objectCounts.emplace_back(objCount);
	totalObjectCount += objCount;
}

void GICudaAllocator::LinkSceneShadowMapArray(const std::vector<GLuint>& shadowMaps)
{
	hipGraphicsResource* resource = nullptr;
	for(unsigned int i = 0; i < shadowMaps.size(); i++)
	{
		hipGraphicsGLRegisterImage(&resource,
									shadowMaps[i],
									GL_TEXTURE_CUBE_MAP,
									hipGraphicsRegisterFlagsReadOnly);
		sceneShadowMapLinks.push_back(resource);
	}
}

void GICudaAllocator::LinkSceneGBuffers(GLuint depthTex,
										GLuint normalTex,
										GLuint lightIntensityTex)
{
	hipGraphicsGLRegisterImage(&depthBuffLink,
								depthTex,
								GL_TEXTURE_2D,
								hipGraphicsRegisterFlagsReadOnly);
	hipGraphicsGLRegisterImage(&normalBuffLink,
								normalTex,
								GL_TEXTURE_2D,
								hipGraphicsRegisterFlagsReadOnly);
	hipGraphicsGLRegisterImage(&lightIntensityLink,
								lightIntensityTex,
								GL_TEXTURE_2D,
								hipGraphicsRegisterFlagsSurfaceLoadStore);
}

void GICudaAllocator::SetupPointersDevicePointers()
{
	hipGraphicsMapResources(static_cast<int>(rTransformLinks.size()), rTransformLinks.data());
	hipGraphicsMapResources(static_cast<int>(transformLinks.size()), transformLinks.data());
	hipGraphicsMapResources(static_cast<int>(aabbLinks.size()), aabbLinks.data());
	hipGraphicsMapResources(static_cast<int>(objectInfoLinks.size()), objectInfoLinks.data());

	hipGraphicsMapResources(static_cast<int>(cacheLinks.size()), cacheLinks.data());
	hipGraphicsMapResources(static_cast<int>(cacheRenderLinks.size()), cacheRenderLinks.data());

	thrust::host_vector<CObjectTransform*> hRelativeTransforms;
	thrust::host_vector<CObjectTransform*> hTransforms;
	thrust::host_vector<CObjectAABB*> hObjectAABB;
	thrust::host_vector<CObjectVoxelInfo*> hObjectInfo;

	thrust::host_vector<CVoxelPacked*> hObjCache;
	thrust::host_vector<CVoxelRender*> hObjRenderCache;

	size_t size = 0;
	for(unsigned int i = 0; i < objectCounts.size(); i++)
	{
		hRelativeTransforms.push_back(nullptr);
		hTransforms.push_back(nullptr);
		hObjectAABB.push_back(nullptr);
		hObjectInfo.push_back(nullptr);

		hObjCache.push_back(nullptr);
		hObjRenderCache.push_back(nullptr);

		hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&hRelativeTransforms.back()), &size, rTransformLinks[i]);
		hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&hTransforms.back()), &size, transformLinks[i]);
		hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&hObjectAABB.back()), &size, aabbLinks[i]);
		hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&hObjectInfo.back()), &size, objectInfoLinks[i]);

		hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&hObjCache.back()), &size, cacheLinks[i]);
		hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&hObjRenderCache.back()), &size, cacheRenderLinks[i]);
	}

	//// Data Sent to GPU
	dRelativeTransforms = hRelativeTransforms;
	dTransforms = hTransforms;
	dObjectAABB = hObjectAABB;
	dObjectInfo = hObjectInfo;

	dObjCache = hObjCache;
	dObjRenderCache = hObjRenderCache;


	// Textures
	hipArray* texArray = nullptr;
	hipResourceDesc resDesc = {};
	hipTextureDesc texDesc = {};

	resDesc.res.array.array = texArray;
	resDesc.resType = hipResourceTypeArray;

	texDesc.addressMode[0] = hipAddressModeWrap;
	texDesc.addressMode[1] = hipAddressModeWrap;
	texDesc.filterMode = hipFilterModeLinear;
	texDesc.readMode = hipReadModeNormalizedFloat;
	texDesc.normalizedCoords = 0;

	hipGraphicsMapResources(static_cast<int>(sceneShadowMapLinks.size()), sceneShadowMapLinks.data());
	for(unsigned int i = 0; i < sceneShadowMapLinks.size(); i++)
	{
		hipGraphicsSubResourceGetMappedArray(&texArray, sceneShadowMapLinks[i], 0, 0);

		shadowMaps.emplace_back();
		hipCreateTextureObject(&shadowMaps.back(), &resDesc, &texDesc, nullptr);
	}

	hipGraphicsMapResources(1, &depthBuffLink);
	hipGraphicsSubResourceGetMappedArray(&texArray, depthBuffLink, 0, 0);
	texDesc.readMode = hipReadModeElementType;
	hipCreateTextureObject(&depthBuffer, &resDesc, &texDesc, nullptr);

	hipGraphicsMapResources(1, &normalBuffLink);
	hipGraphicsSubResourceGetMappedArray(&texArray, normalBuffLink, 0, 0);
	texDesc.readMode = hipReadModeElementType;
	hipCreateTextureObject(&normalBuffer, &resDesc, &texDesc, nullptr);

	hipGraphicsMapResources(1, &lightIntensityLink);
	hipGraphicsSubResourceGetMappedArray(&texArray, lightIntensityLink, 0, 0);
	hipCreateSurfaceObject(&lightIntensityBuffer, &resDesc);

}

void GICudaAllocator::ClearDevicePointers()
{
	dRelativeTransforms.clear();
	dTransforms.clear();
	dObjectAABB.clear();
	dObjectInfo.clear();

	dObjCache.clear();
	dObjRenderCache.clear();

	hipDestroySurfaceObject(lightIntensityBuffer);
	hipDestroyTextureObject(normalBuffer);
	hipDestroyTextureObject(depthBuffer);

	for(unsigned int i = 0; i < shadowMaps.size(); i++)
	{
		hipDestroyTextureObject(shadowMaps[i]);
	}
	shadowMaps.clear();

	hipGraphicsUnmapResources(1, &depthBuffLink);
	hipGraphicsUnmapResources(1, &normalBuffLink);
	hipGraphicsUnmapResources(1, &lightIntensityLink);
	hipGraphicsUnmapResources(static_cast<int>(sceneShadowMapLinks.size()), sceneShadowMapLinks.data());

	hipGraphicsUnmapResources(static_cast<int>(rTransformLinks.size()), rTransformLinks.data());
	hipGraphicsUnmapResources(static_cast<int>(transformLinks.size()), transformLinks.data());
	hipGraphicsUnmapResources(static_cast<int>(aabbLinks.size()), aabbLinks.data());
	hipGraphicsUnmapResources(static_cast<int>(objectInfoLinks.size()), objectInfoLinks.data());

	hipGraphicsUnmapResources(static_cast<int>(cacheLinks.size()), cacheLinks.data());
	hipGraphicsUnmapResources(static_cast<int>(cacheRenderLinks.size()), cacheRenderLinks.data());
}

void GICudaAllocator::AddVoxelPage(size_t count)
{
	for(unsigned int i = 0; i < count; i++)
	{
		// Allocating Page
		hPageData.emplace_back(CVoxelPageData
		{
			thrust::device_vector<CVoxelPacked>(GI_PAGE_SIZE),
			thrust::device_vector<unsigned int>(GI_BLOCK_PER_PAGE)
		});

		CVoxelPage voxData =
		{
			thrust::raw_pointer_cast(hPageData.back().dVoxelPage.data()),
			thrust::raw_pointer_cast(hPageData.back().dEmptySegmentList.data()),
			0
		};
		hVoxelPages.push_back(voxData);
	}
	dVoxelPages = hVoxelPages;
}

void GICudaAllocator::ResetSceneData()
{
	for(unsigned int i = 0; i < rTransformLinks.size(); i++)
	{
		hipGraphicsUnregisterResource(rTransformLinks[i]);
		hipGraphicsUnregisterResource(transformLinks[i]);
		hipGraphicsUnregisterResource(aabbLinks[i]);
		hipGraphicsUnregisterResource(objectInfoLinks[i]);

		hipGraphicsUnregisterResource(cacheLinks[i]);
		hipGraphicsUnregisterResource(cacheRenderLinks[i]);
	}

	for(unsigned int i = 0; i < sceneShadowMapLinks.size(); i++)
	{
		hipGraphicsUnregisterResource(sceneShadowMapLinks[i]);
	}
	hipGraphicsUnregisterResource(depthBuffLink);
	hipGraphicsUnregisterResource(normalBuffLink);
	hipGraphicsUnregisterResource(lightIntensityLink);

	rTransformLinks.clear();
	transformLinks.clear();
	aabbLinks.clear();
	objectInfoLinks.clear();

	cacheLinks.clear();
	cacheRenderLinks.clear();

	sceneShadowMapLinks.clear();
}

//const CObjectTransform** GICudaAllocator::GetRelativeTransformsDevice() 
//{
//	return thrust::raw_pointer_cast(dRelativeTransforms.data());
//}
//
//const CObjectTransform** GICudaAllocator::GetTransformsDevice()
//{
//	return thrust::raw_pointer_cast(dTransforms.data());
//}
//
//const CObjectAABB** GICudaAllocator::GetObjectAABBDevice()
//{
//	return thrust::raw_pointer_cast(dObjectAABB.data());
//}
//
//const CObjectVoxelInfo** GICudaAllocator::GetObjectInfoDevice()
//{
//	return thrust::raw_pointer_cast(dObjectInfo.data());
//}
//
//const CVoxelPacked** GICudaAllocator::GetObjCacheDevice()
//{
//	return thrust::raw_pointer_cast(dObjCache.data());
//}
//
//const CVoxelRender** GICudaAllocator::GetObjRenderCacheDevice()
//{
//	return thrust::raw_pointer_cast(dObjRenderCache.data());
//}

CVoxelPage* GICudaAllocator::GetVoxelPagesDevice()
{
	return thrust::raw_pointer_cast(dVoxelPages.data());
}
