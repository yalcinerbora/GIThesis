#include "hip/hip_runtime.h"
#include "GICudaVoxelScene.h"
#include "GIKernels.cuh"
#include "IEUtility/IEMath.h"
#include "CudaTimer.h"
#include "Macros.h"
#include "VoxelCopyToVAO.cuh"
#include "IEUtility/IEVector3.h"
#include "CDebug.cuh"
#include <cuda_gl_interop.h>

GICudaVoxelScene::GICudaVoxelScene(const IEVector3& intialCenterPos, float span, unsigned int dim)
	: allocator(CVoxelGrid 
				{
					{ 
						intialCenterPos.getX() - (dim * span * 0.5f),
						intialCenterPos.getY() - (dim * span * 0.5f),
						intialCenterPos.getZ() - (dim * span * 0.5f)
					},
					span, 
					{ dim, dim, dim }, 
					static_cast<unsigned int>(log2f(static_cast<float>(dim)))
				})
	, vaoNormPosData(512)
	, vaoColorData(512)
{}

GICudaVoxelScene::~GICudaVoxelScene()
{
	if(vaoNormPosResource) CUDA_CHECK(hipGraphicsUnregisterResource(vaoNormPosResource));
	if (vaoRenderResource) CUDA_CHECK(hipGraphicsUnregisterResource(vaoRenderResource));
}

void GICudaVoxelScene::InitCuda()
{
	// Setting Device
	hipSetDevice(0);

	// Cuda Check
	hipDeviceProp_t props;
	CUDA_CHECK(hipGetDeviceProperties(&props, 0));

	// Info Print
	GI_LOG("Cuda Information...");
	GI_LOG("GPU Name\t\t: %s", props.name);
	GI_LOG("GPU Compute Capability\t: %d%d", props.major, props.minor);
	GI_LOG("GPU Shared Memory(SM)\t: %dKB", props.sharedMemPerMultiprocessor / 1024);
	GI_LOG("GPU Shared Memory(Block): %dKB", props.sharedMemPerBlock / 1024);
	GI_LOG("");

	// Minimum Required Compute Capability
	if(props.major < 3)
	{
		GI_LOG("#######################################################################");
		GI_LOG("UNSUPPORTED GPU, CUDA PORTION WILL NOT WORK. NEEDS ATLEAST SM_30 DEVICE");
		GI_LOG("#######################################################################");
		GI_LOG("");
	}

	// Shared Memory Prep
	// 16 Kb memory is enough for our needs most of the time
	CUDA_CHECK(hipDeviceSetCacheConfig(hipFuncCachePreferL1));

	// Voxel Transform Function needs 48kb memory
	CUDA_CHECK(hipFuncSetCacheConfig(reinterpret_cast<const void*>(VoxelTransform), hipFuncCachePreferShared));

}

void GICudaVoxelScene::LinkOGL(GLuint aabbBuffer,
							   GLuint transformBufferID,
							   GLuint infoBufferID,
							   GLuint voxelCacheNormPos,
							   GLuint voxelCacheIds,
							   GLuint voxelCacheRender,
							   uint32_t objCount,
							   uint32_t voxelCount)
{
	allocator.LinkOGLVoxelCache(aabbBuffer, transformBufferID, infoBufferID, 
								voxelCacheNormPos, voxelCacheIds, voxelCacheRender, 
								objCount, voxelCount);
}

void GICudaVoxelScene::AllocateWRTLinkedData(float coverageRatio)
{
	// Hint Device that we will use already linked resources
	allocator.ReserveForSegments(coverageRatio);
	vaoNormPosData.Resize(allocator.NumPages() * GI_PAGE_SIZE);
	vaoColorData.Resize(allocator.NumPages() * GI_PAGE_SIZE);

	// Cuda Register	
	if(vaoNormPosResource) CUDA_CHECK(hipGraphicsUnregisterResource(vaoNormPosResource));
	if(vaoRenderResource) CUDA_CHECK(hipGraphicsUnregisterResource(vaoRenderResource));
	CUDA_CHECK(hipGraphicsGLRegisterBuffer(&vaoNormPosResource, vaoNormPosData.getGLBuffer(), cudaGraphicsMapFlagsWriteDiscard));
	CUDA_CHECK(hipGraphicsGLRegisterBuffer(&vaoRenderResource, vaoColorData.getGLBuffer(), cudaGraphicsMapFlagsWriteDiscard));
}

void GICudaVoxelScene::Reset()
{
	allocator.ResetSceneData();
}

void GICudaVoxelScene::VoxelUpdate(double& ioTiming,
								   double& updateTiming,
								   double& svoReconTiming,
								   const IEVector3& playerPos)
{
	// Pass if there is not any linked objects
	if(allocator.NumSegments() == 0) return;

	// Main Call Chain Called Every Frame
	// Manages Voxel Pages
	allocator.SetupDevicePointers();
	
	CudaTimer timer(0);
	timer.Start();
	for(unsigned int i = 0; i < allocator.NumObjectBatches(); i++)
	{
		// Call Logic Per Obj Segment
		unsigned int gridSize = (allocator.NumObjectSegments(i) + GI_THREAD_PER_BLOCK_SMALL - 1) /
									GI_THREAD_PER_BLOCK_SMALL;

		// KC ALLOCATE
		VoxelObjectAlloc<<<gridSize, GI_THREAD_PER_BLOCK_SMALL>>>
			(// Voxel System
			 allocator.GetVoxelPagesDevice(),
			 allocator.NumPages(),
			 *allocator.GetVoxelGridDevice(),
			 
			 // Per Object Segment Related
			 allocator.GetSegmentAllocLoc(i),
			 allocator.GetSegmentObjectID(i),
			 allocator.NumObjectSegments(i),
			 
			 // Per Object Related
			 allocator.GetWriteSignals(i),
			 allocator.GetObjectAABBDevice(i),
			 allocator.GetTransformsDevice(i));
		CUDA_KERNEL_CHECK();

		// Call Logic Per Voxel
		gridSize = (allocator.NumVoxels(i) + GI_THREAD_PER_BLOCK - 1) /
					GI_THREAD_PER_BLOCK;
		
		// KC OBJECT VOXEL INCLUDE
		VoxelObjectInclude<<<gridSize, GI_THREAD_PER_BLOCK>>>
			(// Voxel System
			 allocator.GetVoxelPagesDevice(),
			 *allocator.GetVoxelGridDevice(),
			 
			 // Per Object Segment Related
			 allocator.GetSegmentAllocLoc(i),
			 allocator.NumObjectSegments(i),
			 
			 // Per Object Related
			 allocator.GetWriteSignals(i),
			 allocator.GetVoxelStrides(i),
			 allocator.GetObjectAllocationIndexLookup(i),
			 allocator.GetObjectAABBDevice(i),
			 allocator.GetTransformsDevice(i),
			 allocator.GetObjectInfoDevice(i),
			 
			 // Per Voxel Related
			 allocator.GetObjCacheIdsDevice(i),
			 allocator.NumVoxels(i),

			 // Batch(ObjectGroup in terms of OGL) Id
			 i);
		CUDA_KERNEL_CHECK();

		// Clear Write Signals
		CUDA_CHECK(hipMemset(allocator.GetWriteSignals(i), 0, sizeof(char) * allocator.NumObjects(i)));
	}

	for(unsigned int i = 0; i < allocator.NumObjectBatches(); i++)
	{
		//timerSub.Start();

		// Call Logic Per Obj Segment
		unsigned int gridSize = (allocator.NumObjectSegments(i) + GI_THREAD_PER_BLOCK_SMALL - 1) /
			GI_THREAD_PER_BLOCK_SMALL;

		// KC DEALLOCATE
		VoxelObjectDealloc<<<gridSize, GI_THREAD_PER_BLOCK_SMALL>>>
			(// Voxel System
			allocator.GetVoxelPagesDevice(),
			*allocator.GetVoxelGridDevice(),

			// Per Object Segment Related
			allocator.GetSegmentAllocLoc(i),
			allocator.GetSegmentObjectID(i),
			allocator.NumObjectSegments(i),

			// Per Object Related
			allocator.GetWriteSignals(i),
			allocator.GetObjectAABBDevice(i),
			allocator.GetTransformsDevice(i));
		CUDA_KERNEL_CHECK();
	}

	// Call Logic Per Voxel in Page
	unsigned int gridSize = (allocator.NumPages() * GI_PAGE_SIZE + GI_THREAD_PER_BLOCK - 1) /
							GI_THREAD_PER_BLOCK;

	// KC CLEAR MARKED
	VoxelClearMarked<<<gridSize, GI_THREAD_PER_BLOCK>>>(allocator.GetVoxelPagesDevice());
	CUDA_KERNEL_CHECK();

	// Call Logic Per Segment in Page
	gridSize = (allocator.NumPages() * GI_SEGMENT_PER_PAGE + GI_THREAD_PER_BLOCK - 1) /
				GI_THREAD_PER_BLOCK;

	// KC CLEAR SIGNAL
	VoxelClearSignal<<<gridSize, GI_THREAD_PER_BLOCK>>>(allocator.GetVoxelPagesDevice(),
														allocator.NumPages());
	CUDA_KERNEL_CHECK();

	//-----------------------------------------------
	//DEBUG
	// ONLY WORKS IF THERE IS SINGLE SEGMENT IN THE SYSTEM
	// Call Logic Per Obj Segment
	unsigned int gridSize2 = (allocator.NumObjectSegments(0) + GI_THREAD_PER_BLOCK - 1) /
		GI_THREAD_PER_BLOCK;
	// KC DEBUG CHECK UNIQUE ALLOC
	DebugCheckUniqueAlloc<<<gridSize2, GI_THREAD_PER_BLOCK>>>(allocator.GetSegmentAllocLoc(0),
															  allocator.NumObjectSegments(0));
	CUDA_KERNEL_CHECK();
	// KC DEBUG CHECK UNIQUE SEGMENT ALLOC
	DebugCheckSegmentAlloc<<<gridSize2, GI_THREAD_PER_BLOCK>>>
		(*allocator.GetVoxelGridDevice(),
		allocator.GetSegmentAllocLoc(0),
		allocator.GetSegmentObjectID(0),
		allocator.NumObjectSegments(0),
		allocator.GetObjectAABBDevice(0),
		allocator.GetTransformsDevice(0));
	CUDA_KERNEL_CHECK();
	//DEBUG END
	//-----------------------------------------------

	timer.Stop();
	ioTiming = timer.ElapsedMilliS();

	// Now Call Update
	timer.Start();
	IEVector3 gridNewPos = allocator.GetNewVoxelPos(playerPos);

	gridSize = (allocator.NumPages() * GI_PAGE_SIZE + GI_THREAD_PER_BLOCK - 1) /
				GI_THREAD_PER_BLOCK;
	VoxelTransform<<<gridSize, GI_THREAD_PER_BLOCK>>>
	  (// Voxel Pages
	   allocator.GetVoxelPagesDevice(),
	   *allocator.GetVoxelGridDevice(),
	   float3{gridNewPos.getX(), gridNewPos.getY(), gridNewPos.getZ()},
	   
	   // Per Object Segment
	   allocator.GetSegmentAllocLoc2D(),				   

	   // Object Related
	   allocator.GetObjectAllocationIndexLookup2D(),
	   allocator.GetTransformsDevice(),
	   allocator.GetObjCacheNormPosDevice(),
	   allocator.GetObjRenderCacheDevice(),
	   allocator.GetObjectInfoDevice(),
	   allocator.GetObjectAABBDevice());
	CUDA_KERNEL_CHECK();

	allocator.SendNewVoxPosToDevice();
	
	timer.Stop();
	updateTiming = timer.ElapsedMilliS();

	// Then Call SVO Reconstruct
	timer.Start();
	timer.Stop();
	svoReconTiming = timer.ElapsedMilliS();

	// Done
	allocator.ClearDevicePointers();
}

uint64_t GICudaVoxelScene::AllocatorMemoryUsage() const
{
	return allocator.SystemTotalMemoryUsage();
}

uint32_t GICudaVoxelScene::VoxelCountInPage()
{
	// Pass if there is not any linked objects
	if(allocator.NumSegments() == 0) return 0;

	int h_VoxCount;
	int* d_VoxCount = nullptr;

	CUDA_CHECK(hipMalloc(&d_VoxCount, sizeof(int)));
	CUDA_CHECK(hipMemset(d_VoxCount, 0, sizeof(int)));

	uint32_t gridSize = ((allocator.NumPages() * GI_PAGE_SIZE) + GI_THREAD_PER_BLOCK - 1) / GI_THREAD_PER_BLOCK;

	// KC VOXEL COUNT DETERMINE FROM VOXELS
	DetermineTotalVoxCount<<<gridSize, GI_THREAD_PER_BLOCK>>>
		(*d_VoxCount,
		 // Page Related
		 allocator.GetVoxelPagesDevice(),
		 *allocator.GetVoxelGridDevice(),
		 allocator.NumPages());
	CUDA_KERNEL_CHECK();

	CUDA_CHECK(hipMemcpy(&h_VoxCount, d_VoxCount, sizeof(int), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipFree(d_VoxCount));
	return static_cast<uint32_t>(h_VoxCount);
}


VoxelDebugVAO GICudaVoxelScene::VoxelDataForRendering(CVoxelGrid& voxGridData, double& time, uint32_t voxCount)
{
	// Pass if there is not any linked objects
	if(allocator.NumSegments() > 0)
	{
		CudaTimer timer(0);
		timer.Start();

		// Map
		unsigned int* d_atomicCounter = nullptr;
		CVoxelNormPos* vBufferNormPosPtr = nullptr;
		uchar4* vBufferRenderPackedPtr = nullptr;
		size_t size = 0;

		allocator.SetupDevicePointers();

		unsigned int zero = 0;
		glBindBuffer(GL_COPY_WRITE_BUFFER, vaoNormPosData.getGLBuffer());
		glClearBufferData(GL_COPY_WRITE_BUFFER, GL_RG32UI, GL_RED, GL_UNSIGNED_INT, &zero);

		CUDA_CHECK(hipGraphicsMapResources(1, &vaoNormPosResource));
		CUDA_CHECK(hipGraphicsMapResources(1, &vaoRenderResource));
		CUDA_CHECK(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&vBufferNormPosPtr), &size, vaoNormPosResource));
		CUDA_CHECK(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&vBufferRenderPackedPtr), &size, vaoRenderResource));
	
		CUDA_CHECK(hipMalloc(&d_atomicCounter, sizeof(unsigned int)));
		CUDA_CHECK(hipMemset(d_atomicCounter, 0x00, sizeof(unsigned int)));

		// Copy
		// All Pages
		uint32_t gridSize = (allocator.NumPages() * GI_PAGE_SIZE + GI_THREAD_PER_BLOCK - 1) / GI_THREAD_PER_BLOCK;
		VoxelCopyToVAO<<<gridSize, GI_THREAD_PER_BLOCK>>>
			(// Two ogl Buffers for rendering used voxels
			vBufferNormPosPtr,
			vBufferRenderPackedPtr,
			*d_atomicCounter,
			voxCount,

			// Per Obj Segment
			allocator.GetSegmentAllocLoc2D(),

			// Per obj
			allocator.GetObjectAllocationIndexLookup2D(),

			// Per vox
			allocator.GetObjRenderCacheDevice(),

			// Page
			allocator.GetVoxelPagesDevice(),
			allocator.NumPages(),
			*allocator.GetVoxelGridDevice());
		CUDA_KERNEL_CHECK();

		// Unmap
		hipGraphicsUnmapResources(1, &vaoNormPosResource);
		hipGraphicsUnmapResources(1, &vaoRenderResource);
		hipFree(d_atomicCounter);
		allocator.ClearDevicePointers();

		timer.Stop();
		time = timer.ElapsedMilliS();

		voxGridData = allocator.GetVoxelGridHost();
	}
	return VoxelDebugVAO(vaoNormPosData, vaoColorData);
}