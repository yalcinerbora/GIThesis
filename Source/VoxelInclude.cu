#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <assert.h>

#include "GIKernels.cuh"
#include "CVoxel.cuh"
#include "CAxisAlignedBB.cuh"
#include "COpenGLCommon.cuh"

// 1.0f means use max, 0.0f means use min
__device__ static const float3 aabbLookupTable[] =
{
	{ 1.0f, 1.0f, 1.0f },		// V1
	{ 0.0f, 1.0f, 1.0f },		// V2
	{ 1.0f, 0.0f, 1.0f },		// V3
	{ 1.0f, 1.0f, 0.0f },		// V4

	{ 0.0f, 0.0f, 1.0f },		// V5
	{ 0.0f, 1.0f, 0.0f },		// V6
	{ 1.0f, 0.0f, 0.0f },		// V7
	{ 0.0f, 0.0f, 0.0f }		// V8
};

__device__ unsigned int AtomicAllocLoc(unsigned int* gPos)
{
	unsigned int assumed, old = *gPos;
	do
	{
		assumed = old;
		unsigned int result = (assumed == 0) ? 0 : (assumed - 1);
		old = atomicCAS(gPos, assumed, result);
	}
	while(assumed != old);
	return old;
}

__device__ unsigned int AtomicDeallocLoc(unsigned int* gPos)
{
	unsigned int assumed, old = *gPos;
	do
	{
		assumed = old;
		unsigned int result = (assumed == GI_SEGMENT_PER_PAGE) ? GI_SEGMENT_PER_PAGE : (assumed + 1);
		old = atomicCAS(gPos, assumed, result);
	}
	while(assumed != old);
	return old;
}

__device__ void VoxelAdd(// Write Location
						 CVoxelPacked* gVoxelData,

						 // Model Space Voxel
						 const ushort2& objectId,
						 const unsigned int renderLoc,
						 const float3& normal,
						 const uint3& voxPos,
						 const uint3& voxelDim,

						 // Object Related
						 const CObjectTransform& gObjTransform,
						 const CObjectAABB& gObjAABB,
						 const CObjectVoxelInfo& objInfo,
						 const CVoxelGrid& gGridInfo)
{
	// Discard if voxel is too small
	if(voxelDim.x * voxelDim.y * voxelDim.z == 0.0f) return;

	// Generate Model Space Position from voxel
	float4 localPos;
	localPos.x = gObjAABB.min.x + voxPos.x * objInfo.span;
	localPos.y = gObjAABB.min.y + voxPos.y * objInfo.span;
	localPos.z = gObjAABB.min.z + voxPos.z * objInfo.span;
	localPos.w = 1.0f;

	// Convert it to world space
	MultMatrixSelf(localPos, gObjTransform.transform);

	unsigned int voxelSpanRatio = 0;

	// We need to construct additional voxels if this voxel spans multiple gird locations
	uint3 newVoxPos;
	for(unsigned int i = 0; i < voxelDim.x * voxelDim.y * voxelDim.z; i++)
	{
		float3 localPosSubVox;  
		localPosSubVox.x = (localPos.x) + (i % voxelDim.x) * gGridInfo.span;
		localPosSubVox.y = (localPos.y) + (i % voxelDim.y) * gGridInfo.span;
		localPosSubVox.z = (localPos.z) + (i % voxelDim.z) * gGridInfo.span;

		// For each newly introduced voxel
		// Compare world pos with grid
		// Reconstruct Voxel Indices relative to the new pos of the grid
		localPosSubVox.x -= gGridInfo.position.x;
		localPosSubVox.y -= gGridInfo.position.y;
		localPosSubVox.z -= gGridInfo.position.z;
				
		float invSpan = 1.0f / gGridInfo.span;
		newVoxPos.x = static_cast<unsigned int>((localPos.x) * invSpan);
		newVoxPos.y = static_cast<unsigned int>((localPos.y) * invSpan);
		newVoxPos.z = static_cast<unsigned int>((localPos.z) * invSpan);

		// Normal in world space
		float3 normalMult = MultMatrix(normal, gObjTransform.rotation);

		// Write Back
		PackVoxelData(gVoxelData[i], newVoxPos, normalMult, objectId, voxelSpanRatio, renderLoc);
	}
}

__device__ bool CheckGridVoxIntersect(const CVoxelGrid& gGridInfo,
									  const CObjectAABB& gObjectAABB,
									  const CObjectTransform& gObjectTransform)
{
	// Comparing two AABB (Grid Itself is an AABB)
	const CAABB gridAABB =
	{
		{ gGridInfo.position.x, gGridInfo.position.y, gGridInfo.position.z, 1.0f },
		{
			gGridInfo.position.x + gGridInfo.span * gGridInfo.dimension.x,
			gGridInfo.position.y + gGridInfo.span * gGridInfo.dimension.y,
			gGridInfo.position.z + gGridInfo.span * gGridInfo.dimension.z,
			1.0f
		},
	};

	// Construct Transformed AABB
	CAABB transformedAABB =
	{
		{ FLT_MAX, FLT_MAX, FLT_MAX, 1.0f },
		{ -FLT_MAX, -FLT_MAX, -FLT_MAX, 1.0f }
	};

	for(unsigned int i = 0; i < 8; i++)
	{
		float4 data;
		data.x = aabbLookupTable[i].x * gObjectAABB.max.x + (1.0f - aabbLookupTable[i].x) * gObjectAABB.min.x;
		data.y = aabbLookupTable[i].y * gObjectAABB.max.y + (1.0f - aabbLookupTable[i].y) * gObjectAABB.min.y;
		data.z = aabbLookupTable[i].z * gObjectAABB.max.z + (1.0f - aabbLookupTable[i].z) * gObjectAABB.min.z;
		data.w = 1.0f;

		MultMatrixSelf(data, gObjectTransform.transform);
		transformedAABB.max.x = fmax(transformedAABB.max.x, data.x);
		transformedAABB.max.y = fmax(transformedAABB.max.y, data.y);
		transformedAABB.max.z = fmax(transformedAABB.max.z, data.z);

		transformedAABB.min.x = fmin(transformedAABB.min.x, data.x);
		transformedAABB.min.y = fmin(transformedAABB.min.y, data.y);
		transformedAABB.min.z = fmin(transformedAABB.min.z, data.z);
	}
	return Intersects(gridAABB, transformedAABB);
}

__global__ void VoxelObjectDealloc(// Voxel System
								   CVoxelPage* gVoxelData,
								   const unsigned int gPageAmount,
								   const CVoxelGrid& gGridInfo,

								   // Per Object Segment Related
								   ushort2* gObjectAllocLocations,
								   const unsigned int* gSegmentObjectId,
								   const uint32_t totalSegments,

								   // Per Object Related
								   char* gWriteToPages,
								   const CObjectAABB* gObjectAABB,
								   const CObjectTransform* gObjTransforms)
{
	unsigned int globalId = threadIdx.x + blockIdx.x * blockDim.x;

	// Now Thread Scheme changes per objectSegment
	if(globalId >= totalSegments) return;
	
	// Determine Obj Id
	unsigned int objectId = gSegmentObjectId[globalId];
	bool intersects = CheckGridVoxIntersect(gGridInfo, gObjectAABB[objectId], gObjTransforms[objectId]);
	ushort2 objAlloc = gObjectAllocLocations[globalId];

	if(!intersects && objAlloc.x != 0xFFFF)
	{
		// "Dealocate"
		unsigned int location = AtomicDeallocLoc(&(gVoxelData[objAlloc.x].dEmptySegmentIndex)) - 1;
		if(location < GI_SEGMENT_PER_PAGE)
		{
			gVoxelData[objAlloc.x].dEmptySegmentPos[location] = objAlloc.y;
			gVoxelData[objAlloc.x].dIsSegmentOccupied[location] = 0;
			gObjectAllocLocations[globalId] = { 0xFFFF, 0xFFFF };
		}
	}
}



__global__ void VoxelObjectAlloc(// Voxel System
								 CVoxelPage* gVoxelData,
								 const unsigned int gPageAmount,
								 const CVoxelGrid& gGridInfo,

								 // Per Object Segment Related
								 ushort2* gObjectAllocLocations,
								 const unsigned int* gSegmentObjectId,
								 const uint32_t totalSegments,

								 // Per Object Related
								 char* gWriteToPages,
								 const CObjectAABB* gObjectAABB,
								 const CObjectTransform* gObjTransforms)
{
	unsigned int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	if(globalId >= totalSegments) return;
	
	// Determine Obj Id
	unsigned int objectId = gSegmentObjectId[globalId];
	bool intersects = CheckGridVoxIntersect(gGridInfo, gObjectAABB[objectId], gObjTransforms[objectId]);
	ushort2 objAlloc = gObjectAllocLocations[globalId];

	// Check If this object is in
	if(intersects && objAlloc.x == 0xFFFF)
	{
		// "Allocate"
		gWriteToPages[objectId] = 1;
		
		// Check page by page
		for(unsigned int i = 0; i < gPageAmount; i++)
		{
			//unsigned int location = atomicSub(&gVoxelData[i].dEmptySegmentIndex, 1) - 1;
			unsigned int location = AtomicAllocLoc(&(gVoxelData[i].dEmptySegmentIndex)) - 1;
			if(location < GI_SEGMENT_PER_PAGE)
			{
				gObjectAllocLocations[globalId] = 
				{
					static_cast<unsigned short>(i), 
					static_cast<unsigned short>(gVoxelData[i].dEmptySegmentPos[location])
				};
				gVoxelData[i].dIsSegmentOccupied[location] = 1;
				return;
			}
		}
	}
}

__global__ void VoxelObjectInclude(// Voxel System
								   CVoxelPage* gVoxelData,
								   const unsigned int gPageAmount,
								   const CVoxelGrid& gGridInfo,

								   // Per Object Segment Related
								   ushort2* gObjectAllocLocations,
								   const uint32_t segmentCount,
								 
								   // Per Object Related
								   char* gWriteToPages,
								   const unsigned int* gObjectVoxStrides,
								   const unsigned int* gObjectAllocIndexLookup,
								   const CObjectAABB* gObjectAABB,
								   const CObjectTransform* gObjTransforms,
								   const CObjectVoxelInfo* gObjInfo,

								   // Per Voxel Related
								   const CVoxelPacked* gObjectVoxelCache,
								   uint32_t voxCount,

								   // Batch(ObjectGroup in terms of OGL) Id
								   uint32_t batchId)
{
	unsigned int globalId = threadIdx.x + blockIdx.x * blockDim.x;

	// Now Thread Sceheme changes per voxel
	if(globalId >= voxCount) return;
	
	// Mem Fetch
	ushort2 objectId;
	uint3 voxPos;
	float3 normal;
	unsigned int voxelSpanRatio;
	unsigned int renderLoc;
	ExpandVoxelData(voxPos, normal, objectId, renderLoc, voxelSpanRatio, gObjectVoxelCache[globalId]);

	// We need to check if this obj is not already in the page system or not
	if(gWriteToPages[objectId.y] == 1)
	{
		//// We need to check scaling and adjust span
		//// Objects may have different voxel sizes and voxel sizes may change after scaling
		//float3 scaling = ExtractScaleInfo(gObjTransforms[objectId.y].transform);
		//assert(scaling.x == scaling.y);
		//assert(scaling.y == scaling.z);

		//unsigned int voxelDim = static_cast<unsigned int>(gObjInfo[objectId.y].span * scaling.x / gGridInfo.span);
		//unsigned int voxScale = 1; voxelDim == 0 ? 0 : 1;

		//// Determine wich voxel is this thread on that specific object
		unsigned int voxId = globalId - gObjectVoxStrides[objectId.y];
		//unsigned int segment = (voxId * voxScale) / GI_SEGMENT_SIZE;
		//unsigned int segmentStart = gObjectAllocIndexLookup[objectId.y];

		//if(segmentStart < segmentCount)
		//{
		//	ushort2 segmentLoc = gObjectAllocLocations[segmentStart + segment];

		//	//// Finally Actual Voxel Write
		//	//objectId.x = batchId;
		//	//VoxelAdd(&gVoxelData[segmentLoc.x].dGridVoxels[segmentLoc.y],
		//	//			objectId,
		//	//			renderLoc,
		//	//			normal,
		//	//			voxPos,
		//	//			voxelDim,
		//	//			gObjTransforms[objectId.y],
		//	//			gObjectAABB[objectId.y],
		//	//			gObjInfo[objectId.y],
		//	//			gGridInfo);
		//}
		
		// All done stop write signal
		// Determine a leader per object
		if(voxId == 0)
		{
			gWriteToPages[objectId.y] = 0;
		}
	}
}