#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "COpenGLCommon.cuh"
#include "CVoxel.cuh"
#include "GICudaAllocator.h"

__global__ void VoxelTransform(// Voxel Pages
							   CVoxelPage* gVoxelData,
							   const CVoxelGrid& gGridInfo,
							   const float3 hNewGridPosition,

							   // Per Object Segment
							   ushort2** gObjectAllocLocations,

							   // Object Related
							   unsigned int** gObjectAllocIndexLookup,
							   CObjectTransform** gObjTransforms,
							   CVoxelRender** gVoxRenderData,
							   CVoxelPacked** gVoxCacheData,
							   CObjectVoxelInfo** gObjInfo,
							   CObjectAABB** gObjectAABB)
{
	unsigned int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int pageId = globalId / GI_PAGE_SIZE;
	unsigned int pageLocalId = (globalId - pageId * GI_PAGE_SIZE);
	unsigned int pageLocalSegmentId = pageLocalId / GI_SEGMENT_SIZE;
//	if(gVoxelData[pageId].dIsSegmentOccupied[pageLocalSegmentId] == 0) return;
	
	// Mem Fetch and Expand (8 byte per warp, coalesced, 0 stride)
	CVoxelObjectType objType;
	ushort2 objectId;
	unsigned int renderLoc;
	ExpandVoxelIds(renderLoc, objectId, objType, gVoxelData[pageId].dGridVoxIds[pageLocalId]);

	// Skip if this object is not in the grid
//	if(gObjectAllocLocations[objectId.y][gObjectAllocIndexLookup[objectId.y][objectId.x]].x == 0xFFFF) return;

	// Fetch NormalPos Array
	uint3 voxPos;
	float3 normal;
	unsigned int voxelSpanRatio;
	ExpandNormalPos(voxPos, normal, voxelSpanRatio, uint2{gVoxCacheData[objectId.y][renderLoc].x, gVoxCacheData[objectId.y][renderLoc].y});
	
	// Fetch AABB, transform and span
	float4 objAABBMin = gObjectAABB[objectId.y][objectId.x].min;
	CMatrix4x4 transform = gObjTransforms[objectId.y][objectId.x].transform;
	float objSpan = gObjInfo[objectId.y][objectId.x].span;

	// Calculate Span Ratio
	float3 scaling = ExtractScaleInfo(transform);
	assert(scaling.x == scaling.y);
	assert(scaling.y == scaling.z);
	// Calculate Vox Span Ratio (if this object voxel is span higher level)
	// This operation assumes object and voxel span is related (obj is pow of two multiple of grid)
	unsigned int voxRatio = static_cast<unsigned int>(objSpan * scaling.x / gGridInfo.span);
	voxRatio--;
	voxRatio |= voxRatio >> 1;
	voxRatio |= voxRatio >> 2;
	voxRatio |= voxRatio >> 4;
	voxRatio |= voxRatio >> 8;
	voxRatio |= voxRatio >> 16;
	voxRatio++;
	voxelSpanRatio = voxRatio;

	// Generate World Position
	// start with object space position
	float4 worldPos;
	worldPos.x = objAABBMin.x + voxPos.x * objSpan;
	worldPos.y = objAABBMin.y + voxPos.y * objSpan;
	worldPos.z = objAABBMin.z + voxPos.z * objSpan;
	worldPos.w = 1.0f;

	// Transformations
	objType = CVoxelObjectType::STATIC;
	switch(objType)
	{
		case CVoxelObjectType::STATIC:
		case CVoxelObjectType::DYNAMIC:
		{
			// One Transform per voxel
			CMatrix4x4 rotation = gObjTransforms[objectId.y][objectId.x].rotation;

			// Now voxel is in is world space
			MultMatrixSelf(worldPos, transform);
			MultMatrixSelf(normal, rotation);
			break;
		}
		case CVoxelObjectType::SKEL_DYNAMIC:
		{
			// TODO Implement
			break;
		}
		case CVoxelObjectType::MORPH_DYNAMIC:
		{
			// TODO Implement
			break;
		}
		default:
			assert(false);
			break;
	}

	// Reconstruct Voxel Indices relative to the new pos of the grid
	worldPos.x -= hNewGridPosition.x;
	worldPos.y -= hNewGridPosition.y;
	worldPos.z -= hNewGridPosition.z;

	bool outOfBounds;
	outOfBounds = (worldPos.x) < 0 || (worldPos.x > gGridInfo.dimension.x * gGridInfo.span);
	outOfBounds |= (worldPos.y) < 0 || (worldPos.y > gGridInfo.dimension.y * gGridInfo.span);
	outOfBounds |= (worldPos.z) < 0 || (worldPos.z > gGridInfo.dimension.z * gGridInfo.span);

	// Now Write
	// Discard the out of bound voxels
	// will come back into the grid
	if(!outOfBounds)
	{
		float invSpan = 1.0f / (gGridInfo.span);
		voxPos.x = static_cast<unsigned int>(worldPos.x * invSpan);
		voxPos.y = static_cast<unsigned int>(worldPos.y * invSpan);
		voxPos.z = static_cast<unsigned int>(worldPos.z * invSpan);

	/*	voxPos.x = 456; 
		voxPos.y = 256; 
		voxPos.z = 256;*/
		
		// Write to page
		PackVoxelNormPos(gVoxelData[pageId].dGridVoxNormPos[pageLocalId], voxPos, normal, voxelSpanRatio);
	}
}