#include "hip/hip_runtime.h"
#include "GIKernels.cuh"
#include "CSparseVoxelOctree.cuh"
#include "CVoxel.cuh"
#include <hip/hip_runtime.h>

// Lookup table for determining neigbour nodes
// just splitted first 8 values
__device__ static const char3 voxLookup[] =
{
	{0, 0, 0},
	{0, 0, 1},
	{0, 1, 0},
	{0, 1, 1},

	{1, 0, 0},
	{1, 0, 1},
	{1, 1, 0},
	{1, 1, 1},
};

inline __device__ CSVOMaterial Average(const CSVOMaterial& material,
									   const float4& colorUnpack,
									   const float3& normalUnpack)
{
	// Unpack Material
	CSVOColor avgColorPacked;
	CVoxelNorm avgNormalPacked;
	UnpackSVOMaterial(avgColorPacked, avgNormalPacked, material);
	float4 avgColor = UnpackSVOColor(avgColorPacked);
	float4 avgNormal = UnpackNormCount(avgNormalPacked);

	// Averaging (color.w is number of nodes)
	assert(avgNormal.w < 255.0f);
	float ratio = avgNormal.w / (avgNormal.w + 1.0f);

	// New Color Average
	avgColor.x = (ratio * avgColor.x) + (colorUnpack.x / (avgNormal.w + 1.0f));
	avgColor.y = (ratio * avgColor.y) + (colorUnpack.y / (avgNormal.w + 1.0f));
	avgColor.z = (ratio * avgColor.z) + (colorUnpack.z / (avgNormal.w + 1.0f));
	avgColor.w = (ratio * avgColor.w) + (colorUnpack.w / (avgNormal.w + 1.0f));

	// New Normal Average
	avgNormal.x = (ratio * avgNormal.x) + (normalUnpack.x / (avgNormal.w + 1.0f));
	avgNormal.y = (ratio * avgNormal.y) + (normalUnpack.y / (avgNormal.w + 1.0f));
	avgNormal.z = (ratio * avgNormal.z) + (normalUnpack.z / (avgNormal.w + 1.0f));
	avgNormal.w += 1.0f;

	avgColorPacked = PackSVOColor(avgColor);
	avgNormalPacked = PackNormCount(avgNormal);
	return PackSVOMaterial(avgColorPacked, avgNormalPacked);
}

inline __device__ CSVOMaterial AtomicColorNormalAvg(CSVOMaterial* gMaterial,
													CSVOColor color,
													CVoxelNorm voxelNormal)
{
	float4 colorUnpack = UnpackSVOColor(color);
	float4 normalUnpack = ExpandOnlyNormal(voxelNormal);
	CSVOMaterial assumed, old = *gMaterial;
	do
	{
		assumed = old;
		old = atomicCAS(gMaterial, assumed, Average(assumed, 
													colorUnpack, 
													{normalUnpack.x, normalUnpack.y, normalUnpack.z}));
	}
	while(assumed != old);
	return old;
}

inline __device__ unsigned int AtomicAllocateNode(CSVONode* gNode, unsigned int& gLevelAllocator)
{
	// Release Configuration Optimization fucks up the code
	// Prob changes some memory i-o ordering
	// Its fixed but comment is here for future
	// Problem here was cople threads on the same warp waits eachother and
	// after some memory ordering changes by compiler responsible thread waits
	// other threads execution to be done
	// Code becomes something like this after compiler changes some memory orderings
	//
	//	while(old = atomicCAS(gNode, 0xFFFFFFFF, 0xFFFFFFFE) == 0xFFFFFFFE); <-- notice semicolon
	//	 if(old == 0xFFFFFF)
	//		location = allocate();
	//	location = old;
	//	return location;
	//
	// first allocating thread will never return from that loop since 
	// its warp threads are on infinite loop (so deadlock)

	// much cooler version can be warp level exchange intrinsics
	// which slightly reduces atomic pressure on the single node (on lower tree levels atleast)

	CSVONode old = 0xFFFFFFFE;
	while(old == 0xFFFFFFFE)
	{
		old = atomicCAS(gNode, 0xFFFFFFFF, 0xFFFFFFFE);
		if(old == 0xFFFFFFFF)
		{
			// Allocate
			unsigned int location = atomicAdd(&gLevelAllocator, 8);
			//atomicExch(gNode, location);
			*reinterpret_cast<volatile CSVONode*>(gNode) = location;
			old = location;
		}
		__threadfence();	// This is important somehow compiler changes this and makes infinite loop on smae warp threads
	}
	return old;
}

inline __device__ unsigned int FindDenseChildren(const uint3& parentIndex,
												 const unsigned int childId,
												 const unsigned int levelDim)
{
	// Go down 1 lvl
	uint3 childIndex = parentIndex;
	childIndex.x *= 2;
	childIndex.y *= 2;
	childIndex.z *= 2;

	uint3 offsetIndex =
	{
		childId % 2,
		childId / 2,
		childId / 4
	};
	childIndex.x += offsetIndex.x;
	childIndex.y += offsetIndex.y;
	childIndex.z += offsetIndex.z;

	unsigned int childLvlDim = levelDim << 1;
	unsigned int linearChildId = childIndex.z * childLvlDim * childLvlDim +
		childIndex.y * childLvlDim +
		childIndex.z;
	return linearChildId;
}

__global__ void SVOReconstructDetermineNode(CSVONode* gSVODense,
											const CVoxelPage* gVoxelData,

											const unsigned int cascadeNo,
											const CSVOConstants& svoConstants)
{
	unsigned int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int pageId = globalId / GI_PAGE_SIZE;
	unsigned int pageLocalId = globalId % GI_PAGE_SIZE;
	unsigned int pageLocalSegmentId = pageLocalId / GI_SEGMENT_SIZE;

	// Skip Whole segment if necessary
	if(gVoxelData[pageId].dIsSegmentOccupied[pageLocalSegmentId] == SegmentOccupation::EMPTY) return;
	if(gVoxelData[pageId].dIsSegmentOccupied[pageLocalSegmentId] == SegmentOccupation::MARKED_FOR_CLEAR) assert(false);

	// Fetch voxel
	CVoxelPos voxelPosPacked = gVoxelData[pageId].dGridVoxPos[pageLocalId];
	if(voxelPosPacked == 0xFFFFFFFF) return;

	// Local Voxel pos and expand it if its one of the inner cascades
	uint3 voxelUnpacked = ExpandOnlyVoxPos(voxelPosPacked);
	uint3 voxelPos = ExpandToSVODepth(voxelUnpacked, cascadeNo,
									  svoConstants.numCascades,
									  svoConstants.totalDepth);
	uint3 denseIndex = CalculateLevelVoxId(voxelPos, svoConstants.denseDepth,
										   svoConstants.totalDepth);

	assert(denseIndex.x < svoConstants.denseDim &&
		   denseIndex.y < svoConstants.denseDim &&
		   denseIndex.z < svoConstants.denseDim);

	// Signal alloc
	*(gSVODense +
	  svoConstants.denseDim * svoConstants.denseDim * denseIndex.z +
	  svoConstants.denseDim * denseIndex.y +
	  denseIndex.x) = 1;
}

__global__ void SVOReconstructDetermineNode(CSVONode* gSVOSparse,
											hipTextureObject_t tSVODense,
											const CVoxelPage* gVoxelData,
											const unsigned int* gLevelOffsets,

											// Constants
											const unsigned int cascadeNo,
											const unsigned int levelDepth,
											const CSVOConstants& svoConstants)
{
	unsigned int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int pageId = globalId / GI_PAGE_SIZE;
	unsigned int pageLocalId = globalId % GI_PAGE_SIZE;
	unsigned int pageLocalSegmentId = pageLocalId / GI_SEGMENT_SIZE;

	// Skip Whole segment if necessary
	if(gVoxelData[pageId].dIsSegmentOccupied[pageLocalSegmentId] == SegmentOccupation::EMPTY) return;
	if(gVoxelData[pageId].dIsSegmentOccupied[pageLocalSegmentId] == SegmentOccupation::MARKED_FOR_CLEAR) assert(false);

	// Fetch voxel
	CVoxelPos voxelPosPacked = gVoxelData[pageId].dGridVoxPos[pageLocalId];
	if(voxelPosPacked == 0xFFFFFFFF) return;

	// Local Voxel pos and expand it if its one of the inner cascades
	uint3 voxelUnpacked = ExpandOnlyVoxPos(voxelPosPacked);
	uint3 voxelPos = ExpandToSVODepth(voxelUnpacked, cascadeNo,
									  svoConstants.numCascades,
									  svoConstants.totalDepth);

	unsigned int nodeIndex = 0;
	for(unsigned int i = svoConstants.denseDepth; i < levelDepth; i++)
	{
		CSVONode currentNode;
		if(i == svoConstants.denseDepth)
		{
			uint3 denseIndex = CalculateLevelVoxId(voxelPos, svoConstants.denseDepth,
												   svoConstants.totalDepth);

			assert(denseIndex.x < svoConstants.denseDim &&
				   denseIndex.y < svoConstants.denseDim &&
				   denseIndex.z < svoConstants.denseDim);

			currentNode = tex3D<unsigned int>(tSVODense,
											  denseIndex.x,
											  denseIndex.y,
											  denseIndex.z);
		}
		else
		{
			currentNode = gSVOSparse[gLevelOffsets[i - svoConstants.denseDepth] + nodeIndex];
		}

		// Offset according to children
		assert(currentNode != 0xFFFFFFFF);
		unsigned int childIndex = CalculateLevelChildId(voxelPos, i + 1, svoConstants.totalDepth);
		nodeIndex = currentNode + childIndex;
	}

	// Finally Write
	gSVOSparse[gLevelOffsets[levelDepth - svoConstants.denseDepth] + nodeIndex] = 1;
}

__global__ void SVOReconstructAllocateLevel(CSVONode* gSVOLevel,
											unsigned int& gSVONextLevelAllocator,
											const unsigned int& gSVONextLevelTotalSize,
											const unsigned int& gSVOLevelSize,
											const CSVOConstants& svoConstants)
{
	unsigned int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	if(globalId >= gSVOLevelSize) return;

	CSVONode node = gSVOLevel[globalId]; if(node != 1) return;

	// Allocation
	unsigned int location = atomicAdd(&gSVONextLevelAllocator, 8);
	assert(location < gSVONextLevelTotalSize);

	gSVOLevel[globalId] = location;
}

__global__ void SVOReconstructMaterialLeaf(CSVOMaterial* gSVOMat,

										   // Const SVO Data
										   const CSVONode* gSVOSparse,
										   const unsigned int* gLevelOffsets,
										   hipTextureObject_t tSVODense,

										   // Page Data
										   const CVoxelPage* gVoxelData,

										   // For Color Lookup
										   CVoxelRender** gVoxelRenderData,

										   // Constants
										   const unsigned int matSparseOffset,
										   const unsigned int cascadeNo,
										   const CSVOConstants& svoConstants)
{
	unsigned int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int pageId = globalId / GI_PAGE_SIZE;
	unsigned int pageLocalId = globalId % GI_PAGE_SIZE;
	unsigned int pageLocalSegmentId = pageLocalId / GI_SEGMENT_SIZE;

	// Skip Whole segment if necessary
	if(gVoxelData[pageId].dIsSegmentOccupied[pageLocalSegmentId] == SegmentOccupation::EMPTY) return;
	if(gVoxelData[pageId].dIsSegmentOccupied[pageLocalSegmentId] == SegmentOccupation::MARKED_FOR_CLEAR) assert(false);

	// Fetch voxel
	CVoxelPos voxelPosPacked = gVoxelData[pageId].dGridVoxPos[pageLocalId];
	if(voxelPosPacked == 0xFFFFFFFF) return;

	// Local Voxel pos and expand it if its one of the inner cascades
	uint3 voxelUnpacked = ExpandOnlyVoxPos(voxelPosPacked);
	uint3 voxelPos = ExpandToSVODepth(voxelUnpacked,
									  cascadeNo,
									  svoConstants.numCascades,
									  svoConstants.totalDepth);


	unsigned int nodeIndex = 0;
	unsigned int cascadeMaxLevel = svoConstants.totalDepth - (svoConstants.numCascades - cascadeNo);
	for(unsigned int i = svoConstants.denseDepth; i <= cascadeMaxLevel; i++)
	{
		CSVONode currentNode;
		if(i == svoConstants.denseDepth)
		{
			uint3 denseIndex = CalculateLevelVoxId(voxelPos, svoConstants.denseDepth,
												   svoConstants.totalDepth);

			assert(denseIndex.x < svoConstants.denseDim &&
				   denseIndex.y < svoConstants.denseDim &&
				   denseIndex.z < svoConstants.denseDim);

			currentNode = tex3D<unsigned int>(tSVODense,
											  denseIndex.x,
											  denseIndex.y,
											  denseIndex.z);
		}
		else
		{
			currentNode = gSVOSparse[gLevelOffsets[i - svoConstants.denseDepth] + nodeIndex];
		}

		// Offset according to children
		assert(currentNode != 0xFFFFFFFF);
		unsigned int childIndex = CalculateLevelChildId(voxelPos, i + 1, svoConstants.totalDepth);
		nodeIndex = currentNode + childIndex;
	}

	// Finally found location
	// Average color and normal
	// Fetch obj Id to get color
	ushort2 objectId;
	CVoxelObjectType objType;
	unsigned int voxelId;
	ExpandVoxelIds(voxelId, objectId, objType, gVoxelData[pageId].dGridVoxIds[pageLocalId]);

	CVoxelNorm voxelNormPacked = gVoxelData[pageId].dGridVoxNorm[pageLocalId];
	CSVOColor voxelColorPacked = *reinterpret_cast<unsigned int*>(&gVoxelRenderData[objectId.y][voxelId].color);

	// Atomic Average
	AtomicColorNormalAvg(gSVOMat + matSparseOffset +
						 gLevelOffsets[cascadeMaxLevel + 1 - svoConstants.denseDepth] +
						 nodeIndex,
						 voxelColorPacked,
						 voxelNormPacked);

	//gSVOMat[matSparseOffset + gLevelOffsets[cascadeMaxLevel + 1 - 
	//		svoConstants.denseDepth] +
	//		nodeIndex] = PackSVOMaterial(voxelColorPacked, voxelNormPacked);

}

__global__ void SVOReconstructAverageNode(CSVOMaterial* gSVOMat,
										  hipSurfaceObject_t sDenseMat,

										  const CSVONode* gSVODense,
										  const CSVONode* gSVOSparse,
										  
										  const unsigned int& gSVOLevelOffset,
										  const unsigned int& gSVONextLevelOffset,

										  const unsigned int levelNodeCount,
										  const unsigned int matOffset,
										  const unsigned int currentLevel,
										  const CSVOConstants& svoConstants)
{
	unsigned int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int globalParentId = globalId / GI_NODE_THREAD_COUNT;

	// Warp Level Fetch
	unsigned int warpId = threadIdx.x / warpSize;
	unsigned int laneId = threadIdx.x % warpSize;
	unsigned int parentPerWarp = warpSize / GI_NODE_THREAD_COUNT;

	// Linear ID
	unsigned int warpLinearId = (blockIdx.x * (blockDim.x / GI_NODE_THREAD_COUNT)) + warpId * parentPerWarp + laneId;
	unsigned int localNodeChildId = threadIdx.x % GI_NODE_THREAD_COUNT;

	// Cull if out of range
	if(globalParentId > levelNodeCount) return;

	// Read Sibling Materials
	CSVOMaterial parentMat = 0;
	CSVONode node = 0xFFFFFFFF;
	// Coalesced Parent Load (Warp Level)
	if(laneId < parentPerWarp)
	{
		// Coalesced Fetch
		const CSVONode* n = (currentLevel == svoConstants.denseDepth) ? gSVODense : gSVOSparse;
		node = n[gSVOLevelOffset + warpLinearId];

		// Only fetch parent when its contributes to the average
		bool fetchParentMat = ((svoConstants.totalDepth - currentLevel) < svoConstants.numCascades);// || (node != 0xFFFFFFFF);
		parentMat = fetchParentMat ? gSVOMat[matOffset + gSVOLevelOffset + warpLinearId] : 0;
	}

	// Shuffle each parent to actual users
	if(GI_NODE_THREAD_COUNT != 1) node = __shfl(node, laneId / GI_NODE_THREAD_COUNT);
	if(GI_NODE_THREAD_COUNT != 1) parentMat = __shfl(parentMat, laneId / GI_NODE_THREAD_COUNT);

	// Average Portion
	// Material Data
	unsigned int count = 0;
	float4 colorAvg = {0.0f, 0.0f, 0.0f, 0.0f};
	float4 normalAvg = {0.0f, 0.0f, 0.0f, 0.0f};

	// Average Yours
	#pragma unroll
	for(unsigned int i = 0; i < (8 / GI_NODE_THREAD_COUNT); i++)
	{
		unsigned int nodeId = node + localNodeChildId;
		CSVOMaterial mat = (node != 0xFFFFFFFF) ? gSVOMat[matOffset + gSVONextLevelOffset + nodeId + i * GI_NODE_THREAD_COUNT] : 0;
		if(mat != 0)
		{
			CSVOColor colorPacked;
			CVoxelNorm normalPacked;
			UnpackSVOMaterial(colorPacked, normalPacked, mat);
			float4 color = UnpackSVOColor(colorPacked);
			float4 normal = ExpandOnlyNormal(normalPacked);

			colorAvg.x += color.x;
			colorAvg.y += color.y;
			colorAvg.z += color.z;
			colorAvg.w += color.w;
			

			normalAvg.x += normal.x;
			normalAvg.y += normal.y;
			normalAvg.z += normal.z;
			normalAvg.w += (currentLevel == (svoConstants.totalDepth - 1)) ? 1.0f : normalAvg.w;

			count++;
		}
	}

	if(threadIdx.x % GI_NODE_THREAD_COUNT == 0)
	{
		// Parent also may contain color fetch and add it to average
		if(parentMat != 0)
		{
			CSVOColor colorPacked;
			CVoxelNorm normalPacked;
			UnpackSVOMaterial(colorPacked, normalPacked, parentMat);
			float4 color = UnpackSVOColor(colorPacked);
			float4 normal = ExpandOnlyNormal(normalPacked);

			colorAvg.x += 8 * color.x;
			colorAvg.y += 8 * color.y;
			colorAvg.z += 8 * color.z;
			colorAvg.w += 8 * color.w;

			normalAvg.x += 8 * normal.x;
			normalAvg.y += 8 * normal.y;
			normalAvg.z += 8 * normal.z;

			// Wieghted average since this color spans more area (8 times more)
			count += 8;
		}
	}

	// Average Between Threads ( Warp Transfer is ideal here)
	#pragma unroll
	for(int offset = GI_NODE_THREAD_COUNT / 2; offset > 0; offset /= 2)
	{
		colorAvg.x += __shfl_down(colorAvg.x, offset, GI_NODE_THREAD_COUNT);
		colorAvg.y += __shfl_down(colorAvg.y, offset, GI_NODE_THREAD_COUNT);
		colorAvg.z += __shfl_down(colorAvg.z, offset, GI_NODE_THREAD_COUNT);
		colorAvg.w += __shfl_down(colorAvg.w, offset, GI_NODE_THREAD_COUNT);

		normalAvg.x += __shfl_down(normalAvg.x, offset, GI_NODE_THREAD_COUNT);
		normalAvg.y += __shfl_down(normalAvg.y, offset, GI_NODE_THREAD_COUNT);
		normalAvg.z += __shfl_down(normalAvg.z, offset, GI_NODE_THREAD_COUNT);

		count += __shfl_down(count, offset, GI_NODE_THREAD_COUNT);
	}

	// Divide by Count
	if(threadIdx.x % GI_NODE_THREAD_COUNT == 0)
	{
		// Divide by Count
		float countInv = 1.0f / static_cast<float>(count);

		colorAvg.x *= countInv;
		colorAvg.y *= countInv;
		colorAvg.z *= countInv;
		colorAvg.w *= countInv;

		normalAvg.x *= countInv;
		normalAvg.y *= countInv;
		normalAvg.z *= countInv;
		normalAvg.w *= 0.125f;
	}
	if(parentMat != 0) colorAvg.w = 1.0f;	// Opaque
	
	CSVOMaterial matAvg = PackSVOMaterial(PackSVOColor(colorAvg), PackOnlyVoxNorm(normalAvg));
	if(GI_NODE_THREAD_COUNT != 1) matAvg = __shfl(matAvg, laneId * GI_NODE_THREAD_COUNT);
	if(laneId < parentPerWarp && matAvg != 0)
	{
		if(currentLevel == svoConstants.denseDepth)
		{
			int3 dim =
			{
				static_cast<int>(globalId % svoConstants.denseDim),
				static_cast<int>((globalId / svoConstants.denseDim) % svoConstants.denseDim),
				static_cast<int>(globalId / (svoConstants.denseDim * svoConstants.denseDim))
			};
			uint2 data =
			{
				static_cast<unsigned int>(matAvg & 0x00000000FFFFFFFF),
				static_cast<unsigned int>(matAvg >> 32)
			};
			surf3Dwrite(data, sDenseMat, dim.x * sizeof(uint2), dim.y, dim.z);
		}
		else
			gSVOMat[matOffset + gSVOLevelOffset + warpLinearId] = matAvg;
	}
}

__global__ void SVOReconstruct(CSVOMaterial* gSVOMat,
							   CSVONode* gSVOSparse,
							   CSVONode* gSVODense,
							   unsigned int* gLevelAllocators,

							   const unsigned int* gLevelOffsets,
							   const unsigned int* gLevelTotalSizes,

							   // For Color Lookup
							   const CVoxelPage* gVoxelData,
							   CVoxelRender** gVoxelRenderData,

							   const unsigned int matSparseOffset,
							   const unsigned int cascadeNo,
							   const CSVOConstants& svoConstants)
{
	unsigned int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int pageId = globalId / GI_PAGE_SIZE;
	unsigned int pageLocalId = globalId % GI_PAGE_SIZE;
	unsigned int pageLocalSegmentId = pageLocalId / GI_SEGMENT_SIZE;

	// Skip Whole segment if necessary
	if(gVoxelData[pageId].dIsSegmentOccupied[pageLocalSegmentId] == SegmentOccupation::EMPTY) return;
	if(gVoxelData[pageId].dIsSegmentOccupied[pageLocalSegmentId] == SegmentOccupation::MARKED_FOR_CLEAR) assert(false);

	// Fetch voxel
	CVoxelPos voxelPosPacked = gVoxelData[pageId].dGridVoxPos[pageLocalId];
	if(voxelPosPacked == 0xFFFFFFFF) return;

	// Local Voxel pos and expand it if its one of the inner cascades
	uint3 voxelUnpacked = ExpandOnlyVoxPos(voxelPosPacked);
	int3 splitId;
	splitId.x = static_cast<int>(voxelUnpacked.x & 0x00000001) * 2 - 1;
	splitId.y = static_cast<int>(voxelUnpacked.y & 0x00000001) * 2 - 1;
	splitId.z = static_cast<int>(voxelUnpacked.z & 0x00000001) * 2 - 1;

	// Put the color value to the each node corners of the interpolate nodes
	// TODO Mark Dynamic objects and expand those in here
	for(unsigned int i = 0; i < 1; i++)
	{
		int3 voxSigned;
		voxSigned.x = static_cast<int>(voxelUnpacked.x) + (voxLookup[i].x * splitId.x);
		voxSigned.y = static_cast<int>(voxelUnpacked.y) + (voxLookup[i].y * splitId.y);
		voxSigned.z = static_cast<int>(voxelUnpacked.z) + (voxLookup[i].z * splitId.z);
															
		// It may be out of bounds
		int totalDim = 0x1 << (svoConstants.totalDepth - (svoConstants.numCascades - 1));
		if(voxSigned.x <= 0 || voxSigned.x > totalDim ||
		   voxSigned.y <= 0 || voxSigned.y > totalDim ||
		   voxSigned.z <= 0 || voxSigned.z > totalDim)
		   continue;

		uint3 vox;
		vox.x = static_cast<unsigned int>(voxSigned.x);
		vox.y = static_cast<unsigned int>(voxSigned.y);
		vox.z = static_cast<unsigned int>(voxSigned.z);
		uint3 voxelPos = ExpandToSVODepth(vox,
										  cascadeNo,
										  svoConstants.numCascades,
										  svoConstants.totalDepth);

		unsigned int location;
		unsigned int cascadeMaxLevel = svoConstants.totalDepth - (svoConstants.numCascades - cascadeNo);
		for(unsigned int i = svoConstants.denseDepth; i <= cascadeMaxLevel; i++)
		{
			unsigned int levelIndex = i - svoConstants.denseDepth;
			CSVONode* node = nullptr;
			if(i == svoConstants.denseDepth)
			{
				uint3 levelVoxId = CalculateLevelVoxId(voxelPos, i, svoConstants.totalDepth);
				node = gSVODense + svoConstants.denseDim * svoConstants.denseDim * levelVoxId.z +
					svoConstants.denseDim * levelVoxId.y +
					levelVoxId.x;
			}
			else
			{
				node = gSVOSparse + gLevelOffsets[levelIndex] + location;
			}

			// Allocate (or acquire) next location
			location = AtomicAllocateNode(node, gLevelAllocators[levelIndex + 1]);
			assert(location < gLevelTotalSizes[levelIndex + 1]);

			// Offset child
			unsigned int childId = CalculateLevelChildId(voxelPos, i + 1, svoConstants.totalDepth);
			location += childId;
		}

		// We are at bottom of the location can write colors (---)
		ushort2 objectId;
		CVoxelObjectType objType;
		unsigned int voxelId;
		ExpandVoxelIds(voxelId, objectId, objType, gVoxelData[pageId].dGridVoxIds[pageLocalId]);

		CVoxelNorm voxelNormPacked = gVoxelData[pageId].dGridVoxNorm[pageLocalId];
		CSVOColor voxelColorPacked = *reinterpret_cast<unsigned int*>(&gVoxelRenderData[objectId.y][voxelId].color);
		AtomicColorNormalAvg(gSVOMat + matSparseOffset +
							 gLevelOffsets[cascadeMaxLevel + 1 - svoConstants.denseDepth] +
							 location,
							 voxelColorPacked,
							 voxelNormPacked);

		//// Non atmoic overwrite
		//gSVOMat[matSparseOffset + gLevelOffsets[cascadeMaxLevel + 1 -
		//		svoConstants.denseDepth] +
		//		nodeIndex] = PackSVOMaterial(voxelColorPacked, voxelNormPacked);
	}
}