#include "hip/hip_runtime.h"
#include "GISparseVoxelOctree.h"
#include "GICudaAllocator.h"
#include "GIKernels.cuh"
#include "CudaTimer.h"
#include "Macros.h"
#include "Camera.h"
#include "Globals.h"
#include "IEUtility/IEMath.h"

#include <cuda_gl_interop.h>
#include <numeric>
#include <hip/hip_runtime_api.h>

GISparseVoxelOctree::GISparseVoxelOctree()
	: svoNodeBuffer(512)
	, svoMaterialBuffer(512)
	, svoLevelOffsets(32)
	, dSVOConstants(1)
	, tSVODense(0)
	, computeVoxTraceWorld(ShaderType::COMPUTE, "Shaders/VoxTraceWorld.glsl")
	, svoTraceData(1)
	, svoNodeResource(nullptr)
	, svoLevelOffsetResource(nullptr)
	, svoMaterialResource(nullptr)
{
	svoTraceData.AddData({});
}

GISparseVoxelOctree::~GISparseVoxelOctree()
{
	if(svoNodeResource) CUDA_CHECK(hipGraphicsUnregisterResource(svoNodeResource));
	if(svoMaterialResource) CUDA_CHECK(hipGraphicsUnregisterResource(svoMaterialResource));
	if(svoLevelOffsetResource) CUDA_CHECK(hipGraphicsUnregisterResource(svoLevelOffsetResource));
	if(tSVODense) CUDA_CHECK(hipDestroyTextureObject(tSVODense));
	if(denseArray) CUDA_CHECK(hipFreeArray(denseArray));
}

void GISparseVoxelOctree::LinkAllocators(Array32<GICudaAllocator*> newAllocators,
										 uint32_t totalCount,
										 const uint32_t levelCounts[])
{
	allocatorGrids.clear();
	allocators.resize(newAllocators.length);
	allocatorGrids.resize(newAllocators.length);

	assert(newAllocators.length > 0);
	assert(newAllocators.arr != nullptr);

	std::copy(newAllocators.arr, newAllocators.arr + newAllocators.length, allocators.data());
	for(unsigned int i = 0; i < newAllocators.length; i++)
		allocatorGrids[i] = &(newAllocators.arr[i]->GetVoxelGridHost());

	size_t sparseNodeCount = allocatorGrids[0]->depth + newAllocators.length - GI_DENSE_LEVEL;
	uint32_t totalLevel = allocatorGrids[0]->depth + newAllocators.length - 1;
	size_t totalAlloc = totalCount;

	// TODO: More Dynamic Allocation Scheme
	hSVOLevelTotalSizes.resize(sparseNodeCount);
	dSVOLevelTotalSizes.Resize(sparseNodeCount);
	dSVOLevelSizes.Resize(sparseNodeCount);
	hSVOLevelSizes.resize(sparseNodeCount);
	
	svoNodeBuffer.Resize(totalAlloc + GI_DENSE_SIZE_CUBE);
	svoLevelOffsets.Resize(sparseNodeCount);

	dSVODense = nullptr;
	dSVOSparse = nullptr;

	// Mat Tree holds up to level 0
	matSparseOffset = static_cast<unsigned int>((1.0 - std::pow(8.0f, GI_DENSE_LEVEL + 1)) / 
												(1.0f - 8.0f));
	svoMaterialBuffer.Resize(totalAlloc + matSparseOffset);
	
	// Register
	if(svoNodeResource) CUDA_CHECK(hipGraphicsUnregisterResource(svoNodeResource));
	if(svoMaterialResource) CUDA_CHECK(hipGraphicsUnregisterResource(svoMaterialResource));
	if(svoLevelOffsetResource) CUDA_CHECK(hipGraphicsUnregisterResource(svoLevelOffsetResource));
	CUDA_CHECK(hipGraphicsGLRegisterBuffer(&svoNodeResource, 
											svoNodeBuffer.getGLBuffer(), 
											cudaGLMapFlagsWriteDiscard));

	CUDA_CHECK(hipGraphicsGLRegisterBuffer(&svoMaterialResource, 
											svoMaterialBuffer.getGLBuffer(), 
											cudaGLMapFlagsWriteDiscard));

	CUDA_CHECK(hipGraphicsGLRegisterBuffer(&svoLevelOffsetResource,
											svoLevelOffsets.getGLBuffer(),
											cudaGLMapFlagsReadOnly));

	// Actual Data Init
	GLuint allOne = 0xFFFFFFFF;
	GLuint zero = 0;

	glBindBuffer(GL_COPY_WRITE_BUFFER, svoNodeBuffer.getGLBuffer());
	glClearBufferData(GL_COPY_WRITE_BUFFER, GL_R32UI, GL_RED_INTEGER, GL_UNSIGNED_INT, &allOne);

	glBindBuffer(GL_COPY_WRITE_BUFFER, svoMaterialBuffer.getGLBuffer());
	glClearBufferData(GL_COPY_WRITE_BUFFER, GL_R32UI, GL_RED_INTEGER, GL_UNSIGNED_INT, &zero);

	
	dSVOLevelSizes.Memset(0x00, 0, dSVOLevelSizes.Size());
	std::fill(hSVOLevelSizes.begin(), hSVOLevelSizes.end(), 0);
	std::copy(levelCounts + GI_DENSE_LEVEL, 
			  levelCounts + GI_DENSE_LEVEL + sparseNodeCount, 
			  hSVOLevelTotalSizes.data());
	hSVOLevelTotalSizes[0] = GI_DENSE_SIZE_CUBE;
	dSVOLevelTotalSizes = hSVOLevelTotalSizes;

	// SVO Constants set
	hSVOConstants.denseDepth = GI_DENSE_LEVEL;
	hSVOConstants.denseDim = GI_DENSE_SIZE;
	hSVOConstants.totalDepth = totalLevel;
	hSVOConstants.numCascades = newAllocators.length;

	// Offset Set
	uint32_t levelOffset = 0;
	svoLevelOffsets.CPUData().clear();
	for(unsigned int i = GI_DENSE_LEVEL; i <= totalLevel; i++)
	{
		svoLevelOffsets.AddData(levelOffset);
		levelOffset += (i != GI_DENSE_LEVEL) ? levelCounts[i] : 0;
	}
	svoLevelOffsets.SendData();
	assert(levelOffset <= totalCount);

	// Copy to device
	CUDA_CHECK(hipMemcpy(dSVOConstants.Data(), 
						  &hSVOConstants, 
						  sizeof(CSVOConstants), 
						  hipMemcpyHostToDevice));


	hipChannelFormatDesc fd = hipCreateChannelDesc<unsigned int>();
	if(denseArray) CUDA_CHECK(hipFreeArray(denseArray));
	CUDA_CHECK(hipMalloc3DArray(&denseArray,
								 &fd,
								 {GI_DENSE_SIZE, GI_DENSE_SIZE, GI_DENSE_SIZE},
								 hipArrayDefault));

	// Texture of SVO Dense
	hipResourceDesc resDesc = {};
	hipTextureDesc texDesc = {};
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = denseArray;
	
	texDesc.addressMode[0] = hipAddressModeWrap;
	texDesc.addressMode[1] = hipAddressModeWrap;
	texDesc.addressMode[2] = hipAddressModeWrap;
	texDesc.filterMode = hipFilterModePoint;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = 0;

	if(tSVODense != 0) CUDA_CHECK(hipDestroyTextureObject(tSVODense));
	CUDA_CHECK(hipCreateTextureObject(&tSVODense, &resDesc, &texDesc, nullptr));
}

void GISparseVoxelOctree::ConstructDense()
{
	// Level 0 does not gurantee lower cascade parents
	// Each Allocator tries to allocate its parent
	for(unsigned int i = 0; i < allocators.size(); i++)
	{
		uint32_t gridSize = ((allocators[i]->NumPages() * GI_PAGE_SIZE) + 
								GI_THREAD_PER_BLOCK - 1) /
								GI_THREAD_PER_BLOCK;
		SVOReconstructDetermineNode<<<gridSize, GI_THREAD_PER_BLOCK>>>
		(
			dSVODense,
			allocators[i]->GetVoxelPagesDevice(),
		
			i,
			*dSVOConstants.Data()
		);
		CUDA_KERNEL_CHECK();
	}

	uint32_t gridSize = ((GI_DENSE_SIZE_CUBE) + GI_THREAD_PER_BLOCK - 1) / GI_THREAD_PER_BLOCK;
	SVOReconstructAllocateLevel<<<gridSize, GI_THREAD_PER_BLOCK>>>
	(
		dSVODense,
		*(dSVOLevelSizes.Data() + 1),
		*(dSVOLevelTotalSizes.Data() + 1),
		*(dSVOLevelTotalSizes.Data()),
		*dSVOConstants.Data()
	);
	CUDA_KERNEL_CHECK();
}

void GISparseVoxelOctree::ConstructLevel(unsigned int currentLevel,
										 unsigned int allocatorOffset)
{
	// Early Bail check 
	unsigned int currentLevelIndex = currentLevel - GI_DENSE_LEVEL;
	CUDA_CHECK(hipMemcpy(hSVOLevelSizes.data() + currentLevelIndex,
						  dSVOLevelSizes.Data() + currentLevelIndex,
						  sizeof(unsigned int),
						  hipMemcpyDeviceToHost));
	if(hSVOLevelSizes[currentLevelIndex] == 0) return;

	// ChildBitSet your Level (with next level's child)
	// Allocate next level
	// Memcopy next level start location to array
	// Only ChildBitSet Upper Level
	// Then Allocate your level
	// Average Color to the level
	for(unsigned int i = allocatorOffset; i < allocators.size(); i++)
	{
		uint32_t gridSize = ((allocators[i]->NumPages() * GI_PAGE_SIZE) + 
							 GI_THREAD_PER_BLOCK - 1) /
							 GI_THREAD_PER_BLOCK;

		SVOReconstructDetermineNode<<<gridSize, GI_THREAD_PER_BLOCK>>>
		(
			dSVOSparse,
			tSVODense,
			allocators[i]->GetVoxelPagesDevice(),
			dSVOOffsets,

			i,
			currentLevel,
			*dSVOConstants.Data()
		);
		CUDA_KERNEL_CHECK();
	}
	
	uint32_t gridSize = (hSVOLevelSizes[currentLevelIndex] + GI_THREAD_PER_BLOCK - 1) / GI_THREAD_PER_BLOCK;
	SVOReconstructAllocateLevel<<<gridSize, GI_THREAD_PER_BLOCK>>>
	(
		dSVOSparse + svoLevelOffsets.CPUData()[currentLevelIndex],
		*(dSVOLevelSizes.Data() + currentLevelIndex + 1),
		*(dSVOLevelTotalSizes.Data() + currentLevelIndex + 1),
		*(dSVOLevelSizes.Data() + currentLevelIndex),
		*dSVOConstants.Data()
	);
	CUDA_KERNEL_CHECK();
}

void GISparseVoxelOctree::ConstructFullAtomic()
{
	// Fully Atomic Version
	for(unsigned int i = 0; i < allocators.size(); i++)
	{
		uint32_t gridSize = (allocators[i]->NumPages() * GI_PAGE_SIZE + GI_THREAD_PER_BLOCK - 1) /
							GI_THREAD_PER_BLOCK;
		SVOReconstruct<<<gridSize, GI_THREAD_PER_BLOCK>>>
		(
			dSVOMaterial,
			dSVOSparse,
			dSVODense,
			dSVOLevelSizes.Data(),

			dSVOOffsets,
			dSVOLevelTotalSizes.Data(),
				
			// VoxelSystem Data
			allocators[i]->GetVoxelPagesDevice(),
			allocators[i]->GetObjRenderCacheDevice(),

			matSparseOffset,
			i,
			*dSVOConstants.Data()
		);
		CUDA_KERNEL_CHECK();
	}
	// Copy Level Sizes
	CUDA_CHECK(hipMemcpy(hSVOLevelSizes.data(),
						  dSVOLevelSizes.Data(),
						  hSVOLevelSizes.size() * sizeof(uint32_t),
						  hipMemcpyDeviceToHost));

	// Full atomic reconst does not use cuda3d tex
	//// Copy Dense to Texture
	//hipMemcpy3DParms params = { 0 };
	//params.dstArray = denseArray;
	//params.srcPtr =
	//{
	//	dSVODense,
	//	GI_DENSE_SIZE * sizeof(unsigned int),
	//	GI_DENSE_SIZE,
	//	GI_DENSE_SIZE
	//};
	//params.extent = { GI_DENSE_SIZE, GI_DENSE_SIZE, GI_DENSE_SIZE };
	//params.kind = hipMemcpyDeviceToDevice;
	//CUDA_CHECK(hipMemcpy3D(&params));
}

void GISparseVoxelOctree::ConstructLevelByLevel()
{
	// Start with constructing dense
	ConstructDense();

	// Copy Dense to Texture
	hipMemcpy3DParms params = { 0 };
	params.dstArray = denseArray;
	params.srcPtr =
	{
		dSVODense,
		GI_DENSE_SIZE * sizeof(unsigned int),
		GI_DENSE_SIZE,
		GI_DENSE_SIZE
	};
	params.extent = { GI_DENSE_SIZE, GI_DENSE_SIZE, GI_DENSE_SIZE };
	params.kind = hipMemcpyDeviceToDevice;
	CUDA_CHECK(hipMemcpy3D(&params));

	// Construct Levels
	for(unsigned int i = GI_DENSE_LEVEL + 1; i < allocatorGrids[0]->depth; i++)
	{
		ConstructLevel(i, 0);
	}

	// Now adding cascade levels
	for(unsigned int i = 1; i < allocators.size(); i++)
	{
		unsigned int currentLevel = allocatorGrids[0]->depth + i - 1;
		ConstructLevel(currentLevel, i);
	}

	// Memcpy Last Total Size
	CUDA_CHECK(hipMemcpy(hSVOLevelSizes.data() + (hSVOConstants.totalDepth - GI_DENSE_LEVEL),
						  dSVOLevelSizes.Data() + (hSVOConstants.totalDepth - GI_DENSE_LEVEL),
						  sizeof(uint32_t),
						  hipMemcpyDeviceToHost));
}

void GISparseVoxelOctree::AverageNodes(bool skipLeaf)
{
	// First Average Leafs atomically
	if(!skipLeaf)
	for(unsigned int i = 0; i < allocators.size(); i++)
	{
		assert(allocators[i]->IsGLMapped() == true);
		uint32_t gridSize = (allocators[i]->NumPages() * GI_PAGE_SIZE +  GI_THREAD_PER_BLOCK - 1) / 
							GI_THREAD_PER_BLOCK;
				
		// Average Leaf Node
		SVOReconstructMaterialLeaf<<<gridSize, GI_THREAD_PER_BLOCK>>>
		(
			dSVOMaterial,

			// Const SVO Data
			dSVOSparse,
			dSVOOffsets,
			tSVODense,

			// Page Data
			allocators[i]->GetVoxelPagesDevice(),
										  
			// For Color Lookup
			allocators[i]->GetObjRenderCacheDevice(),

			// Constants
			matSparseOffset,
			i,
			*dSVOConstants.Data()
		);
		CUDA_KERNEL_CHECK();
	}

	// Now use leaf nodes to average upper nodes
	// Start bottom up
	for(int i = hSVOConstants.totalDepth - 1; i >= static_cast<int>(hSVOConstants.denseDepth); i--)
	{
		unsigned int arrayIndex = i - GI_DENSE_LEVEL;
		unsigned int levelDim = GI_DENSE_SIZE >> (GI_DENSE_LEVEL - i);
		unsigned int levelSize = (i > GI_DENSE_LEVEL) ? hSVOLevelSizes[arrayIndex] : 
														levelDim * levelDim * levelDim;
		if(levelSize == 0) continue;

		uint32_t gridSize = ((levelSize * GI_NODE_THREAD_COUNT) + GI_THREAD_PER_BLOCK - 1) /
							GI_THREAD_PER_BLOCK;
		
		// Average Level
		SVOReconstructAverageNode<<<gridSize, GI_THREAD_PER_BLOCK>>>
		(
			dSVOMaterial,
			dSVODense,
			dSVOSparse,

			*(dSVOOffsets + arrayIndex),
			*(dSVOOffsets + arrayIndex + 1),

			levelSize,
			matSparseOffset,
			i,
			*dSVOConstants.Data()
		);
		CUDA_KERNEL_CHECK();
	}
	// Call once for all lower levels
}

double GISparseVoxelOctree::UpdateSVO()
{
	CUDA_CHECK(hipProfilerStart());

	CUDA_CHECK(hipGraphicsMapResources(1, &svoMaterialResource));
	CUDA_CHECK(hipGraphicsMapResources(1, &svoNodeResource));
	CUDA_CHECK(hipGraphicsMapResources(1, &svoLevelOffsetResource));
	
	size_t size;
	CUDA_CHECK(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&dSVODense), 
													 &size, svoNodeResource));
	CUDA_CHECK(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&dSVOMaterial),
													 &size, svoMaterialResource));
	CUDA_CHECK(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&dSVOOffsets),
													&size, svoLevelOffsetResource));
	dSVOSparse = dSVODense + GI_DENSE_SIZE_CUBE;

	CudaTimer timer;
	timer.Start();

	// Reset Atomic Counter since we reconstruct every frame
	uint32_t usedNodeCount = hSVOLevelSizes.back() + svoLevelOffsets.CPUData().back();
	CUDA_CHECK(hipMemset(dSVODense, 0xFF, sizeof(CSVONode) * (usedNodeCount + GI_DENSE_SIZE_CUBE)));
	CUDA_CHECK(hipMemset(dSVOMaterial, 0x00, sizeof(CSVOMaterial) * (usedNodeCount + matSparseOffset)));

	dSVOLevelSizes.Memset(0x00, 0, dSVOLevelSizes.Size());
	std::fill(hSVOLevelSizes.begin(), hSVOLevelSizes.end(), 0);

	// Maxwell is faster with fully atomic code (CAS Locks etc.)
	// However kepler sucks(660ti) (100ms compared to 5ms) 
	if(CudaInit::CapabilityMajor() >= 6)
	{
		ConstructFullAtomic();
		AverageNodes(true);
	}
		
	else
	{
		ConstructLevelByLevel();
		AverageNodes(false);
	}
	
	//// DEBUG
	//GI_LOG("-------------------------------------------");
	//GI_LOG("Tree Node Data");
	//unsigned int i;
	//for(i = 0; i <= allocatorGrids[0]->depth - GI_DENSE_LEVEL + allocators.size() - 1; i++)
	//{
	//	if(i == 0) GI_LOG("#%d Dense : %d", GI_DENSE_LEVEL + i, GI_DENSE_SIZE_CUBE);
	//	else GI_LOG("#%d Level : %d", GI_DENSE_LEVEL + i, hSVOLevelSizes[i]);
	//}
	//unsigned int total = std::accumulate(hSVOLevelSizes.begin(),
	//									 hSVOLevelSizes.end(), 0);
	//GI_LOG("Total : %d", total);

	timer.Stop();
	CUDA_CHECK(hipGraphicsUnmapResources(1, &svoMaterialResource));
	CUDA_CHECK(hipGraphicsUnmapResources(1, &svoNodeResource));
	CUDA_CHECK(hipGraphicsUnmapResources(1, &svoLevelOffsetResource));

	CUDA_CHECK(hipProfilerStop());
	return timer.ElapsedMilliS();
}

double GISparseVoxelOctree::ConeTrace(GLuint depthBuffer,
									  GLuint normalBuffer,
									  GLuint colorBuffer,
									  const Camera& camera)
{
	return 0.0;
}

double GISparseVoxelOctree::DebugTraceSVO(GLuint writeImage,
										  StructuredBuffer<InvFrameTransform>& invFT,
										  FrameTransformBuffer& ft,
										  const uint2& imgDim,
										  uint32_t renderLevel)
{
	// Timing Voxelization Process
	GLuint queryID;
	glGenQueries(1, &queryID);
	glBeginQuery(GL_TIME_ELAPSED, queryID);

	// Set Cascade Trace Data
	float3 pos = allocatorGrids[0]->position;
	uint32_t dim = allocatorGrids[0]->dimension.x * (0x1 << (allocators.size() - 1));
	uint32_t depth = allocatorGrids[0]->depth + static_cast<uint32_t>(allocators.size()) - 1;
	svoTraceData.CPUData()[0] = 
	{
		{pos.x, pos.y, pos.z, allocatorGrids.back()->span},
		{dim, depth, GI_DENSE_SIZE, GI_DENSE_LEVEL},
		{
			static_cast<unsigned int>(allocators.size()), 
			GI_DENSE_SIZE * GI_DENSE_SIZE * GI_DENSE_SIZE,
			matSparseOffset,
			renderLevel
		}
	};
	svoTraceData.SendData();

	// Shaders
	computeVoxTraceWorld.Bind();

	// Buffers
	svoNodeBuffer.BindAsShaderStorageBuffer(LU_SVO_NODE);
	svoMaterialBuffer.BindAsShaderStorageBuffer(LU_SVO_MATERIAL);
	svoLevelOffsets.BindAsShaderStorageBuffer(LU_SVO_LEVEL_OFFSET);
	invFT.BindAsUniformBuffer(U_INVFTRANSFORM);
	ft.Bind();
	svoTraceData.BindAsUniformBuffer(U_SVO_CONSTANTS);

	// Images
	glBindImageTexture(I_COLOR_FB, writeImage, 0, false, 0, GL_WRITE_ONLY, GL_RGBA8);

	// Dispatch
	uint2 gridSize;
	gridSize.x = (imgDim.x + 16 - 1) / 16;
	gridSize.y = (imgDim.y + 16 - 1) / 16;
	glDispatchCompute(gridSize.x, gridSize.y, 1);
	
	// Timer
	GLuint64 timeElapsed = 0;
	glEndQuery(GL_TIME_ELAPSED);
	glGetQueryObjectui64v(queryID, GL_QUERY_RESULT, &timeElapsed);
	glMemoryBarrier(GL_SHADER_IMAGE_ACCESS_BARRIER_BIT);

	// I have to unbind the compute shader or weird things happen
	Shader::Unbind(ShaderType::COMPUTE);
	return timeElapsed / 1000000.0;
}

uint64_t GISparseVoxelOctree::MemoryUsage() const
{
	uint64_t totalBytes = 0;
	totalBytes += svoNodeBuffer.Capacity() * sizeof(CSVONode);
	totalBytes += svoMaterialBuffer.Capacity() * sizeof(CSVOMaterial);
	totalBytes += dSVOLevelSizes.Size() * sizeof(unsigned int);
	totalBytes += sizeof(unsigned int);
	return totalBytes;
}

const CSVOConstants& GISparseVoxelOctree::SVOConsts() const
{
	return hSVOConstants;
}