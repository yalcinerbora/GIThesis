#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "COpenGLCommon.cuh"
#include "CVoxel.cuh"
#include "GICudaAllocator.h"
#include "CHash.cuh"

#define GI_MAX_JOINT_COUNT GI_MAX_SHARED_COUNT

inline __device__ void LoadTransformData(// Shared Mem
										 CMatrix4x4* sTransformMatrices,
										 CMatrix3x3* sRotationMatrices,
										 uint8_t* sMatrixLookup,

										 // Object Transform Matrix
										 CObjectTransform** gObjTransforms,
										 CObjectTransform** gJointTransforms,
										 uint32_t** gObjTransformIds,

										 // Current Voxel Weight
										 const uchar4& voxelWeightIndex,

										 // Object Type that will be broadcasted
										 const CVoxelObjectType& objType,
										 const uint16_t& objId,
										 const uint16_t& batchId)
{
	unsigned int blockLocalId = threadIdx.x;

	// transform Id Fetched only by first warp
	unsigned int transformId = 0;
	if(blockLocalId < warpSize)
		transformId = gObjTransformIds[batchId][objId];

	// Here we will load transform and rotation matrices
	// Each thread will load 1 float. There is two 4x4 matrix
	// 32 floats will be loaded
	// Just enough for a warp to do the work
	// Load matrices (4 byte load by each thread sequential no bank conflict)
	if(blockLocalId < 16)
	{
		unsigned int columnId = blockLocalId / 4;
		unsigned int rowId = blockLocalId % 4;
		reinterpret_cast<float*>(&sTransformMatrices[0].column[columnId])[rowId] =
			reinterpret_cast<float*>(&gObjTransforms[batchId][transformId].transform.column[columnId])[rowId];
	}
	else if(blockLocalId < 28)
	{
		unsigned int rotationId = blockLocalId - 16;
		unsigned int columnId = rotationId / 3;
		unsigned int rowId = rotationId % 3;
		reinterpret_cast<float*>(&sRotationMatrices[0].column[columnId])[rowId] =
			reinterpret_cast<float*>(&gObjTransforms[batchId][transformId].rotation.column[columnId])[rowId];
	}

	// Load Joint Transforms if Skeletal Object
	if(objType == CVoxelObjectType::SKEL_DYNAMIC)
	{
		// All valid objects will request matrix load
		// then entire block will try to load it
		// Max skeleton bone count is 64
		// Worst case 64 * 16 = 1024 float will be loaded to sMem
		// Some blocks will load twice
		// However its extremely rare (even impossible case)
		// In a realistic scenario (and if a segment holds adjacent voxels)
		// And if max bone influence per vertex is around 4 
		// there should be at most 8

		// Matrix Lookup Initialize
		if(blockLocalId < GI_MAX_JOINT_COUNT)
			sMatrixLookup[blockLocalId] = 0;
		__syncthreads();

		if(voxelWeightIndex.x != 0xFF) sMatrixLookup[voxelWeightIndex.x] = 1;
		if(voxelWeightIndex.y != 0xFF) sMatrixLookup[voxelWeightIndex.y] = 1;
		if(voxelWeightIndex.z != 0xFF) sMatrixLookup[voxelWeightIndex.z] = 1;
		if(voxelWeightIndex.w != 0xFF) sMatrixLookup[voxelWeightIndex.w] = 1;
		__syncthreads();

		// Lookup Tables are Loaded
		// Theorethical 63 Matrices will be loaded
		//	Each thread will load 1 float we need 1024 threads
		unsigned int iterationCount = (GI_MAX_JOINT_COUNT * 16) / blockDim.x;
		unsigned int matricesPerIteration = blockDim.x / 16;
		for(unsigned int i = 0; i < iterationCount; i++)
		{
			if(blockLocalId + (matricesPerIteration * i) < (GI_MAX_JOINT_COUNT * 16))
			{
				unsigned int sharedLoc = (blockLocalId / 16) + matricesPerIteration * i + 1;
				if(sMatrixLookup[sharedLoc - 1] == 1)
				{
					unsigned int column = (blockLocalId / 4) % 4;
					unsigned int row = blockLocalId % 4;

					// Transform
					reinterpret_cast<float*>(&sTransformMatrices[sharedLoc].column[column])[row] =
						reinterpret_cast<float*>(&gJointTransforms[batchId][sharedLoc - 1].transform.column[column])[row];
				}
			}
			// Rotation
			if(blockLocalId + (matricesPerIteration * i) < (GI_MAX_JOINT_COUNT * 9))
			{
				unsigned int sharedLoc = (blockLocalId / 9) + matricesPerIteration * i + 1;
				if(sMatrixLookup[sharedLoc - 1] == 1)
				{
					unsigned int column = (blockLocalId / 3) % 3;
					unsigned int row = blockLocalId % 3;

					reinterpret_cast<float*>(&sRotationMatrices[sharedLoc].column[column])[row] =
						reinterpret_cast<float*>(&gJointTransforms[batchId][sharedLoc - 1].rotation.column[column])[row];
				}
			}
		}

		//// Inefficient Test Code
		//if(blockLocalId < GI_MAX_JOINT_COUNT)
		//{
		//	if(sMatrixLookup[blockLocalId] == 1)
		//	{
		//		sTransformMatrices[blockLocalId + 1] = gJointTransforms[batchId][blockLocalId].transform;

		//		//sTransformMatrices[blockLocalId + 1] = CMatrix4x4
		//		//{{
		//		//	{1.0f, 0.0f, 0.0f, 0.0f},
		//		//	{0.0f, 1.0f, 0.0f, 0.0f},
		//		//	{0.0f, 0.0f, 1.0f, 0.0f},
		//		//	{0.0f, 0.0f, 0.0f, 1.0f}
		//		//}};

		//		//sRotationMatrices[blockLocalId + 1] = gJointTransforms[objectId.y][blockLocalId].rotation;
		//	}
		//}
	}

	// We write to shared mem sync between warps
	__syncthreads();
}

__global__ void VoxelTransform(// Voxel Pages
							   CVoxelPage* gVoxelData,
							   const CVoxelGrid& gGridInfo,
							   const float3 hNewGridPosition,

							   // Object Related
							   CObjectTransform** gObjTransforms,
							   CObjectTransform** gJointTransforms,
							   uint32_t** gObjTransformIds,

							   // Cache
							   CVoxelNormPos** gVoxNormPosCacheData,
							   CVoxelColor** gVoxRenderData,
							   CVoxelWeight** gVoxWeightData,

							   CObjectVoxelInfo** gObjInfo,	
							   CObjectAABB** gObjectAABB)
{
	// Cache Loading
	// Shared Memory which used for transform rendering
	__shared__ CMatrix4x4 sTransformMatrices[GI_MAX_JOINT_COUNT + 1];	// First index holds model matrix
	__shared__ CMatrix3x3 sRotationMatrices[GI_MAX_JOINT_COUNT + 1];
	__shared__ uint8_t sMatrixLookup[GI_MAX_JOINT_COUNT];

	unsigned int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int pageId = globalId / GI_PAGE_SIZE;
	unsigned int pageLocalId = globalId % GI_PAGE_SIZE;
	unsigned int pageLocalSegmentId = pageLocalId / GI_SEGMENT_SIZE;
	unsigned int segmentLocalVoxId = pageLocalId % GI_SEGMENT_SIZE;

	// Get Segments Obj Information Struct
	SegmentObjData segObj = gVoxelData[pageId].dSegmentObjData[pageLocalSegmentId];
	CVoxelObjectType objType;
	uint16_t segLoad;
	SegmentOccupation segOccup;
	ExpandSegmentPacked(objType, segOccup, segLoad, segObj.packed);

	if(segOccup == SegmentOccupation::EMPTY) return;
	assert(segOccup != SegmentOccupation::MARKED_FOR_CLEAR);
	
	// Calculate your Object VoxelId
	unsigned int cacheVoxelId = segObj.voxStride + segmentLocalVoxId;
	
	CVoxelWeight weights = {{0x00, 0x00, 0x00, 0x00}, {0xFF, 0xFF, 0xFF, 0xFF}};
	if(segmentLocalVoxId < segLoad &&
	   objType == CVoxelObjectType::SKEL_DYNAMIC)
	   weights = gVoxWeightData[segObj.batchId][cacheVoxelId];
	
	// Segment is occupied so load matrices before culling unused warps
	LoadTransformData(// Shared Mem
					  sTransformMatrices,
					  sRotationMatrices,
					  sMatrixLookup,

					  // Object Transform Matrix
					  gObjTransforms,
					  gJointTransforms,
					  gObjTransformIds,

					  // Weight Index
					  weights.weightIndex,

					  // Object Type that will be broadcasted
					  objType,
					  segObj.objId,
					  segObj.batchId);

	// Now we can cull unused threads
	if(segmentLocalVoxId >= segLoad) return;

	// Fetch NormalPos from cache
	uint3 voxPos;
	float3 normal;
	float4 normalWithOcc;
	bool isMip;
	ExpandNormalPos(voxPos, normalWithOcc, isMip, gVoxNormPosCacheData[segObj.batchId][cacheVoxelId]);
	normal = {normalWithOcc.x, normalWithOcc.y, normalWithOcc.z};

	// Fetch AABB min, transform and span
	float4 objAABBMin = gObjectAABB[segObj.batchId][segObj.objId].min;
	float objSpan = gObjInfo[segObj.batchId][segObj.objId].span;

	// Generate World Position
	// start with object space position
	float3 worldPos;
	worldPos.x = objAABBMin.x + voxPos.x * objSpan;
	worldPos.y = objAABBMin.y + voxPos.y * objSpan;
	worldPos.z = objAABBMin.z + voxPos.z * objSpan;

	// Transformations
	// Model multiplication
	MultMatrixSelf(worldPos, sTransformMatrices[0]);
	MultMatrixSelf(normal, sRotationMatrices[0]);
	//// Unoptimized Matrix Load
	//CMatrix4x4 transform = gObjTransforms[segObj.batchId][gObjTransformIds[segObj.batchId][segObj.objId]].transform;
	//CMatrix4x4 rotation = gObjTransforms[segObj.batchId][gObjTransformIds[segObj.batchId][segObj.objId]].transform;
	//MultMatrixSelf(worldPos, transform);
	//MultMatrixSelf(normal, rotation);

	if(objType == CVoxelObjectType::SKEL_DYNAMIC)
	{
		float4 weightUnorm;
		weightUnorm.x = static_cast<float>(weights.weight.x) / 255.0f;
		weightUnorm.y = static_cast<float>(weights.weight.y) / 255.0f;
		weightUnorm.z = static_cast<float>(weights.weight.z) / 255.0f;
		weightUnorm.w = static_cast<float>(weights.weight.w) / 255.0f;

		//if(threadIdx.x == 0)
		//	printf("x %d, y %d, z %d, w %d\n",
		//	weights.weightIndex.x,
		//	weights.weightIndex.y,
		//	weights.weightIndex.z,
		//	weights.weightIndex.w);

		assert(weights.weightIndex.x <= 24);
		assert(weights.weightIndex.y <= 24);
		assert(weights.weightIndex.z <= 24);
		assert(weights.weightIndex.w <= 24);

		float3 pos = {0.0f, 0.0f, 0.0f};
		float3 p = MultMatrix(worldPos, sTransformMatrices[weights.weightIndex.x + 1]);
		pos.x += weightUnorm.x * p.x;
		pos.y += weightUnorm.x * p.y;
		pos.z += weightUnorm.x * p.z;

		p = MultMatrix(worldPos, sTransformMatrices[weights.weightIndex.y + 1]);
		pos.x += weightUnorm.y * p.x;
		pos.y += weightUnorm.y * p.y;
		pos.z += weightUnorm.y * p.z;

		p = MultMatrix(worldPos, sTransformMatrices[weights.weightIndex.z + 1]);
		pos.x += weightUnorm.z * p.x;
		pos.y += weightUnorm.z * p.y;
		pos.z += weightUnorm.z * p.z;

		p = MultMatrix(worldPos, sTransformMatrices[weights.weightIndex.w + 1]);
		pos.x += weightUnorm.w * p.x;
		pos.y += weightUnorm.w * p.y;
		pos.z += weightUnorm.w * p.z;

		worldPos = pos;


		float3 norm = {0.0f, 0.0f, 0.0f};
		float3 n = MultMatrix(normal, sRotationMatrices[weights.weightIndex.x + 1]);
		norm.x += weightUnorm.x * n.x;
		norm.y += weightUnorm.x * n.y;
		norm.z += weightUnorm.x * n.z;

		n = MultMatrix(normal, sRotationMatrices[weights.weightIndex.y + 1]);
		norm.x += weightUnorm.y * n.x;
		norm.y += weightUnorm.y * n.y;
		norm.z += weightUnorm.y * n.z;

		n = MultMatrix(normal, sRotationMatrices[weights.weightIndex.z + 1]);
		norm.x += weightUnorm.z * n.x;
		norm.y += weightUnorm.z * n.y;
		norm.z += weightUnorm.z * n.z;

		n = MultMatrix(normal, sRotationMatrices[weights.weightIndex.w + 1]);
		norm.x += weightUnorm.w * n.x;
		norm.y += weightUnorm.w * n.y;
		norm.z += weightUnorm.w * n.z;

		normal = norm;
	}

	// Reconstruct Voxel Indices relative to the new pos of the grid
	worldPos.x -= hNewGridPosition.x;
	worldPos.y -= hNewGridPosition.y;
	worldPos.z -= hNewGridPosition.z;

	bool outOfBounds;
	outOfBounds = (worldPos.x < 0.0f) || (worldPos.x >= gGridInfo.dimension.x * gGridInfo.span);
	outOfBounds |= (worldPos.y < 0.0f) || (worldPos.y >= gGridInfo.dimension.y * gGridInfo.span);
	outOfBounds |= (worldPos.z < 0.0f) || (worldPos.z >= gGridInfo.dimension.z * gGridInfo.span);

	// If its mip dont update inner cascade
	bool inInnerCascade = false;
	if(isMip)
	{
		inInnerCascade = (worldPos.x > gGridInfo.dimension.x * gGridInfo.span * 0.25f) &&
						 (worldPos.x < gGridInfo.dimension.x * gGridInfo.span * 0.75f);
		inInnerCascade &= (worldPos.y > gGridInfo.dimension.y * gGridInfo.span * 0.25f) &&
						  (worldPos.y < gGridInfo.dimension.y * gGridInfo.span * 0.75f);
		inInnerCascade &= (worldPos.z > gGridInfo.dimension.z * gGridInfo.span * 0.25f) &&
						  (worldPos.z < gGridInfo.dimension.z * gGridInfo.span * 0.75f);
	}
	outOfBounds |= inInnerCascade;

	// Now Write
	// Discard the out of bound voxels
	if(!outOfBounds)
	{
		float invSpan = 1.0f / gGridInfo.span;
		voxPos.x = static_cast<unsigned int>(worldPos.x * invSpan + 0.5f);
		voxPos.y = static_cast<unsigned int>(worldPos.y * invSpan + 0.5f);
		voxPos.z = static_cast<unsigned int>(worldPos.z * invSpan + 0.5f);

		// Write to page
		uint2 packedVoxNormPos;
		normalWithOcc = {normal.x, normal.y, normal.z, 0.0f};
		PackVoxelNormPos(packedVoxNormPos, voxPos, normalWithOcc, isMip);
		gVoxelData[pageId].dGridVoxPos[pageLocalId] = packedVoxNormPos.x;
		gVoxelData[pageId].dGridVoxNorm[pageLocalId] = packedVoxNormPos.y;
	}
	else
	{
		gVoxelData[pageId].dGridVoxPos[pageLocalId] = 0xFFFFFFFF;
		gVoxelData[pageId].dGridVoxNorm[pageLocalId] = 0xFFFFFFFF;
	}
}