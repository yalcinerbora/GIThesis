#include "hip/hip_runtime.h"
#include "GICudaAllocator.h"
#include "GICudaStructMatching.h"
#include <cuda_gl_interop.h>
#include "CudaTimer.h"
#include "Macros.h"

// Small Helper Kernel That used to init inital obj Pages
// Logic is per segment
__global__ void EmptyPageInit(unsigned int* gPageEmptySegmentPos)
{
	unsigned int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	if(globalId >= GI_SEGMENT_PER_PAGE) return;
	gPageEmptySegmentPos[globalId] = globalId;
}

__global__ void SegmentAllocLocInit(ushort2* gSegments,
									const uint32_t segmentCount)
{
	unsigned int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	if(globalId >= segmentCount) return;
	gSegments[globalId].x = 0xFFFF;
	gSegments[globalId].y = 0xFFFF;
}

// Small Helper Kernel That used to determine total segment size used by the object batch
// Logic per object in batch
__global__ void DetermineTotalSegment(int* dTotalSegmentCount,

									  // Grid Related
									  const CVoxelGrid& gGridInfo,

									  // Per object Related
									  unsigned int* gObjectVoxStrides,
									  unsigned int* gObjectAllocIndexLookup,
									  const CObjectVoxelInfo* gVoxelInfo,
									  const CObjectTransform* gObjTransforms,
									  const uint32_t objCount)
{
	unsigned int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	if(globalId >= objCount) return;

	// Determine Segment Count and add do total segment counter

	// We need to check scaling and adjust span
	// Objects may have different voxel sizes and voxel sizes may change after scaling
	float3 scaling = ExtractScaleInfo(gObjTransforms[globalId].transform);
	assert(scaling.x == scaling.y);
	assert(scaling.y == scaling.z);

	unsigned int voxelDim = static_cast<unsigned int>(gVoxelInfo[globalId].span * scaling.x / gGridInfo.span);
	unsigned int voxScale = voxelDim == 0 ? 0 : 1;
	unsigned int segmentCount = ((gVoxelInfo[globalId].voxelCount * voxScale) + GI_SEGMENT_SIZE - 1) / GI_SEGMENT_SIZE;
	
	// Determine Strides
	// Here this implementation is slow and does redundant work 
	// but its most easily written version
	unsigned int objStirde = 0, objIndexLookup = 0;
	for(unsigned int i = 0; i < globalId; i++)
	{
		float3 scalingObj = ExtractScaleInfo(gObjTransforms[i].transform);
		assert(scalingObj.x == scalingObj.y);
		assert(scalingObj.y == scalingObj.z);

		unsigned int voxelDim = static_cast<unsigned int>(gVoxelInfo[i].span * scaling.x / gGridInfo.span);
		unsigned int voxScaleObj = voxelDim == 0 ? 0 : 1;
		
		objStirde += gVoxelInfo[i].voxelCount * voxScaleObj;
		objIndexLookup += ((gVoxelInfo[i].voxelCount * voxScaleObj) + GI_SEGMENT_SIZE - 1) / GI_SEGMENT_SIZE;
	}

	if(globalId == objCount - 1)
		dTotalSegmentCount[0] = objIndexLookup + segmentCount;

	gObjectVoxStrides[globalId] = objStirde;
	gObjectAllocIndexLookup[globalId] = objIndexLookup;
}

// Used to populate segment object id's
// Logic per object in batch
__global__ void DetermineSegmentObjId(unsigned int* gSegmentObjectId,

									  // Grid Related
									  const CVoxelGrid& gGridInfo,

									  const unsigned int* gObjectAllocIndexLookup,
									  const CObjectVoxelInfo* gVoxelInfo,
									  const CObjectTransform* gObjTransforms,
									  const uint32_t objCount)
{
	unsigned int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	if(globalId >= objCount) return;

	float3 scaling = ExtractScaleInfo(gObjTransforms[globalId].transform);
	assert(scaling.x == scaling.y);
	assert(scaling.y == scaling.z);
	unsigned int voxelDim = static_cast<unsigned int>(gVoxelInfo[globalId].span * scaling.x / gGridInfo.span);
	unsigned int voxScale = voxelDim == 0 ? 0 : 1;

	unsigned int segmentCount = ((gVoxelInfo[globalId].voxelCount * voxScale) + GI_SEGMENT_SIZE - 1) / GI_SEGMENT_SIZE;
	for(unsigned int i = 0; i < segmentCount; i++)
	{
		gSegmentObjectId[gObjectAllocIndexLookup[globalId] + i] = globalId;
	}
}


GICudaAllocator::GICudaAllocator(const CVoxelGrid& gridInfo)
	: totalObjectCount(0)
	, dVoxelGridInfo(1)
	, hVoxelGridInfo(gridInfo)
{
	cudaGLSetGLDevice(0);
	dVoxelGridInfo.Assign(0, hVoxelGridInfo);
}

void GICudaAllocator::LinkOGLVoxelCache(GLuint batchAABBBuffer,
										GLuint batchTransformBuffer,
										GLuint relativeTransformBuffer,
										GLuint infoBuffer,
										GLuint voxelBuffer,
										GLuint voxelRenderBuffer,
										uint32_t objCount,
										uint32_t voxelCount)
{
	hipError_t cudaErr;
	CudaTimer timer(0);
	timer.Start();

	rTransformLinks.emplace_back();
	transformLinks.emplace_back();
	aabbLinks.emplace_back();
	objectInfoLinks.emplace_back();
	cacheLinks.emplace_back();
	cacheRenderLinks.emplace_back();

	hipGraphicsGLRegisterBuffer(&rTransformLinks.back(), relativeTransformBuffer, cudaGraphicsMapFlagsReadOnly);
	hipGraphicsGLRegisterBuffer(&transformLinks.back(), batchTransformBuffer, cudaGraphicsMapFlagsReadOnly);
	hipGraphicsGLRegisterBuffer(&aabbLinks.back(), batchAABBBuffer, cudaGraphicsMapFlagsReadOnly);
	hipGraphicsGLRegisterBuffer(&objectInfoLinks.back(), infoBuffer, cudaGraphicsMapFlagsReadOnly);

	hipGraphicsGLRegisterBuffer(&cacheLinks.back(), voxelBuffer, cudaGraphicsMapFlagsReadOnly);
	hipGraphicsGLRegisterBuffer(&cacheRenderLinks.back(), voxelRenderBuffer, cudaGraphicsMapFlagsReadOnly);

	objectCounts.emplace_back(objCount);
	voxelCounts.emplace_back(voxelCount);
	totalObjectCount += objCount;

	// Allocate Helper Data
	dVoxelStrides.emplace_back(objCount);
	dObjectAllocationIndexLookup.emplace_back(objCount);
	dWriteSignals.emplace_back(objCount);

	dVoxelStrides.back().Memset(0, 0, dVoxelStrides.back().Size());
	dObjectAllocationIndexLookup.back().Memset(0, 0, dObjectAllocationIndexLookup.back().Size());
	dWriteSignals.back().Memset(0, 0, dWriteSignals.back().Size());
	
	// Populate Helper Data
	// Determine object segement sizes
	int* dTotalCount = nullptr;
	int hTotalCount = 0;
	cudaErr = hipMalloc(reinterpret_cast<void**>(&dTotalCount), sizeof(int));
	cudaErr = hipMemcpy(dTotalCount, &hTotalCount, sizeof(int), hipMemcpyHostToDevice);

	// Mapping Pointer
	CObjectVoxelInfo* dVoxelInfo = nullptr;
	CObjectTransform* dObjTransform = nullptr;
	size_t size = 0;
	hipGraphicsMapResources(1, &objectInfoLinks.back());
	hipGraphicsMapResources(1, &transformLinks.back());
	hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&dVoxelInfo), &size, objectInfoLinks.back());
	hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&dObjTransform), &size, transformLinks.back());
	
	uint32_t gridSize = static_cast<uint32_t>((objCount + GI_THREAD_PER_BLOCK - 1) / GI_THREAD_PER_BLOCK);
	DetermineTotalSegment<<<gridSize, GI_THREAD_PER_BLOCK>>>(dTotalCount,
															 *dVoxelGridInfo.Data(),

															 dVoxelStrides.back().Data(),
															 dObjectAllocationIndexLookup.back().Data(),
															 dVoxelInfo,
															 dObjTransform,
															 objCount);
	
	// Allocation after determining total index count
	hipMemcpy(&hTotalCount, dTotalCount, sizeof(int), hipMemcpyDeviceToHost);
	dSegmentObjecId.emplace_back(hTotalCount);
	dSegmentAllocLoc.emplace_back(hTotalCount);
	dSegmentAllocLoc.back().Memset(0xFF, 0, dSegmentAllocLoc.back().Size());

	gridSize = static_cast<uint32_t>((objCount + GI_THREAD_PER_BLOCK - 1) / GI_THREAD_PER_BLOCK);
	DetermineSegmentObjId<<<gridSize, GI_THREAD_PER_BLOCK>>>(dSegmentObjecId.back().Data(),
															 *dVoxelGridInfo.Data(),

															 dObjectAllocationIndexLookup.back().Data(),
															 dVoxelInfo,
															 dObjTransform,
															 objCount);
	hipGraphicsUnmapResources(1, &objectInfoLinks.back());
	hipGraphicsMapResources(1, &transformLinks.back());

	dObjectAllocationIndexLookup2D.InsertEnd(dObjectAllocationIndexLookup.back().Data());
	dSegmentAllocLoc2D.InsertEnd(dSegmentAllocLoc.back().Data());
	dObjectVoxStrides2D.InsertEnd(dVoxelStrides.back().Data());
	hipFree(dTotalCount);
	timer.Stop();
	GI_LOG("Linked Object Batch to CUDA. Elaped time %f ms", timer.ElapsedMilliS());

	// Dump Initial Helper Files
	//dSegmentAllocLoc.back().DumpToFile("segAllocLoc");
	//dSegmentObjecId.back().DumpToFile("segObjId");

	//dVoxelStrides.back().DumpToFile("voxStride");
	//dObjectAllocationIndexLookup.back().DumpToFile("allocIndexLookup");
	//dWriteSignals.back().DumpToFile("writeSignals");

	assert(rTransformLinks.size() == transformLinks.size());
	assert(transformLinks.size() == aabbLinks.size());
	assert(aabbLinks.size() == transformLinks.size());
	assert(rTransformLinks.size() == objectInfoLinks.size());
	assert(objectInfoLinks.size() == cacheLinks.size());
	assert(cacheLinks.size() == cacheRenderLinks.size());
	assert(cacheRenderLinks.size() == dVoxelStrides.size());
	assert(dVoxelStrides.size() == dObjectAllocationIndexLookup.size());
	assert(dObjectAllocationIndexLookup.size() == dWriteSignals.size());
	assert(dWriteSignals.size() == dSegmentObjecId.size());
	assert(dSegmentObjecId.size() == dSegmentAllocLoc.size());
}

void GICudaAllocator::LinkSceneShadowMapArray(GLuint shadowMapArray)
{
	hipError_t cudaErr;
	cudaErr = hipGraphicsGLRegisterImage(&sceneShadowMapLink,
										  shadowMapArray,
										  GL_TEXTURE_2D_ARRAY,
										  hipGraphicsRegisterFlagsReadOnly);
}

void GICudaAllocator::LinkSceneGBuffers(GLuint depthTex,
										GLuint normalTex,
										GLuint lightIntensityTex)
{
	hipGraphicsGLRegisterImage(&depthBuffLink,
								depthTex,
								GL_TEXTURE_2D,
								hipGraphicsRegisterFlagsReadOnly);
	hipGraphicsGLRegisterImage(&normalBuffLink,
								normalTex,
								GL_TEXTURE_2D,
								hipGraphicsRegisterFlagsReadOnly);
	hipGraphicsGLRegisterImage(&lightIntensityLink,
								lightIntensityTex,
								GL_TEXTURE_2D,
								hipGraphicsRegisterFlagsSurfaceLoadStore);
}

void GICudaAllocator::UnLinkGBuffers()
{
	hipGraphicsUnregisterResource(depthBuffLink);
	hipGraphicsUnregisterResource(normalBuffLink);
	hipGraphicsUnregisterResource(lightIntensityLink);
}

void GICudaAllocator::SetupDevicePointers()
{
	hipGraphicsMapResources(static_cast<int>(rTransformLinks.size()), rTransformLinks.data());
	hipGraphicsMapResources(static_cast<int>(transformLinks.size()), transformLinks.data());
	hipGraphicsMapResources(static_cast<int>(aabbLinks.size()), aabbLinks.data());
	hipGraphicsMapResources(static_cast<int>(objectInfoLinks.size()), objectInfoLinks.data());

	hipGraphicsMapResources(static_cast<int>(cacheLinks.size()), cacheLinks.data());
	hipGraphicsMapResources(static_cast<int>(cacheRenderLinks.size()), cacheRenderLinks.data());

	size_t size = 0;
	for(unsigned int i = 0; i < objectCounts.size(); i++)
	{
		hRelativeTransforms.push_back(nullptr);
		hTransforms.push_back(nullptr);
		hObjectAABB.push_back(nullptr);
		hObjectInfo.push_back(nullptr);

		hObjCache.push_back(nullptr);
		hObjRenderCache.push_back(nullptr);

		hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&hRelativeTransforms.back()), &size, rTransformLinks[i]);
		hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&hTransforms.back()), &size, transformLinks[i]);
		hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&hObjectAABB.back()), &size, aabbLinks[i]);
		hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&hObjectInfo.back()), &size, objectInfoLinks[i]);

		hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&hObjCache.back()), &size, cacheLinks[i]);
		hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&hObjRenderCache.back()), &size, cacheRenderLinks[i]);
	}

	//// Data Sent to GPU
	dRelativeTransforms = hRelativeTransforms;
	dTransforms = hTransforms;
	dObjectAABB = hObjectAABB;
	dObjectInfo = hObjectInfo;

	dObjCache = hObjCache;
	dObjRenderCache = hObjRenderCache;

	// Textures
	//hipArray_t texArray;
	//hipMipmappedArray_t mipArray;
	//hipResourceDesc resDesc = {};
	//hipTextureDesc texDesc = {};

	//resDesc.resType = hipResourceTypeMipmappedArray;

	//texDesc.addressMode[0] = hipAddressModeWrap;
	//texDesc.addressMode[1] = hipAddressModeWrap;
	//texDesc.filterMode = hipFilterModePoint;
	//texDesc.readMode = hipReadModeElementType;
	//texDesc.normalizedCoords = 1;

	//hipError_t cerr;
	//cerr = hipGraphicsMapResources(1, &sceneShadowMapLink);
	//cerr = cudaGraphicsResourceGetMappedMipmappedArray(&mipArray, sceneShadowMapLink);
	//resDesc.res.mipmap.mipmap = mipArray;
	//cerr = hipCreateTextureObject(&shadowMaps, &resDesc, &texDesc, nullptr);

	//texDesc.normalizedCoords = 1;
	//resDesc.resType = hipResourceTypeArray;

	//cerr = hipGraphicsMapResources(1, &depthBuffLink);
	//cerr = hipGraphicsSubResourceGetMappedArray(&texArray, depthBuffLink, 0, 0);
	//resDesc.res.array.array = texArray;
	//cerr = hipCreateTextureObject(&depthBuffer, &resDesc, &texDesc, nullptr);

	//cerr = hipGraphicsMapResources(1, &normalBuffLink);
	//hipGraphicsSubResourceGetMappedArray(&texArray, normalBuffLink, 0, 0);
	//resDesc.res.array.array = texArray;
	//hipCreateTextureObject(&normalBuffer, &resDesc, &texDesc, nullptr);

	//hipGraphicsMapResources(1, &lightIntensityLink);
	//hipGraphicsSubResourceGetMappedArray(&texArray, lightIntensityLink, 0, 0);
	//resDesc.res.array.array = texArray;
	//hipCreateSurfaceObject(&lightIntensityBuffer, &resDesc);
}

void GICudaAllocator::ClearDevicePointers()
{
	dRelativeTransforms.Clear();
	dTransforms.Clear();
	dObjectAABB.Clear();
	dObjectInfo.Clear();

	dObjCache.Clear();
	dObjRenderCache.Clear();

	hRelativeTransforms.clear();
	hTransforms.clear();
	hObjectAABB.clear();
	hObjectInfo.clear();

	hObjCache.clear();
	hObjRenderCache.clear();
	
	hipError_t cerr;
	//cerr = hipDestroySurfaceObject(lightIntensityBuffer);
	//cerr = hipDestroyTextureObject(normalBuffer);
	//cerr = hipDestroyTextureObject(depthBuffer);
	//cerr = hipDestroyTextureObject(shadowMaps);

	//cerr = hipGraphicsUnmapResources(1, &lightIntensityLink);
	//cerr = hipGraphicsUnmapResources(1, &normalBuffLink);
	//cerr = hipGraphicsUnmapResources(1, &depthBuffLink);
	//cerr = hipGraphicsUnmapResources(1, &sceneShadowMapLink);

	cerr = hipGraphicsUnmapResources(static_cast<int>(rTransformLinks.size()), rTransformLinks.data());
	cerr = hipGraphicsUnmapResources(static_cast<int>(transformLinks.size()), transformLinks.data());
	cerr = hipGraphicsUnmapResources(static_cast<int>(aabbLinks.size()), aabbLinks.data());
	cerr = hipGraphicsUnmapResources(static_cast<int>(objectInfoLinks.size()), objectInfoLinks.data());

	cerr = hipGraphicsUnmapResources(static_cast<int>(cacheLinks.size()), cacheLinks.data());
	cerr = hipGraphicsUnmapResources(static_cast<int>(cacheRenderLinks.size()), cacheRenderLinks.data());
}

void GICudaAllocator::AddVoxelPage(size_t count)
{
	hPageData.reserve(hPageData.size() + count);
	for(unsigned int i = 0; i < count; i++)
	{
		// Allocating Page
		hPageData.emplace_back(GI_PAGE_SIZE, GI_SEGMENT_PER_PAGE);
		EmptyPageInit<<<(GI_SEGMENT_PER_PAGE + GI_THREAD_PER_BLOCK - 1) / GI_THREAD_PER_BLOCK, GI_THREAD_PER_BLOCK>> >
		(
			hPageData.back().dEmptySegmentList.Data()
		);
		hPageData.back().dIsSegmentOccupied.Memset(0, 0, hPageData.back().dIsSegmentOccupied.Size());
		hPageData.back().dVoxelPage.Memset(0, 0, hPageData.back().dVoxelPage.Size());

		CVoxelPage voxData =
		{
			hPageData.back().dVoxelPage.Data(),
			hPageData.back().dEmptySegmentList.Data(),
			hPageData.back().dIsSegmentOccupied.Data(),
			GI_SEGMENT_PER_PAGE
		};
		hVoxelPages.push_back(voxData);

		//if(i == 0)
		//{
		//	hPageData.back().dEmptySegmentList.DumpToFile("pageEmpty");
		//	hPageData.back().dIsSegmentOccupied.DumpToFile("pageOccp");
		//}
	}
	dVoxelPages = hVoxelPages;
}

void GICudaAllocator::ResetSceneData()
{
	for(unsigned int i = 0; i < rTransformLinks.size(); i++)
	{
		hipGraphicsUnregisterResource(rTransformLinks[i]);
		hipGraphicsUnregisterResource(transformLinks[i]);
		hipGraphicsUnregisterResource(aabbLinks[i]);
		hipGraphicsUnregisterResource(objectInfoLinks[i]);

		hipGraphicsUnregisterResource(cacheLinks[i]);
		hipGraphicsUnregisterResource(cacheRenderLinks[i]);
	}	
	hipGraphicsUnregisterResource(sceneShadowMapLink);

	rTransformLinks.clear();
	transformLinks.clear();
	aabbLinks.clear();
	objectInfoLinks.clear();

	cacheLinks.clear();
	cacheRenderLinks.clear();

	dSegmentObjecId.clear();
	dSegmentAllocLoc.clear();

	dVoxelStrides.clear();
	dObjectAllocationIndexLookup.clear();
	dWriteSignals.clear();
	
	dObjectAllocationIndexLookup2D.Clear();
	dSegmentAllocLoc2D.Clear();

	objectCounts.clear();
	voxelCounts.clear();

	totalObjectCount = 0;
}

void GICudaAllocator::Reserve(uint32_t pageAmount)
{
	if(dVoxelPages.Size() < pageAmount)
	{
		AddVoxelPage(pageAmount - dVoxelPages.Size());
	}
}

uint32_t GICudaAllocator::NumObjectBatches() const
{
	return static_cast<uint32_t>(rTransformLinks.size());
}

uint32_t GICudaAllocator::NumObjects(uint32_t batchIndex) const
{
	return static_cast<uint32_t>(objectCounts[batchIndex]);
}

uint32_t GICudaAllocator::NumObjectSegments(uint32_t batchIndex) const
{
	return static_cast<uint32_t>(dSegmentObjecId[batchIndex].Size());
}

uint32_t GICudaAllocator::NumVoxels(uint32_t batchIndex) const
{
	return static_cast<uint32_t>(voxelCounts[batchIndex]);
}

uint32_t GICudaAllocator::NumPages() const
{
	return static_cast<uint32_t>(hVoxelPages.size());
}

CVoxelGrid* GICudaAllocator::GetVoxelGridDevice()
{
	return dVoxelGridInfo.Data();
}

CVoxelGrid GICudaAllocator::GetVoxelGridHost()
{
	return hVoxelGridInfo;
}

IEVector3 GICudaAllocator::GetNewVoxelPos(const IEVector3& playerPos)
{
	hVoxelGridInfo.position.x = playerPos.getX() - hVoxelGridInfo.span * hVoxelGridInfo.dimension.x * 0.5f;
	hVoxelGridInfo.position.y = playerPos.getY() - hVoxelGridInfo.span * hVoxelGridInfo.dimension.y * 0.5f;
	hVoxelGridInfo.position.z = playerPos.getZ() - hVoxelGridInfo.span * hVoxelGridInfo.dimension.z * 0.5f;

	//TODO: Dummy update, remove this when you call actual update
	dVoxelGridInfo.Assign(0, hVoxelGridInfo);
	
	return {hVoxelGridInfo.position.x, hVoxelGridInfo.position.y, hVoxelGridInfo.position.z};
}

CObjectTransform** GICudaAllocator::GetRelativeTransformsDevice() 
{
	return dRelativeTransforms.Data();
}

CObjectTransform** GICudaAllocator::GetTransformsDevice()
{
	return dTransforms.Data();
}

CObjectAABB** GICudaAllocator::GetObjectAABBDevice()
{
	return dObjectAABB.Data();
}

CObjectVoxelInfo** GICudaAllocator::GetObjectInfoDevice()
{
	return dObjectInfo.Data();
}

CVoxelPacked** GICudaAllocator::GetObjCacheDevice()
{
	return dObjCache.Data();
}

CVoxelRender** GICudaAllocator::GetObjRenderCacheDevice()
{
	return dObjRenderCache.Data();
}

CObjectTransform* GICudaAllocator::GetRelativeTransformsDevice(uint32_t index)
{
	return hRelativeTransforms[index];
}

CObjectTransform* GICudaAllocator::GetTransformsDevice(uint32_t index)
{
	return hTransforms[index];
}

CObjectAABB* GICudaAllocator::GetObjectAABBDevice(uint32_t index)
{
	return hObjectAABB[index];
}

CObjectVoxelInfo* GICudaAllocator::GetObjectInfoDevice(uint32_t index)
{
	return hObjectInfo[index];
}


CVoxelPacked* GICudaAllocator::GetObjCacheDevice(uint32_t index)
{
	return hObjCache[index];
}

CVoxelRender* GICudaAllocator::GetObjRenderCacheDevice(uint32_t index)
{
	return hObjRenderCache[index];
}

CVoxelPage* GICudaAllocator::GetVoxelPagesDevice()
{
	return dVoxelPages.Data();
}

unsigned int* GICudaAllocator::GetSegmentObjectID(uint32_t index)
{
	return dSegmentObjecId[index].Data();
}

ushort2* GICudaAllocator::GetSegmentAllocLoc(uint32_t index)
{
	return dSegmentAllocLoc[index].Data();
}

unsigned int* GICudaAllocator::GetVoxelStrides(uint32_t index)
{
	return dVoxelStrides[index].Data();
}

unsigned int* GICudaAllocator::GetObjectAllocationIndexLookup(uint32_t index)
{
	return dObjectAllocationIndexLookup[index].Data();
}

char* GICudaAllocator::GetWriteSignals(uint32_t index)
{
	return dWriteSignals[index].Data();
}

unsigned int** GICudaAllocator::GetObjectAllocationIndexLookup2D()
{
	return dObjectAllocationIndexLookup2D.Data();
}

unsigned int** GICudaAllocator::GetObjectVoxStrides2D()
{
	return dObjectVoxStrides2D.Data();
}

ushort2** GICudaAllocator::GetSegmentAllocLoc2D()
{
	return dSegmentAllocLoc2D.Data();
}