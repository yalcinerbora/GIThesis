#include "GISparseVoxelOctree.h"

//void GICudaAllocator::LinkSceneShadowMapArray(GLuint shadowMapArray)
//{
//	//CUDA_CHECK(hipGraphicsGLRegisterImage(&sceneShadowMapLink,
//	//									   shadowMapArray,
//	//									   GL_TEXTURE_2D_ARRAY,
//	//									   hipGraphicsRegisterFlagsReadOnly));
//}
//
//void GICudaAllocator::LinkSceneGBuffers(GLuint depthTex,
//										GLuint normalTex,
//										GLuint lightIntensityTex)
//{
//	//CUDA_CHECK(hipGraphicsGLRegisterImage(&depthBuffLink,
//	//										depthTex,
//	//										GL_TEXTURE_2D,
//	//										hipGraphicsRegisterFlagsReadOnly));
//	//CUDA_CHECK(hipGraphicsGLRegisterImage(&normalBuffLink,
//	//										normalTex,
//	//										GL_TEXTURE_2D,
//	//										hipGraphicsRegisterFlagsReadOnly));
//	//CUDA_CHECK(hipGraphicsGLRegisterImage(&lightIntensityLink,
//	//										lightIntensityTex,
//	//										GL_TEXTURE_2D,
//	//										hipGraphicsRegisterFlagsSurfaceLoadStore));
//}
//
//void GICudaAllocator::UnLinkGBuffers()
//{
//	//CUDA_CHECK(hipGraphicsUnregisterResource(depthBuffLink));
//	//CUDA_CHECK(hipGraphicsUnregisterResource(normalBuffLink));
//	//CUDA_CHECK(hipGraphicsUnregisterResource(lightIntensityLink));
//}

// Textures
//hipArray_t texArray;
//hipMipmappedArray_t mipArray;
//hipResourceDesc resDesc = {};
//hipTextureDesc texDesc = {};

//resDesc.resType = hipResourceTypeMipmappedArray;

//texDesc.addressMode[0] = hipAddressModeWrap;
//texDesc.addressMode[1] = hipAddressModeWrap;
//texDesc.filterMode = hipFilterModePoint;
//texDesc.readMode = hipReadModeElementType;
//texDesc.normalizedCoords = 1;

//CUDA_CHECK(hipGraphicsMapResources(1, &sceneShadowMapLink));
//CUDA_CHECK(cudaGraphicsResourceGetMappedMipmappedArray(&mipArray, sceneShadowMapLink));
//resDesc.res.mipmap.mipmap = mipArray;
//CUDA_CHECK(hipCreateTextureObject(&shadowMaps, &resDesc, &texDesc, nullptr));

//texDesc.normalizedCoords = 1;
//resDesc.resType = hipResourceTypeArray;

//CUDA_CHECK(hipGraphicsMapResources(1, &depthBuffLink));
//CUDA_CHECK(hipGraphicsSubResourceGetMappedArray(&texArray, depthBuffLink, 0, 0));
//resDesc.res.array.array = texArray;
//CUDA_CHECK(hipCreateTextureObject(&depthBuffer, &resDesc, &texDesc, nullptr));

//CUDA_CHECK(hipGraphicsMapResources(1, &normalBuffLink));
//CUDA_CHECK(hipGraphicsSubResourceGetMappedArray(&texArray, normalBuffLink, 0, 0));
//resDesc.res.array.array = texArray;
//CUDA_CHECK(hipCreateTextureObject(&normalBuffer, &resDesc, &texDesc, nullptr));

//CUDA_CHECK(hipGraphicsMapResources(1, &lightIntensityLink));
//CUDA_CHECK(hipGraphicsSubResourceGetMappedArray(&texArray, lightIntensityLink, 0, 0));
//resDesc.res.array.array = texArray;
//CUDA_CHECK(hipCreateSurfaceObject(&lightIntensityBuffer, &resDesc));