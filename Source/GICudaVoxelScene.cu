#include "hip/hip_runtime.h"
#include "GICudaVoxelScene.h"
#include "GIKernels.cuh"
#include "IEUtility/IEMath.h"
#include "CudaTimer.h"
#include "Macros.h"
#include "VoxelCopyToVAO.cuh"
#include "IEUtility/IEVector3.h"
#include "CDebug.cuh"
#include <cuda_gl_interop.h>

GICudaVoxelScene::GICudaVoxelScene(const IEVector3& intialCenterPos, float span, unsigned int dim)
	: allocator(CVoxelGrid 
				{
					{ 
						intialCenterPos.getX() - (dim * span * 0.5f),
						intialCenterPos.getY() - (dim * span * 0.5f),
						intialCenterPos.getZ() - (dim * span * 0.5f)
					},
					span, 
					{ dim, dim, dim }, 
					static_cast<unsigned int>(log2f(static_cast<float>(dim)))
				})
	, vaoData(512)
	, vaoColorData(512)
{

	// Shared Memory Prep
	// 16 Kb memory is enough for our needs most of the time
	hipDeviceSetCacheConfig(hipFuncCachePreferL1);

	// Voxel Transform Function needs 48kb memory
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(VoxelTransform), hipFuncCachePreferShared);

}

GICudaVoxelScene::~GICudaVoxelScene()
{
	hipGraphicsUnregisterResource(vaoResource);
	hipGraphicsUnregisterResource(vaoRenderResource);
}

void GICudaVoxelScene::LinkOGL(GLuint aabbBuffer,
							   GLuint transformBufferID,
							   GLuint infoBufferID,
							   GLuint voxelCache,
							   GLuint voxelCacheRender,
							   uint32_t objCount,
							   uint32_t voxelCount)
{
	allocator.LinkOGLVoxelCache(aabbBuffer, transformBufferID, infoBufferID, 
								voxelCache, voxelCacheRender, objCount, voxelCount);
}

void GICudaVoxelScene::LinkSceneTextures(GLuint shadowMapArray)
{
	allocator.LinkSceneShadowMapArray(shadowMapArray);
}

void GICudaVoxelScene::UnLinkDeferredRendererBuffers()
{
	allocator.UnLinkGBuffers();
}

void GICudaVoxelScene::LinkDeferredRendererBuffers(GLuint depthBuffer,
												   GLuint normalGBuff,
												   GLuint lightIntensityTex)
{
	allocator.LinkSceneGBuffers(depthBuffer, normalGBuff, lightIntensityTex);
}

void GICudaVoxelScene::AllocateInitialPages(uint32_t approxVoxCount)
{
	// Hint Device that we will use already linked resources
	uint32_t voxCount = IEMath::UpperPowTwo(static_cast<unsigned int>(approxVoxCount));
	uint32_t pageCount = (voxCount + (GI_PAGE_SIZE - 1)) / GI_PAGE_SIZE;
	allocator.Reserve(pageCount);
	vaoData.Resize(voxCount);
	vaoColorData.Resize(voxCount);

	// Cuda Register	
	hipGraphicsGLRegisterBuffer(&vaoResource, vaoData.getGLBuffer(), cudaGraphicsMapFlagsWriteDiscard);
	hipGraphicsGLRegisterBuffer(&vaoRenderResource, vaoColorData.getGLBuffer(), cudaGraphicsMapFlagsWriteDiscard);
}

void GICudaVoxelScene::Reset()
{
	allocator.ResetSceneData();
}

void GICudaVoxelScene::Voxelize(double& ioTiming,
								double& updateTiming,
								double& svoReconTiming,
								const IEVector3& playerPos)
{
	CudaTimer timer(0);
	timer.Start();

	// Main Call Chain Called Every Frame
	// Manages Voxel Pages
	allocator.SetupDevicePointers();

	for(unsigned int i = 0; i < allocator.NumObjectBatches(); i++)
	{
		// Call Logic Per Obj Segment
		unsigned int gridSize = (allocator.NumObjectSegments(i) + GI_THREAD_PER_BLOCK - 1) /
			GI_THREAD_PER_BLOCK;

		// KC ALLOCATE
		VoxelObjectAlloc<<<gridSize, GI_THREAD_PER_BLOCK>>>
			(// Voxel System
			 allocator.GetVoxelPagesDevice(),
			 allocator.NumPages(),
			 *allocator.GetVoxelGridDevice(),
			 
			 // Per Object Segment Related
			 allocator.GetSegmentAllocLoc(i),
			 allocator.GetSegmentObjectID(i),
			 allocator.NumObjectSegments(i),
			 
			 // Per Object Related
			 allocator.GetWriteSignals(i),
			 allocator.GetObjectAABBDevice(i),
			 allocator.GetTransformsDevice(i));

		// Call Logic Per Voxel
		gridSize = (allocator.NumVoxels(i) + GI_THREAD_PER_BLOCK - 1) /
					GI_THREAD_PER_BLOCK;
		
		// KC OBJECT VOXEL INCLUDE
		VoxelObjectInclude<<<gridSize, GI_THREAD_PER_BLOCK>>>
			(// Voxel System
			 allocator.GetVoxelPagesDevice(),
			 *allocator.GetVoxelGridDevice(),
			 
			 // Per Object Segment Related
			 allocator.GetSegmentAllocLoc(i),
			 allocator.NumObjectSegments(i),
			 
			 // Per Object Related
			 allocator.GetWriteSignals(i),
			 allocator.GetVoxelStrides(i),
			 allocator.GetObjectAllocationIndexLookup(i),
			 allocator.GetObjectAABBDevice(i),
			 allocator.GetTransformsDevice(i),
			 allocator.GetObjectInfoDevice(i),
			 
			 // Per Voxel Related
			 allocator.GetObjCacheDevice(i),
			 allocator.NumVoxels(i),

			 // Batch(ObjectGroup in terms of OGL) Id
			 i);

		// Clear Write Signals
		hipMemset(allocator.GetWriteSignals(i), 0, sizeof(char) * allocator.NumObjects(i));
	}

	for(unsigned int i = 0; i < allocator.NumObjectBatches(); i++)
	{
		// Call Logic Per Obj Segment
		unsigned int gridSize = (allocator.NumObjectSegments(i) + GI_THREAD_PER_BLOCK - 1) /
			GI_THREAD_PER_BLOCK;

		// KC DEALLOCATE
		VoxelObjectDealloc<<<gridSize, GI_THREAD_PER_BLOCK>>>
			(// Voxel System
			allocator.GetVoxelPagesDevice(),
			*allocator.GetVoxelGridDevice(),

			// Per Object Segment Related
			allocator.GetSegmentAllocLoc(i),
			allocator.GetSegmentObjectID(i),
			allocator.NumObjectSegments(i),

			// Per Object Related
			allocator.GetWriteSignals(i),
			allocator.GetObjectAABBDevice(i),
			allocator.GetTransformsDevice(i));
	}

	// Call Logic Per Voxel in Page
	unsigned int gridSize = (allocator.NumPages() * GI_PAGE_SIZE + GI_THREAD_PER_BLOCK - 1) /
							GI_THREAD_PER_BLOCK;

	// KC CLEAR MARKED
	VoxelClearMarked<<<gridSize, GI_THREAD_PER_BLOCK>>>(allocator.GetVoxelPagesDevice());

	// Call Logic Per Segment in Page
	gridSize = (allocator.NumPages() * GI_SEGMENT_PER_PAGE + GI_THREAD_PER_BLOCK - 1) /
				GI_THREAD_PER_BLOCK;

	// KC CLEAR SIGNAL
	VoxelClearSignal<<<gridSize, GI_THREAD_PER_BLOCK>>>(allocator.GetVoxelPagesDevice());


	////DEBUG
	//// ONLY WORKS IF THERE IS SINGLE SEGMENT IN THE SYSTEM
	//// Call Logic Per Obj Segment
	//unsigned int gridSize2 = (allocator.NumObjectSegments(0) + GI_THREAD_PER_BLOCK - 1) /
	//	GI_THREAD_PER_BLOCK;
	//// KC DEBUG CHECK UNIQUE ALLOC
	//DebugCheckUniqueAlloc<<<gridSize2, GI_THREAD_PER_BLOCK>>>(allocator.GetSegmentAllocLoc(0),
	//														  allocator.NumObjectSegments(0));
	//// KC DEBUG CHECK UNIQUE SEGMENT ALLOC
	//DebugCheckSegmentAlloc<<<gridSize2, GI_THREAD_PER_BLOCK>>>
	//	(*allocator.GetVoxelGridDevice(),
	//	allocator.GetSegmentAllocLoc(0),
	//	allocator.GetSegmentObjectID(0),
	//	allocator.NumObjectSegments(0),
	//	allocator.GetObjectAABBDevice(0),
	//	allocator.GetTransformsDevice(0));
	////DEBUG END






	timer.Stop();
	ioTiming = timer.ElapsedMilliS();

	// Now Call Update
	timer.Start();
	IEVector3 gridNewPos = allocator.GetNewVoxelPos(playerPos);

	gridSize = (allocator.NumPages() * GI_PAGE_SIZE + GI_THREAD_PER_BLOCK - 1) /
				GI_THREAD_PER_BLOCK;
	VoxelTransform <<<gridSize, GI_THREAD_PER_BLOCK>>>
	  (// Voxel Pages
	   allocator.GetVoxelPagesDevice(),
	   *allocator.GetVoxelGridDevice(),
	   float3{gridNewPos.getX(), gridNewPos.getY(), gridNewPos.getZ()},
	   
	   // Per Object Segment
	   allocator.GetSegmentAllocLoc2D(),				   

	   // Object Related
	   allocator.GetObjectAllocationIndexLookup2D(),
	   allocator.GetTransformsDevice(),
	   allocator.GetObjRenderCacheDevice(),
	   allocator.GetObjCacheDevice(),
	   allocator.GetObjectInfoDevice(),
	   allocator.GetObjectAABBDevice());

	allocator.SendNewVoxPosToDevice();
	
	timer.Stop();
	updateTiming = timer.ElapsedMilliS();

	// Then Call SVO Reconstruct
	timer.Start();
	timer.Stop();
	svoReconTiming = timer.ElapsedMilliS();

	// Done
	allocator.ClearDevicePointers();
}

uint32_t GICudaVoxelScene::VoxelCountInPage()
{
	int h_VoxCount;
	int* d_VoxCount = nullptr;

	hipMalloc(&d_VoxCount, sizeof(int));
	hipMemset(d_VoxCount, 0, sizeof(int));

	uint32_t gridSize = ((allocator.NumPages() * GI_PAGE_SIZE) + GI_THREAD_PER_BLOCK - 1) / GI_THREAD_PER_BLOCK;

	// KC VOXEL COUNT DETERMINE FROM VOXELS
	DetermineTotalVoxCount<<<gridSize, GI_THREAD_PER_BLOCK>>>
		(*d_VoxCount,
		 // Page Related
		 allocator.GetVoxelPagesDevice(),
		 *allocator.GetVoxelGridDevice(),
		 allocator.NumPages());

	hipMemcpy(&h_VoxCount, d_VoxCount, sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_VoxCount);
	return static_cast<uint32_t>(h_VoxCount);
}


VoxelDebugVAO GICudaVoxelScene::VoxelDataForRendering(CVoxelGrid& voxGridData, double& time, uint32_t voxCount)
{
	CudaTimer timer(0);
	timer.Start();

	// Map
	unsigned int* d_atomicCounter = nullptr;
	CVoxelPacked* vBufferPackedPtr = nullptr;
	uchar4* vBufferRenderPackedPtr = nullptr;
	size_t size = 0;

	allocator.SetupDevicePointers();

	unsigned int zero = 0;
	glBindBuffer(GL_COPY_WRITE_BUFFER, vaoData.getGLBuffer());
	glClearBufferData(GL_COPY_WRITE_BUFFER, GL_RG32UI, GL_RED, GL_UNSIGNED_INT, &zero);

	hipGraphicsMapResources(1, &vaoResource);
	hipGraphicsMapResources(1, &vaoRenderResource);
	hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&vBufferPackedPtr), &size, vaoResource);
	hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&vBufferRenderPackedPtr), &size, vaoRenderResource);
	
	hipMalloc(&d_atomicCounter, sizeof(unsigned int));
	hipMemset(d_atomicCounter, 0x00, sizeof(unsigned int));

	// Copy
	// All Pages
	uint32_t gridSize = (allocator.NumPages() * GI_PAGE_SIZE + GI_THREAD_PER_BLOCK - 1) / GI_THREAD_PER_BLOCK;
	VoxelCopyToVAO<<<gridSize, GI_THREAD_PER_BLOCK>>>
		(// Two ogl Buffers for rendering used voxels
		vBufferPackedPtr,
		vBufferRenderPackedPtr,
		*d_atomicCounter,

		// Per Obj Segment
		allocator.GetSegmentAllocLoc2D(),

		// Per obj
		allocator.GetObjectAllocationIndexLookup2D(),

		// Per vox
		allocator.GetObjRenderCacheDevice(),

		// Page
		allocator.GetVoxelPagesDevice(),
		allocator.NumPages(),
		*allocator.GetVoxelGridDevice());
		
	voxGridData = allocator.GetVoxelGridHost();

	// Unmap
	hipGraphicsUnmapResources(1, &vaoResource);
	hipGraphicsUnmapResources(1, &vaoRenderResource);
	hipFree(d_atomicCounter);
	allocator.ClearDevicePointers();

	timer.Stop();
	time = timer.ElapsedMilliS();

	return VoxelDebugVAO(vaoData, vaoColorData);
}