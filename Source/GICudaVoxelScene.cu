#include "hip/hip_runtime.h"
#include "GICudaVoxelScene.h"
#include "GIKernels.cuh"

GICudaVoxelScene::GICudaVoxelScene()
	: dVoxGrid(nullptr)
{
	hipMalloc(reinterpret_cast<void**>(dVoxGrid), sizeof(CVoxelGrid));
}

GICudaVoxelScene::~GICudaVoxelScene()
{
	hipFree(dVoxGrid);
}

// Determines and Allocates the initial Page Size for the first frame
void GICudaVoxelScene::LinkOGL(GLuint aabbBuffer,
							   GLuint transformBufferID,
							   GLuint relativeTransformBufferID,
							   GLuint voxelCache,
							   GLuint voxelCacheRender)
{

}

void GICudaVoxelScene::AllocateInitialPages()
{
	// Hint Device that we will use already linked resources
}

void GICudaVoxelScene::Voxelize(const IEVector3& playerPos)
{
	// Main Call Chain Called Every Frame
	// Manages Voxel Pages


	//// Introduce Cull KC
	//VoxelObjectCull(unsigned int* gObjectIndices,
	//				unsigned int& gIndicesIndex,
	//				const CObjectAABB* gObjectAABB,
	//				const CObjectTransform* gObjTransforms,
	//				const CVoxelGrid& gGridInfo);

	//// Introduce KC
	//VoxelIntroduce(CVoxelData* gVoxelData,
	//			   const unsigned int gPageAmount,
	//			   const CVoxelPacked* gObjectVoxelCache,
	//			   const CVoxelRender* gObjectVoxelRenderCache,
	//			   const CObjectTransform& gObjTransform,
	//			   const CObjectAABB& objAABB,
	//			   const CVoxelGrid& gGridInfo);

	//// Voxel Transform KC
	//void VoxelTransform(CVoxelData* gVoxelData,
	//					CVoxelGrid& gGridInfo,
	//					const float3 newGridPos,
	//					const CObjectTransform* gObjTransformsRelative);

	////


}

GLuint GICudaVoxelScene::VoxelDataForRendering()
{
	return 0;
}