#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "COpenGLCommon.cuh"
#include "CVoxel.cuh"
#include "GICudaAllocator.h"
#include "CHash.cuh"

__device__ void LoadTransformData(// Shared Mem
								  unsigned int* sHashIndex,
								  CMatrix4x4* sTransformMatrices,
								  CMatrix4x4* sRotationMatrices,

								  // Object Transform Matrix
								  CObjectTransform** gObjTransforms,

								  // Object Type that will be broadcasted
								  const CVoxelObjectType& objType,
								  const ushort2& objectId)

{
	__shared__ CVoxelObjectType sObjType;
	unsigned int blockLocalId = threadIdx.x;
	
	// Broadcast objType
	if(blockLocalId == 0)
	{
		sObjType = objType;
	}
	__syncthreads();
	
	// Init Index Cache if this object type is morph dynamic or skeleton dynamic
	if((sObjType != CVoxelObjectType::STATIC || 
		sObjType != CVoxelObjectType::DYNAMIC) &&
		blockLocalId < GI_MAX_SHARED_COUNT_PRIME)
	{
		sHashIndex[blockLocalId] = 0;
	}
	__syncthreads();


	// Each Voxel Type Has Different Deformation(Animation)
	switch(objType)
	{
		case CVoxelObjectType::STATIC:
		case CVoxelObjectType::DYNAMIC:
		{
			// Static or Dynamic Objects have single transformation matrix to animate
			// they also have rotation only matrix for normal manipulation

			// Here we will load transform and rotation matrices
			// Each thread will load 1 float. There is two 4x4 matrix
			// 32 floats will be loaded
			// Just enough for a warp to do the work
			// Because of that we will broadcast obj id using the first warp
			// Pack objId to int
			unsigned int objIdShuffle;
			objIdShuffle = static_cast<unsigned int>(objectId.y) << 16;
			objIdShuffle |= static_cast<unsigned int>(objectId.x);

			// Broadcast
			objIdShuffle = __shfl(objIdShuffle, 0);
			
			// Unpack broadcasted objId to ushort2
			ushort2 objIdAfterShuffle;
			objIdAfterShuffle.x = (objIdShuffle & 0x0000FFFF);
			objIdAfterShuffle.y = (objIdShuffle & 0xFFFF0000) >> 16;
		
			// Load matrices
			if(blockLocalId < 16)
			//if(blockLocalId < 4)
			{
				if(objIdAfterShuffle.x == 244)
				{
					assert(blockLocalId != 15);
				}

				reinterpret_cast<float*>(&sTransformMatrices[0].column[blockLocalId / 4])[blockLocalId % 4] =
					reinterpret_cast<float*>(&gObjTransforms[objIdAfterShuffle.y][objIdAfterShuffle.x].transform.column[blockLocalId / 4])[blockLocalId % 4];
				//sTransformMatrices[0].column[blockLocalId % 4] = gObjTransforms[objIdAfterShuffle.y][objIdAfterShuffle.x].transform.column[blockLocalId % 4];
			}
			else if(blockLocalId < 32)
			//else if(blockLocalId < 8)
			{
				blockLocalId -= 16;
				reinterpret_cast<float*>(&sRotationMatrices[0].column[blockLocalId / 4])[blockLocalId % 4] =
					reinterpret_cast<float*>(&gObjTransforms[objIdAfterShuffle.y][objIdAfterShuffle.x].rotation.column[blockLocalId / 4])[blockLocalId % 4];
				//sRotationMatrices[0].column[blockLocalId % 4] = gObjTransforms[objIdAfterShuffle.y][objIdAfterShuffle.x].rotation.column[blockLocalId % 4];
			}
			
			//if(blockLocalId == 0)
			//{
			//	sTransformMatrices[0] = gObjTransforms[objIdAfterShuffle.y][objIdAfterShuffle.x].transform;
			//	//{{
			//	//	{0.19f, 0.0f, 0.0f, 0.0f},
			//	//	{0.0f, 0.19f, 0.0f, 0.0f},
			//	//	{0.0f, 0.0f, 0.19f, 0.0f},
			//	//	{0.0f, 0.0f, 0.0f, 0.19f},
			//	//}};
			//}
			//else if(blockLocalId == 1)
			//{
			//	sRotationMatrices[0] = gObjTransforms[objIdAfterShuffle.y][objIdAfterShuffle.x].rotation;
			//	//{{
			//	//	{1.0f, 0.0f, 0.0f, 0.0f},
			//	//	{0.0f, 1.0f, 0.0f, 0.0f},
			//	//	{0.0f, 0.0f, 1.0f, 0.0f},
			//	//	{0.0f, 0.0f, 0.0f, 1.0f},
			//	//}};
			//}
			break;
		}
		case CVoxelObjectType::SKEL_DYNAMIC:
		{
			// TODO Implement
			//
			break;
		}
		case CVoxelObjectType::MORPH_DYNAMIC:
		{
			// TODO Implement
			//
			break;
		}
		default:
			assert(false);
			break;
	}
	// We write to shared mem sync between warps
	__syncthreads();
}

__global__ void VoxelTransform(// Voxel Pages
							   CVoxelPage* gVoxelData,
							   const CVoxelGrid& gGridInfo,
							   const float3 hNewGridPosition,

							   // Per Object Segment
							   ushort2** gObjectAllocLocations,

							   // Object Related
							   unsigned int** gObjectAllocIndexLookup,
							   CObjectTransform** gObjTransforms,
							   CVoxelRender** gVoxRenderData,
							   CVoxelPacked** gVoxCacheData,
							   CObjectVoxelInfo** gObjInfo,	
							   CObjectAABB** gObjectAABB)
{
	// CacheLoading
	// Shared Memory which used for transform rendering
	__shared__ unsigned int sHashIndex[GI_MAX_SHARED_COUNT_PRIME];
	__shared__ CMatrix4x4 sTransformMatrices[GI_MAX_SHARED_COUNT_PRIME];
	__shared__ CMatrix4x4 sRotationMatrices[GI_MAX_SHARED_COUNT_PRIME];
		
	unsigned int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int pageId = globalId / GI_PAGE_SIZE;
	unsigned int pageLocalId = globalId % GI_PAGE_SIZE;
	unsigned int pageLocalSegmentId = pageLocalId / GI_SEGMENT_SIZE;

	if(gVoxelData[pageId].dIsSegmentOccupied[pageLocalSegmentId] == SegmentOccupation::EMPTY) return;
	if(gVoxelData[pageId].dIsSegmentOccupied[pageLocalSegmentId] == SegmentOccupation::MARKED_FOR_CLEAR) assert(false);
	
	// Fetch this voxel's id chunk from page
	CVoxelObjectType objType;
	ushort2 objectId;
	unsigned int renderLoc;

	CVoxelIds voxIdPacked = gVoxelData[pageId].dGridVoxIds[pageLocalId];
	ExpandVoxelIds(renderLoc, objectId, objType, voxIdPacked);

	// Segment is occupied so load matrices before culling unused warps
	LoadTransformData(// Shared Mem
					  sHashIndex,
					  sTransformMatrices,
					  sRotationMatrices,

					  // Object Transform Matrix
					  gObjTransforms,

					  // Object Type that will be broadcasted
					  objType,
					  objectId);

	// Cull unused warps
	if(voxIdPacked.x == 0xFFFFFFFF && voxIdPacked.y == 0xFFFFFFFF) return;

	// Fetch NormalPos from cache
	uint3 voxPos;
	float3 normal;
	unsigned int voxelSpanRatio;
	ExpandNormalPos(voxPos, normal, voxelSpanRatio, uint2{gVoxCacheData[objectId.y][renderLoc].x, gVoxCacheData[objectId.y][renderLoc].y});

	// Fetch AABB min, transform and span
	float4 objAABBMin = gObjectAABB[objectId.y][objectId.x].min;
	float objSpan = gObjInfo[objectId.y][objectId.x].span;

	// Generate World Position
	// start with object space position
	float4 worldPos;
	worldPos.x = objAABBMin.x + voxPos.x * objSpan;
	worldPos.y = objAABBMin.y + voxPos.y * objSpan;
	worldPos.z = objAABBMin.z + voxPos.z * objSpan;
	worldPos.w = 1.0f;

	// Transformations
	switch(objType)
	{
		case CVoxelObjectType::STATIC:
		case CVoxelObjectType::DYNAMIC:
		{
			// Entire block will do the assertion
			if(objectId.x != 7 &&
			   objectId.x != 16 )
			{
				//assert(fabs(sTransformMatrices[0].column[0].x - 0.18954435f) < 0.001f);
				//assert(fabs(sTransformMatrices[0].column[0].y - 0.0f) < 0.001f);
				//assert(fabs(sTransformMatrices[0].column[0].z - 0.0f) < 0.001f);
				//assert(fabs(sTransformMatrices[0].column[0].w - 0.0f) < 0.001f);

				//assert(fabs(sTransformMatrices[0].column[1].x - 0.0f) < 0.001f);
				//assert(fabs(sTransformMatrices[0].column[1].y - 0.18954435f) < 0.001f);
				//assert(fabs(sTransformMatrices[0].column[1].z - 0.0f) < 0.001f);
				//assert(fabs(sTransformMatrices[0].column[1].w - 0.0f) < 0.001f);

				//assert(fabs(sTransformMatrices[0].column[2].x - 0.0f) < 0.001f);
				//assert(fabs(sTransformMatrices[0].column[2].y - 0.0f) < 0.001f);
				//assert(fabs(sTransformMatrices[0].column[2].z - 0.18954435f) < 0.001f);
				//assert(fabs(sTransformMatrices[0].column[2].w - 0.0f) < 0.001f);

				//assert(sTransformMatrices[0].column[3].x == 0.0f);
				//assert(sTransformMatrices[0].column[3].y == 0.0f);
				//assert(sTransformMatrices[0].column[3].z == 0.0f);
				//assert(sTransformMatrices[0].column[3].w == 1.0f);


				//assert(gObjTransforms[objectId.y][objectId.x].transform.column[0].x == 0.18954435f);
				//assert(gObjTransforms[objectId.y][objectId.x].transform.column[0].y == 0.0f);
				//assert(gObjTransforms[objectId.y][objectId.x].transform.column[0].z == 0.0f);
				//assert(gObjTransforms[objectId.y][objectId.x].transform.column[0].w == 0.0f);

				//assert(gObjTransforms[objectId.y][objectId.x].transform.column[1].x == 0.0f);
				//assert(gObjTransforms[objectId.y][objectId.x].transform.column[1].y == 0.18954435f);
				//assert(gObjTransforms[objectId.y][objectId.x].transform.column[1].z == 0.0f);
				//assert(gObjTransforms[objectId.y][objectId.x].transform.column[1].w == 0.0f);

				//assert(gObjTransforms[objectId.y][objectId.x].transform.column[2].x == 0.0f);
				//assert(gObjTransforms[objectId.y][objectId.x].transform.column[2].y == 0.0f);
				//assert(gObjTransforms[objectId.y][objectId.x].transform.column[2].z == 0.18954435f);
				//assert(gObjTransforms[objectId.y][objectId.x].transform.column[2].w == 0.0f);

				//assert(gObjTransforms[objectId.y][objectId.x].transform.column[3].x == 0.0f);
				//assert(gObjTransforms[objectId.y][objectId.x].transform.column[3].y == 0.0f);
				//assert(gObjTransforms[objectId.y][objectId.x].transform.column[3].z == 0.0f);
				//assert(gObjTransforms[objectId.y][objectId.x].transform.column[3].w == 1.0f);
			}
			// Now voxel is in is world space
			MultMatrixSelf(worldPos, sTransformMatrices[0]);
			MultMatrixSelf(normal, sRotationMatrices[0]);
			break;
		}
		case CVoxelObjectType::SKEL_DYNAMIC:
		{
			// TODO Implement
			//
			break;
		}
		case CVoxelObjectType::MORPH_DYNAMIC:
		{
			// TODO Implement
			//
			break;
		}
		default:
			assert(false);
			break;
	}

	// Reconstruct Voxel Indices relative to the new pos of the grid
	worldPos.x -= hNewGridPosition.x;
	worldPos.y -= hNewGridPosition.y;
	worldPos.z -= hNewGridPosition.z;

	bool outOfBounds;
	outOfBounds = (worldPos.x < 0.0f) || (worldPos.x > gGridInfo.dimension.x * gGridInfo.span);
	outOfBounds |= (worldPos.y < 0.0f) || (worldPos.y > gGridInfo.dimension.y * gGridInfo.span);
	outOfBounds |= (worldPos.z < 0.0f) || (worldPos.z > gGridInfo.dimension.z * gGridInfo.span);

	// Now Write
	// Discard the out of bound voxels
	if(!outOfBounds)
	{
		float invSpan = 1.0f / gGridInfo.span;
		voxPos.x = static_cast<unsigned int>(worldPos.x * invSpan);
		voxPos.y = static_cast<unsigned int>(worldPos.y * invSpan);
		voxPos.z = static_cast<unsigned int>(worldPos.z * invSpan);

		// Write to page
		PackVoxelNormPos(gVoxelData[pageId].dGridVoxNormPos[pageLocalId], voxPos, normal, voxelSpanRatio);
	}
	else
	{
		gVoxelData[pageId].dGridVoxNormPos[pageLocalId] = uint2{0, 0};
	}
}