#include "GICudaAllocator.h"
#include "GICudaStructMatching.h"
#include <cuda_gl_interop.h>

GICudaAllocator::GICudaAllocator()
	: totalObjectCount(0)
{
	cudaGLSetGLDevice(0);
}

void GICudaAllocator::LinkOGLVoxelCache(GLuint batchAABBBuffer,
											GLuint batchTransformBuffer,
											GLuint relativeTransformBuffer,
											GLuint infoBuffer,
											GLuint voxelBuffer,
											GLuint voxelRenderBuffer,
											size_t objCount)
{
	rTransformLinks.emplace_back(nullptr);
	transformLinks.emplace_back(nullptr);
	aabbLinks.emplace_back(nullptr);
	objectInfoLinks.emplace_back(nullptr);
	cacheLinks.emplace_back(nullptr);
	cacheRenderLinks.emplace_back(nullptr);

	hipGraphicsGLRegisterBuffer(&rTransformLinks.back(), relativeTransformBuffer, cudaGraphicsMapFlagsReadOnly);
	hipGraphicsGLRegisterBuffer(&transformLinks.back(), batchTransformBuffer, cudaGraphicsMapFlagsReadOnly);
	hipGraphicsGLRegisterBuffer(&aabbLinks.back(), batchAABBBuffer, cudaGraphicsMapFlagsReadOnly);
	hipGraphicsGLRegisterBuffer(&objectInfoLinks.back(), infoBuffer, cudaGraphicsMapFlagsReadOnly);

	hipGraphicsGLRegisterBuffer(&cacheLinks.back(), voxelBuffer, cudaGraphicsMapFlagsReadOnly);
	hipGraphicsGLRegisterBuffer(&cacheRenderLinks.back(), voxelRenderBuffer, cudaGraphicsMapFlagsReadOnly);

	objectCounts.emplace_back(objCount);
	totalObjectCount += objCount;
}

void GICudaAllocator::SetupPointersDevicePointers()
{
	hipGraphicsMapResources(rTransformLinks.size(), rTransformLinks.data());
	hipGraphicsMapResources(transformLinks.size(), transformLinks.data());
	hipGraphicsMapResources(aabbLinks.size(), aabbLinks.data());
	hipGraphicsMapResources(objectInfoLinks.size(), objectInfoLinks.data());

	hipGraphicsMapResources(cacheLinks.size(), cacheLinks.data());
	hipGraphicsMapResources(cacheRenderLinks.size(), cacheRenderLinks.data());

	thrust::host_vector<CObjectTransform*> hRelativeTransforms;
	thrust::host_vector<CObjectTransform*> hTransforms;
	thrust::host_vector<CObjectAABB*> hObjectAABB;
	thrust::host_vector<CObjectVoxelInfo*> hObjectInfo;
			
	thrust::host_vector<CVoxelPacked*> hObjCache;
	thrust::host_vector<CVoxelRender*> hObjRenderCache;

	size_t size = 0;
	for(unsigned int i = 0; i < objectCounts.size(); i++)
	{
		hRelativeTransforms.push_back(nullptr);
		hTransforms.push_back(nullptr);
		hObjectAABB.push_back(nullptr);
		hObjectInfo.push_back(nullptr);

		hObjCache.push_back(nullptr);
		hObjRenderCache.push_back(nullptr);

		hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&hRelativeTransforms.back()), &size, rTransformLinks[i]);
		hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&hTransforms.back()), &size, transformLinks[i]);
		hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&hObjectAABB.back()), &size, aabbLinks[i]);
		hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&hObjectInfo.back()), &size, objectInfoLinks[i]);

		hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&hObjCache.back()), &size, cacheLinks[i]);
		hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&hObjRenderCache.back()), &size, cacheRenderLinks[i]);
	}

	// Dat Sent to GPU
	dRelativeTransforms = hRelativeTransforms;
	dTransforms = hTransforms;
	dObjectAABB = hObjectAABB;
	dObjectInfo = hObjectInfo;

	dObjCache = hObjCache;
	dObjRenderCache = hObjRenderCache;
}

void GICudaAllocator::ClearDevicePointers()
{
	dRelativeTransforms.clear();
	dTransforms.clear();
	dObjectAABB.clear();
	dObjectInfo.clear();

	dObjCache.clear();
	dObjRenderCache.clear();

	hipGraphicsUnmapResources(rTransformLinks.size(), rTransformLinks.data());
	hipGraphicsUnmapResources(transformLinks.size(), transformLinks.data());
	hipGraphicsUnmapResources(aabbLinks.size(), aabbLinks.data());
	hipGraphicsUnmapResources(objectInfoLinks.size(), objectInfoLinks.data());

	hipGraphicsUnmapResources(cacheLinks.size(), cacheLinks.data());
	hipGraphicsUnmapResources(cacheRenderLinks.size(), cacheRenderLinks.data());
}


void GICudaAllocator::AddVoxelPage(size_t count)
{

	for(unsigned int i = 0; i < count; i++)
	{
		// Allocating Page
		hPageData.emplace_back(thrust::device_vector<CVoxelPacked>({0}, GI_PAGE_SIZE),
							   thrust::device_vector<CVoxelRender>({0}, GI_PAGE_SIZE),
							   thrust::device_vector<unsigned int>({0}, GI_PAGE_SIZE),
							   thrust::device_vector<unsigned int>(0, GI_BLOCK_PER_PAGE));

		CVoxelPage voxData =
		{
			thrust::raw_pointer_cast(hPageData.back().dVoxelPage.data()),
			thrust::raw_pointer_cast(hPageData.back().dVoxelPageRender.data()),
			thrust::raw_pointer_cast(hPageData.back().dVoxelState.data()),
			thrust::raw_pointer_cast(hPageData.back().dEmptySegmentPos.data()),
			0
		};
		hVoxelPages.push_back(voxData);
		dVoxelPages.push_back(voxData);
	}
}

const CObjectTransform** GICudaAllocator::GetRelativeTransformsDevice() 
{
	return thrust::raw_pointer_cast(dRelativeTransforms.data());
}

const CObjectTransform** GICudaAllocator::GetTransformsDevice()
{
	return thrust::raw_pointer_cast(dTransforms.data());
}

const CObjectAABB** GICudaAllocator::GetObjectAABBDevice()
{
	return thrust::raw_pointer_cast(dObjectAABB.data());
}

const CObjectVoxelInfo** GICudaAllocator::GetObjectInfoDevice()
{
	return thrust::raw_pointer_cast(dObjectInfo.data());
}

const CVoxelPacked** GICudaAllocator::GetObjCacheDevice()
{
	return thrust::raw_pointer_cast(dObjCache.data());
}

const CVoxelRender** GICudaAllocator::GetObjRenderCacheDevice()
{
	return thrust::raw_pointer_cast(dObjRenderCache.data());
}

CVoxelPage* GICudaAllocator::GetVoxelPagesDevice()
{
	return thrust::raw_pointer_cast(dVoxelPages.data());
}
