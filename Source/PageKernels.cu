#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <assert.h>

#include "PageKernels.cuh"
#include "CVoxelFunctions.cuh"
#include "CMatrixFunctions.cuh"
#include "CAABBFunctions.cuh"
#include "COpenGLTypes.h"
#include "CAtomicAlloc.cuh"
#include "GIVoxelPages.h"

#define GI_MAX_JOINT_COUNT 64

__global__ void InitializePage(unsigned char* emptySegments, const size_t pageCount)
{
	size_t sizePerPage = GIVoxelPages::PageSize *
						 (sizeof(CVoxelPos) +
						  sizeof(CVoxelNorm) +
						  sizeof(CVoxelOccupancy))
						 +
						 GIVoxelPages::SegmentSize *
						 (sizeof(unsigned char) +
						  sizeof(CSegmentInfo));

	unsigned int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int pageLocalSegmentId = globalId % GIVoxelPages::SegmentPerPage;
	unsigned int pageId = globalId / GIVoxelPages::SegmentPerPage;

	// Cull if out of bounds
	if(globalId >= pageCount * GIVoxelPages::SegmentPerPage) return;
	emptySegments[pageId * sizePerPage + pageLocalSegmentId] = GIVoxelPages::SegmentSize - pageLocalSegmentId - 1;
}

__global__ void VoxelObjectDealloc(// Voxel System
								   CVoxelPage* gVoxelData,
								   const CVoxelGrid* gGridInfos,
								   // Helper Structures								  
								   ushort2* gSegmentAllocInfo,
								   const CSegmentInfo* gSegmentInfo,
								   // Per Object Related
								   const BatchOGLData* gBatchOGLData,
								   // Limits
								   const uint32_t totalSegments)
{
	unsigned int globalId = threadIdx.x + blockIdx.x * blockDim.x;

	// Now Thread Scheme changes per objectSegment
	if(globalId >= totalSegments) return;

	// Determine Obj Id (-1 Id means this object is too small for this grid)
	const CSegmentInfo segInfo = gSegmentInfo[globalId];
	if(segInfo.objId == 0xFFFF) return;

	// Unpack segmentInfo
	uint8_t cascadeNo = ExpandOnlyCascadeNo(segInfo.packed);
	const CVoxelGrid cascadeGrid = gGridInfos[cascadeNo];

	// Intersection Check
	const uint32_t transformId = gBatchOGLData[segInfo.batchId].dModelTransformIndices[segInfo.objId];
	const CMatrix4x4 transform = gBatchOGLData[segInfo.batchId].dModelTransforms[transformId].transform;
	const CAABB objAABB = gBatchOGLData[segInfo.batchId].dAABBs[segInfo.objId];
	bool intersects = CheckGridVoxIntersect(cascadeGrid, objAABB, transform);

	// Check if this object is not allocated
	ushort2 objAlloc = gSegmentAllocInfo[globalId];
	if(!intersects && objAlloc.x != 0xFFFF)
	{
		// "Dealocate"
		assert(ExpandOnlyOccupation(gVoxelData[objAlloc.x].dSegmentInfo[objAlloc.y].packed) == CSegmentOccupation::OCCUPIED);
		unsigned int size = AtomicDealloc(&(gVoxelData[objAlloc.x].dEmptySegmentStackSize), GIVoxelPages::SegmentPerPage);
		assert(size != GIVoxelPages::SegmentPerPage);
		if(size != GIVoxelPages::SegmentPerPage)
		{
			unsigned int location = size;
			gVoxelData[objAlloc.x].dEmptySegmentPos[location] = objAlloc.y;

			CSegmentInfo segObjId = {0};
			gVoxelData[objAlloc.x].dSegmentInfo[objAlloc.y] = segObjId;
			gSegmentAllocInfo[globalId] = ushort2{0xFFFF, 0xFFFF};
		}
	}
}

__global__ void VoxelObjectAlloc(// Voxel System
								 CVoxelPage* gVoxelData,
								 const CVoxelGrid* gGridInfos,
								 // Helper Structures
								 ushort2* gSegmentAllocInfo,
								 const CSegmentInfo* gSegmentInfo,
								 // Per Object Related
								 const BatchOGLData* gBatchOGLData,
								 // Limits
								 const uint32_t totalSegments,
								 const uint32_t pageAmount)
{
	unsigned int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	if(globalId >= totalSegments) return;

	// Determine Obj Id (-1 Id means this object is too small for this grid)
	const CSegmentInfo segInfo = gSegmentInfo[globalId];
	if(segInfo.objId == 0xFFFF) return;

	// Unpack segmentInfo
	uint8_t cascadeNo = ExpandOnlyCascadeNo(segInfo.packed);
	const CVoxelGrid cascadeGrid = gGridInfos[cascadeNo];
	
	// Intersection Check
	const uint32_t transformId = gBatchOGLData[segInfo.batchId].dModelTransformIndices[segInfo.objId];
	const CMatrix4x4 transform = gBatchOGLData[segInfo.batchId].dModelTransforms[transformId].transform;
	const CAABB objAABB = gBatchOGLData[segInfo.batchId].dAABBs[segInfo.objId];
	bool intersects = CheckGridVoxIntersect(cascadeGrid, objAABB, transform);

	// Check if this object already allocated
	ushort2 objAlloc = gSegmentAllocInfo[globalId];
	if(intersects && objAlloc.x == 0xFFFF)
	{
		// "Allocate"
		// Check page by page
		for(unsigned int i = 0; i < pageAmount; i++)
		{
			unsigned int size = AtomicAlloc(&(gVoxelData[i].dEmptySegmentStackSize));
			if(size != 0)
			{
				unsigned int location = gVoxelData[i].dEmptySegmentPos[size - 1];
				assert(ExpandOnlyOccupation(gVoxelData[i].dSegmentInfo[location].packed) == CSegmentOccupation::EMPTY);
				gSegmentAllocInfo[globalId] = ushort2
				{
					static_cast<unsigned short>(i),
					static_cast<unsigned short>(location)
				};
				gVoxelData[i].dSegmentInfo[location] = segInfo;
				return;
			}
		}
	}
}

inline __device__ void LoadTransformData(// Shared Mem
										 CMatrix4x4* sTransformMatrices,
										 CMatrix3x3* sRotationMatrices,
										 uint8_t* sMatrixLookup,

										 // Object Transform Matrix
										 const BatchOGLData& gBatchOGLData,

										 // Current Voxel Weight
										 const uchar4& voxelWeightIndex,

										 // Object Type that will be broadcasted
										 const CObjectType& objType,
										 const uint16_t& objId,
										 const uint16_t& transformId)
{
	unsigned int blockLocalId = threadIdx.x;

	// Here we will load transform and rotation matrices
	// Each thread will load 1 float. There is two 4x4 matrix
	// 32 floats will be loaded
	// Just enough for a warp to do the work
	// Load matrices (4 byte load by each thread sequential no bank conflict)
	const CModelTransform& objectMT = gBatchOGLData.dModelTransforms[transformId];
	float* sTrans = reinterpret_cast<float*>(&sTransformMatrices[0]);
	float* sRot = reinterpret_cast<float*>(&sRotationMatrices[0]);
	if(blockLocalId < 16)
	{
		const float* objectTransform = reinterpret_cast<const float*>(&objectMT.transform);
		sTrans[blockLocalId] = objectTransform[blockLocalId];
	}
	else if(blockLocalId < 25)
	{
		unsigned int rotationId = blockLocalId - 16;
		unsigned int columnId = rotationId / 3;
		unsigned int rowId = rotationId % 3;

		const float* objectRotation = reinterpret_cast<const float*>(&objectMT.rotation);
		sRot[columnId * 3 + rowId] = objectRotation[columnId * 4 + rowId];
	}

	// Load Joint Transforms if Skeletal Object
	if(objType == CObjectType::SKEL_DYNAMIC)
	{
		// All valid objects will request matrix load
		// then entire block will try to load it
		// Max skeleton bone count is 64
		// Worst case 64 * 16 = 1024 float will be loaded to sMem
		// Some blocks will load twice
		// However its extremely rare (even impossible case)
		// In a realistic scenario (and if a segment holds adjacent voxels)
		// And if max bone influence per vertex is around 4 
		// there should be at most 8

		// Matrix Lookup Initialize
		if(blockLocalId < GI_MAX_JOINT_COUNT)
			sMatrixLookup[blockLocalId] = 0;
		__syncthreads();

		if(voxelWeightIndex.x != 0xFF) sMatrixLookup[voxelWeightIndex.x] = 1;
		if(voxelWeightIndex.y != 0xFF) sMatrixLookup[voxelWeightIndex.y] = 1;
		if(voxelWeightIndex.z != 0xFF) sMatrixLookup[voxelWeightIndex.z] = 1;
		if(voxelWeightIndex.w != 0xFF) sMatrixLookup[voxelWeightIndex.w] = 1;
		__syncthreads();

		// Lookup Tables are Loaded
		// Theorethical 63 Matrices will be loaded
		//	Each thread will load 1 float we need 1024 threads
		unsigned int iterationCount = (GI_MAX_JOINT_COUNT * 16) / blockDim.x;
		for(unsigned int i = 0; i < iterationCount; i++)
		{
			// Transformation
			unsigned int floatCount = GI_MAX_JOINT_COUNT * 16;
			unsigned int floatId = blockLocalId + (blockDim.x * i);
			if(floatId < floatCount)
			{
				unsigned int matrixId = (floatId / 16);
				unsigned int matrixLocalFloatId = floatId % 16;				
				if(sMatrixLookup[matrixId] == 1)
				{
					const CMatrix4x4& jointT = gBatchOGLData.dJointTransforms[matrixId].transform;
					const float* jointTFloat = reinterpret_cast<const float*>(&jointT);
					float* sTrans = reinterpret_cast<float*>(&sTransformMatrices[matrixId + 1]);

					sTrans[matrixLocalFloatId] = jointTFloat[matrixLocalFloatId];
				}
			}
			// Rotation
			floatCount = GI_MAX_JOINT_COUNT * 9;
			if(floatId < floatCount)
			{
				unsigned int matrixId = (floatId / 9);
				unsigned int matrixLocalFloatId = floatId % 9;
				if(sMatrixLookup[matrixId] == 1)
				{
					const CMatrix4x4& jointRot = gBatchOGLData.dJointTransforms[matrixId].rotation;
					const float* jointRotFloat = reinterpret_cast<const float*>(&jointRot);
					float* sRot = reinterpret_cast<float*>(&sRotationMatrices[matrixId + 1]);

					unsigned int column = matrixLocalFloatId / 3;
					unsigned int row = matrixLocalFloatId % 3;
					sRot[column * 3 + row] = jointRotFloat[column * 4 + row];		
				}
			}
		}
	}
	// We write to shared mem sync between warps
	__syncthreads();
}

__global__ void VoxelTransform(// Voxel Pages
							   CVoxelPage* gVoxelPages,
							   const CVoxelGrid& gGridInfo,
							   const float3 hNewGridPosition,
							   // OGL Related
							   const BatchOGLData* gBatchOGLData,
							   // Voxel Cache Related
							   const BatchVoxelCache* gBatchVoxelCache,
							   
							   const float baseSpan,
							   const uint32_t batchCount)
{
	// Cache Loading
	// Shared Memory which used for transform rendering
	__shared__ CMatrix4x4 sTransformMatrices[GI_MAX_JOINT_COUNT + 1];	// First index holds model matrix
	__shared__ CMatrix3x3 sRotationMatrices[GI_MAX_JOINT_COUNT + 1];
	__shared__ uint8_t sMatrixLookup[GI_MAX_JOINT_COUNT];	
	// Shared Memory which is used by
	__shared__ CMeshVoxelInfo sMeshVoxelInfo;
	__shared__ CSegmentInfo sSegInfo;
	__shared__ uint16_t	sObjTransformId;
	
	unsigned int blockLocalId = threadIdx.x;
	unsigned int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int pageId = globalId / GIVoxelPages::PageSize;
	unsigned int pageLocalId = globalId % GIVoxelPages::PageSize;
	unsigned int pageLocalSegmentId = pageLocalId / GIVoxelPages::SegmentSize;
	unsigned int segmentLocalVoxId = pageLocalId % GIVoxelPages::SegmentSize;
	
	// Get Segments Obj Information Struct
	CObjectType objType;
	CSegmentOccupation occupation;
	uint8_t cascadeId;
	if(blockLocalId == 0)
	{
		// Load to smem
		sSegInfo = gVoxelPages[pageId].dSegmentInfo[pageLocalSegmentId];
		ExpandSegmentInfo(cascadeId, objType, occupation, sSegInfo.packed);
		sObjTransformId = gBatchOGLData[sSegInfo.batchId].dModelTransformIndices[sSegInfo.objId];
		sMeshVoxelInfo = gBatchVoxelCache[cascadeId * batchCount + sSegInfo.batchId].dMeshVoxelInfo[sSegInfo.objId];
	}
	__syncthreads();
	if(blockLocalId != 0)
	{
		ExpandSegmentInfo(cascadeId, objType, occupation, sSegInfo.packed);
	}
	// Full Block Cull
	if(occupation == CSegmentOccupation::EMPTY) return;
	assert(occupation != CSegmentOccupation::MARKED_FOR_CLEAR);

	// Find your opengl data and voxel cache
	const uint16_t& batchId = sSegInfo.batchId;
	const uint16_t& objectId = sSegInfo.objId;
	const BatchOGLData& batchOGLData = gBatchOGLData[batchId];
	const BatchVoxelCache& batchCache = gBatchVoxelCache[cascadeId * batchCount + batchId];
	float cascadeSpan = baseSpan * static_cast<float>(1 << cascadeId);

	// Voxel Ids
	const uint32_t objectLocalVoxelId = sSegInfo.objectSegmentId * GIVoxelPages::SegmentSize + segmentLocalVoxId;
	const uint32_t batchLocalVoxelId = objectLocalVoxelId + sMeshVoxelInfo.voxOffset;

	// Load weights if necessary
	CVoxelWeights weights = {{0x00, 0x00, 0x00, 0x00},{0xFF, 0xFF, 0xFF, 0xFF}};
	if(objectLocalVoxelId < sMeshVoxelInfo.voxCount && objType == CObjectType::SKEL_DYNAMIC)
	{
		weights = batchCache.dVoxelWeight[batchLocalVoxelId];
	}

	// Segment is occupied so load matrices before culling unused warps
	LoadTransformData(// Shared Mem
					  sTransformMatrices,
					  sRotationMatrices,
					  sMatrixLookup,
					  // OGL
					  batchOGLData,
					  // Weight Index
					  weights.weightIndex,
					  // Object Type that will be broadcasted
					  objType,
					  objectId,
					  sObjTransformId);

	// Cull threads
	// Edge case where last segment do not always full
	if(objectLocalVoxelId >= sMeshVoxelInfo.voxCount)
	{
		gVoxelPages[pageId].dGridVoxPos[pageLocalId] = 0xFFFFFFFF;
		gVoxelPages[pageId].dGridVoxNorm[pageLocalId] = 0xFFFFFFFF;
		return;
	}

	// Fetch NormalPos from cache
	uint4 voxPos;
	float3 normal;
	voxPos = ExpandVoxPos(batchCache.dVoxelPos[batchLocalVoxelId]);
	normal = ExpandVoxNormal(batchCache.dVoxelNorm[batchLocalVoxelId]);

	// Fetch AABB min, transform and span
	float4 objAABBMin = batchOGLData.dAABBs[objectId].min;

	// Generate World Position
	// start with object space position
	float3 worldPos;
	worldPos.x = objAABBMin.x + voxPos.x * cascadeSpan;
	worldPos.y = objAABBMin.y + voxPos.y * cascadeSpan;
	worldPos.z = objAABBMin.z + voxPos.z * cascadeSpan;

	// Joint Transformations
	if(objType == CObjectType::SKEL_DYNAMIC)
	{
		float4 weightUnorm;
		weightUnorm.x = static_cast<float>(weights.weight.x) / 255.0f;
		weightUnorm.y = static_cast<float>(weights.weight.y) / 255.0f;
		weightUnorm.z = static_cast<float>(weights.weight.z) / 255.0f;
		weightUnorm.w = static_cast<float>(weights.weight.w) / 255.0f;

		//if(threadIdx.x == 0)
		//	printf("x %d, y %d, z %d, w %d\n",
		//	weights.weightIndex.x,
		//	weights.weightIndex.y,
		//	weights.weightIndex.z,
		//	weights.weightIndex.w);

		// Nyra Char Related Assert
		assert(weights.weightIndex.x <= 24);
		assert(weights.weightIndex.y <= 24);
		assert(weights.weightIndex.z <= 24);
		assert(weights.weightIndex.w <= 24);

		float3 pos = {0.0f, 0.0f, 0.0f};
		float3 p = MultMatrix(worldPos, sTransformMatrices[weights.weightIndex.x + 1]);
		//float3 p = MultMatrix(worldPos, gJointTransforms[segObj.batchId][weights.weightIndex.x].transform);

		pos.x += weightUnorm.x * p.x;
		pos.y += weightUnorm.x * p.y;
		pos.z += weightUnorm.x * p.z;

		p = MultMatrix(worldPos, sTransformMatrices[weights.weightIndex.y + 1]);
		//p = MultMatrix(worldPos, gJointTransforms[segObj.batchId][weights.weightIndex.y].transform);
		pos.x += weightUnorm.y * p.x;
		pos.y += weightUnorm.y * p.y;
		pos.z += weightUnorm.y * p.z;

		p = MultMatrix(worldPos, sTransformMatrices[weights.weightIndex.z + 1]);
		//p = MultMatrix(worldPos, gJointTransforms[segObj.batchId][weights.weightIndex.z].transform);
		pos.x += weightUnorm.z * p.x;
		pos.y += weightUnorm.z * p.y;
		pos.z += weightUnorm.z * p.z;

		p = MultMatrix(worldPos, sTransformMatrices[weights.weightIndex.w + 1]);
		//p = MultMatrix(worldPos, gJointTransforms[segObj.batchId][weights.weightIndex.w].transform);
		pos.x += weightUnorm.w * p.x;
		pos.y += weightUnorm.w * p.y;
		pos.z += weightUnorm.w * p.z;

		worldPos = pos;

		float3 norm = {0.0f, 0.0f, 0.0f};
		float3 n = MultMatrix(normal, sRotationMatrices[weights.weightIndex.x + 1]);
		norm.x += weightUnorm.x * n.x;
		norm.y += weightUnorm.x * n.y;
		norm.z += weightUnorm.x * n.z;

		n = MultMatrix(normal, sRotationMatrices[weights.weightIndex.y + 1]);
		norm.x += weightUnorm.y * n.x;
		norm.y += weightUnorm.y * n.y;
		norm.z += weightUnorm.y * n.z;

		n = MultMatrix(normal, sRotationMatrices[weights.weightIndex.z + 1]);
		norm.x += weightUnorm.z * n.x;
		norm.y += weightUnorm.z * n.y;
		norm.z += weightUnorm.z * n.z;

		n = MultMatrix(normal, sRotationMatrices[weights.weightIndex.w + 1]);
		norm.x += weightUnorm.w * n.x;
		norm.y += weightUnorm.w * n.y;
		norm.z += weightUnorm.w * n.z;

		normal = norm;
	}

	// Model Transformations
	MultMatrixSelf(worldPos, sTransformMatrices[0]);
	MultMatrixSelf(normal, sRotationMatrices[0]);
	//// Unoptimized Matrix Load
	//CMatrix4x4 transform = gObjTransforms[segObj.batchId][gObjTransformIds[segObj.batchId][segObj.objId]].transform;
	//CMatrix4x4 rotation = gObjTransforms[segObj.batchId][gObjTransformIds[segObj.batchId][segObj.objId]].transform;
	//MultMatrixSelf(worldPos, transform);
	//MultMatrixSelf(normal, rotation);

	// Reconstruct Voxel Indices relative to the new pos of the grid
	worldPos.x -= hNewGridPosition.x;
	worldPos.y -= hNewGridPosition.y;
	worldPos.z -= hNewGridPosition.z;

	bool outOfBounds;
	outOfBounds = (worldPos.x < 0.0f) || (worldPos.x >= gGridInfo.dimension.x * gGridInfo.span);
	outOfBounds |= (worldPos.y < 0.0f) || (worldPos.y >= gGridInfo.dimension.y * gGridInfo.span);
	outOfBounds |= (worldPos.z < 0.0f) || (worldPos.z >= gGridInfo.dimension.z * gGridInfo.span);

	// If its mip dont update inner cascade
	bool inInnerCascade = false;
	if(cascadeId > 0)
	{
		inInnerCascade = (worldPos.x > gGridInfo.dimension.x * gGridInfo.span * 0.25f) &&
			(worldPos.x < gGridInfo.dimension.x * gGridInfo.span * 0.75f);
		inInnerCascade &= (worldPos.y > gGridInfo.dimension.y * gGridInfo.span * 0.25f) &&
			(worldPos.y < gGridInfo.dimension.y * gGridInfo.span * 0.75f);
		inInnerCascade &= (worldPos.z > gGridInfo.dimension.z * gGridInfo.span * 0.25f) &&
			(worldPos.z < gGridInfo.dimension.z * gGridInfo.span * 0.75f);
	}
	outOfBounds |= inInnerCascade;

	// Voxel Space
	float invSpan = 1.0f / gGridInfo.span;
	voxPos.x = static_cast<unsigned int>(worldPos.x * invSpan);
	voxPos.y = static_cast<unsigned int>(worldPos.y * invSpan);
	voxPos.z = static_cast<unsigned int>(worldPos.z * invSpan);

	// Calculate VoxelWeights
	float3 volumeWeight;
	volumeWeight.x = worldPos.x * invSpan;
	volumeWeight.y = worldPos.y * invSpan;
	volumeWeight.z = worldPos.z * invSpan;

	volumeWeight.x = volumeWeight.x - static_cast<float>(voxPos.x);
	volumeWeight.y = volumeWeight.y - static_cast<float>(voxPos.y);
	volumeWeight.z = volumeWeight.z - static_cast<float>(voxPos.z);

	//volumeWeight.x = 1.0f;
	//volumeWeight.y = 1.0f;
	//volumeWeight.z = 1.0f;

	uint3 neigbourBits;
	neigbourBits.x = (volumeWeight.x > 0) ? 1 : 0;
	neigbourBits.y = (volumeWeight.y > 0) ? 1 : 0;
	neigbourBits.z = (volumeWeight.z > 0) ? 1 : 0;

	// Outer Bound Check
	outOfBounds |= (voxPos.x >= gGridInfo.dimension.x);
	outOfBounds |= (voxPos.y >= gGridInfo.dimension.y);
	outOfBounds |= (voxPos.z >= gGridInfo.dimension.z);

	// Now Write
	// Discard the out of bound voxels
	//outOfBounds = false;
	if(!outOfBounds)
	{
		// Write to page
		gVoxelPages[pageId].dGridVoxPos[pageLocalId] = PackVoxPos({voxPos.x, voxPos.y, voxPos.z}, cascadeId);
		gVoxelPages[pageId].dGridVoxNorm[pageLocalId] = PackVoxNormal(normal);
		gVoxelPages[pageId].dGridVoxOccupancy[pageLocalId] = PackOccupancy(neigbourBits, volumeWeight);
	}
	else
	{
		gVoxelPages[pageId].dGridVoxPos[pageLocalId] = 0xFFFFFFFF;
		gVoxelPages[pageId].dGridVoxNorm[pageLocalId] = 0xFFFFFFFF;
	}
}

__global__ void VoxelClearMarked(CVoxelPage* gVoxelData)
{
	unsigned int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int pageId = globalId / GIVoxelPages::PageSize;
	unsigned int pageLocalId = globalId % GIVoxelPages::PageSize;
	unsigned int pageLocalSegmentId = pageLocalId / GIVoxelPages::SegmentSize;

	// Check if segment is marked for clear
	if(ExpandOnlyOccupation(gVoxelData[pageId].dSegmentInfo[pageLocalSegmentId].packed) == CSegmentOccupation::MARKED_FOR_CLEAR)
	{
		// Segment is marked for clear, clear it
		gVoxelData[pageId].dGridVoxPos[pageLocalId] = 0xFFFFFFFF;
		gVoxelData[pageId].dGridVoxNorm[pageLocalId] = 0xFFFFFFFF;
		gVoxelData[pageId].dGridVoxOccupancy[pageLocalId] = 0xFFFFFFFF;
	}
}

__global__ void VoxelClearSignal(CVoxelPage* gVoxelData,
								 const uint32_t numPages)
{
	unsigned int globalId = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int pageId = globalId / GIVoxelPages::SegmentPerPage;
	unsigned int pageLocalSegmentId = globalId % GIVoxelPages::SegmentPerPage;

	// Check if segment is marked for clear
	if(globalId >= numPages * GIVoxelPages::SegmentPerPage) return;
	if(ExpandOnlyOccupation(gVoxelData[pageId].dSegmentInfo[pageLocalSegmentId].packed) == CSegmentOccupation::MARKED_FOR_CLEAR)
	{
		gVoxelData[pageId].dSegmentInfo[pageLocalSegmentId] = {0};
	}
}
