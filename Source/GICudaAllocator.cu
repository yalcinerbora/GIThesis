#include "GICudaAllocator.h"
#include <cuda_gl_interop.h>

GICudaAllocator::GICudaAllocator()
{
	cudaGLSetGLDevice(0);
}

void GICudaAllocator::LinkOGLVoxelCache(GLuint batchAABBBuffer,
										GLuint batchTransformBuffer,
										GLuint relativeTransformBuffer,
										GLuint infoBuffer,
										GLuint voxelBuffer,
										GLuint voxelRenderBuffer)
{
	rTransformLinks.emplace_back(nullptr);
	transformLinks.emplace_back(nullptr);
	aabbLinks.emplace_back(nullptr);
	objectInfoLinks.emplace_back(nullptr);
	cacheLinks.emplace_back(nullptr);
	cacheRenderLinks.emplace_back(nullptr);

	hipGraphicsGLRegisterBuffer(&rTransformLinks.back(), relativeTransformBuffer, cudaGraphicsMapFlagsReadOnly);
	hipGraphicsGLRegisterBuffer(&transformLinks.back(), batchTransformBuffer, cudaGraphicsMapFlagsReadOnly);
	hipGraphicsGLRegisterBuffer(&aabbLinks.back(), batchAABBBuffer, cudaGraphicsMapFlagsReadOnly);
	hipGraphicsGLRegisterBuffer(&objectInfoLinks.back(), infoBuffer, cudaGraphicsMapFlagsReadOnly);

	hipGraphicsGLRegisterBuffer(&cacheLinks.back(), voxelBuffer, cudaGraphicsMapFlagsReadOnly);
	hipGraphicsGLRegisterBuffer(&cacheRenderLinks.back(), voxelRenderBuffer, cudaGraphicsMapFlagsReadOnly);
}

void GICudaAllocator::SetupPointersDevicePointers()
{

}

void GICudaAllocator::ClearDevicePointers()
{

}


void GICudaAllocator::AddVoxelPage(size_t count)
{

}

void GICudaAllocator::ShrinkVoxelPages(size_t pageCount)
{

}