#include "hip/hip_runtime.h"
#include "GIVoxelPages.h"
#include "PageKernels.cuh"
#include "DrawBuffer.h"
#include "CudaInit.h"
#include "CudaTimer.h"
#include "GIVoxelCache.h"
#include "GISparseVoxelOctree.h"
#include "MeshBatchSkeletal.h"
#include "OGLTimer.h"
#include "IEUtility/IEMath.h"
#include "GLSLBindPoints.h"
#include "Camera.h"
#include <cuda_gl_interop.h>
#include "IEUtility/IEAxisAalignedBB.h"

inline static std::ostream& operator<<(std::ostream& ostr, const CSegmentInfo& segObj)
{
	uint16_t cascadeNo = (segObj.packed >> 14) & 0x0003;
	uint16_t objType = (segObj.packed >> 12) & 0x0003;
	uint16_t occupation = (segObj.packed >> 10) & 0x0003;

	ostr << cascadeNo << ", ";
	ostr << segObj.batchId << ", ";
	ostr << segObj.objId << " | ";

	ostr << segObj.objectSegmentId << " | ";
	ostr << objType << " | ";
	ostr << occupation << " | ";
	return ostr;
}

GIVoxelPages::FastVoxelizer::FastVoxelizer()
	: denseResource(nullptr)
	, octreeParams(nullptr)
{}

GIVoxelPages::FastVoxelizer::FastVoxelizer(OctreeParameters* octreeParams)
	: denseResource(nullptr)
	, octreeParams(octreeParams)
{
	size_t offset = 0;
	
	// Grid Transform
	offset += DeviceOGLParameters::UBOAlignOffset(offset);
	gridTransformOffset = static_cast<GLuint>(offset);
	offset += sizeof(FrameTransformData);
	// Dense
	offset += DeviceOGLParameters::SSBOAlignOffset(offset);
	denseOffset = static_cast<GLuint>(offset);
	offset += octreeParams->CascadeBaseLevelSize *
			  octreeParams->CascadeBaseLevelSize *
			  octreeParams->CascadeBaseLevelSize * sizeof(uint2);
	// Allocator
	allocatorOffset = static_cast<GLuint>(offset);
	offset += sizeof(uint32_t);

	// Gen OGL Buffers
	oglData.Resize(offset, false);
	
	CUDA_CHECK(hipGraphicsGLRegisterBuffer(&denseResource,
											oglData.getGLBuffer(),
											hipGraphicsRegisterFlagsReadOnly));
}

GIVoxelPages::FastVoxelizer::FastVoxelizer(FastVoxelizer&& other)
	: oglData(std::move(other.oglData))
	, denseResource(other.denseResource)
	, octreeParams(other.octreeParams)
{
	other.denseResource = nullptr;
}

GIVoxelPages::FastVoxelizer& GIVoxelPages::FastVoxelizer::operator=(FastVoxelizer&& other)
{
	assert(this != &other);
	if(denseResource) CUDA_CHECK(hipGraphicsUnregisterResource(denseResource));
	oglData = std::move(other.oglData);
	denseResource = other.denseResource;
	octreeParams = other.octreeParams;
	other.denseResource = nullptr;
	return *this;
}

GIVoxelPages::FastVoxelizer::~FastVoxelizer()
{
	if(denseResource) CUDA_CHECK(hipGraphicsUnregisterResource(denseResource));
}

double GIVoxelPages::FastVoxelizer::Voxelize(const std::vector<MeshBatchI*>& batches,
										   const IEVector3& gridCenter,
										   const IEVector3& gridCorner,
										   bool doTiming)
{
	OGLTimer t;
	if(doTiming) t.Start();

	// States
	glDisable(GL_DEPTH_TEST);
	glDisable(GL_CULL_FACE);
	glEnable(GL_MULTISAMPLE);
	//glEnable(GL_CONSERVATIVE_RASTERIZATION_NV);
	glDepthMask(false);
	glStencilMask(0x0000);
	glColorMask(false, false, false, false);

	GLsizei totalSize = static_cast<GLsizei>(octreeParams->CascadeBaseLevelSize);
	glViewport(0, 0, totalSize, totalSize);

	// Generate GridAABB
	float worldWidthHeight = gridCenter[0] - gridCorner[0];
	IEVector3 gridMin = gridCorner;
	IEVector3 gridMax = gridMin + (worldWidthHeight * 2.0f);
	IEAxisAlignedBB3 gridAABB(gridMin, gridMax);

	// Generate Ortho Projection and View
	IEMatrix4x4 projectionAndView[2];
	projectionAndView[0] = IEMatrix4x4::Ortogonal(worldWidthHeight, worldWidthHeight,
												  worldWidthHeight * 0.5f,
												  worldWidthHeight * 0.5f);
	projectionAndView[1] = IEMatrix4x4::LookAt(gridCenter,
											   gridCenter - IEVector3::ZAxis,
											   IEVector3::YAxis);
	oglData.SendSubData(reinterpret_cast<const uint8_t*>(projectionAndView),
						gridTransformOffset,
						sizeof(FrameTransformData));

	
	// Dense Buffer & GridTransform buffer
	oglData.BindAsUniformBuffer(U_GRID_TRANSFORM, gridTransformOffset, sizeof(FrameTransformData));
	oglData.BindAsShaderStorageBuffer(LU_VOXEL_RENDER, denseOffset,
									  octreeParams->CascadeBaseLevelSize *
									  octreeParams->CascadeBaseLevelSize *
									  octreeParams->CascadeBaseLevelSize);

	// Shaders
	Shader::Unbind(ShaderType::GEOMETRY);
	fragVoxelizeFast.Bind();

	for(MeshBatchI* batch : batches)
	{
		DrawBuffer& drawBuffer = batch->getDrawBuffer();
		VertexBuffer& vertexBuffer = batch->getVertexBuffer();

		// Batch Binds
		vertexBuffer.Bind();
		drawBuffer.BindModelTransform(LU_MTRANSFORM);
		drawBuffer.BindAsDrawIndirectBuffer();
		if(batch->MeshType() == MeshBatchType::SKELETAL)
		{
			MeshBatchSkeletal* batchPtr = static_cast<MeshBatchSkeletal*>(batch);
			batchPtr->getJointTransforms().BindAsShaderStorageBuffer(LU_JOINT_TRANS);
			vertVoxelizeFastSkeletal.Bind();
		}
		else vertVoxelizeFast.Bind();
		
		// For each object
		for(uint32_t drawId = 0; drawId < batch->DrawCount(); drawId++)
		{
			// Do a AABB check with grid and skip if out of bounds
			const auto& aabbData = drawBuffer.getAABB(drawId);
			IEAxisAlignedBB3 objectAABB(aabbData.min, aabbData.max);
			if(!objectAABB.Intersects(gridAABB)) continue;
			
			// Bind material and draw
			drawBuffer.BindMaterialForDraw(drawId);
			drawBuffer.DrawCallSingle(drawId);
		}		
	}

	if(doTiming)
	{
		t.Stop();
		return t.ElapsedMS();
	}
	return 0.0;
}

double GIVoxelPages::FastVoxelizer::Filter(uint32_t& offset, CVoxelPage* dVoxelPages,
										   uint32_t pageCapacity, uint32_t cascadeId,
										   bool doTiming)
{
	CudaTimer t;
	if(doTiming) t.Start();

	// Map to CUDA
	uint8_t* oglDataCUDA; size_t size = 0;
	CUDA_CHECK(hipGraphicsMapResources(1, &denseResource));
	CUDA_CHECK(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&oglDataCUDA),
													&size, denseResource));
	uint2* dDenseData = reinterpret_cast<uint2*>(oglDataCUDA + denseOffset);
	uint32_t& dAllocator = reinterpret_cast<uint32_t&>(oglDataCUDA[allocatorOffset]);

	// Filter valid voxels to page system
	int totalSize = octreeParams->CascadeBaseLevelSize *
					octreeParams->CascadeBaseLevelSize *
					octreeParams->CascadeBaseLevelSize;
	int gridSize = CudaInit::GenBlockSize(totalSize);
	int blockSize = CudaInit::TBP;

	// KC
	FilterVoxels<<<gridSize, blockSize>>>(// Voxel System
										  dVoxelPages,
										  // Dense Data from OGL
										  dAllocator,
										  dDenseData,
										  // Limits
										  cascadeId,
										  offset);

	// Fetch Allocator for total voxel count
	// then determine offset
	//offset += ??;

	// Unmap
	CUDA_CHECK(hipGraphicsUnmapResources(1, &denseResource));

	if(doTiming)
	{
		t.Stop();
		return t.ElapsedMilliS();
	}
	return 0.0;
}

double GIVoxelPages::FastVoxelizer::FastVoxelize(CVoxelPage* dVoxelPages, uint32_t pageCount,
												 const CVoxelGrid* dVoxelGrids, uint32_t gridCount,
												 const std::vector<MeshBatchI*>& batches,
												 bool doTiming)
{
	assert(denseResource);

	// Copy new positions
	std::vector<IEVector3> gridPos(gridCount, IEVector3::ZeroVector);
	CUDA_CHECK(hipMemcpy2D(gridPos.data(), sizeof(IEVector3),
							dVoxelPages, sizeof(CVoxelGrid),
							sizeof(IEVector3), gridCount,
							hipMemcpyDeviceToHost));

	double totalTime = 0.0f;

	// Find Center Pos
	uint32_t usedSegmentCount = 0;
	for(uint32_t i = 0; i < gridCount; i++)
	{
		// Clear oglData
		oglData.Memset(static_cast<uint32_t>(0x0));

		// Voxelize to Center Grid
		IEVector3 gridCenter = gridPos[i] + (octreeParams->BaseSpan *
											 octreeParams->CascadeBaseLevel *
											 (0x1 << (gridCount - i - 1)));
		totalTime += Voxelize(batches, gridCenter, gridPos[i], doTiming);

		// Filter valid voxel to page system
		totalTime += Filter(usedSegmentCount, dVoxelPages, pageCount, i, doTiming);
	}
	return totalTime;
}

GIVoxelPages::PageRenderer::PageRenderer()
	: debugBufferResource(nullptr)
	, debugBufferCUDA(nullptr)
	, drawParameterOffset(0)
	, atomicIndexOffset(0)
	, gridInfoOffset(0)
	, voxelPositionOffset(0)
	, voxelRenderOffset(0)
{}

GIVoxelPages::PageRenderer::PageRenderer(const GIVoxelPages& pages)
	: vRenderWorldVoxel(ShaderType::VERTEX, "Shaders/VoxRenderWorld.vert")
	, fRenderWorldVoxel(ShaderType::FRAGMENT, "Shaders/VoxRender.frag")
	, debugBufferResource(nullptr)
	, debugBufferCUDA(nullptr)
	, drawParameterOffset(0)
	, atomicIndexOffset(0)
	, gridInfoOffset(0)
	, voxelPositionOffset(0)
	, voxelRenderOffset(0)
{
	VoxelVAO::CubeOGL cube = VoxelVAO::LoadCubeDataFromGFG();
	size_t maxVoxelCount = pages.dPages.Size() * PageSize;

	// Since Grid info will be bound as SSBO it inneds to be properly aligned
	size_t cubeOffset = cube.data.size();
	size_t cubeVertexOffset = cube.drawCount * sizeof(uint32_t);

	// Grid Info
	size_t offset = cubeOffset;
	offset = DeviceOGLParameters::SSBOAlignOffset(offset);
	gridInfoOffset = offset;
	offset += pages.svoParams->CascadeCount * sizeof(CVoxelGrid);
	
	// Atomic Index
	drawParameterOffset = offset;
	atomicIndexOffset = offset + offsetof(DrawPointIndexed, instanceCount);
	offset += sizeof(DrawPointIndexed);
	// Voxel Positions
	voxelPositionOffset = offset;
	offset += maxVoxelCount * sizeof(VoxelPosition);
	// Voxel Albedo or Normal
	voxelRenderOffset = offset;
	offset += maxVoxelCount * sizeof(VoxelNormal);
	static_assert(sizeof(VoxelNormal) == sizeof(VoxelAlbedo), "Implementation assumes all debug render types has the same size");

	// Allocate
	debugDrawBuffer.Resize(offset, false);

	// Now Register
	size_t bufferSize = 0;
	CUDA_CHECK(hipGraphicsGLRegisterBuffer(&debugBufferResource, debugDrawBuffer.getGLBuffer(),
											cudaGraphicsMapFlagsWriteDiscard));
	CUDA_CHECK(hipGraphicsMapResources(1, &debugBufferResource));
	CUDA_CHECK(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&debugBufferCUDA),
													&bufferSize,
													debugBufferResource));
	assert(bufferSize == debugDrawBuffer.Capacity());

	// Copy Cube Vertex and Indices
	CUDA_CHECK(hipMemcpy(debugBufferCUDA,
						  cube.data.data(),
						  cube.data.size(),
						  hipMemcpyHostToDevice));

	// Copy Grid Info
	CUDA_CHECK(hipMemcpy(debugBufferCUDA + gridInfoOffset,
						  pages.dVoxelGrids,
						  pages.svoParams->CascadeCount * sizeof(CVoxelGrid),
						  hipMemcpyDeviceToDevice));

	// Copy Draw Point
	DrawPointIndexed dp =
	{
		cube.drawCount,
		0,	// Instance count will be filled each frame
		0,
		0,
		0
	};
	CUDA_CHECK(hipMemcpy(debugBufferCUDA + drawParameterOffset,
						  &dp, sizeof(DrawPointIndexed),
						  hipMemcpyHostToDevice));

	// All Done! (Unmap and continue)
	CUDA_CHECK(hipGraphicsUnmapResources(1, &debugBufferResource));
	debugBufferCUDA = nullptr;

	// Finally Generate VAO
	debugDrawVao = VoxelVAO(debugDrawBuffer,
							cubeVertexOffset,
							voxelPositionOffset,
							voxelRenderOffset);
}

GIVoxelPages::PageRenderer::PageRenderer(PageRenderer&& other)
	: vRenderWorldVoxel(std::move(other.vRenderWorldVoxel))
	, fRenderWorldVoxel(std::move(other.fRenderWorldVoxel))
	, debugBufferResource(other.debugBufferResource)
	, debugDrawBuffer(std::move(other.debugDrawBuffer))
	, debugBufferCUDA(other.debugBufferCUDA)
	, debugDrawVao(std::move(other.debugDrawVao))
	, drawParameterOffset(other.drawParameterOffset)
	, atomicIndexOffset(other.atomicIndexOffset)
	, gridInfoOffset(other.gridInfoOffset)
	, voxelPositionOffset(other.voxelPositionOffset)
	, voxelRenderOffset(other.voxelRenderOffset)
{
	other.debugBufferResource = nullptr;
	other.debugBufferCUDA = nullptr;
}

GIVoxelPages::PageRenderer& GIVoxelPages::PageRenderer::operator=(PageRenderer&& other)
{
	if(debugBufferResource)
		CUDA_CHECK(hipGraphicsUnregisterResource(debugBufferResource));

	vRenderWorldVoxel = std::move(other.vRenderWorldVoxel);
	fRenderWorldVoxel = std::move(other.fRenderWorldVoxel);
	debugBufferResource = other.debugBufferResource;
	debugDrawBuffer = std::move(other.debugDrawBuffer);
	debugBufferCUDA = other.debugBufferCUDA;
	debugDrawVao = std::move(other.debugDrawVao);
	drawParameterOffset = other.drawParameterOffset;
	atomicIndexOffset = other.atomicIndexOffset;
	gridInfoOffset = other.gridInfoOffset;
	voxelPositionOffset = other.voxelPositionOffset;
	voxelRenderOffset = other.voxelRenderOffset;

	other.debugBufferResource = nullptr;
	other.debugBufferCUDA = nullptr;
	return *this;
}

GIVoxelPages::PageRenderer::~PageRenderer()
{
	if(debugBufferResource)
		CUDA_CHECK(hipGraphicsUnregisterResource(debugBufferResource));
}

double GIVoxelPages::PageRenderer::Draw(bool doTiming,
										uint32_t cascade,
										VoxelRenderType renderType,
										const Camera& camera,
										const GIVoxelCache& cache,
										const GIVoxelPages& pages)
{
	// Skip if not allocated
	if(!Allocated()) return 0.0;

	CudaTimer cT;
	if(doTiming) cT.Start();

	// Map Buffer
	size_t bufferSize = 0;
	CUDA_CHECK(hipGraphicsMapResources(1, &debugBufferResource));
	CUDA_CHECK(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&debugBufferCUDA),
													&bufferSize,
													debugBufferResource));
	assert(bufferSize == debugDrawBuffer.Capacity());

	// Copy Requested Data
	// Gen pointers
	VoxelPosition* voxelPosition = reinterpret_cast<VoxelPosition*>(debugBufferCUDA + voxelPositionOffset);
	unsigned int* voxelRender = reinterpret_cast<unsigned int*>(debugBufferCUDA + voxelRenderOffset);
	unsigned int* atomicIndex = reinterpret_cast<unsigned int*>(debugBufferCUDA + atomicIndexOffset);

	// Clear atomic counter
	CUDA_CHECK(hipMemset(atomicIndex, 0x00, sizeof(unsigned int)));

	// Load new Grid Positions
	// Copy Grid Info
	CUDA_CHECK(hipMemcpy2D(debugBufferCUDA + gridInfoOffset, sizeof(CVoxelGrid),
							pages.dVoxelGrids, sizeof(CVoxelGrid),
							sizeof(float3), pages.svoParams->CascadeCount,
							hipMemcpyDeviceToDevice));

	// KC
	int gridSize = CudaInit::GenBlockSize(static_cast<int>(pages.dPages.Size() * PageSize));
	int blockSize = CudaInit::TBP;
	CopyPage<<<gridSize, blockSize>>>(// OGL Buffer
								      voxelPosition,
								      voxelRender,
								      *atomicIndex,
								      // Voxel Cache
								      cache.getDeviceCascadePointersDevice().Data(),
								      // Voxel Pages
								      reinterpret_cast<const CVoxelPageConst*>(pages.dPages.Data()),
								      //
								      static_cast<uint32_t>(pages.batches->size()),
								      cascade,
								      renderType);
	CUDA_KERNEL_CHECK();

	//// DEBUG
	//uint32_t nodesInCirculation = 0;
	//CUDA_CHECK(hipMemcpy(&nodesInCirculation, atomicIndex, sizeof(uint32_t), hipMemcpyDeviceToHost));
	//GI_LOG("Total Valid node count in pages : %d", nodesInCirculation);

	// Unmap buffer and continue
	CUDA_CHECK(hipGraphicsUnmapResources(1, &debugBufferResource));
	debugBufferCUDA = nullptr;

	// Timing
	OGLTimer t;
	if(doTiming)
	{
		cT.Stop();
		t.Start();
	}
	
	// Now render
	// Framebuffer
	glBindFramebuffer(GL_FRAMEBUFFER, 0);
	glViewport(0, 0,
			   static_cast<GLsizei>(camera.width),
			   static_cast<GLsizei>(camera.height));

	// State
	glDisable(GL_MULTISAMPLE);
	glEnable(GL_DEPTH_TEST);
	glEnable(GL_CULL_FACE);
	glDepthFunc(GL_LEQUAL);
	glDepthMask(true);
	glColorMask(true, true, true, true);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	// Shaders
	Shader::Unbind(ShaderType::GEOMETRY);
	vRenderWorldVoxel.Bind();
	glUniform1ui(U_RENDER_TYPE, static_cast<GLuint>(renderType));
	fRenderWorldVoxel.Bind();

	// Uniforms
	debugDrawBuffer.BindAsShaderStorageBuffer(LU_VOXEL_GRID_INFO, 
											  static_cast<GLuint>(gridInfoOffset), 
											  static_cast<GLuint>(pages.svoParams->CascadeCount * sizeof(CVoxelGrid)));

	// Draw Indirect Buffer
	debugDrawBuffer.BindAsDrawIndirectBuffer();

	// VAO
	debugDrawVao.Bind();
	debugDrawVao.Draw(static_cast<GLuint>(drawParameterOffset));

	// Timer
	if(doTiming)
	{
		t.Stop();
		return t.ElapsedMS() + cT.ElapsedMilliS();
	}
	return 0.0;
}

bool GIVoxelPages::PageRenderer::Allocated() const
{
	return vRenderWorldVoxel.IsValid();
}

GIVoxelPages::MultiPage::MultiPage(size_t pageCount)
{
	assert(pageCount != 0);
	size_t sizePerPage = GIVoxelPages::PageSize *
						 (sizeof(CVoxelPos) +
						  sizeof(CVoxelNorm) +
						  sizeof(CVoxelOccupancy))
						 +
						 GIVoxelPages::SegmentSize *
						 (sizeof(unsigned char) +
						  sizeof(CSegmentInfo));

	size_t totalSize = sizePerPage * pageCount;
	pageData.Resize(totalSize);
	pageData.Memset(0x00, 0, totalSize);
	
	uint8_t* dPtr = pageData.Data();
	ptrdiff_t offset = 0;
	for(size_t i = 0; i < pageCount; i++)
	{
		CVoxelPage page = {};

		page.dGridVoxPos = reinterpret_cast<CVoxelPos*>(dPtr + offset);
		offset += GIVoxelPages::PageSize * sizeof(CVoxelPos);

		page.dGridVoxNorm = reinterpret_cast<CVoxelNorm*>(dPtr + offset);
		offset += GIVoxelPages::PageSize * sizeof(CVoxelNorm);
		
		page.dGridVoxOccupancy = reinterpret_cast<CVoxelOccupancy*>(dPtr + offset);
		offset += GIVoxelPages::PageSize * sizeof(CVoxelOccupancy);

		page.dEmptySegmentPos = reinterpret_cast<unsigned char*>(dPtr + offset);
		offset += GIVoxelPages::SegmentSize * sizeof(unsigned char);

		page.dSegmentInfo = reinterpret_cast<CSegmentInfo*>(dPtr + offset);
		offset += GIVoxelPages::SegmentSize * sizeof(CSegmentInfo);

		page.dEmptySegmentStackSize = GIVoxelPages::SegmentPerPage;
		pages.push_back(page);
	}
	assert(offset == pageData.Size());

	// KC to Initialize Empty Segment Stack
	int gridSize = CudaInit::GenBlockSizeSmall(static_cast<uint32_t>(pageCount * GIVoxelPages::SegmentPerPage));
	int blockSize = CudaInit::TBP;
	InitializePage<<<gridSize, blockSize>>>(pages.front().dEmptySegmentPos, pageCount);
	CUDA_KERNEL_CHECK();
}

GIVoxelPages::MultiPage::MultiPage(MultiPage&& other)
	: pageData(std::move(other.pageData))
	, pages(std::move(other.pages))
{}

size_t GIVoxelPages::MultiPage::PageCount() const
{
	return pages.size();
}

const std::vector<CVoxelPage>& GIVoxelPages::MultiPage::Pages() const
{
	return pages;
}

uint16_t GIVoxelPages::PackSegmentInfo(const uint8_t cascadeId,
									   const CObjectType type,
									   const CSegmentOccupation occupation,
									   const bool firstOccurance)
{
	// MSB to LSB 
	// 2 bit cascadeId
	// 2 bit object type 
	// 2 bit segment occupation
	uint16_t packed = 0;
	packed |= (static_cast<uint16_t>(cascadeId) & 0x0003) << 14;
	packed |= (static_cast<uint16_t>(type) & 0x0003) << 12;
	packed |= (static_cast<uint16_t>(occupation) & 0x0003) << 10;
	packed |= (static_cast<uint16_t>(firstOccurance) & 0x0001) << 9;
	return packed;
}

void GIVoxelPages::GenerateGPUData(const GIVoxelCache& cache)
{
	// Generate SegInfos
	std::vector<CVoxelGrid> grids;
	std::vector<CSegmentInfo> segInfos;
	std::vector<std::vector<bool>> checkBase(batches->size());

	for(uint32_t cascadeId = 0; cascadeId < svoParams->CascadeCount; cascadeId++)
	{
		CVoxelGrid grid = {};
		grid.depth = svoParams->CascadeBaseLevel + svoParams->CascadeCount - cascadeId - 1;
		grid.dimension = 
		{
			svoParams->CascadeBaseLevelSize,
			svoParams->CascadeBaseLevelSize,
			svoParams->CascadeBaseLevelSize
		};
		grid.position = {0.0f, 0.0f, 0.0f};
		grid.span = svoParams->BaseSpan * static_cast<float>(1 << cascadeId);
		grids.push_back(grid);
		
		for(uint32_t batchId = 0; batchId < batches->size(); batchId++)
		{
			if((*batches)[batchId]->DrawCount() == 0) continue;
			if(cascadeId == 0) checkBase[batchId].resize((*batches)[batchId]->DrawCount(), true);
			
			bool nonRigid = (*batches)[batchId]->MeshType() == MeshBatchType::SKELETAL;
			const std::vector<CMeshVoxelInfo> voxInfo = cache.CopyMeshObjectInfo(cascadeId, batchId);

			for(uint32_t objId = 0; objId < voxInfo.size(); objId++)
			{
				const CMeshVoxelInfo& info = voxInfo[objId];
				bool firstOccurance = false;
				if(info.voxCount != 0 && checkBase[batchId][objId] == true)
				{
					checkBase[batchId][objId] = false;
					firstOccurance = true;
				}

				uint32_t segmentCount = (info.voxCount + SegmentSize - 1) / SegmentSize;
				for(uint32_t segId = 0; segId < segmentCount; segId++)
				{
					CObjectType objType = (nonRigid) ? CObjectType::SKEL_DYNAMIC : CObjectType::DYNAMIC;

					CSegmentInfo segInfo;
					segInfo.batchId = static_cast<uint16_t>(batchId);
					segInfo.objectSegmentId = static_cast<uint16_t>(segId);
					segInfo.objId = static_cast<uint16_t>(objId);
					segInfo.packed = PackSegmentInfo(static_cast<uint8_t>(cascadeId), objType,
													 CSegmentOccupation::OCCUPIED, 
													 firstOccurance);

					segInfos.push_back(segInfo);
				}
			}
		}
	}

	// Determine Buffer Size
	size_t bufferSize = segInfos.size() * (sizeof(CSegmentInfo) +
										   sizeof(ushort2));
	bufferSize += batches->size() * sizeof(BatchOGLData);
	bufferSize += svoParams->CascadeCount * sizeof(CVoxelGrid);

	// Send Data to Buffer
	gpuData.Resize(bufferSize);
	size_t bufferOffset = 0;
	// Grids
	CUDA_CHECK(hipMemcpy(gpuData.Data() + bufferOffset,
						  reinterpret_cast<void*>(grids.data()),
						  grids.size() * sizeof(CVoxelGrid),
						  hipMemcpyHostToDevice));
	dVoxelGrids = reinterpret_cast<CVoxelGrid*>(gpuData.Data() + bufferOffset);
	bufferOffset += grids.size() * sizeof(CVoxelGrid);
	// OGL Data
	CUDA_CHECK(hipMemset(gpuData.Data() + bufferOffset, 0,
						  batches->size() * sizeof(BatchOGLData)));
	dBatchOGLData = reinterpret_cast<BatchOGLData*>(gpuData.Data() + bufferOffset);
	bufferOffset += batches->size() * sizeof(BatchOGLData);
	// Segments Alloc
	CUDA_CHECK(hipMemset(gpuData.Data() + bufferOffset, 0xFFFFFFFF,
						  segInfos.size() * sizeof(ushort2)));
	dSegmentAllocInfo = reinterpret_cast<ushort2*>(gpuData.Data() + bufferOffset);
	bufferOffset += segInfos.size() * sizeof(ushort2);
	// Segments
	CUDA_CHECK(hipMemcpy(gpuData.Data() + bufferOffset,
						  reinterpret_cast<void*>(segInfos.data()),
						  segInfos.size() * sizeof(CSegmentInfo),
						  hipMemcpyHostToDevice));
	dSegmentInfo = reinterpret_cast<CSegmentInfo*>(gpuData.Data() + bufferOffset);
	bufferOffset += segInfos.size() * sizeof(CSegmentInfo);
	assert(bufferOffset == gpuData.Size());
	segmentAmount = static_cast<uint32_t>(segInfos.size());
}

void GIVoxelPages::AllocatePages(size_t voxelCapacity)
{
	size_t pageCount = (voxelCapacity + PageSize - 1) / PageSize;
	size_t oldSize = dPages.Size();

	hPages.emplace_back(pageCount);
	dPages.Resize(oldSize + hPages.back().PageCount());
	dPages.Assign(oldSize, hPages.back().PageCount(), hPages.back().Pages().data());
}

void GIVoxelPages::MapOGLResources()
{
	CUDA_CHECK(hipGraphicsMapResources(static_cast<int>(batchOGLResources.size()), batchOGLResources.data()));

	std::vector<BatchOGLData> newOGLData;
	size_t batchIndex = 0;
	for(size_t i = 0; i < batches->size(); i++)
	{
		MeshBatchI& currentBatch = *(*batches)[i];
		if(currentBatch.DrawCount() == 0)
		{
			newOGLData.push_back({});
			continue;
		}

		size_t size;
		uint8_t* glPointer = nullptr;
		CUDA_CHECK(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&glPointer),
														&size, batchOGLResources[batchIndex]));

		size_t aabbByteOffset = (*batches)[i]->getDrawBuffer().getAABBOffset();
		size_t modelTransformByteOffset = (*batches)[i]->getDrawBuffer().getModelTransformOffset();
		size_t modelTransformIndexByteOffset = (*batches)[i]->getDrawBuffer().getModelTransformIndexOffset();

		BatchOGLData batchGL = {};
		batchGL.dAABBs = reinterpret_cast<CAABB*>(glPointer + aabbByteOffset);
		batchGL.dModelTransforms = reinterpret_cast<CModelTransform*>(glPointer + modelTransformByteOffset);
		batchGL.dModelTransformIndices = reinterpret_cast<uint32_t*>(glPointer + modelTransformIndexByteOffset);
		
		batchIndex++;
		if((*batches)[i]->MeshType() == MeshBatchType::SKELETAL)
		{
			CUDA_CHECK(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&glPointer),
															&size, batchOGLResources[batchIndex]));
			batchGL.dJointTransforms = reinterpret_cast<CJointTransform*>(glPointer);
			batchIndex++;
		}
		newOGLData.push_back(batchGL);
	}

	// Copy generated pointers to GPU
	CUDA_CHECK(hipMemcpy(dBatchOGLData,
						  newOGLData.data(),
						  batches->size() * sizeof(BatchOGLData),
						  hipMemcpyHostToDevice));
}

void GIVoxelPages::UnmapOGLResources()
{
	CUDA_CHECK(hipGraphicsUnmapResources(static_cast<int>(batchOGLResources.size()), batchOGLResources.data()));
}

double GIVoxelPages::VoxelizeFast()
{
	return 0.0f;
}

void GIVoxelPages::Update(double& ioTime,
											   double& transTime,
											   const GIVoxelCache& caches,
											   const IEVector3& camPos,
											   bool doTiming,
											   bool useCache)
{
	UpdateGridPositions(camPos);
	if(useCache)
	{
		MapOGLResources();
		ioTime = VoxelIO(doTiming);
		transTime = Transform(caches, doTiming);
		UnmapOGLResources();
	}
	else
	{
		transTime = VoxelizeFast();
	}
}

GIVoxelPages::GIVoxelPages()
	: batches(nullptr)
	, svoParams(nullptr)
	, segmentAmount(0)
	, dVoxelGrids(nullptr)
	, dBatchOGLData(nullptr)
	, dSegmentInfo(nullptr)
	, dSegmentAllocInfo(nullptr)

{}

GIVoxelPages::GIVoxelPages(const GIVoxelCache& cache, 
						   const std::vector<MeshBatchI*>* batches,
						   const OctreeParameters& octreeParams)
	: batches(batches)
	, svoParams(&octreeParams)
	, segmentAmount(0)
	, dVoxelGrids(nullptr)
	, dBatchOGLData(nullptr)
	, dSegmentInfo(nullptr)
	, dSegmentAllocInfo(nullptr)
{
	for(uint32_t i = 0; i < batches->size(); i++)
	{	
		MeshBatchI& batch = *(*batches)[i];
		if(batch.DrawCount() == 0) continue;

		GLuint bufferId = batch.getDrawBuffer().getGLBuffer();
		hipGraphicsResource_t glResource;
		CUDA_CHECK(hipGraphicsGLRegisterBuffer(&glResource, 
												bufferId,
												cudaGraphicsMapFlagsReadOnly));
		batchOGLResources.push_back(glResource);

		if(batch.MeshType() == MeshBatchType::SKELETAL)
		{
			GLuint jointBuffer = static_cast<MeshBatchSkeletal&>(batch).getJointTransforms().getGLBuffer();
			CUDA_CHECK(hipGraphicsGLRegisterBuffer(&glResource,
													jointBuffer,
													cudaGraphicsMapFlagsReadOnly));
			batchOGLResources.push_back(glResource);
		}
	}
	GenerateGPUData(cache);
	AllocatePages(segmentAmount * SegmentSize);
}

GIVoxelPages::GIVoxelPages(GIVoxelPages&& other)
	: batches(other.batches)
	, svoParams(other.svoParams)
	, segmentAmount(other.segmentAmount)
	, outermostGridPosition(other.outermostGridPosition)
	, gpuData(std::move(other.gpuData))
	, dVoxelGrids(other.dVoxelGrids)
	, dBatchOGLData(other.dBatchOGLData)
	, dSegmentInfo(other.dSegmentInfo)
	, dSegmentAllocInfo(other.dSegmentAllocInfo)
	, hPages(std::move(other.hPages))
	, dPages(std::move(other.dPages))
	, batchOGLResources(std::move(other.batchOGLResources))
	, pageRenderer(std::move(other.pageRenderer))
{
	assert(other.batchOGLResources.empty());
}

GIVoxelPages& GIVoxelPages::operator=(GIVoxelPages&& other)
{
	assert(&other != this);
	for(hipGraphicsResource_t resc : batchOGLResources)
	{
		CUDA_CHECK(hipGraphicsUnregisterResource(resc));
	}

	batches = other.batches;
	svoParams = other.svoParams;
	segmentAmount = other.segmentAmount;
	outermostGridPosition = other.outermostGridPosition;
	gpuData = std::move(other.gpuData);
	dVoxelGrids = other.dVoxelGrids;
	dBatchOGLData = other.dBatchOGLData;
	dSegmentInfo = other.dSegmentInfo;
	dSegmentAllocInfo = other.dSegmentAllocInfo;
	hPages = std::move(other.hPages);
	dPages = std::move(other.dPages);
	batchOGLResources = std::move(other.batchOGLResources);
	pageRenderer = std::move(other.pageRenderer);
	return *this;
}

GIVoxelPages::~GIVoxelPages()
{
	for(hipGraphicsResource_t resc : batchOGLResources)
	{
		CUDA_CHECK(hipGraphicsUnregisterResource(resc));
	}
}

void GIVoxelPages::UpdateGridPositions(const IEVector3& cameraPos)
{
	// Calculate outermost span position
	float outerSpan = svoParams->BaseSpan * static_cast<float>(1 << (svoParams->CascadeCount - 1));
	IEVector3 voxelCornerPos = cameraPos - outerSpan * (svoParams->CascadeBaseLevelSize - 1) * 0.5f;

	// Align outermost cascade
	// TODO: Better solution for higher level voxel jittering
    float rootSnapLevelMultiplier = static_cast<float>(0x1 << 3);

	// Removes Jitterin on base cascade level
	float snapSpan = outerSpan * rootSnapLevelMultiplier;
	voxelCornerPos[0] -= std::fmod(voxelCornerPos[0] + snapSpan * 0.5f, snapSpan);
	voxelCornerPos[1] -= std::fmod(voxelCornerPos[1] + snapSpan * 0.5f, snapSpan);
	voxelCornerPos[2] -= std::fmod(voxelCornerPos[2] + snapSpan * 0.5f, snapSpan);

	//// Grid Aligned Center
	//IEVector3 voxelCenter = voxelCornerPos + outerSpan * (svoParams->CascadeBaseLevelSize - 1) * 0.5f;
	//std::vector<IEVector3> positions(svoParams->CascadeCount);
	//for(uint32_t i = 0; i < svoParams->CascadeCount; i++)
	//{
	//	float multiplier = (0x1 << i) * (svoParams->CascadeBaseLevelSize - 1) * 0.5f;
	//	positions[i] = voxelCenter - multiplier;
	//}

	// Now align inner cascades according to outermost
	// In all system cacades and its data lied from inner to outer
	float baseHalf = svoParams->BaseSpan * 0.5f * svoParams->CascadeBaseLevelSize;
	float seriesTotal = IEMathFunctions::GeomSeries(svoParams->CascadeCount - 2, 2.0f);
	std::vector<IEVector3> positions(svoParams->CascadeCount);
	for(uint32_t i = 0; i < svoParams->CascadeCount; i++)
	{
		int32_t termLast = i - 1;
		float lastTermSum = (termLast >= 0) ?  IEMathFunctions::GeomSeries(termLast, 2.0f) : 0;
		float subSeries = seriesTotal - lastTermSum;
		float displacement = subSeries * baseHalf;
		positions[i] = voxelCornerPos + displacement;
	}

	outermostGridPosition = positions.back();

	// Copy new positions
	CUDA_CHECK(hipMemcpy2D(dVoxelGrids, sizeof(CVoxelGrid),
							positions.data(), sizeof(IEVector3),
							sizeof(IEVector3), svoParams->CascadeCount,
							hipMemcpyHostToDevice));
}

double GIVoxelPages::VoxelIO(bool doTiming)
{
	CudaTimer t;
	if(doTiming) t.Start();
	
	// KC
	int gridSize = CudaInit::GenBlockSizeSmall(static_cast<int>(segmentAmount));
	int blockSize = CudaInit::TBPSmall;
	// Voxel I-O (Deallocate first then allocate)
	VoxelDeallocate<<<gridSize, blockSize>>>(// Voxel System
										     dPages.Data(),
										     dVoxelGrids,
										     // Helper Structures		
										     dSegmentAllocInfo,
										     dSegmentInfo,
										     // Per Object Related
										     dBatchOGLData,
										     // Limits
										     segmentAmount);

	VoxelAllocate<<<gridSize, blockSize>>>(// Voxel System
										   dPages.Data(),
										   dVoxelGrids,
										   // Helper Structures		
										   dSegmentAllocInfo,
										   dSegmentInfo,
										   // Per Object Related
										   dBatchOGLData,
										   // Limits
										   segmentAmount,
										   static_cast<uint32_t>(dPages.Size()));
	CUDA_KERNEL_CHECK();
	if(doTiming)
	{
		t.Stop();
		return t.ElapsedMilliS();
	}
	return 0.0;
}

double GIVoxelPages::Transform(const GIVoxelCache& cache,
							   bool doTiming)
{
	CudaTimer t;
	if(doTiming) t.Start();

	// KC
	int gridSize = CudaInit::GenBlockSizeSmall(static_cast<int>(dPages.Size() * PageSize));
	int blockSize = CudaInit::TBPSmall;
	VoxelTransform<<<gridSize, blockSize>>>(// Voxel Pages
										    dPages.Data(),
										    dVoxelGrids,
										    // OGL Related
										    dBatchOGLData,
										    // Voxel Cache Related
										    cache.getDeviceCascadePointersDevice().Data(),
										    // Limits
										    static_cast<uint32_t>(batches->size()));
	hipDeviceSynchronize();
	CUDA_KERNEL_CHECK();
	if(doTiming)
	{
		t.Stop();
		return t.ElapsedMilliS();
	}
	return 0.0;
}

uint64_t GIVoxelPages::MemoryUsage() const
{
	size_t totalSize = gpuData.Size();
	totalSize += dPages.Size() * sizeof(CVoxelPage);
	totalSize += dPages.Size() * PageSize * (sizeof(CVoxelPos) +
											 sizeof(CVoxelNorm) +
											 sizeof(CVoxelOccupancy));
	totalSize += dPages.Size() * SegmentPerPage * (sizeof(unsigned char) +
												   sizeof(CSegmentInfo));
	return totalSize;
}

uint32_t GIVoxelPages::PageCount() const
{
	return static_cast<uint32_t>(dPages.Size());
}

void GIVoxelPages::DumpPageSegments(const char* fileName, size_t offset, size_t pageCount) const
{
	if(pageCount == 0) pageCount = dPages.Size() - offset;
	assert(offset + pageCount <= dPages.Size());

	std::vector<CVoxelPage> pages(pageCount);
	CUDA_CHECK(hipMemcpy(pages.data(), dPages.Data() + offset, 
						  pageCount * sizeof(CVoxelPage),
						  hipMemcpyDeviceToHost));

	std::vector<CSegmentInfo> infos(pageCount * SegmentPerPage);
	for(size_t i = 0; i < pageCount; i++)
	{
		const CVoxelPage& p = pages[i];
		CUDA_CHECK(hipMemcpy(infos.data() + i * SegmentPerPage,
							  p.dSegmentInfo,
							  SegmentPerPage * sizeof(CSegmentInfo),
							  hipMemcpyDeviceToHost));
	}


	std::ofstream fOut;
	fOut.open(fileName);
	for(const CSegmentInfo& data : infos)
	{
		fOut << std::uppercase << std::hex << data;
		fOut << "\t\t\t" << std::nouppercase << std::dec << data;
		fOut << std::endl;
	}
}

void GIVoxelPages::DumpPageEmptyPositions(const char* fileName, size_t offset, size_t pageCount) const
{
	if(pageCount == 0) pageCount = dPages.Size() - offset;
	assert(offset + pageCount <= dPages.Size());

	std::vector<CVoxelPage> pages(pageCount);
	CUDA_CHECK(hipMemcpy(pages.data(), dPages.Data() + offset,
						  pageCount * sizeof(CVoxelPage),
						  hipMemcpyDeviceToHost));

	std::vector<unsigned char> emptySpots(pageCount * SegmentPerPage);
	for(size_t i = 0; i < pageCount; i++)
	{
		const CVoxelPage& p = pages[i];
		CUDA_CHECK(hipMemcpy(emptySpots.data() + i * SegmentPerPage,
							  p.dEmptySegmentPos,
							  SegmentPerPage * sizeof(unsigned char),
							  hipMemcpyDeviceToHost));
	}

	std::ofstream fOut;
	fOut.open(fileName);
	for(const unsigned char& data : emptySpots)
	{
		fOut << std::uppercase << std::hex << static_cast<int>(data);
		fOut << "\t\t\t" << std::nouppercase << std::dec << static_cast<int>(data);
		fOut << std::endl;
	}
}

void GIVoxelPages::DumpSegmentAllocation(const char* fileName, size_t offset, size_t segmentCount) const
{
	if(segmentCount == 0) segmentCount = segmentAmount - offset;
	assert(offset + segmentCount <= segmentAmount);

	std::vector<ushort2> segments(segmentCount);
	CUDA_CHECK(hipMemcpy(segments.data(), dSegmentInfo + offset,
						  segmentCount * sizeof(ushort2),
						  hipMemcpyDeviceToHost));

	std::ofstream fOut;
	fOut.open(fileName);
	for(const ushort2& data : segments)
	{
		fOut << std::uppercase << std::hex << data;
		fOut << "\t\t\t" << std::nouppercase << std::dec << data;
		fOut << std::endl;
	}
}

void GIVoxelPages::DumpSegmentInfo(const char* fileName, size_t offset, size_t segmentCount) const
{
	if(segmentCount == 0) segmentCount = segmentAmount - offset;
	assert(offset + segmentCount <= segmentAmount);

	std::vector<CSegmentInfo> segments(segmentCount);
	CUDA_CHECK(hipMemcpy(segments.data(), dSegmentInfo + offset,
						  segmentCount * sizeof(CSegmentInfo),
						  hipMemcpyDeviceToHost));

	std::ofstream fOut;
	fOut.open(fileName);
	for(const CSegmentInfo& data : segments)
	{
		fOut << std::uppercase << std::hex << data;
		fOut << "\t\t\t" << std::nouppercase << std::dec << data;
		fOut << std::endl;
	}
}

void GIVoxelPages::AllocateDraw()
{
	if(!pageRenderer.Allocated())
	{
		pageRenderer = PageRenderer(*this);
	}
}

double GIVoxelPages::Draw(bool doTiming,
						  uint32_t cascadeCount,
						  VoxelRenderType renderType,
						  const Camera& camera,
						  const GIVoxelCache& cache)
{
	return pageRenderer.Draw(doTiming, cascadeCount, renderType, camera, cache, *this);
}

void GIVoxelPages::DeallocateDraw()
{
	if(pageRenderer.Allocated())
	{
		pageRenderer = PageRenderer();
	}
}

const CVoxelPageConst* GIVoxelPages::getVoxelPagesDevice() const
{
	return reinterpret_cast<const CVoxelPageConst*>(dPages.Data());
}

const CVoxelGrid* GIVoxelPages::getVoxelGridsDevice() const
{
	return dVoxelGrids;
}

const IEVector3& GIVoxelPages::getOutermostGridPosition() const
{
	return outermostGridPosition;
}